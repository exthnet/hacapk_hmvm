#include "hip/hip_runtime.h"
// -*- C++ -*-

// hybrid4
// 考えてはみたが、いまひとつピンとこないため破棄

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <typeinfo>
#include <omp.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_cooperative_groups.h>

#include "hacapk.h"

#define CHECK_DO(act,msg) {ret=act; if(ret!=hipSuccess){printf("%s failed\n",msg);exit(-1);};}

namespace cg = cooperative_groups;

#if __CUDA_ARCH__ < 600
__device__ static double myAtomicAdd(double* address, double val)
{
  unsigned long long int* address_as_ull =
	(unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;

  do {
	assumed = old;
	old = atomicCAS(address_as_ull, assumed,
					__double_as_longlong(val +
										 __longlong_as_double(assumed)));

	// Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
  } while (assumed != old);

  return __longlong_as_double(old);
}
__device__ static inline float myAtomicAdd(float* address, float val)
{
  atomicAdd(address, val);
}
#else
#define myAtomicAdd atomicAdd
#endif

// ######## ######## ######## ######## ######## ######## ######## ########
/*
  hybrid4
  複数WARP単一GEMV単一行GEMVカーネル
  1PMVをmulTBで担当
  PMV内の1行を1/div WARPが担当
  <<<napprox+ndense, 32*mul>>>
  1 PMV by mul TB
  1 line by 1/div WARP
  バリエーション
  - mul：立ち上げるブロック数スレッド数の係数、1つのmat-mat-vecまたはmat-vecを1WARPで実行、mul=1,2,3,...,
  - a2t：a2を転置版で計算するか否か
  - a2interchange：a2のループを入れ替えるか否か
  - aatomic：approxの計算をatomic優先にするかwarp shuffle併用するか
  - datomic：denseの計算をatomic優先にするかwarp shuffle併用するか
*/
template <class T, int atomic>
__global__ void hmvm_cuda_hybrid4
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, T *rowmat, T *rowmat_t,
 int napprox, int *approx, int ndense, int *dense, int mul)
{
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid4 : begin\n");
#endif
  int gid  = blockIdx.x;
  int tid  = threadIdx.x;
  int tlen = blockDim.x;
  int ndl, ndt, nstrtl, nstrtt, ltmtx;
  int ip, kt, il, it, itt, itl, ill;
  size_t head;
  T tmp = 0.0;
  extern __shared__ __align__(sizeof(T)) unsigned char my_smem[];
  T *tmp2 = reinterpret_cast<T *>(my_smem);
  cg::thread_block_tile<32> g = cg::tiled_partition<32>(cg::this_thread_block());

#if 1
  if(gid<ndense){
#ifndef _SKIP_APPROX
	ip = dense[gid];
    ndl = _ndl[ip];
    ndt = _ndt[ip];
    nstrtl = _nstrtl[ip];
    nstrtt = _nstrtt[ip];
    ltmtx = _ltmtx[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif

	head = a1[ip];
	for(il=0; il<ndl; il++){
	  tmp = 0.0;
	  ill=il+nstrtl-1;
	  for(it=tid; it<ndt; it+=tlen){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp += rowmat[head+itl]*d_zu[itt];
	  }
	  //for (int offset = warpSize/2; offset > 0; offset /= 2)tmp += __shfl_down_sync(tmp, offset, warpSize);
	  for (int offset = g.size()/2; offset > 0; offset /= 2)tmp += g.shfl_down(tmp, offset);
	  if(tid%32==0){
		atomicAdd(&d_zaut[ill], tmp);
	  }
	}
#endif
  }
#endif

#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid4 : end\n");
#endif
}

template <class T>
void hmvm_cuda_hybrid4_proxy
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *ltmtx, int *ndt, int *ndl, int *nstrtl, int *nstrtt, int *kt, int *a1, int *a2, T *rowmat, T *rowmat_t,
 int napprox, int *approx, int ndense, int *dense,
 int blocks, int threads, int shms, int mul, int atomic, T *v, T *b, int nd, char *fname, int bench)
{
#if 1
  const int L=10, M=5;
  FILE *F;
  int i, l, lmax;
  double d1, d2, dtimes[L], dmin, dmax, davg;
  hipError_t ret;
  if(bench==0){lmax=1;}else{lmax=L;}
  for(l=0;l<lmax;l++){
	for(i=0;i<nd;i++)v[i] = (T)0.0;
	CHECK_DO(hipMemcpy(d_zaut, v, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy v to d_v");
	CHECK_DO(hipMemcpy(d_zu, b, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy b to d_b");
	hipDeviceSynchronize();
	d1 = omp_get_wtime();
	switch(atomic){
	case 0:
	  hmvm_cuda_hybrid4<T,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, rowmat_t, napprox, approx, ndense, dense, mul);
	  break;
	case 1:
	  hmvm_cuda_hybrid4<T,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, rowmat_t, napprox, approx, ndense, dense, mul);
	  break;
	}
	hipDeviceSynchronize();
	d2 = omp_get_wtime();
	dtimes[l] = d2-d1;
  }
  if(bench==0){
	CHECK_DO(hipMemcpy(v, d_zaut, sizeof(T)*nd, hipMemcpyDeviceToHost),"hipMemcpy d_v to v");
	printf("write to %s\n", fname);
	F = fopen(fname, "w");
	for(i=0;i<nd;i++)fprintf(F, "%.3E\n", v[i]);
	fclose(F);
  }else{
	dmin = 9999.99;
	dmax = 0.0;
	davg = 0.0;
	for(i=M;i<L;i++){
	  davg += dtimes[i];
	  if(dmin>dtimes[i])dmin=dtimes[i];
	  if(dmax<dtimes[i])dmax=dtimes[i];
	}
	davg /= (L-M);
	printf("TIME %d hmvm_cuda1_hybrid4%s min %e max %e avg %e\n", L-M, typeid(T).name(), dmin, dmax, davg);
  }
#endif
}

template<class T>
void hmvm_cuda4(matrix2<T> *mat2, T *b, int kernel, int dump_result)
{
  const int L=10, M=5;
  FILE *F;
  matrix2<T> d_sm;
  int i, l, nd = mat2->nd, ktmax = mat2->ktmax, nlf = mat2->nlf;
  double d1, d2, dtimes[L], dmin, dmax, davg;
  T *v=NULL, *tmp=NULL, *zero;
  T *d_b, *d_v;//, *d_zaut, *d_zbut;
  int ip;
  int len;
  hipError_t ret;
  printf("hmvm_cuda4_%s: begin\n", typeid(T).name()); fflush(stdout);
  v    = new T[nd];    //(double*)malloc(sizeof(double)*mat2->nd);
  tmp  = new T[ktmax]; //(double*)malloc(sizeof(double)*mat2->ktmax);
  zero = new T[ktmax]; //(double*)malloc(sizeof(double)*mat2->ktmax);
  for(i=0;i<nd;i++){
	v[i] = (T)0.0;
  }
  for(i=0;i<ktmax;i++){
	zero[i] = (T)0.0;
  }
  //CHECK_DO(hipMalloc((void**)&d_zaut, sizeof(T)*mat2->nd),"hipMalloc z_aut");
  //CHECK_DO(hipMalloc((void**)&d_zbut, sizeof(T)*mat2->ktmax),"hipMalloc zbut");
  CHECK_DO(hipMalloc((void**)&d_b, sizeof(T)*nd),"hipMalloc d_b");
  CHECK_DO(hipMalloc((void**)&d_v, sizeof(T)*nd),"hipMalloc d_v");
  //for(i=0;i<mat2->nd;i++){d_b[i]=NULL;d_v[i]=NULL;}

  printf("nd = %d\n", nd);												\
  len = mat2->len;
  printf("total length = %d\n", len);
  // host alloc
  // device alloc
  d_sm.nd    = nd;
  d_sm.nlf   = nlf;
  d_sm.ktmax = ktmax;
  hipMalloc((void**)&d_sm.ltmtx, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.ndl, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.ndt, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.nstrtl, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.nstrtt, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.kt, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.a1, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.a2, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.rowmat,sizeof(T)*mat2->len);
  hipMalloc((void**)&d_sm.rowmat_t,sizeof(T)*mat2->len);
  // memcpy
  CHECK_DO(hipMemcpy(d_sm.ltmtx, mat2->ltmtx, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.ltmtx");
  CHECK_DO(hipMemcpy(d_sm.ndt, mat2->ndt, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.ndt");
  CHECK_DO(hipMemcpy(d_sm.ndl, mat2->ndl, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.ndl");
  CHECK_DO(hipMemcpy(d_sm.nstrtl, mat2->nstrtl, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.nstrtl");
  CHECK_DO(hipMemcpy(d_sm.nstrtt, mat2->nstrtt, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.nstrtt");
  CHECK_DO(hipMemcpy(d_sm.kt, mat2->kt, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.kt");
  CHECK_DO(hipMemcpy(d_sm.a1, mat2->a1, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.a1");
  CHECK_DO(hipMemcpy(d_sm.a2, mat2->a2, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.a2");
  CHECK_DO(hipMemcpy(d_sm.rowmat, mat2->rowmat, sizeof(T)*mat2->len, hipMemcpyHostToDevice),"d_sm.rowmat");
  CHECK_DO(hipMemcpy(d_sm.rowmat_t, mat2->rowmat_t, sizeof(T)*mat2->len, hipMemcpyHostToDevice),"d_sm.rowmat_t");

  // 分離
  printf("begin splitting\n");
  mat2->approx = (int*)malloc(sizeof(int)*nlf);
  mat2->dense  = (int*)malloc(sizeof(int)*nlf);
  mat2->napprox = mat2->ndense = 0;
  for(ip=0; ip<nlf; ip++){
	if(mat2->ltmtx[ip]==1){
	  mat2->approx[mat2->napprox++] = ip;
	}else{
	  mat2->dense[mat2->ndense++] = ip;
	}
  }
  hipMalloc((void**)&d_sm.approx,sizeof(int)*mat2->napprox);
  hipMalloc((void**)&d_sm.dense,sizeof(int)*mat2->ndense);
  CHECK_DO(hipMemcpy(d_sm.approx, mat2->approx, sizeof(int)*mat2->napprox, hipMemcpyHostToDevice),"d_sm.approx");
  CHECK_DO(hipMemcpy(d_sm.dense, mat2->dense, sizeof(int)*mat2->ndense, hipMemcpyHostToDevice),"d_sm.dense");
  d_sm.napprox = mat2->napprox;
  d_sm.ndense  = mat2->ndense;
  printf("end splitting (napprox=%d, ndense=%d)\n", mat2->napprox, mat2->ndense);

#if 1
/*
  hybrid4
  複数WARP単一GEMV単一行GEMVカーネル
  <<<napprox+ndense, 32*mul>>>
  1 GEMV by mul TB
  1 line by 1/mul WARP
  バリエーション
  - mul：
  - a2t：a2を転置版で計算するか否か
  - a2interchange：a2のループを入れ替えるか否か
  - aatomic：approxの計算をatomic優先にするかwarp shuffle併用するか
  - datomic：denseの計算をatomic優先にするかwarp shuffle併用するか
*/
  if((kernel>=4000)&&(kernel<4192)){
	int subkernel = kernel-4000;
	int MUL, ATOMIC;
	ATOMIC = subkernel/16;
	MUL = subkernel%16;
	ATOMIC = 1; MUL = 1; // test
	char name[32], fname[64];
	snprintf(name,32,"hybrid4_mul%d_atomic%d_%s", MUL, ATOMIC, typeid(T).name());
	snprintf(fname,64,"result_cuda4_%s.txt", name);
	printf("fname = %s\n", fname);
	printf("MUL = %d, ATOMIC = %d\n", MUL, ATOMIC);
	// EXEC
	{
    hmvm_cuda_hybrid4_proxy<T>(d_v, d_b, d_sm.nlf, d_sm.ktmax,
	d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt,
	d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat, d_sm.rowmat_t,
	d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense,
	d_sm.napprox+d_sm.ndense,32*MUL,d_sm.ktmax*sizeof(T)*MUL,MUL,ATOMIC, v, b, nd, fname, 0);
	//(d_sm.napprox+MUL-1)/MUL,32*MUL,d_sm.ktmax*sizeof(T)*MUL,DIV,MUL,ATOMIC, v, b, nd, fname, 0);
	//(d_sm.ndense+MUL-1)/MUL,32*MUL,d_sm.ktmax*sizeof(T),DIV,MUL,ATOMIC, v, b, nd, fname, 0);
	}
	// BENCH
	{
	}
  }
#endif

  // ######## ######## ######## ######## ######## ######## ######## ########
  hipFree(d_sm.ltmtx);
  hipFree(d_sm.ndl);
  hipFree(d_sm.ndt);
  hipFree(d_sm.nstrtl);
  hipFree(d_sm.nstrtt);
  hipFree(d_sm.kt);
  hipFree(d_sm.a1);
  hipFree(d_sm.a2);
  hipFree(d_sm.rowmat);
  hipFree(d_sm.rowmat_t);
  //hipFree(d_zaut);
  //hipFree(d_zbut);
  hipFree(d_b);
  hipFree(d_v);

  delete [] v; delete [] tmp; delete [] zero;
  //free(v); free(tmp); free(zero);
  printf("hmvm_cuda4: end\n");
}


// ######## ######## ######## ######## ######## ######## ######## ########
// template関数の実体化のための宣言
// ######## ######## ######## ######## ######## ######## ######## ########
template void hmvm_cuda4<float>(matrix2<float>  *mat2, float *b, int kernel, int dump_result);
template void hmvm_cuda4<double>(matrix2<double> *mat2, double *b, int kernel, int dump_result);
