#include "hip/hip_runtime.h"
// -*- C++ -*-
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <typeinfo>
#include <omp.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_cooperative_groups.h>

#include "hacapk.h"

#define CHECK_DO(act,msg) {ret=act; if(ret!=hipSuccess){printf("%s failed\n",msg);exit(-1);};}

namespace cg = cooperative_groups;

#if __CUDA_ARCH__ < 600
__device__ static double myAtomicAdd(double* address, double val)
{
  unsigned long long int* address_as_ull =
	(unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;

  do {
	assumed = old;
	old = atomicCAS(address_as_ull, assumed,
					__double_as_longlong(val +
										 __longlong_as_double(assumed)));

	// Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
  } while (assumed != old);

  return __longlong_as_double(old);
}
__device__ static inline float myAtomicAdd(float* address, float val)
{
  atomicAdd(address, val);
}
#else
#define myAtomicAdd atomicAdd
#endif

// ######## ######## ######## ######## ######## ######## ######## ########
/*
  hybrid1
  TBあたりスレッド数は32に固定
  1TBが1つのPMVを担当
  PMV内の1行を1/div WARPが担当
  <<<napprox+ndense,32>>>
  1 PMV by 1 TB (1 TB = 1 WARP)
  1 line by 1/div WARP
  バリエーション
  - div：1行を1/divのWARPで計算する、div=1,2,4,8,16,32
  - a2t：a2を転置版で計算するか否か(0,1)
  - a2interchange：a2のループを入れ替えるか否か(0,1)
  - aatomic：approxの計算をatomic優先にするかwarp shuffle併用するか(0,1)
  - datomic：denseの計算をatomic優先にするかwarp shuffle併用するか(0,1)
  6x2x2x2x2=96通り
*/
template <class T, int div, int a2t, int a2i, int aatomic, int datomic>
__global__ void hmvm_cuda_hybrid1
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt,
 int *a1, int *a2, T *rowmat, T *rowmat_t,
 int napprox, int *approx, int ndense, int *dense
 )
{
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid1 : begin\n");
#endif
  int gid   = blockIdx.x;
  int bid   = threadIdx.x/(32/div);
  int blen  = div;
  int xid   = threadIdx.x%(32/div);
  int xlen  = (32/div);
  int ndl, ndt, nstrtl, nstrtt;
  int ip, kt, il, it, itt, itl, ill;
  size_t head;
  T tmp1 = (T)0.0;
  extern __shared__ __align__(sizeof(T)) unsigned char my_smem[];
  T *tmp2 = reinterpret_cast<T *>(my_smem);
  cg::thread_block_tile<32/div> g = cg::tiled_partition<32/div>(cg::this_thread_block());

  if(gid<napprox){
#ifndef _SKIP_APPROX
	// approx
	ip = approx[gid];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	kt = _kt[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt);
#endif
	head = a1[ip];
	for(il=bid; il<kt; il+=blen){
	  if(xid==0)tmp2[il] = (T)0.0;
	  tmp1 = (T)0.0;
	  for(it=xid; it<ndt; it+=xlen){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		if(a2t==0){
		  tmp1 += rowmat[head+itl]*d_zu[itt];
		}else{
		  tmp1 += rowmat_t[head+itl]*d_zu[itt];
		}
	  }
	  for (int offset = g.size()/2; offset > 0; offset /= 2)tmp1 += g.shfl_down(tmp1, offset);
	  if(xid==0)tmp2[il] = tmp1;
	}
	__syncwarp();
	head = a2[ip];
	if(a2t==0){ // a2t==0
	  if(a2i==0){ // a2i==0
		for(il=bid; il<kt; il+=blen){
		  for(it=xid; it<ndl; it+=xlen){
			ill=it+nstrtl-1;
			itl=it+il*ndl;
			myAtomicAdd(&d_zaut[ill], rowmat[head+itl]*tmp2[il]);
		  }
		}
	  }else{ // a2i==1
		if(aatomic==0){ // aatomic==0
		  for(it=bid; it<ndl; it+=blen){
			ill=it+nstrtl-1;
			tmp1 = (T)0.0;
			for(il=xid; il<kt; il+=xlen){
			  itl=it+il*ndl;
			  tmp1 += rowmat[head+itl]*tmp2[il];
			}
			for (int offset = g.size()/2; offset > 0; offset /= 2)tmp1 += g.shfl_down(tmp1, offset);
			if(xid==0){
			  myAtomicAdd(&d_zaut[ill], tmp1);
			}
		  }
		}else{ // aatomic==1
		  for(it=bid; it<ndl; it+=blen){
			ill=it+nstrtl-1;
			tmp1 = (T)0.0;
			for(il=xid; il<kt; il+=xlen){
			  itl=it+il*ndl;
			  tmp1 += rowmat[head+itl]*tmp2[il];
			}
			myAtomicAdd(&d_zaut[ill], tmp1);
		  }
		}
	  }
	}else{ // a2t==1
	  if(a2i==0){ // a2i==0
		for(il=bid; il<kt; il+=blen){
		  for(it=xid; it<ndl; it+=xlen){
			ill=it+nstrtl-1;
			itl=it*kt+il;
			myAtomicAdd(&d_zaut[ill], rowmat_t[head+itl]*tmp2[il]);
		  }
		}
	  }else{ // a2i==1
		if(aatomic==0){ // aatomic==0
		  for(it=bid; it<ndl; it+=blen){
			ill=it+nstrtl-1;
			tmp1 = (T)0.0;
			for(il=xid; il<kt; il+=xlen){
			  itl=it*kt+il;
			  tmp1 += rowmat_t[head+itl]*tmp2[il];
			}
			for (int offset = g.size()/2; offset > 0; offset /= 2)tmp1 += g.shfl_down(tmp1, offset);
			if(xid==0){
			  myAtomicAdd(&d_zaut[ill], tmp1);
			}
		  }
		}else{ // aatomic==1
		  for(it=bid; it<ndl; it+=blen){
			ill=it+nstrtl-1;
			tmp1 = (T)0.0;
			for(il=xid; il<kt; il+=xlen){
			  itl=it*kt+il;
			  tmp1 += rowmat_t[head+itl]*tmp2[il];
			}
			myAtomicAdd(&d_zaut[ill], tmp1);
		  }
		}
	  }
	}
#endif // approx
  }else{
#ifndef _SKIP_DENSE
	// dense
	ip = dense[gid-napprox];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt);
#endif
	head = a1[ip];
	for(il=bid; il<ndl; il+=blen){
	  ill = il+nstrtl-1;
	  tmp1 = (T)0.0;
	  for(it=xid; it<ndt; it+=xlen){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		if(a2t==0){ // a2t==0
		  tmp1 += rowmat[head+itl]*d_zu[itt];
		}else{ // a2t==1
		  tmp1 += rowmat_t[head+itl]*d_zu[itt];
		}
	  }
	  if(datomic==0){ // datomic==0
		for (int offset = g.size()/2; offset > 0; offset /= 2)tmp1 += g.shfl_down(tmp1, offset);
		if(xid==0){
		  myAtomicAdd(&d_zaut[ill], tmp1);
		}
	  }else{ // datomic==1
		myAtomicAdd(&d_zaut[ill], tmp1);
	  }
	}
#endif // dense
  }

#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid1 : end\n");
#endif
}

template <class T, int div, int a2t, int a2i, int aa, int da>
void hmvm_cuda_hybrid1_proxy
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *ltmtx, int *ndt, int *ndl, int *nstrtl, int *nstrtt, int *kt,
 int *a1, int *a2, T *rowmat, T *rowmat_t,
 int napprox, int *approx, int ndense, int *dense,
 int blocks, int threads, int shms,
 T *v, T *b, int nd, char *fname, int bench)
{
  int M=5, L=M+bench;
  FILE *F;
  int i, l, lmax;
  double d1, d2, *dtimes, dmin, dmax, davg;
  hipError_t ret;
  dtimes = new double[L];
  if(bench==0){lmax=1;}else{lmax=L;}
  for(l=0;l<lmax;l++){
	for(i=0;i<nd;i++)v[i] = (T)0.0;
	CHECK_DO(hipMemcpy(d_zaut, v, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy v to d_v");
	CHECK_DO(hipMemcpy(d_zu, b, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy b to d_b");
	hipDeviceSynchronize();
	d1 = omp_get_wtime();
	hmvm_cuda_hybrid1<T,div,a2t,a2i,aa,da><<<blocks,threads,shms>>>
	  (d_zaut, d_zu, nlf, ktmax, ltmtx,
	   ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, rowmat_t,
	   napprox, approx, ndense, dense);
	hipDeviceSynchronize();
	d2 = omp_get_wtime();
	dtimes[l] = d2-d1;
  }
  if(bench==0){
	CHECK_DO(hipMemcpy(v, d_zaut, sizeof(T)*nd, hipMemcpyDeviceToHost),"hipMemcpy d_v to v");
	printf("write to %s\n", fname);
	F = fopen(fname, "w");
	for(i=0;i<nd;i++)fprintf(F, "%.3E\n", v[i]);
	fclose(F);
  }else{
	dmin = 9999.99;
	dmax = 0.0;
	davg = 0.0;
	for(i=M;i<L;i++){
	  davg += dtimes[i];
	  if(dmin>dtimes[i])dmin=dtimes[i];
	  if(dmax<dtimes[i])dmax=dtimes[i];
	}
	davg /= (L-M);
	printf("TIME %d hmvm_cuda_hybrid1%s min %e max %e avg %e\n", L-M, typeid(T).name(), dmin, dmax, davg);
  }
  delete [] dtimes;
}
#include "template_hybrid1.hpp"
// ######## ######## ######## ######## ######## ######## ######## ########

template<class T>
void hmvm_cuda1(matrix2<T> *mat2, T *b, int kernel, int dump_result)
{
  matrix2<T> d_sm;
  int i, nd = mat2->nd, ktmax = mat2->ktmax, nlf = mat2->nlf;
  T *v=NULL;
  T *d_b, *d_v;
  int ip;
  int len;
  hipError_t ret;
  printf("hmvm_cuda1_%s: begin\n", typeid(T).name()); fflush(stdout);
  v    = new T[nd];
  for(i=0;i<nd;i++){
	v[i] = (T)0.0;
  }
  CHECK_DO(hipMalloc((void**)&d_b, sizeof(T)*nd),"hipMalloc d_b");
  CHECK_DO(hipMalloc((void**)&d_v, sizeof(T)*nd),"hipMalloc d_v");

  printf("nd = %d\n", nd);												\
  len = mat2->len;
  printf("total length = %d\n", len);
  // host alloc
  // device alloc
  d_sm.nd    = nd;
  d_sm.nlf   = nlf;
  d_sm.ktmax = ktmax;
  hipMalloc((void**)&d_sm.ltmtx, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.ndl, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.ndt, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.nstrtl, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.nstrtt, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.kt, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.a1, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.a2, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.rowmat,sizeof(T)*mat2->len);
  hipMalloc((void**)&d_sm.rowmat_t,sizeof(T)*mat2->len);
  // memcpy
  CHECK_DO(hipMemcpy(d_sm.ltmtx, mat2->ltmtx, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.ltmtx");
  CHECK_DO(hipMemcpy(d_sm.ndt, mat2->ndt, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.ndt");
  CHECK_DO(hipMemcpy(d_sm.ndl, mat2->ndl, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.ndl");
  CHECK_DO(hipMemcpy(d_sm.nstrtl, mat2->nstrtl, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.nstrtl");
  CHECK_DO(hipMemcpy(d_sm.nstrtt, mat2->nstrtt, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.nstrtt");
  CHECK_DO(hipMemcpy(d_sm.kt, mat2->kt, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.kt");
  CHECK_DO(hipMemcpy(d_sm.a1, mat2->a1, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.a1");
  CHECK_DO(hipMemcpy(d_sm.a2, mat2->a2, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.a2");
  CHECK_DO(hipMemcpy(d_sm.rowmat, mat2->rowmat, sizeof(T)*mat2->len, hipMemcpyHostToDevice),"d_sm.rowmat");
  CHECK_DO(hipMemcpy(d_sm.rowmat_t, mat2->rowmat_t, sizeof(T)*mat2->len, hipMemcpyHostToDevice),"d_sm.rowmat_t");

  // 分離
  printf("begin splitting\n");
  mat2->approx = (int*)malloc(sizeof(int)*nlf);
  mat2->dense  = (int*)malloc(sizeof(int)*nlf);
  mat2->napprox = mat2->ndense = 0;
  for(ip=0; ip<nlf; ip++){
	if(mat2->ltmtx[ip]==1){
	  mat2->approx[mat2->napprox++] = ip;
	}else{
	  mat2->dense[mat2->ndense++] = ip;
	}
  }
  hipMalloc((void**)&d_sm.approx,sizeof(int)*mat2->napprox);
  hipMalloc((void**)&d_sm.dense,sizeof(int)*mat2->ndense);
  CHECK_DO(hipMemcpy(d_sm.approx, mat2->approx, sizeof(int)*mat2->napprox, hipMemcpyHostToDevice),"d_sm.approx");
  CHECK_DO(hipMemcpy(d_sm.dense, mat2->dense, sizeof(int)*mat2->ndense, hipMemcpyHostToDevice),"d_sm.dense");
  d_sm.napprox = mat2->napprox;
  d_sm.ndense  = mat2->ndense;
  printf("end splitting (napprox=%d, ndense=%d)\n", mat2->napprox, mat2->ndense);

/*
  hybrid1
  基本スレッド並列化カーネル
  1PMVを1TB(32Thread=1WARP)で計算する
  1行を1/div WARPで計算する
  <<<napprox+ndense,32>>>
  1 GEMV by 1 TB
  1 line by 1/div WARP
  バリエーション
  - div：1行を1/divのWARPで計算する、div=1,2,4,8,16,32
  - a2t：a2を転置版で計算するか否か(0,1)
  - a2interchange：a2のループを入れ替えるか否か(0,1)
  - aatomic：approxの計算をatomic優先にするかwarp shuffle併用するか(0,1)
  - datomic：denseの計算をatomic優先にするかwarp shuffle併用するか(0,1)
  6x2x2x2x2=96通り

  何故かdivが8or16の時にa2i=1,aa=1の結果がおかしい
  div8は誤差の範囲かも知れない、div16はよりおかしい
*/
  if(kernel>=1000&&kernel<1096){
	int subkernel = kernel-1000;
	int div, a2t, a2i, aa, da;
	div = subkernel%6;
	a2t = (subkernel/6)%2;
	a2i = ((subkernel/6)/2)%2;
	aa = (((subkernel/6)/2)/2)%2;
	da = ((((subkernel/6)/2)/2)/2)%2;
	div = pow(2,div);
	char name[0xff], fname[0xff];
	snprintf(name,0xff,"hybrid1_div%d_a2t%d_a2i%d_aa%d_da%d_%s", div, a2t, a2i, aa, da, typeid(T).name());
	snprintf(fname,0xff,"result_cuda1_%s.txt", name);
	printf("fname = %s\n", fname);
	// EXEC
	hmvm_cuda_hybrid1_proxy<T>
	  (d_v, d_b, d_sm.nlf, d_sm.ktmax,
	   d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt, d_sm.kt,
	   d_sm.a1, d_sm.a2, d_sm.rowmat, d_sm.rowmat_t,
	   d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense,
	   d_sm.napprox+d_sm.ndense, 32, d_sm.ktmax*sizeof(T),
	   v, b, nd, fname, 0,
	   div, a2t, a2i, aa, da);
	// BENCH
	hmvm_cuda_hybrid1_proxy<T>
	  (d_v, d_b, d_sm.nlf, d_sm.ktmax,
	   d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt, d_sm.kt,
	   d_sm.a1, d_sm.a2, d_sm.rowmat, d_sm.rowmat_t,
	   d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense,
	   d_sm.napprox+d_sm.ndense, 32, d_sm.ktmax*sizeof(T),
	   v, b, nd, fname, 5,
	   div, a2t, a2i, aa, da);
  }

  // ######## ######## ######## ######## ######## ######## ######## ########
  hipFree(d_sm.ltmtx);
  hipFree(d_sm.ndl);
  hipFree(d_sm.ndt);
  hipFree(d_sm.nstrtl);
  hipFree(d_sm.nstrtt);
  hipFree(d_sm.kt);
  hipFree(d_sm.a1);
  hipFree(d_sm.a2);
  hipFree(d_sm.rowmat);
  hipFree(d_sm.rowmat_t);
  hipFree(d_b);
  hipFree(d_v);

  delete [] v;
  printf("hmvm_cuda1: end\n");
}


// ######## ######## ######## ######## ######## ######## ######## ########
// template関数の実体化のための宣言
// ######## ######## ######## ######## ######## ######## ######## ########
template void hmvm_cuda1<float>(matrix2<float>  *mat2, float *b, int kernel, int dump_result);
template void hmvm_cuda1<double>(matrix2<double> *mat2, double *b, int kernel, int dump_result);
