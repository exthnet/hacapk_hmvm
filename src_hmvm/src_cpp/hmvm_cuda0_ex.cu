#include "hip/hip_runtime.h"
// -*- C++ -*-
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <typeinfo>
#include <omp.h>
#include <hip/hip_runtime_api.h>

#include "hacapk.h"

#define CHECK_DO(act,msg) {ret=act; if(ret!=hipSuccess){printf("%s failed\n",msg);exit(-1);};}

//namespace cg = cooperative_groups;

#if __CUDA_ARCH__ < 600
__device__ static double myAtomicAdd(double* address, double val)
{
  unsigned long long int* address_as_ull =
	(unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;

  do {
	assumed = old;
	old = atomicCAS(address_as_ull, assumed,
					__double_as_longlong(val +
										 __longlong_as_double(assumed)));

	// Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
  } while (assumed != old);

  return __longlong_as_double(old);
}
__device__ static inline float myAtomicAdd(float* address, float val)
{
  atomicAdd(address, val);
}
#else
#define myAtomicAdd atomicAdd
#endif

// ######## ######## ######## ######## ######## ######## ######## ########
/*
  完全逐次
  バリエーション：
  - a2trans：approxy2を転置版で計算するか否か(0,1)
  - a2interchange：approxy2のループを入れ替えるか否か(0,1)
*/
template <class T, int a2t, int a2i>
__global__ void hmvm_cuda_seq
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt,
 int *a1, int *a2, T *rowmat, T *rowmat_t,
 int napprox, int *approx, int ndense, int *dense)
{
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_seq : begin\n");
#endif
  int ndl, ndt, nstrtl, nstrtt;
  int ip, kt, il, it, itt, itl, ill;
  size_t head;
  T tmp1;
  //extern __shared__ T tmp2[];
  extern __shared__ __align__(sizeof(T)) unsigned char my_smem[];
  T *tmp2 = reinterpret_cast<T *>(my_smem);
  int i;

  // approx
  for(i=0; i<napprox; i++){
#ifndef _SKIP_APPROX
	ip = approx[i];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	kt = _kt[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt);
#endif
	head = a1[ip];
	if(a2t==0){
	  for(il=0; il<kt; il++){
		tmp1 = (T)0.0;
		for(it=0; it<ndt; it++){
		  itt=it+nstrtt-1;
		  itl=it+il*ndt;
		  tmp1 += rowmat[head+itl]*d_zu[itt];
		}
		tmp2[il] = tmp1;
	  }
	}else{
	  for(il=0; il<kt; il++){
		tmp1 = (T)0.0;
		for(it=0; it<ndt; it++){
		  itt=it+nstrtt-1;
		  itl=it+il*ndt;
		  tmp1 += rowmat_t[head+itl]*d_zu[itt];
		}
		tmp2[il] = tmp1;
	  }
	}
	head = a2[ip];
	if(a2t==0){
	  if(a2i==0){
		for(il=0; il<kt; il++){
		  for(it=0; it<ndl; it++){
			ill=it+nstrtl-1;
			itl=it+il*ndl;
			//myAtomicAdd(&d_zaut[ill], rowmat[head+itl]*tmp2[il]);
			d_zaut[ill] += rowmat[head+itl]*tmp2[il];
		  }
		}
	  }else{ // a2i==1
		for(it=0; it<ndl; it++){
		  ill=it+nstrtl-1;
		  tmp1 = (T)0.0;
		  for(il=0; il<kt; il++){
			itl=it+il*ndl;
			//myAtomicAdd(&d_zaut[ill], rowmat[head+itl]*tmp2[il]);
			tmp1 += rowmat[head+itl]*tmp2[il];
		  }
		  d_zaut[ill] += tmp1;
		}
	  }
	}else{ // a2t==1
	  if(a2i==0){
		for(il=0; il<kt; il++){
		  for(it=0; it<ndl; it++){
			ill=it+nstrtl-1;
			itl=it*kt+il;
			//myAtomicAdd(&d_zaut[ill], rowmat_t[head+itl]*tmp2[il]);
			d_zaut[ill] += rowmat_t[head+itl]*tmp2[il];
		  }
		}
	  }else{ // a2i==1
		for(it=0; it<ndl; it++){
		  ill=it+nstrtl-1;
		  tmp1 = (T)0.0;
		  for(il=0; il<kt; il++){
			itl=it*kt+il;
			//myAtomicAdd(&d_zaut[ill], rowmat_t[head+itl]*tmp2[il]);
			tmp1 += rowmat_t[head+itl]*tmp2[il];
		  }
		  d_zaut[ill] += tmp1;
		}
	  }
	}
#endif
  }

  // dense
  for(i=0; i<ndense; i++){
#ifndef _SKIP_DENSE
	ip = dense[i];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt);
#endif
	head = a1[ip];
	for(il=0; il<ndl; il++){
	  T tmp = (T)0.0;
	  ill=il+nstrtl-1;
	  for(it=0; it<ndt; it++){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		if(a2t==0){
		  tmp += rowmat[head+itl]*d_zu[itt];
		}else{
		  tmp += rowmat_t[head+itl]*d_zu[itt];
		}
	  }
	  //myAtomicAdd(&d_zaut[ill], tmp);
	  d_zaut[ill] += tmp;
	}
#endif
  }
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_seq : end\n");
#endif
}

template <class T, int a2t, int a2i>
void hmvm_cuda_seq_proxy
(T *d_zaut, T *d_zu,
 matrix2<T> *h_mat, matrix2<T> *d_mat,
 T *v, T *b, char *fname, int bench)
{
  int M=5, L=M+bench;
  FILE *F;
  int i, l, lmax;
  double d1, d2, *dtimes, dmin, dmax, davg;
  hipError_t ret;
  dtimes = new double[L];
  if(bench==0){lmax=1;}else{lmax=L;}
  for(l=0;l<lmax;l++){
	for(i=0;i<h_mat->nd;i++)v[i] = (T)0.0;
	CHECK_DO(hipMemcpy(d_zaut, v, sizeof(T)*h_mat->nd, hipMemcpyHostToDevice),"hipMemcpy v to d_v");
	CHECK_DO(hipMemcpy(d_zu, b, sizeof(T)*h_mat->nd, hipMemcpyHostToDevice),"hipMemcpy b to d_b");
	hipDeviceSynchronize();
	d1 = omp_get_wtime();
	hmvm_cuda_seq<T,a2t,a2i><<<1,1,sizeof(T)*h_mat->ktmax>>>
	  (d_zaut, d_zu, d_mat->nlf, d_mat->ktmax,
	   d_mat->ltmtx, d_mat->ndt, d_mat->ndl, d_mat->nstrtl, d_mat->nstrtt,
	   d_mat->kt, d_mat->a1, d_mat->a2, d_mat->rowmat, d_mat->rowmat_t,
	   d_mat->napprox, d_mat->approx, d_mat->ndense, d_mat->dense);
	hipDeviceSynchronize();
	d2 = omp_get_wtime();
	dtimes[l] = d2-d1;
  }
  if(bench==0){
	CHECK_DO(hipMemcpy(v, d_zaut, sizeof(T)*h_mat->nd, hipMemcpyDeviceToHost),"hipMemcpy d_v to v");
	printf("write to %s\n", fname);
	F = fopen(fname, "w");
	for(i=0;i<h_mat->nd;i++)fprintf(F, "%.3E\n", v[i]);
	fclose(F);
  }else{
	dmin = 9999.99;
	dmax = 0.0;
	davg = 0.0;
	for(i=M;i<L;i++){
	  davg += dtimes[i];
	  if(dmin>dtimes[i])dmin=dtimes[i];
	  if(dmax<dtimes[i])dmax=dtimes[i];
	}
	davg /= (L-M);
	printf("TIME %d hmvm_cuda0_seq%s min %e max %e avg %e\n", L-M, typeid(T).name(), dmin, dmax, davg);
  }
  delete [] dtimes;
}

template <class T>
void hmvm_cuda_seq_proxy
(T *d_zaut, T *d_zu,
 matrix2<T> *h_mat, matrix2<T> *d_mat,
 T *v, T *b, char *fname, int bench,
 int a2t, int a2i)
{
  if(a2t==0 && a2i==0)hmvm_cuda_seq_proxy<T,0,0>(d_zaut, d_zu, h_mat, d_mat, v, b, fname, bench);
  if(a2t==0 && a2i==1)hmvm_cuda_seq_proxy<T,0,1>(d_zaut, d_zu, h_mat, d_mat, v, b, fname, bench);
  if(a2t==1 && a2i==0)hmvm_cuda_seq_proxy<T,1,0>(d_zaut, d_zu, h_mat, d_mat, v, b, fname, bench);
  if(a2t==1 && a2i==1)hmvm_cuda_seq_proxy<T,1,1>(d_zaut, d_zu, h_mat, d_mat, v, b, fname, bench);
}

// ######## ######## ######## ######## ######## ######## ######## ########
/*
  block並列化
  ThreadBlockごとに1つの部分行列積(mat-mat-vecまたはmat-vec)を行う
  ThreadBlock内部は逐次
  - a2trans：approxy2を転置版で計算するか否か(0,1)
  - a2interchange：approxy2のループを入れ替えるか否か(0,1)
 */
template <class T, int a2t, int a2i>
__global__ void hmvm_cuda_block
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt,
 int *a1, int *a2, T *rowmat, T *rowmat_t,
 int napprox, int *approx, int ndense, int *dense)
{
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cudaD_block : begin\n");
#endif
  int ndl, ndt, nstrtl, nstrtt;
  int ip, kt, il, it, itt, itl, ill;
  size_t head;
  T tmp1;
  extern __shared__ __align__(sizeof(T)) unsigned char my_smem[];
  T *tmp2 = reinterpret_cast<T *>(my_smem);

  if(blockIdx.x<napprox){
#ifndef _SKIP_APPROX
	// approx
	//for(i=0; i<napprox; i++){
	ip = approx[blockIdx.x];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	kt = _kt[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt);
#endif
	head = a1[ip];
	if(a2t==0){
	  for(il=0; il<kt; il++){
		tmp1 = (T)0.0;
		for(it=0; it<ndt; it++){
		  itt=it+nstrtt-1;
		  itl=it+il*ndt;
		  tmp1 += rowmat[head+itl]*d_zu[itt];
		}
		tmp2[il] = tmp1;
	  }
	}else{
	  for(il=0; il<kt; il++){
		tmp1 = (T)0.0;
		for(it=0; it<ndt; it++){
		  itt=it+nstrtt-1;
		  itl=it+il*ndt;
		  tmp1 += rowmat_t[head+itl]*d_zu[itt];
		}
		tmp2[il] = tmp1;
	  }
	}
	head = a2[ip];
	if(a2t==0){
	  if(a2i==0){
		for(il=0; il<kt; il++){
		  for(it=0; it<ndl; it++){
			ill=it+nstrtl-1;
			itl=it+il*ndl;
			myAtomicAdd(&d_zaut[ill], rowmat[head+itl]*tmp2[il]);
		  }
		}
	  }else{
		for(it=0; it<ndl; it++){
		  ill=it+nstrtl-1;
		  tmp1 = (T)0.0;
		  for(il=0; il<kt; il++){
			itl=it+il*ndl;
			tmp1 += rowmat[head+itl]*tmp2[il];
		  }
		  myAtomicAdd(&d_zaut[ill], tmp1);
		}
	  }
	}else{
	  if(a2i==0){
		for(il=0; il<kt; il++){
		  for(it=0; it<ndl; it++){
			ill=it+nstrtl-1;
			itl=it*kt+il;
			myAtomicAdd(&d_zaut[ill], rowmat_t[head+itl]*tmp2[il]);
		  }
		}
	  }else{
		for(it=0; it<ndl; it++){
		  ill=it+nstrtl-1;
		  tmp1 = (T)0.0;
		  for(il=0; il<kt; il++){
			itl=it*kt+il;
			tmp1 += rowmat_t[head+itl]*tmp2[il];
		  }
		  myAtomicAdd(&d_zaut[ill], tmp1);
		}
	  }
	}
#endif
  }else{
#ifndef _SKIP_DENSE
	// dense
	//for(i=0; i<ndense; i++){
	ip = dense[blockIdx.x - napprox];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt);
#endif
	head = a1[ip];
	for(il=0; il<ndl; il++){
	  tmp1 = (T)0.0;
	  ill=il+nstrtl-1;
	  for(it=0; it<ndt; it++){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		if(a2t==0){
		  tmp1 += rowmat[head+itl]*d_zu[itt];
		}else{
		  tmp1 += rowmat_t[head+itl]*d_zu[itt];
		}
	  }
	  myAtomicAdd(&d_zaut[ill], tmp1);
	}
#endif
  }
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cudaD_block : end\n");
#endif
}

template <class T, int a2t, int a2i>
void hmvm_cuda_block_proxy
(T *d_zaut, T *d_zu,
 matrix2<T> *h_mat, matrix2<T> *d_mat,
 T *v, T *b, char *fname, int bench)
{
  int M=5, L=M+bench;
  FILE *F;
  int i, l, lmax;
  double d1, d2, *dtimes, dmin, dmax, davg;
  hipError_t ret;
  dtimes = new double[L];
  if(bench==0){lmax=1;}else{lmax=L;}
  for(l=0;l<lmax;l++){
	for(i=0;i<h_mat->nd;i++)v[i] = (T)0.0;
	CHECK_DO(hipMemcpy(d_zaut, v, sizeof(T)*h_mat->nd, hipMemcpyHostToDevice),"hipMemcpy v to d_v");
	CHECK_DO(hipMemcpy(d_zu, b, sizeof(T)*h_mat->nd, hipMemcpyHostToDevice),"hipMemcpy b to d_b");
	hipDeviceSynchronize();
	d1 = omp_get_wtime();
	hmvm_cuda_block<T,a2t,a2i><<<h_mat->napprox+h_mat->ndense,1,sizeof(T)*h_mat->ktmax>>>
	  (d_zaut, d_zu, d_mat->nlf, d_mat->ktmax,
	   d_mat->ltmtx, d_mat->ndt, d_mat->ndl, d_mat->nstrtl, d_mat->nstrtt,
	   d_mat->kt, d_mat->a1, d_mat->a2, d_mat->rowmat, d_mat->rowmat_t,
	   d_mat->napprox, d_mat->approx, d_mat->ndense, d_mat->dense);
	hipDeviceSynchronize();
	d2 = omp_get_wtime();
	dtimes[l] = d2-d1;
  }
  if(bench==0){
	CHECK_DO(hipMemcpy(v, d_zaut, sizeof(T)*h_mat->nd, hipMemcpyDeviceToHost),"hipMemcpy d_v to v");
	printf("write to %s\n", fname);
	F = fopen(fname, "w");
	for(i=0;i<h_mat->nd;i++)fprintf(F, "%.3E\n", v[i]);
	fclose(F);
  }else{
	dmin = 9999.99;
	dmax = 0.0;
	davg = 0.0;
	for(i=M;i<L;i++){
	  davg += dtimes[i];
	  if(dmin>dtimes[i])dmin=dtimes[i];
	  if(dmax<dtimes[i])dmax=dtimes[i];
	}
	davg /= (L-M);
	printf("TIME %d hmvm_cuda0_block%s min %e max %e avg %e\n", L-M, typeid(T).name(), dmin, dmax, davg);
  }
  delete [] dtimes;
}

template <class T>
void hmvm_cuda_block_proxy
(T *d_zaut, T *d_zu,
 matrix2<T> *h_mat, matrix2<T> *d_mat,
 T *v, T *b, char *fname, int bench,
 int a2t, int a2i)
{
  if(a2t==0 && a2i==0)hmvm_cuda_block_proxy<T,0,0>(d_zaut, d_zu, h_mat, d_mat, v, b, fname, bench);
  if(a2t==0 && a2i==1)hmvm_cuda_block_proxy<T,0,1>(d_zaut, d_zu, h_mat, d_mat, v, b, fname, bench);
  if(a2t==1 && a2i==0)hmvm_cuda_block_proxy<T,1,0>(d_zaut, d_zu, h_mat, d_mat, v, b, fname, bench);
  if(a2t==1 && a2i==1)hmvm_cuda_block_proxy<T,1,1>(d_zaut, d_zu, h_mat, d_mat, v, b, fname, bench);
}

// ######## ######## ######## ######## ######## ######## ######## ########
template<class T>
void hmvm_cuda0(matrix2<T> *mat2, T *b, int kernel, int dump_result, int nbench)
{
  matrix2<T> d_sm;
  int i, nd = mat2->nd, ktmax = mat2->ktmax, nlf = mat2->nlf;
  T *v=NULL;
  T *d_b, *d_v;
  int ip;
  int len;
  hipError_t ret;
  printf("hmvm_cuda0_%s: begin\n", typeid(T).name()); fflush(stdout);
  v    = new T[nd];
  for(i=0;i<nd;i++){
	v[i] = (T)0.0;
  }
  CHECK_DO(hipMalloc((void**)&d_b, sizeof(T)*nd),"hipMalloc d_b");
  CHECK_DO(hipMalloc((void**)&d_v, sizeof(T)*nd),"hipMalloc d_v");

  printf("nd = %d\n", nd);												\
  len = mat2->len;
  printf("total length = %d\n", len);
  // host alloc
  // device alloc
  d_sm.nd    = nd;
  d_sm.nlf   = nlf;
  d_sm.ktmax = ktmax;
  hipMalloc((void**)&d_sm.ltmtx, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.ndl, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.ndt, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.nstrtl, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.nstrtt, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.kt, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.a1, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.a2, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.rowmat,sizeof(T)*mat2->len);
  hipMalloc((void**)&d_sm.rowmat_t,sizeof(T)*mat2->len);
  // memcpy
  CHECK_DO(hipMemcpy(d_sm.ltmtx, mat2->ltmtx, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.ltmtx");
  CHECK_DO(hipMemcpy(d_sm.ndt, mat2->ndt, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.ndt");
  CHECK_DO(hipMemcpy(d_sm.ndl, mat2->ndl, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.ndl");
  CHECK_DO(hipMemcpy(d_sm.nstrtl, mat2->nstrtl, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.nstrtl");
  CHECK_DO(hipMemcpy(d_sm.nstrtt, mat2->nstrtt, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.nstrtt");
  CHECK_DO(hipMemcpy(d_sm.kt, mat2->kt, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.kt");
  CHECK_DO(hipMemcpy(d_sm.a1, mat2->a1, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.a1");
  CHECK_DO(hipMemcpy(d_sm.a2, mat2->a2, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.a2");
  CHECK_DO(hipMemcpy(d_sm.rowmat, mat2->rowmat, sizeof(T)*mat2->len, hipMemcpyHostToDevice),"d_sm.rowmat");
  CHECK_DO(hipMemcpy(d_sm.rowmat_t, mat2->rowmat_t, sizeof(T)*mat2->len, hipMemcpyHostToDevice),"d_sm.rowmat_t");

  // 分離
  printf("begin splitting\n");
  mat2->approx = (int*)malloc(sizeof(int)*nlf);
  mat2->dense  = (int*)malloc(sizeof(int)*nlf);
  mat2->napprox = mat2->ndense = 0;
  for(ip=0; ip<nlf; ip++){
	if(mat2->ltmtx[ip]==1){
	  mat2->approx[mat2->napprox++] = ip;
	}else{
	  mat2->dense[mat2->ndense++] = ip;
	}
  }
  hipMalloc((void**)&d_sm.approx,sizeof(int)*mat2->napprox);
  hipMalloc((void**)&d_sm.dense,sizeof(int)*mat2->ndense);
  CHECK_DO(hipMemcpy(d_sm.approx, mat2->approx, sizeof(int)*mat2->napprox, hipMemcpyHostToDevice),"d_sm.approx");
  CHECK_DO(hipMemcpy(d_sm.dense, mat2->dense, sizeof(int)*mat2->ndense, hipMemcpyHostToDevice),"d_sm.dense");
  d_sm.napprox = mat2->napprox;
  d_sm.ndense  = mat2->ndense;
  printf("end splitting (napprox=%d, ndense=%d)\n", mat2->napprox, mat2->ndense);

  /*
	完全逐次
	バリエーション：
	- a2trans：approxy2を転置版で計算するか否か(0,1)
	- a2interchange：approxy2のループを入れ替えるか否か(0,1)
  */
  if(kernel>=0 && kernel<4)
  {
	int subkernel = kernel;
	int a2t, a2i;
	a2t = subkernel%2;
	a2i = (subkernel/2)%2;
	char name[0xff], fname[0xff];
	snprintf(name,0xff,"seq_a2t%d_a2i%d%s", a2t, a2i, typeid(T).name());
	snprintf(fname,0xff,"result_cuda0_%s.txt", name);
	printf("fname = %s\n", fname);
	// EXEC
	if(dump_result)hmvm_cuda_seq_proxy<T>(d_v, d_b, mat2, &d_sm, v, b, fname, 0, a2t, a2i);
	// BENCH
	if(nbench>0)hmvm_cuda_seq_proxy<T>(d_v, d_b, mat2, &d_sm, v, b, fname, nbench, a2t, a2i);
  }

  /*
	block並列化
	ThreadBlockごとに1つの部分行列積(mat-mat-vecまたはmat-vec)を行う
	ThreadBlock内部は逐次
	- a2trans：approxy2を転置版で計算するか否か(0,1)
	- a2interchange：approxy2のループを入れ替えるか否か(0,1)
  */
  if(kernel>=10 && kernel<14)
  {
	int subkernel = kernel-10;
	int a2t, a2i;
	a2t = subkernel%2;
	a2i = (subkernel/2)%2;
	char name[0xff], fname[0xff];
	snprintf(name,0xff,"block_a2t%d_a2i%d%s", a2t, a2i, typeid(T).name());
	snprintf(fname,0xff,"result_cuda0_%s.txt", name);
	printf("fname = %s\n", fname);
	// EXEC
	if(dump_result)hmvm_cuda_block_proxy<T>(d_v, d_b, mat2, &d_sm, v, b, fname, 0, a2t, a2i);
	// BENCH
	if(nbench>0)hmvm_cuda_block_proxy<T>(d_v, d_b, mat2, &d_sm, v, b, fname, nbench, a2t, a2i);
  }

  // ######## ######## ######## ######## ######## ######## ######## ########
  hipFree(d_sm.ltmtx);
  hipFree(d_sm.ndl);
  hipFree(d_sm.ndt);
  hipFree(d_sm.nstrtl);
  hipFree(d_sm.nstrtt);
  hipFree(d_sm.kt);
  hipFree(d_sm.a1);
  hipFree(d_sm.a2);
  hipFree(d_sm.rowmat);
  hipFree(d_sm.rowmat_t);
  hipFree(d_b);
  hipFree(d_v);

  delete [] v;
  printf("hmvm_cuda0: end\n");
}

// ######## ######## ######## ######## ######## ######## ######## ########
// template関数の実体化のための宣言
// ######## ######## ######## ######## ######## ######## ######## ########
template void hmvm_cuda0<float>(matrix2<float>  *mat2, float *b, int kernel, int dump_result, int nbench);
template void hmvm_cuda0<double>(matrix2<double> *mat2, double *b, int kernel, int dump_result, int nbench);
