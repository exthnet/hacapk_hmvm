#include "hip/hip_runtime.h"
// -*- c++ -*-
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <typeinfo>
#include <omp.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_cooperative_groups.h>

#include "hacapk.h"

// できれば重複計算していないかチェックもしたい

#define CHECK_DO(act,msg) {ret=act; if(ret!=hipSuccess){printf("%s failed\n",msg);exit(-1);};}

namespace cg = cooperative_groups;

#if __CUDA_ARCH__ < 600
__device__ double myAtomicAdd(double* address, double val)
{
  unsigned long long int* address_as_ull =
	(unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;

  do {
	assumed = old;
	old = atomicCAS(address_as_ull, assumed,
					__double_as_longlong(val +
										 __longlong_as_double(assumed)));

	// Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
  } while (assumed != old);

  return __longlong_as_double(old);
}
__device__ float myAtomicAdd(float* address, float val)
{
  atomicAdd(address, val);
}
#else
#define myAtomicAdd atomicAdd
#endif

#if 1
// ######## ######## ######## ######## ######## ######## ######## ########
/*
  完全逐次
  バリエーション：
  - a2trans：approxy2を転置版で計算するか否か(0,1)
  - a2interchange：approxy2のループを入れ替えるか否か(0,1)
*/
template <class T, int a2t, int a2i>
__global__ void hmvm_cuda_seq
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt,
 int *a1, int *a2, T *rowmat, T *rowmat_t,
 int napprox, int *approx, int ndense, int *dense)
{
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_seq : begin\n");
#endif
  int ndl, ndt, nstrtl, nstrtt;
  int ip, kt, il, it, itt, itl, ill;
  size_t head;
  //extern __shared__ T tmp2[];
  extern __shared__ __align__(sizeof(T)) unsigned char my_smem[];
  T *tmp2 = reinterpret_cast<T *>(my_smem);
  int i;

  // approx
  for(i=0; i<napprox; i++){
#ifndef _SKIP_APPROX
	ip = approx[i];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	kt = _kt[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt);
#endif
	head = a1[ip];
	for(il=0; il<kt; il++){
	  tmp2[il] = 0.0;
	  for(it=0; it<ndt; it++){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		if(a2t==0){
		  tmp2[il] += rowmat[head+itl]*d_zu[itt];
		}else{
		  tmp2[il] += rowmat_t[head+itl]*d_zu[itt];
		}
	  }
	}
	head = a2[ip];
	if(a2t==0){
	  if(a2i==0){
		for(il=0; il<kt; il++){
		  for(it=0; it<ndl; it++){
			ill=it+nstrtl-1;
			itl=it+il*ndl;
			myAtomicAdd(&d_zaut[ill], rowmat[head+itl]*tmp2[il]);
		  }
		}
	  }else{
		for(it=0; it<ndl; it++){
		  ill=it+nstrtl-1;
		  for(il=0; il<kt; il++){
			itl=it+il*ndl;
			myAtomicAdd(&d_zaut[ill], rowmat[head+itl]*tmp2[il]);
		  }
		}
	  }
	}else{
	  if(a2i==0){
		for(il=0; il<kt; il++){
		  for(it=0; it<ndl; it++){
			ill=it+nstrtl-1;
			itl=it*kt+il;
			myAtomicAdd(&d_zaut[ill], rowmat_t[head+itl]*tmp2[il]);
		  }
		}
	  }else{
		for(it=0; it<ndl; it++){
		  ill=it+nstrtl-1;
		  for(il=0; il<kt; il++){
			itl=it*kt+il;
			myAtomicAdd(&d_zaut[ill], rowmat_t[head+itl]*tmp2[il]);
		  }
		}
	  }
	}
#endif
  }

  // dense
  for(i=0; i<ndense; i++){
#ifndef _SKIP_DENSE
	ip = dense[i];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt);
#endif
	head = a1[ip];
	for(il=0; il<ndl; il++){
	  T tmp = (T)0.0;
	  ill=il+nstrtl-1;
	  for(it=0; it<ndt; it++){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		if(a2t==0){
		  tmp += rowmat[head+itl]*d_zu[itt];
		}else{
		  tmp += rowmat_t[head+itl]*d_zu[itt];
		}
	  }
	  myAtomicAdd(&d_zaut[ill], tmp);
	}
#endif
  }
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_seq : end\n");
#endif
}

template <class T, int a2t, int a2i>
void hmvm_cuda_seq_proxy
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *ltmtx, int *ndt, int *ndl, int *nstrtl, int *nstrtt, int *kt,
 int *a1, int *a2, T *rowmat, T *rowmat_t,
 int napprox, int *approx, int ndense, int *dense,
 T *v, T *b, int nd, char *fname, int bench)
{
#if 1
  int M=5, L=M+bench;
  FILE *F;
  int i, l, lmax;
  double d1, d2, *dtimes, dmin, dmax, davg;
  hipError_t ret;
  dtimes = new double[L];
  if(bench==0){lmax=1;}else{lmax=L;}
  for(l=0;l<lmax;l++){
	for(i=0;i<nd;i++)v[i] = (T)0.0;
	CHECK_DO(hipMemcpy(d_zaut, v, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy v to d_v");
	CHECK_DO(hipMemcpy(d_zu, b, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy b to d_b");
	hipDeviceSynchronize();
	d1 = omp_get_wtime();
	hmvm_cuda_seq<T,a2t,a2i><<<1,1,sizeof(T)*ktmax>>>
	  (d_zaut, d_zu, nlf, ktmax,
	   ltmtx, ndt, ndl, nstrtl, nstrtt, kt,
	   a1, a2, rowmat, rowmat_t,
	   napprox, approx, ndense, dense);
	hipDeviceSynchronize();
	d2 = omp_get_wtime();
	dtimes[l] = d2-d1;
  }
  if(bench==0){
	CHECK_DO(hipMemcpy(v, d_zaut, sizeof(T)*nd, hipMemcpyDeviceToHost),"hipMemcpy d_v to v");
	printf("write to %s\n", fname);
	F = fopen(fname, "w");
	for(i=0;i<nd;i++)fprintf(F, "%.3E\n", v[i]);
	fclose(F);
  }else{
	dmin = 9999.99;
	dmax = 0.0;
	davg = 0.0;
	for(i=M;i<L;i++){
	  davg += dtimes[i];
	  if(dmin>dtimes[i])dmin=dtimes[i];
	  if(dmax<dtimes[i])dmax=dtimes[i];
	}
	davg /= (L-M);
	printf("TIME %d hmvm_cuda1_seq%s min %e max %e avg %e\n", L-M, typeid(T).name(), dmin, dmax, davg);
  }
  delete [] dtimes;
#endif
}

void hmvm_cuda_seq_proxy
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt,
 int *a1, int *a2, float *rowmat, float *rowmat_t,
 int napprox, int *approx, int ndense, int *dense,
 float *v, float *b, int nd, char *fname, int bench,
 int a2t, int a2i){
  if(a2t==0 && a2i==0)
	hmvm_cuda_seq_proxy<float,0,0>
	  (d_zaut, d_zu, nlf, ktmax,
	   _ltmtx, _ndt, _ndl, _nstrtl, _nstrtt, _kt,
	   a1, a2, rowmat, rowmat_t,
	   napprox, approx, ndense, dense,
	   v, b, nd, fname, bench);
  if(a2t==0 && a2i==1)
	hmvm_cuda_seq_proxy<float,0,1>
	  (d_zaut, d_zu, nlf, ktmax,
	   _ltmtx, _ndt, _ndl, _nstrtl, _nstrtt, _kt,
	   a1, a2, rowmat, rowmat_t,
	   napprox, approx, ndense, dense,
	   v, b, nd, fname, bench);
  if(a2t==1 && a2i==0)
	hmvm_cuda_seq_proxy<float,1,0>
	  (d_zaut, d_zu, nlf, ktmax,
	   _ltmtx, _ndt, _ndl, _nstrtl, _nstrtt, _kt,
	   a1, a2, rowmat, rowmat_t,
	   napprox, approx, ndense, dense,
	   v, b, nd, fname, bench);
  if(a2t==1 && a2i==1)
	hmvm_cuda_seq_proxy<float,1,1>
	  (d_zaut, d_zu, nlf, ktmax,
	   _ltmtx, _ndt, _ndl, _nstrtl, _nstrtt, _kt,
	   a1, a2, rowmat, rowmat_t,
	   napprox, approx, ndense, dense,
	   v, b, nd, fname, bench);
}
void hmvm_cuda_seq_proxy
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt,
 int *a1, int *a2, double *rowmat, double *rowmat_t,
 int napprox, int *approx, int ndense, int *dense,
 double *v, double *b, int nd, char *fname, int bench,
 int a2t, int a2i){
  if(a2t==0 && a2i==0)
	hmvm_cuda_seq_proxy<double,0,0>
	  (d_zaut, d_zu, nlf, ktmax,
	   _ltmtx, _ndt, _ndl, _nstrtl, _nstrtt, _kt,
	   a1, a2, rowmat, rowmat_t,
	   napprox, approx, ndense, dense,
	   v, b, nd, fname, bench);
  if(a2t==0 && a2i==1)
	hmvm_cuda_seq_proxy<double,0,1>
	  (d_zaut, d_zu, nlf, ktmax,
	   _ltmtx, _ndt, _ndl, _nstrtl, _nstrtt, _kt,
	   a1, a2, rowmat, rowmat_t,
	   napprox, approx, ndense, dense,
	   v, b, nd, fname, bench);
  if(a2t==1 && a2i==0)
	hmvm_cuda_seq_proxy<double,1,0>
	  (d_zaut, d_zu, nlf, ktmax,
	   _ltmtx, _ndt, _ndl, _nstrtl, _nstrtt, _kt,
	   a1, a2, rowmat, rowmat_t,
	   napprox, approx, ndense, dense,
	   v, b, nd, fname, bench);
  if(a2t==1 && a2i==1)
	hmvm_cuda_seq_proxy<double,1,1>
	  (d_zaut, d_zu, nlf, ktmax,
	   _ltmtx, _ndt, _ndl, _nstrtl, _nstrtt, _kt,
	   a1, a2, rowmat, rowmat_t,
	   napprox, approx, ndense, dense,
	   v, b, nd, fname, bench);
}
#endif
// ######## ######## ######## ######## ######## ######## ######## ########

// ######## ######## ######## ######## ######## ######## ######## ########
/*
  block並列化
  ThreadBlockごとに1つの部分行列積(mat-mat-vecまたはmat-vec)を行う
  ThreadBlock内部は逐次
  - a2trans：approxy2を転置版で計算するか否か(0,1)
  - a2interchange：approxy2のループを入れ替えるか否か(0,1)
 */
template <class T, int a2t, int a2i>
__global__ void hmvm_cuda_block
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt,
 int *a1, int *a2, T *rowmat, T *rowmat_t,
 int napprox, int *approx, int ndense, int *dense)
{
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cudaD_block : begin\n");
#endif
  int ndl, ndt, nstrtl, nstrtt;
  int ip, kt, il, it, itt, itl, ill;
  size_t head;
  extern __shared__ __align__(sizeof(T)) unsigned char my_smem[];
  T *tmp2 = reinterpret_cast<T *>(my_smem);

  if(blockIdx.x<napprox){
#ifndef _SKIP_APPROX
	// approx
	//for(i=0; i<napprox; i++){
	ip = approx[blockIdx.x];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	kt = _kt[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt);
#endif
	head = a1[ip];
	for(il=0; il<kt; il++){
	  tmp2[il] = 0.0;
	  for(it=0; it<ndt; it++){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		if(a2t==0){
		  tmp2[il] += rowmat[head+itl]*d_zu[itt];
		}else{
		  tmp2[il] += rowmat_t[head+itl]*d_zu[itt];
		}
	  }
	}
	head = a2[ip];
	if(a2t==0){
	  if(a2i==0){
		for(il=0; il<kt; il++){
		  for(it=0; it<ndl; it++){
			ill=it+nstrtl-1;
			itl=it+il*ndl;
			myAtomicAdd(&d_zaut[ill], rowmat[head+itl]*tmp2[il]);
		  }
		}
	  }else{
		for(it=0; it<ndl; it++){
		  ill=it+nstrtl-1;
		  for(il=0; il<kt; il++){
			itl=it+il*ndl;
			myAtomicAdd(&d_zaut[ill], rowmat[head+itl]*tmp2[il]);
		  }
		}
	  }
	}else{
	  if(a2i==0){
		for(il=0; il<kt; il++){
		  for(it=0; it<ndl; it++){
			ill=it+nstrtl-1;
			itl=it*kt+il;
			myAtomicAdd(&d_zaut[ill], rowmat_t[head+itl]*tmp2[il]);
		  }
		}
	  }else{
		for(it=0; it<ndl; it++){
		  ill=it+nstrtl-1;
		  for(il=0; il<kt; il++){
			itl=it*kt+il;
			myAtomicAdd(&d_zaut[ill], rowmat_t[head+itl]*tmp2[il]);
		  }
		}
	  }
	}
#endif
  }else{
#ifndef _SKIP_DENSE
	// dense
	//for(i=0; i<ndense; i++){
	ip = dense[blockIdx.x - napprox];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt);
#endif
	head = a1[ip];
	for(il=0; il<ndl; il++){
	  T tmp = 0.0;
	  ill=il+nstrtl-1;
	  for(it=0; it<ndt; it++){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		if(a2t==0){
		  tmp += rowmat[head+itl]*d_zu[itt];
		}else{
		  tmp += rowmat_t[head+itl]*d_zu[itt];
		}
	  }
	  myAtomicAdd(&d_zaut[ill], tmp);
	}
#endif
  }
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cudaD_block : end\n");
#endif
}

template <class T, int a2t, int a2i>
void hmvm_cuda_block_proxy
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *ltmtx, int *ndt, int *ndl, int *nstrtl, int *nstrtt, int *kt,
 int *a1, int *a2, T *rowmat, T *rowmat_t,
 int napprox, int *approx, int ndense, int *dense,
 T *v, T *b, int nd, char *fname, int bench)
{
#if 1
  int M=5, L=M+bench;
  FILE *F;
  int i, l, lmax;
  double d1, d2, *dtimes, dmin, dmax, davg;
  hipError_t ret;
  dtimes = new double[L];
  if(bench==0){lmax=1;}else{lmax=L;}
  for(l=0;l<lmax;l++){
	for(i=0;i<nd;i++)v[i] = (T)0.0;
	CHECK_DO(hipMemcpy(d_zaut, v, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy v to d_v");
	CHECK_DO(hipMemcpy(d_zu, b, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy b to d_b");
	hipDeviceSynchronize();
	d1 = omp_get_wtime();
	hmvm_cuda_block<T,a2t,a2i><<<napprox+ndense,1,sizeof(T)*ktmax>>>
	  (d_zaut, d_zu, nlf, ktmax,
	   ltmtx, ndt, ndl, nstrtl, nstrtt,
	   kt, a1, a2, rowmat, rowmat_t,
	   napprox, approx, ndense, dense);
	hipDeviceSynchronize();
	d2 = omp_get_wtime();
	dtimes[l] = d2-d1;
  }
  if(bench==0){
	CHECK_DO(hipMemcpy(v, d_zaut, sizeof(T)*nd, hipMemcpyDeviceToHost),"hipMemcpy d_v to v");
	printf("write to %s\n", fname);
	F = fopen(fname, "w");
	for(i=0;i<nd;i++)fprintf(F, "%.3E\n", v[i]);
	fclose(F);
  }else{
	dmin = 9999.99;
	dmax = 0.0;
	davg = 0.0;
	for(i=M;i<L;i++){
	  davg += dtimes[i];
	  if(dmin>dtimes[i])dmin=dtimes[i];
	  if(dmax<dtimes[i])dmax=dtimes[i];
	}
	davg /= (L-M);
	printf("TIME %d hmvm_cuda1_block%s min %e max %e avg %e\n", L-M, typeid(T).name(), dmin, dmax, davg);
  }
  delete [] dtimes;
#endif
}

void hmvm_cuda_block_proxy
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt,
 int *a1, int *a2, float *rowmat, float *rowmat_t,
 int napprox, int *approx, int ndense, int *dense,
 float *v, float *b, int nd, char *fname, int bench,
 int a2t, int a2i){
  if(a2t==0 && a2i==0)
	hmvm_cuda_block_proxy<float,0,0>
	  (d_zaut, d_zu, nlf, ktmax,
	   _ltmtx, _ndt, _ndl, _nstrtl, _nstrtt,
	   _kt, a1, a2, rowmat, rowmat_t,
	   napprox, approx, ndense, dense,
	   v, b, nd, fname, bench);
  if(a2t==0 && a2i==1)
	hmvm_cuda_block_proxy<float,0,1>
	  (d_zaut, d_zu, nlf, ktmax,
	   _ltmtx, _ndt, _ndl, _nstrtl, _nstrtt,
	   _kt, a1, a2, rowmat, rowmat_t,
	   napprox, approx, ndense, dense,
	   v, b, nd, fname, bench);
  if(a2t==1 && a2i==0)
	hmvm_cuda_block_proxy<float,1,0>
	  (d_zaut, d_zu, nlf, ktmax,
	   _ltmtx, _ndt, _ndl, _nstrtl, _nstrtt,
	   _kt, a1, a2, rowmat, rowmat_t,
	   napprox, approx, ndense, dense,
	   v, b, nd, fname, bench);
  if(a2t==1 && a2i==1)
	hmvm_cuda_block_proxy<float,1,1>
	  (d_zaut, d_zu, nlf, ktmax,
	   _ltmtx, _ndt, _ndl, _nstrtl, _nstrtt,
	   _kt, a1, a2, rowmat, rowmat_t,
	   napprox, approx, ndense, dense,
	   v, b, nd, fname, bench);
}
void hmvm_cuda_block_proxy
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt,
 int *a1, int *a2, double *rowmat, double *rowmat_t,
 int napprox, int *approx, int ndense, int *dense,
 double *v, double *b, int nd, char *fname, int bench,
 int a2t, int a2i){
  if(a2t==0 && a2i==0)
	hmvm_cuda_block_proxy<double,0,0>
	  (d_zaut, d_zu, nlf, ktmax,
	   _ltmtx, _ndt, _ndl, _nstrtl, _nstrtt,
	   _kt, a1, a2, rowmat, rowmat_t,
	   napprox, approx, ndense, dense,
	   v, b, nd, fname, bench);
  if(a2t==0 && a2i==1)
	hmvm_cuda_block_proxy<double,0,1>
	  (d_zaut, d_zu, nlf, ktmax,
	   _ltmtx, _ndt, _ndl, _nstrtl, _nstrtt,
	   _kt, a1, a2, rowmat, rowmat_t,
	   napprox, approx, ndense, dense,
	   v, b, nd, fname, bench);
  if(a2t==1 && a2i==0)
	hmvm_cuda_block_proxy<double,1,0>
	  (d_zaut, d_zu, nlf, ktmax,
	   _ltmtx, _ndt, _ndl, _nstrtl, _nstrtt,
	   _kt, a1, a2, rowmat, rowmat_t,
	   napprox, approx, ndense, dense,
	   v, b, nd, fname, bench);
  if(a2t==1 && a2i==1)
	hmvm_cuda_block_proxy<double,1,1>
	  (d_zaut, d_zu, nlf, ktmax,
	   _ltmtx, _ndt, _ndl, _nstrtl, _nstrtt,
	   _kt, a1, a2, rowmat, rowmat_t,
	   napprox, approx, ndense, dense,
	   v, b, nd, fname, bench);
}
// ######## ######## ######## ######## ######## ######## ######## ########

// ######## ######## ######## ######## ######## ######## ######## ########
#if 1
/*
  hybrid1
  TBあたりスレッド数は32に固定
  1TBが1つのPMVを担当
  PMV内の1行を1/div WARPが担当
  <<<napprox+ndense,32>>>
  1 PMV by 1 TB(=1WARP)
  1 line by 1/div WARP
  バリエーション
  - div：1行を1/divのWARPで計算する、div=1,2,4,8,16,32
  - a2t：a2を転置版で計算するか否か(0,1)
  - a2interchange：a2のループを入れ替えるか否か(0,1)
  - aatomic：approxの計算をatomic優先にするかwarp shuffle併用するか(0,1)
  - datomic：denseの計算をatomic優先にするかwarp shuffle併用するか(0,1)
  6x2x2x2x2=96通り
  divが大きな時にちょっとおかしいかも？
*/
template <class T, int div, int a2t, int a2i, int aatomic, int datomic>
__global__ void hmvm_cuda_hybrid1
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt,
 int *a1, int *a2, T *rowmat, T *rowmat_t,
 int napprox, int *approx, int ndense, int *dense)
{
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid1 : begin\n");
#endif
  int gid   = blockIdx.x;
  int bid   = threadIdx.x/(32/div);
  int blen  = div;
  int xid   = threadIdx.x%(32/div);
  int xlen  = (32/div);
  int ndl, ndt, nstrtl, nstrtt;
  int ip, kt, il, it, itt, itl, ill;
  size_t head;
  T tmp = 0.0;
  extern __shared__ __align__(sizeof(T)) unsigned char my_smem[];
  T *tmp2 = reinterpret_cast<T *>(my_smem);
  cg::thread_block_tile<32/div> g = cg::tiled_partition<32/div>(cg::this_thread_block());

  if(gid<napprox){
#ifndef _SKIP_APPROX
	// approx
	ip = approx[gid];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	kt = _kt[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt);
#endif
	head = a1[ip];
	for(il=bid; il<kt; il+=blen){
	  if(xid==0)tmp2[il] = 0.0;
	  tmp = 0.0;
	  for(it=xid; it<ndt; it+=xlen){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		if(a2t==0){
		  tmp += rowmat[head+itl]*d_zu[itt];
		}else{
		  tmp += rowmat_t[head+itl]*d_zu[itt];
		}
	  }
	  for (int offset = g.size()/2; offset > 0; offset /= 2)tmp += g.shfl_down(tmp, offset);
	  if(xid==0)tmp2[il] = tmp;
	}
	head = a2[ip];
	if(a2t==0){
	  if(a2i==0){
		for(il=bid; il<kt; il+=blen){
		  for(it=xid; it<ndl; it+=xlen){
			ill=it+nstrtl-1;
			itl=it+il*ndl;
			myAtomicAdd(&d_zaut[ill], rowmat[head+itl]*tmp2[il]);
		  }
		}
	  }else{ // a2i==1
		if(aatomic==0){
		  for(it=bid; it<ndl; it+=blen){
			ill=it+nstrtl-1;
			tmp = 0.0;
			for(il=xid; il<kt; il+=xlen){
			  itl=it+il*ndl;
			  tmp += rowmat[head+itl]*tmp2[il];
			}
			for (int offset = g.size()/2; offset > 0; offset /= 2)tmp += g.shfl_down(tmp, offset);
			if(xid==0){
			  myAtomicAdd(&d_zaut[ill], tmp);
			}
		  }
		}else{ // aatomic==1
		  for(it=bid; it<ndl; it+=blen){
			ill=it+nstrtl-1;
			for(il=xid; il<kt; il+=xlen){
			  itl=it+il*ndl;
			  myAtomicAdd(&d_zaut[ill], rowmat[head+itl]*tmp2[il]);
			}
		  }
		}
	  }
	}else{ // a2t==1
	  if(a2i==0){
		for(il=bid; il<kt; il+=blen){
		  for(it=xid; it<ndl; it+=xlen){
			ill=it+nstrtl-1;
			itl=it*kt+il;
			myAtomicAdd(&d_zaut[ill], rowmat_t[head+itl]*tmp2[il]);
		  }
		}
	  }else{ // a2i==1
		if(aatomic==0){
		  for(it=bid; it<ndl; it+=blen){
			ill=it+nstrtl-1;
			tmp = 0.0;
			for(il=xid; il<kt; il+=xlen){
			  itl=it*kt+il;
			  tmp += rowmat_t[head+itl]*tmp2[il];
			}
			for (int offset = g.size()/2; offset > 0; offset /= 2)tmp += g.shfl_down(tmp, offset);
			if(xid==0){
			  myAtomicAdd(&d_zaut[ill], tmp);
			}
		  }
		}else{ // aatomic==1
		  for(it=bid; it<ndl; it+=blen){
			ill=it+nstrtl-1;
			for(il=xid; il<kt; il+=xlen){
			  itl=it*kt+il;
			  myAtomicAdd(&d_zaut[ill], rowmat_t[head+itl]*tmp2[il]);
			}
		  }
		}
	  }
	}
#endif // approx
  }else{
#ifndef _SKIP_DENSE
	// dense
	ip = dense[gid-napprox];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt);
#endif
	head = a1[ip];
	for(il=bid; il<ndl; il+=blen){
	  tmp = 0.0;
	  ill=il+nstrtl-1;
	  for(it=xid; it<ndt; it+=xlen){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		if(a2t==0){
		  tmp += rowmat[head+itl]*d_zu[itt];
		}else{
		  tmp += rowmat_t[head+itl]*d_zu[itt];
		}
	  }
	  if(datomic==0){
		for (int offset = g.size()/2; offset > 0; offset /= 2)tmp += g.shfl_down(tmp, offset);
		if(xid==0){
		  myAtomicAdd(&d_zaut[ill], tmp);
		}
	  }else{
		myAtomicAdd(&d_zaut[ill], tmp);
	  }
	}
#endif // dense
  }

#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid1 : end\n");
#endif
}

template <class T, int div, int a2t, int a2i, int aa, int da>
void hmvm_cuda_hybrid1_proxy
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *ltmtx, int *ndt, int *ndl, int *nstrtl, int *nstrtt, int *kt,
 int *a1, int *a2, T *rowmat, T *rowmat_t,
 int napprox, int *approx, int ndense, int *dense,
 int blocks, int threads, int shms,
 T *v, T *b, int nd, char *fname, int bench)
{
#if 1
  int M=5, L=M+bench;
  FILE *F;
  int i, l, lmax;
  double d1, d2, *dtimes, dmin, dmax, davg;
  hipError_t ret;
  dtimes = new double[L];
  if(bench==0){lmax=1;}else{lmax=L;}
  for(l=0;l<lmax;l++){
	for(i=0;i<nd;i++)v[i] = (T)0.0;
	CHECK_DO(hipMemcpy(d_zaut, v, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy v to d_v");
	CHECK_DO(hipMemcpy(d_zu, b, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy b to d_b");
	hipDeviceSynchronize();
	d1 = omp_get_wtime();
	hmvm_cuda_hybrid1<T,div,a2t,a2i,aa,da><<<blocks,threads,shms>>>
	  (d_zaut, d_zu, nlf, ktmax, ltmtx,
	   ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, rowmat_t,
	   napprox, approx, ndense, dense);
	hipDeviceSynchronize();
	d2 = omp_get_wtime();
	dtimes[l] = d2-d1;
  }
  if(bench==0){
	CHECK_DO(hipMemcpy(v, d_zaut, sizeof(T)*nd, hipMemcpyDeviceToHost),"hipMemcpy d_v to v");
	printf("write to %s\n", fname);
	F = fopen(fname, "w");
	for(i=0;i<nd;i++)fprintf(F, "%.3E\n", v[i]);
	fclose(F);
  }else{
	dmin = 9999.99;
	dmax = 0.0;
	davg = 0.0;
	for(i=M;i<L;i++){
	  davg += dtimes[i];
	  if(dmin>dtimes[i])dmin=dtimes[i];
	  if(dmax<dtimes[i])dmax=dtimes[i];
	}
	davg /= (L-M);
	printf("TIME %d hmvm_cuda1_hybrid1%s min %e max %e avg %e\n", L-M, typeid(T).name(), dmin, dmax, davg);
  }
  delete [] dtimes;
#endif
}
#include "template_hybrid1.hpp"
// ######## ######## ######## ######## ######## ######## ######## ########
#endif

#if 1
// ######## ######## ######## ######## ######## ######## ######## ########
/*
  hybrid2
  複数WARP単一GEMV個別行カーネル
  1PMVを担当するのは1TBのまま
  1TBあたりスレッド数を32*mulに増やす
  PMV内の1行を1/div WARPが担当
  <<<napprox+ndense,32*mul>>>
  1 PMV by 1 TB
  1 line by 1/div WARP
  バリエーション
  - div：1行を1/divのWARPで計算する、div=1,2,4,8,16,32
  - mul：立ち上げるスレッド数(mul*32)、1つのmat-mat-vecまたはmat-vecをmul TBで実行、mul=1,2,3,...,16
  - a2t：a2を転置版で計算するか否か(0,1)
  - a2interchange：a2のループを入れ替えるか否か(0,1)
  - aatomic：approxの計算をatomic優先にするかwarp shuffle併用するか(0,1)
  - datomic：denseの計算をatomic優先にするかwarp shuffle併用するか(0,1)
  6x16*2x2x2x2=1536通り
*/
template <class T, int div, int mul, int a2t, int a2i, int aatomic, int datomic>
__global__ void hmvm_cuda_hybrid2
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt,
 int *a1, int *a2, T *rowmat, T *rowmat_t,
 int napprox, int *approx, int ndense, int *dense)
{
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid2 : begin\n");
#endif
  int gid   = blockIdx.x;
  //int tid   = threadIdx.x;
  int bid   = threadIdx.x/(32/div);
  int blen  = mul*div;
  int xid   = threadIdx.x%(32/div);
  int xlen  = (32/div);
  int ndl, ndt, nstrtl, nstrtt;
  int ip, kt, il, it, itt, itl, ill;
  size_t head;
  T tmp = 0.0;
  extern __shared__ __align__(sizeof(T)) unsigned char my_smem[];
  T *tmp2 = reinterpret_cast<T *>(my_smem);
  cg::thread_block_tile<32/div> g = cg::tiled_partition<32/div>(cg::this_thread_block());

  if(gid<napprox){
#ifndef _SKIP_APPROX
	// approx
	ip = approx[gid];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	kt = _kt[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt);
#endif
	head = a1[ip];
	for(il=bid; il<kt; il+=blen){
	  if(xid==0)tmp2[il] = 0.0;
	  tmp = 0.0;
	  for(it=xid; it<ndt; it+=xlen){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp += rowmat[head+itl]*d_zu[itt];
	  }
	  //for (int offset = warpSize/(2*div); offset > 0; offset /= 2)tmp += __shfl_down_sync(0xffff, tmp, offset, warpSize);
	  for (int offset = g.size()/2; offset > 0; offset /= 2)tmp += g.shfl_down(tmp, offset);
	  if(xid==0)tmp2[il] = tmp;
	}
	head = a2[ip];
	for(il=bid; il<kt; il+=blen){
	  for(it=xid; it<ndl; it+=xlen){
		ill=it+nstrtl-1;
		itl=it+il*ndl;
		myAtomicAdd(&d_zaut[ill], rowmat[head+itl]*tmp2[il]);
	  }
	}
#endif // approx
  }else if(gid-napprox<ndense){
#ifndef _SKIP_DENSE
	// dense
	ip = dense[gid-napprox];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt);
#endif
	head = a1[ip];
	for(il=bid; il<ndl; il+=blen){
	  tmp = 0.0;
	  ill=il+nstrtl-1;
	  for(it=xid; it<ndt; it+=xlen){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp += rowmat[head+itl]*d_zu[itt];
	  }
	  if(datomic==0){
		//for (int offset = warpSize/(2*div); offset > 0; offset /= 2)tmp += __shfl_down_sync(tmp, offset, warpSize);
		for (int offset = g.size()/2; offset > 0; offset /= 2)tmp += g.shfl_down(tmp, offset);
		if(xid==0){
		  myAtomicAdd(&d_zaut[ill], tmp);
		}
	  }else{
		myAtomicAdd(&d_zaut[ill], tmp);
	  }
	}
#endif // dense
  }

#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid2 : end\n");
#endif
}

#if 0
template <class T, int div>
__global__ void hmvm_cuda_hybrid2
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt,
 int *a1, int *a2, T *rowmat, T *rowmat_t,
 int napprox, int *approx, int ndense, int *dense,
 int mul, int a2t, int a2i, int aatomic, int datomic)
{
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid2 : begin\n");
#endif
  int gid   = blockIdx.x;
  //int tid   = threadIdx.x;
  int bid   = threadIdx.x/(32/div);
  int blen  = mul*div;
  int xid   = threadIdx.x%(32/div);
  int xlen  = (32/div);
  int ndl, ndt, nstrtl, nstrtt;
  int ip, kt, il, it, itt, itl, ill;
  size_t head;
  T tmp = 0.0;
  extern __shared__ __align__(sizeof(T)) unsigned char my_smem[];
  T *tmp2 = reinterpret_cast<T *>(my_smem);
  cg::thread_block_tile<32/div> g = cg::tiled_partition<32/div>(cg::this_thread_block());

  if(gid<napprox){
#ifndef _SKIP_APPROX
	// approx
	ip = approx[gid];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	kt = _kt[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt);
#endif
	head = a1[ip];
	for(il=bid; il<kt; il+=blen){
	  if(xid==0)tmp2[il] = 0.0;
	  tmp = 0.0;
	  for(it=xid; it<ndt; it+=xlen){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp += rowmat[head+itl]*d_zu[itt];
	  }
	  //for (int offset = warpSize/(2*div); offset > 0; offset /= 2)tmp += __shfl_down_sync(0xffff, tmp, offset, warpSize);
	  for (int offset = g.size()/2; offset > 0; offset /= 2)tmp += g.shfl_down(tmp, offset);
	  if(xid==0)tmp2[il] = tmp;
	}
	head = a2[ip];
	for(il=bid; il<kt; il+=blen){
	  for(it=xid; it<ndl; it+=xlen){
		ill=it+nstrtl-1;
		itl=it+il*ndl;
		myAtomicAdd(&d_zaut[ill], rowmat[head+itl]*tmp2[il]);
	  }
	}
#endif // approx
  }else if(gid-napprox<ndense){
#ifndef _SKIP_DENSE
	// dense
	ip = dense[gid-napprox];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt);
#endif
	head = a1[ip];
	for(il=bid; il<ndl; il+=blen){
	  tmp = 0.0;
	  ill=il+nstrtl-1;
	  for(it=xid; it<ndt; it+=xlen){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp += rowmat[head+itl]*d_zu[itt];
	  }
	  if(datomic==0){
		//for (int offset = warpSize/(2*div); offset > 0; offset /= 2)tmp += __shfl_down_sync(tmp, offset, warpSize);
		for (int offset = g.size()/2; offset > 0; offset /= 2)tmp += g.shfl_down(tmp, offset);
		if(xid==0){
		  myAtomicAdd(&d_zaut[ill], tmp);
		}
	  }else{
		myAtomicAdd(&d_zaut[ill], tmp);
	  }
	}
#endif // dense
  }

#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid2 : end\n");
#endif
}

template <class T>
void hmvm_cuda_hybrid2_proxy
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *ltmtx, int *ndt, int *ndl, int *nstrtl, int *nstrtt, int *kt,
 int *a1, int *a2, T *rowmat, T *rowmat_t,
 int napprox, int *approx, int ndense, int *dense,
 int blocks, int threads, int shms,
 T *v, T *b, int nd, char *fname, int bench,
 int div, int mul, int a2t, int a2i, int aa, int da)
{
#if 1
  int M=5, L=M+bench;
  FILE *F;
  int i, l, lmax;
  double d1, d2, *dtimes, dmin, dmax, davg;
  hipError_t ret;
  dtimes = new double[L];
  if(bench==0){lmax=1;}else{lmax=L;}
  for(l=0;l<lmax;l++){
	for(i=0;i<nd;i++)v[i] = (T)0.0;
	CHECK_DO(hipMemcpy(d_zaut, v, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy v to d_v");
	CHECK_DO(hipMemcpy(d_zu, b, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy b to d_b");
	hipDeviceSynchronize();
	d1 = omp_get_wtime();
	switch(div){
	case 1:
	  hmvm_cuda_hybrid2<T,1><<<blocks,threads,shms>>>
		(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt,
		 a1, a2, rowmat, rowmat_t, napprox, approx, ndense, dense , mul, a2t, a2i, aa, da);
	  break;
	case 2:
	  hmvm_cuda_hybrid2<T,2><<<blocks,threads,shms>>>
		(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt,
		 a1, a2, rowmat, rowmat_t, napprox, approx, ndense, dense , mul, a2t, a2i, aa, da);
	  break;
	case 4:
	  hmvm_cuda_hybrid2<T,4><<<blocks,threads,shms>>>
		(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt,
		 a1, a2, rowmat, rowmat_t, napprox, approx, ndense, dense , mul, a2t, a2i, aa, da);
	  break;
	case 8:
	  hmvm_cuda_hybrid2<T,8><<<blocks,threads,shms>>>
		(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt,
		 a1, a2, rowmat, rowmat_t, napprox, approx, ndense, dense , mul, a2t, a2i, aa, da);
	  break;
	case 16:
	  hmvm_cuda_hybrid2<T,16><<<blocks,threads,shms>>>
		(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt,
		 a1, a2, rowmat, rowmat_t, napprox, approx, ndense, dense , mul, a2t, a2i, aa, da);
	  break;
	case 32:
	  hmvm_cuda_hybrid2<T,32><<<blocks,threads,shms>>>
		(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt,
		 a1, a2, rowmat, rowmat_t, napprox, approx, ndense, dense , mul, a2t, a2i, aa, da);
	  break;
	}
	hipDeviceSynchronize();
	d2 = omp_get_wtime();
	dtimes[l] = d2-d1;
  }
  if(bench==0){
	CHECK_DO(hipMemcpy(v, d_zaut, sizeof(T)*nd, hipMemcpyDeviceToHost),"hipMemcpy d_v to v");
	printf("write to %s\n", fname);
	F = fopen(fname, "w");
	for(i=0;i<nd;i++)fprintf(F, "%.3E\n", v[i]);
	fclose(F);
  }else{
	dmin = 9999.99;
	dmax = 0.0;
	davg = 0.0;
	for(i=M;i<L;i++){
	  davg += dtimes[i];
	  if(dmin>dtimes[i])dmin=dtimes[i];
	  if(dmax<dtimes[i])dmax=dtimes[i];
	}
	davg /= (L-M);
	printf("TIME %d hmvm_cuda1_hybrid2%s min %e max %e avg %e\n", L-M, typeid(T).name(), dmin, dmax, davg);
  }
  delete [] dtimes;
#endif
}
#else
template <class T, int div, int mul, int a2t, int a2i, int aa, int da>
void hmvm_cuda_hybrid2_proxy
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *ltmtx, int *ndt, int *ndl, int *nstrtl, int *nstrtt, int *kt,
 int *a1, int *a2, T *rowmat, T *rowmat_t,
 int napprox, int *approx, int ndense, int *dense,
 int blocks, int threads, int shms,
 T *v, T *b, int nd, char *fname, int bench)
{
#if 1
  int M=5, L=M+bench;
  FILE *F;
  int i, l, lmax;
  double d1, d2, *dtimes, dmin, dmax, davg;
  hipError_t ret;
  dtimes = new double[L];
  if(bench==0){lmax=1;}else{lmax=L;}
  for(l=0;l<lmax;l++){
	for(i=0;i<nd;i++)v[i] = (T)0.0;
	CHECK_DO(hipMemcpy(d_zaut, v, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy v to d_v");
	CHECK_DO(hipMemcpy(d_zu, b, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy b to d_b");
	hipDeviceSynchronize();
	d1 = omp_get_wtime();
	hmvm_cuda_hybrid2<T,div,mul,a2t,a2i,aa,da><<<blocks,threads,shms>>>
	  (d_zaut, d_zu, nlf, ktmax, ltmtx,
	   ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, rowmat_t,
	   napprox, approx, ndense, dense);
	hipDeviceSynchronize();
	d2 = omp_get_wtime();
	dtimes[l] = d2-d1;
  }
  if(bench==0){
	CHECK_DO(hipMemcpy(v, d_zaut, sizeof(T)*nd, hipMemcpyDeviceToHost),"hipMemcpy d_v to v");
	printf("write to %s\n", fname);
	F = fopen(fname, "w");
	for(i=0;i<nd;i++)fprintf(F, "%.3E\n", v[i]);
	fclose(F);
  }else{
	dmin = 9999.99;
	dmax = 0.0;
	davg = 0.0;
	for(i=M;i<L;i++){
	  davg += dtimes[i];
	  if(dmin>dtimes[i])dmin=dtimes[i];
	  if(dmax<dtimes[i])dmax=dtimes[i];
	}
	davg /= (L-M);
	printf("TIME %d hmvm_cuda1_hybrid2%s min %e max %e avg %e\n", L-M, typeid(T).name(), dmin, dmax, davg);
  }
  delete [] dtimes;
#endif
}
#include "template_hybrid2.hpp"
#endif
// ######## ######## ######## ######## ######## ######## ######## ########
#endif

#if 1
// ######## ######## ######## ######## ######## ######## ######## ########
/*
  hybrid3
  複数WARP個別GEMVカーネル
  1PMVを1WARPが担当
  1TBあたりスレッド数は32*mul(mul WARP)
  PMV内の1行を1/div WARPが担当
  <<<napprox/mul+ndense/mul, 32*mul>>>
  1 PMV by 1 WARP
  1 line by 1/div WARP
  バリエーション
  - div：1行を1/divのWARPで計算する、div=1,2,4,8,16,32
  - mul：立ち上げるブロック数スレッド数の係数、1つのmat-mat-vecまたはmat-vecを1WARPで実行、mul=1,2,3,...,
  - a2t：a2を転置版で計算するか否か
  - a2interchange：a2のループを入れ替えるか否か
  - aatomic：approxの計算をatomic優先にするかwarp shuffle併用するか
  - datomic：denseの計算をatomic優先にするかwarp shuffle併用するか
*/
template <class T, int div, int mul, int a2t, int a2i, int aatomic, int datomic>
__global__ void hmvm_cuda_hybrid3
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt,
 int *a1, int *a2, T *rowmat, T *rowmat_t,
 int napprox, int *approx, int ndense, int *dense)
{
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid3 : begin\n");
#endif
  int gid   = blockIdx.x*mul+threadIdx.x/32;
  int bid   = ((threadIdx.x%32)/(32/div));
  int blen  = (32/(32/div));
  int xid   = (threadIdx.x%(32/div));
  int xlen  = (32/div);
  int ndl, ndt, nstrtl, nstrtt, ltmtx;
  int ip, kt, il, it, itt, itl, ill;
  size_t head;
  T tmp = 0.0;
  extern __shared__ __align__(sizeof(T)) unsigned char my_smem[];
  T *tmp2 = reinterpret_cast<T *>(my_smem);
  cg::thread_block_tile<32/div> g = cg::tiled_partition<32/div>(cg::this_thread_block());

#if 0
  if(gid<napprox){
	// approx
	ip = approx[gid];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	ltmtx = _ltmtx[ip];
	kt = _kt[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	head = a1[ip];
	for(il=bid; il<kt; il+=blen){
	  if(xid==0)tmp2[(threadIdx.x/32)*ktmax+il] = 0.0;
	  tmp = 0.0;
	  for(it=xid; it<ndt; it+=xlen){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp += rowmat[head+itl]*d_zu[itt];
	  }
	  //for (int offset = warpSize/(2*div); offset > 0; offset /= 2)tmp += __shfl_down_sync(0xffff, tmp, offset, warpSize);
	  for (int offset = g.size()/2; offset > 0; offset /= 2)tmp += g.shfl_down(tmp, offset);
	  if(xid==0)tmp2[(threadIdx.x/32)*ktmax+il] = tmp;
	}
	head = a2[ip];
	for(il=bid; il<kt; il+=blen){
	  for(it=xid; it<ndl; it+=xlen){
		ill=it+nstrtl-1;
		itl=it+il*ndl;
		myAtomicAdd(&d_zaut[ill], rowmat[head+itl]*tmp2[(threadIdx.x/32)*ktmax+il]);
	  }
	}
  }
#endif

#if 0
  //ip = dense[gid-((napprox+mul-1)/mul)];
  if(gid < ndense){
	ip = dense[gid];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	ltmtx = _ltmtx[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	head = a1[ip];
	for(il=bid; il<ndl; il+=blen){
      tmp = 0.0;
	  ill=il+nstrtl-1;
	  for(it=xid; it<ndt; it+=xlen){
	    itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp += rowmat[head+itl]*d_zu[itt];
      }
	  if(atomic==0){
	    //for (int offset = warpSize/(2*div); offset > 0; offset /= 2)tmp += __shfl_down(tmp, offset);
	    for (int offset = g.size()/2; offset > 0; offset /= 2)tmp += g.shfl_down(tmp, offset);
		if(xid==0){
	      atomicAdd(&d_zaut[ill], tmp);
        }
      }else{
  	    atomicAdd(&d_zaut[ill], tmp);
      }
    }
  }
#endif

#if 1
  if(gid<((napprox+mul-1)/mul)*mul){
#ifndef _SKIP_APPROX
	if(gid<napprox){
	  // approx
	  ip = approx[gid];
	  ndl = _ndl[ip];
	  ndt = _ndt[ip];
	  nstrtl = _nstrtl[ip];
	  nstrtt = _nstrtt[ip];
	  ltmtx = _ltmtx[ip];
	  kt = _kt[ip];
#if _DEBUG_LEVEL >= 3
	  printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	  head = a1[ip];
	  for(il=bid; il<kt; il+=blen){
  	    if(xid==0)tmp2[(threadIdx.x/32)*ktmax+il] = 0.0;
		tmp = 0.0;
		for(it=xid; it<ndt; it+=xlen){
	      itt=it+nstrtt-1;
		  itl=it+il*ndt;
		  tmp += rowmat[head+itl]*d_zu[itt];
        }
		//for (int offset = warpSize/(2*div); offset > 0; offset /= 2)tmp += __shfl_down_sync(0xffff, tmp, offset, warpSize);
		for (int offset = g.size()/2; offset > 0; offset /= 2)tmp += g.shfl_down(tmp, offset);
		if(xid==0)tmp2[(threadIdx.x/32)*ktmax+il] = tmp;
      }
	  head = a2[ip];
	  for(il=bid; il<kt; il+=blen){
	    for(it=xid; it<ndl; it+=xlen){
	      ill=it+nstrtl-1;
		  itl=it+il*ndl;
		  myAtomicAdd(&d_zaut[ill], rowmat[head+itl]*tmp2[(threadIdx.x/32)*ktmax+il]);
        }
      }
    }
#endif // approx
  }else{
#ifndef _SKIP_DENSE
	ip = gid-((napprox+mul-1)/mul)*mul;
    if(ip<ndense){
	  ip = dense[ip];
	  ndl = _ndl[ip];
	  ndt = _ndt[ip];
	  nstrtl = _nstrtl[ip];
	  nstrtt = _nstrtt[ip];
	  ltmtx = _ltmtx[ip];
#if _DEBUG_LEVEL >= 3
	  printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	  head = a1[ip];
	  for(il=bid; il<ndl; il+=blen){
		tmp = 0.0;
		ill=il+nstrtl-1;
		for(it=xid; it<ndt; it+=xlen){
		  itt=it+nstrtt-1;
		  itl=it+il*ndt;
		  tmp += rowmat[head+itl]*d_zu[itt];
		}
		if(datomic==0){
		  //for (int offset = warpSize/(2*div); offset > 0; offset /= 2)tmp += __shfl_down(tmp, offset);
		  for (int offset = g.size()/2; offset > 0; offset /= 2)tmp += g.shfl_down(tmp, offset);
		  if(xid==0){
			atomicAdd(&d_zaut[ill], tmp);
		  }
		}else{
		  atomicAdd(&d_zaut[ill], tmp);
		}
	  }
	}
#endif // dense
  }
#endif

#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid3 : end\n");
#endif
}

#if 1
template <class T, int div>
__global__ void hmvm_cuda_hybrid3
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt,
 int *a1, int *a2, T *rowmat, T *rowmat_t,
 int napprox, int *approx, int ndense, int *dense,
 int mul, int a2t, int a2i, int aatomic, int datomic)
{
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid3 : begin\n");
#endif
  int gid   = blockIdx.x*mul+threadIdx.x/32;
  int bid   = ((threadIdx.x%32)/(32/div));
  int blen  = (32/(32/div));
  int xid   = (threadIdx.x%(32/div));
  int xlen  = (32/div);
  int ndl, ndt, nstrtl, nstrtt, ltmtx;
  int ip, kt, il, it, itt, itl, ill;
  size_t head;
  T tmp = 0.0;
  extern __shared__ __align__(sizeof(T)) unsigned char my_smem[];
  T *tmp2 = reinterpret_cast<T *>(my_smem);
  cg::thread_block_tile<32/div> g = cg::tiled_partition<32/div>(cg::this_thread_block());

#if 0
  if(gid<napprox){
	// approx
	ip = approx[gid];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	ltmtx = _ltmtx[ip];
	kt = _kt[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	head = a1[ip];
	for(il=bid; il<kt; il+=blen){
	  if(xid==0)tmp2[(threadIdx.x/32)*ktmax+il] = 0.0;
	  tmp = 0.0;
	  for(it=xid; it<ndt; it+=xlen){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp += rowmat[head+itl]*d_zu[itt];
	  }
	  //for (int offset = warpSize/(2*div); offset > 0; offset /= 2)tmp += __shfl_down_sync(0xffff, tmp, offset, warpSize);
	  for (int offset = g.size()/2; offset > 0; offset /= 2)tmp += g.shfl_down(tmp, offset);
	  if(xid==0)tmp2[(threadIdx.x/32)*ktmax+il] = tmp;
	}
	head = a2[ip];
	for(il=bid; il<kt; il+=blen){
	  for(it=xid; it<ndl; it+=xlen){
		ill=it+nstrtl-1;
		itl=it+il*ndl;
		myAtomicAdd(&d_zaut[ill], rowmat[head+itl]*tmp2[(threadIdx.x/32)*ktmax+il]);
	  }
	}
  }
#endif

#if 0
  //ip = dense[gid-((napprox+mul-1)/mul)];
  if(gid < ndense){
	ip = dense[gid];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	ltmtx = _ltmtx[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	head = a1[ip];
	for(il=bid; il<ndl; il+=blen){
      tmp = 0.0;
	  ill=il+nstrtl-1;
	  for(it=xid; it<ndt; it+=xlen){
	    itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp += rowmat[head+itl]*d_zu[itt];
      }
	  if(datomic==0){
	    //for (int offset = warpSize/(2*div); offset > 0; offset /= 2)tmp += __shfl_down(tmp, offset);
	    for (int offset = g.size()/2; offset > 0; offset /= 2)tmp += g.shfl_down(tmp, offset);
		if(xid==0){
	      atomicAdd(&d_zaut[ill], tmp);
        }
      }else{
  	    atomicAdd(&d_zaut[ill], tmp);
      }
    }
  }
#endif

#if 1
  if(gid<((napprox+mul-1)/mul)*mul){
#ifndef _SKIP_APPROX
	if(gid<napprox){
	  // approx
	  ip = approx[gid];
	  ndl = _ndl[ip];
	  ndt = _ndt[ip];
	  nstrtl = _nstrtl[ip];
	  nstrtt = _nstrtt[ip];
	  ltmtx = _ltmtx[ip];
	  kt = _kt[ip];
#if _DEBUG_LEVEL >= 3
	  printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	  head = a1[ip];
	  for(il=bid; il<kt; il+=blen){
  	    if(xid==0)tmp2[(threadIdx.x/32)*ktmax+il] = 0.0;
		tmp = 0.0;
		for(it=xid; it<ndt; it+=xlen){
	      itt=it+nstrtt-1;
		  itl=it+il*ndt;
		  tmp += rowmat[head+itl]*d_zu[itt];
        }
		//for (int offset = warpSize/(2*div); offset > 0; offset /= 2)tmp += __shfl_down_sync(0xffff, tmp, offset, warpSize);
		for (int offset = g.size()/2; offset > 0; offset /= 2)tmp += g.shfl_down(tmp, offset);
		if(xid==0)tmp2[(threadIdx.x/32)*ktmax+il] = tmp;
      }
	  head = a2[ip];
	  for(il=bid; il<kt; il+=blen){
	    for(it=xid; it<ndl; it+=xlen){
	      ill=it+nstrtl-1;
		  itl=it+il*ndl;
		  myAtomicAdd(&d_zaut[ill], rowmat[head+itl]*tmp2[(threadIdx.x/32)*ktmax+il]);
        }
      }
    }
#endif // approx
  }else{
#ifndef _SKIP_DENSE
	ip = gid-((napprox+mul-1)/mul)*mul;
    if(ip<ndense){
	  ip = dense[ip];
	  ndl = _ndl[ip];
	  ndt = _ndt[ip];
	  nstrtl = _nstrtl[ip];
	  nstrtt = _nstrtt[ip];
	  ltmtx = _ltmtx[ip];
#if _DEBUG_LEVEL >= 3
	  printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	  head = a1[ip];
	  for(il=bid; il<ndl; il+=blen){
		tmp = 0.0;
		ill=il+nstrtl-1;
		for(it=xid; it<ndt; it+=xlen){
		  itt=it+nstrtt-1;
		  itl=it+il*ndt;
		  tmp += rowmat[head+itl]*d_zu[itt];
		}
		if(datomic==0){
		  //for (int offset = warpSize/(2*div); offset > 0; offset /= 2)tmp += __shfl_down(tmp, offset);
		  for (int offset = g.size()/2; offset > 0; offset /= 2)tmp += g.shfl_down(tmp, offset);
		  if(xid==0){
			atomicAdd(&d_zaut[ill], tmp);
		  }
		}else{
		  atomicAdd(&d_zaut[ill], tmp);
		}
	  }
	}
#endif // dense
  }
#endif

#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid3 : end\n");
#endif
}

template <class T>
void hmvm_cuda_hybrid3_proxy
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *ltmtx, int *ndt, int *ndl, int *nstrtl, int *nstrtt, int *kt,
 int *a1, int *a2, T *rowmat, T *rowmat_t,
 int napprox, int *approx, int ndense, int *dense,
 int blocks, int threads, int shms,
 T *v, T *b, int nd, char *fname, int bench,
 int div, int mul, int a2t, int a2i, int aa, int da)
{
#if 1
  int M=5, L=M+bench;
  FILE *F;
  int i, l, lmax;
  double d1, d2, *dtimes, dmin, dmax, davg;
  hipError_t ret;
  dtimes = new double[L];
  if(bench==0){lmax=1;}else{lmax=L;}
  for(l=0;l<lmax;l++){
	for(i=0;i<nd;i++)v[i] = (T)0.0;
	CHECK_DO(hipMemcpy(d_zaut, v, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy v to d_v");
	CHECK_DO(hipMemcpy(d_zu, b, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy b to d_b");
	hipDeviceSynchronize();
	d1 = omp_get_wtime();
	switch(div){
	case  1: hmvm_cuda_hybrid3<T, 1><<<blocks,threads,shms>>>
		(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt,
		 a1, a2, rowmat, rowmat_t, napprox, approx, ndense, dense, mul, a2t, a2i, aa, da);
	  break;
	case  2: hmvm_cuda_hybrid3<T, 2><<<blocks,threads,shms>>>
		(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt,
		 a1, a2, rowmat, rowmat_t, napprox, approx, ndense, dense, mul, a2t, a2i, aa, da);
	  break;
	case  4: hmvm_cuda_hybrid3<T, 4><<<blocks,threads,shms>>>
		(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt,
		 a1, a2, rowmat, rowmat_t, napprox, approx, ndense, dense, mul, a2t, a2i, aa, da);
	  break;
	case  8: hmvm_cuda_hybrid3<T, 8><<<blocks,threads,shms>>>
		(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt,
		 a1, a2, rowmat, rowmat_t, napprox, approx, ndense, dense, mul, a2t, a2i, aa, da);
	  break;
	case 16: hmvm_cuda_hybrid3<T,16><<<blocks,threads,shms>>>
		(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt,
		 a1, a2, rowmat, rowmat_t, napprox, approx, ndense, dense, mul, a2t, a2i, aa, da);
	  break;
	case 32: hmvm_cuda_hybrid3<T,32><<<blocks,threads,shms>>>
		(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt,
		 a1, a2, rowmat, rowmat_t, napprox, approx, ndense, dense, mul, a2t, a2i, aa, da);
	  break;
	}
	hipDeviceSynchronize();
	d2 = omp_get_wtime();
	dtimes[l] = d2-d1;
  }
  if(bench==0){
	CHECK_DO(hipMemcpy(v, d_zaut, sizeof(T)*nd, hipMemcpyDeviceToHost),"hipMemcpy d_v to v");
	printf("write to %s\n", fname);
	F = fopen(fname, "w");
	for(i=0;i<nd;i++)fprintf(F, "%.3E\n", v[i]);
	fclose(F);
  }else{
	dmin = 9999.99;
	dmax = 0.0;
	davg = 0.0;
	for(i=M;i<L;i++){
	  davg += dtimes[i];
	  if(dmin>dtimes[i])dmin=dtimes[i];
	  if(dmax<dtimes[i])dmax=dtimes[i];
	}
	davg /= (L-M);
	printf("TIME %d hmvm_cuda1_hybrid3%s min %e max %e avg %e\n", L-M, typeid(T).name(), dmin, dmax, davg);
  }
  delete [] dtimes;
#endif
}
#else
template <class T, int div, int mul, int a2t, int a2i, int aa, int da>
void hmvm_cuda_hybrid3_proxy
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *ltmtx, int *ndt, int *ndl, int *nstrtl, int *nstrtt, int *kt,
 int *a1, int *a2, T *rowmat, T *rowmat_t,
 int napprox, int *approx, int ndense, int *dense,
 int blocks, int threads, int shms,
 T *v, T *b, int nd, char *fname, int bench)
{
#if 1
  int M=5, L=M+bench;
  FILE *F;
  int i, l, lmax;
  double d1, d2, *dtimes, dmin, dmax, davg;
  hipError_t ret;
  dtimes = new double[L];
  if(bench==0){lmax=1;}else{lmax=L;}
  for(l=0;l<lmax;l++){
	for(i=0;i<nd;i++)v[i] = (T)0.0;
	CHECK_DO(hipMemcpy(d_zaut, v, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy v to d_v");
	CHECK_DO(hipMemcpy(d_zu, b, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy b to d_b");
	hipDeviceSynchronize();
	d1 = omp_get_wtime();
	hmvm_cuda_hybrid3<T,div,mul,a2t,a2i,aa,da><<<blocks,threads,shms>>>
	  (d_zaut, d_zu, nlf, ktmax, ltmtx,
	   ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, rowmat_t,
	   napprox, approx, ndense, dense);
	hipDeviceSynchronize();
	d2 = omp_get_wtime();
	dtimes[l] = d2-d1;
  }
  if(bench==0){
	CHECK_DO(hipMemcpy(v, d_zaut, sizeof(T)*nd, hipMemcpyDeviceToHost),"hipMemcpy d_v to v");
	printf("write to %s\n", fname);
	F = fopen(fname, "w");
	for(i=0;i<nd;i++)fprintf(F, "%.3E\n", v[i]);
	fclose(F);
  }else{
	dmin = 9999.99;
	dmax = 0.0;
	davg = 0.0;
	for(i=M;i<L;i++){
	  davg += dtimes[i];
	  if(dmin>dtimes[i])dmin=dtimes[i];
	  if(dmax<dtimes[i])dmax=dtimes[i];
	}
	davg /= (L-M);
	printf("TIME %d hmvm_cuda1_hybrid3%s min %e max %e avg %e\n", L-M, typeid(T).name(), dmin, dmax, davg);
  }
  delete [] dtimes;
#endif
}
#include "template_hybrid3.hpp"
#endif
// ######## ######## ######## ######## ######## ######## ######## ########
#endif

#if 1
// ######## ######## ######## ######## ######## ######## ######## ########
/*
  hybrid4
  複数WARP単一GEMV単一行GEMVカーネル
  <<<napprox+ndense, 32*mul>>>
  1 GEMV by mul TB
  1 line by 1/mul WARP
  バリエーション
  - mul：立ち上げるブロック数スレッド数の係数、1つのmat-mat-vecまたはmat-vecを1WARPで実行、mul=1,2,3,...,
  - a2t：a2を転置版で計算するか否か
  - a2interchange：a2のループを入れ替えるか否か
  - aatomic：approxの計算をatomic優先にするかwarp shuffle併用するか
  - datomic：denseの計算をatomic優先にするかwarp shuffle併用するか
*/
template <class T, int atomic>
__global__ void hmvm_cuda_hybrid4
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, T *rowmat, T *rowmat_t,
 int napprox, int *approx, int ndense, int *dense, int mul)
{
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid4 : begin\n");
#endif
  int gid  = blockIdx.x;
  int tid  = threadIdx.x;
  int tlen = blockDim.x;
  int ndl, ndt, nstrtl, nstrtt, ltmtx;
  int ip, kt, il, it, itt, itl, ill;
  size_t head;
  T tmp = 0.0;
  extern __shared__ __align__(sizeof(T)) unsigned char my_smem[];
  T *tmp2 = reinterpret_cast<T *>(my_smem);
  cg::thread_block_tile<32> g = cg::tiled_partition<32>(cg::this_thread_block());

#if 1
  if(gid<ndense){
#ifndef _SKIP_APPROX
	ip = dense[gid];
    ndl = _ndl[ip];
    ndt = _ndt[ip];
    nstrtl = _nstrtl[ip];
    nstrtt = _nstrtt[ip];
    ltmtx = _ltmtx[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif

	head = a1[ip];
	for(il=0; il<ndl; il++){
	  tmp = 0.0;
	  ill=il+nstrtl-1;
	  for(it=tid; it<ndt; it+=tlen){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp += rowmat[head+itl]*d_zu[itt];
	  }
	  //for (int offset = warpSize/2; offset > 0; offset /= 2)tmp += __shfl_down_sync(tmp, offset, warpSize);
	  for (int offset = g.size()/2; offset > 0; offset /= 2)tmp += g.shfl_down(tmp, offset);
	  if(tid%32==0){
		atomicAdd(&d_zaut[ill], tmp);
	  }
	}
#endif
  }
#endif

#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid4 : end\n");
#endif
}

template <class T>
void hmvm_cuda_hybrid4_proxy
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *ltmtx, int *ndt, int *ndl, int *nstrtl, int *nstrtt, int *kt, int *a1, int *a2, T *rowmat, T *rowmat_t,
 int napprox, int *approx, int ndense, int *dense,
 int blocks, int threads, int shms, int mul, int atomic, T *v, T *b, int nd, char *fname, int bench)
{
#if 1
  const int L=10, M=5;
  FILE *F;
  int i, l, lmax;
  double d1, d2, dtimes[L], dmin, dmax, davg;
  hipError_t ret;
  if(bench==0){lmax=1;}else{lmax=L;}
  for(l=0;l<lmax;l++){
	for(i=0;i<nd;i++)v[i] = (T)0.0;
	CHECK_DO(hipMemcpy(d_zaut, v, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy v to d_v");
	CHECK_DO(hipMemcpy(d_zu, b, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy b to d_b");
	hipDeviceSynchronize();
	d1 = omp_get_wtime();
	switch(atomic){
	case 0:
	  hmvm_cuda_hybrid4<T,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, rowmat_t, napprox, approx, ndense, dense, mul);
	  break;
	case 1:
	  hmvm_cuda_hybrid4<T,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, rowmat_t, napprox, approx, ndense, dense, mul);
	  break;
	}
	hipDeviceSynchronize();
	d2 = omp_get_wtime();
	dtimes[l] = d2-d1;
  }
  if(bench==0){
	CHECK_DO(hipMemcpy(v, d_zaut, sizeof(T)*nd, hipMemcpyDeviceToHost),"hipMemcpy d_v to v");
	printf("write to %s\n", fname);
	F = fopen(fname, "w");
	for(i=0;i<nd;i++)fprintf(F, "%.3E\n", v[i]);
	fclose(F);
  }else{
	dmin = 9999.99;
	dmax = 0.0;
	davg = 0.0;
	for(i=M;i<L;i++){
	  davg += dtimes[i];
	  if(dmin>dtimes[i])dmin=dtimes[i];
	  if(dmax<dtimes[i])dmax=dtimes[i];
	}
	davg /= (L-M);
	printf("TIME %d hmvm_cuda1_hybrid4%s min %e max %e avg %e\n", L-M, typeid(T).name(), dmin, dmax, davg);
  }
#endif
}

template
void hmvm_cuda_hybrid4_proxy<float>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat, float *rowmat_t,
 int napprox, int *approx, int ndense, int *dense,
 int blocks, int threads, int shms, int mul, int atomic, float *v, float *b, int nd, char *fname, int bench);
template
void hmvm_cuda_hybrid4_proxy<double>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat, double *rowmat_t,
 int napprox, int *approx, int ndense, int *dense,
 int blocks, int threads, int shms, int mul, int atomic, double *v, double *b, int nd, char *fname, int bench);
// ######## ######## ######## ######## ######## ######## ######## ########
#endif
