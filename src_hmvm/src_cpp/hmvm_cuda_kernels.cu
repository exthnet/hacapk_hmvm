#include "hip/hip_runtime.h"
// -*- c++ -*-
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <omp.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_cooperative_groups.h>

#include "hacapk.h"

namespace cg = cooperative_groups;

#if __CUDA_ARCH__ < 600
__device__ double myAtomicAdd(double* address, double val)
{
  unsigned long long int* address_as_ull =
	(unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;

  do {
	assumed = old;
	old = atomicCAS(address_as_ull, assumed,
					__double_as_longlong(val +
										 __longlong_as_double(assumed)));

	// Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
  } while (assumed != old);

  return __longlong_as_double(old);
}
__device__ float myAtomicAdd(float* address, float val)
{
  atomicAdd(address, val);
}
#else
#define myAtomicAdd atomicAdd
#endif

// ######## ######## ######## ######## ######## ######## ######## ########
// 0: 完全逐次
#if 1
template <class T>
__global__ void hmvm_cuda_seq
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, T *rowmat,
 int napprox, int *approx, int ndense, int *dense)
{
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_seq : begin\n");
#endif
  int ndl, ndt, nstrtl, nstrtt, ltmtx;
  int ip, kt, il, it, itt, itl, ill;
  size_t head;
  //extern __shared__ T tmp2[];
  extern __shared__ __align__(sizeof(T)) unsigned char my_smem[];
  T *tmp2 = reinterpret_cast<T *>(my_smem);
  int i;

  // approx
  for(i=0; i<napprox; i++){
#if 1
	ip = approx[i];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	ltmtx = _ltmtx[ip];
	kt = _kt[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	head = a1[ip];
	for(il=0; il<kt; il++){
	  tmp2[il] = 0.0;
	  for(it=0; it<ndt; it++){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp2[il] += rowmat[head+itl]*d_zu[itt];
	  }
	}
	head = a2[ip];
	for(il=0; il<kt; il++){
	  for(it=0; it<ndl; it++){
		ill=it+nstrtl-1;
		itl=it+il*ndl;
		myAtomicAdd(&d_zaut[ill], rowmat[head+itl]*tmp2[il]);
	  }
	}
#endif
  }
  // dense
  for(i=0; i<ndense; i++){
#if 1
	ip = dense[i];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	ltmtx = _ltmtx[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	head = a1[ip];
	for(il=0; il<ndl; il++){
	  T tmp = (T)0.0;
	  ill=il+nstrtl-1;
	  for(it=0; it<ndt; it++){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp += rowmat[head+itl]*d_zu[itt];
	  }
	  myAtomicAdd(&d_zaut[ill], tmp);
	}
#endif
  }
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_seq : end\n");
#endif
}
#endif
// ######## ######## ######## ######## ######## ######## ######## ########

// block並列化カーネル
template <class T>
__global__ void hmvm_cuda_block
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, T *rowmat,
 int napprox, int *approx, int ndense, int *dense)
{
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cudaD_block : begin\n");
#endif
  int ndl, ndt, nstrtl, nstrtt, ltmtx;
  int ip, kt, il, it, itt, itl, ill;
  size_t head;
  extern __shared__ __align__(sizeof(T)) unsigned char my_smem[];
  T *tmp2 = reinterpret_cast<T *>(my_smem);
  int i;

  if(blockIdx.x<napprox){
#if 1
	// approx
	//for(i=0; i<napprox; i++){
	ip = approx[blockIdx.x];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	ltmtx = _ltmtx[ip];
	kt = _kt[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	head = a1[ip];
	for(il=0; il<kt; il++){
	  tmp2[il] = 0.0;
	  for(it=0; it<ndt; it++){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp2[il] += rowmat[head+itl]*d_zu[itt];
	  }
	}
	head = a2[ip];
	for(il=0; il<kt; il++){
	  for(it=0; it<ndl; it++){
		ill=it+nstrtl-1;
		itl=it+il*ndl;
		myAtomicAdd(&d_zaut[ill], rowmat[head+itl]*tmp2[il]);
	  }
	}
#endif
  }else{
#if 1
	// dense
	//for(i=0; i<ndense; i++){
	ip = dense[blockIdx.x - napprox];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	ltmtx = _ltmtx[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	head = a1[ip];
	for(il=0; il<ndl; il++){
	  T tmp = 0.0;
	  ill=il+nstrtl-1;
	  for(it=0; it<ndt; it++){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp += rowmat[head+itl]*d_zu[itt];
	  }
	  myAtomicAdd(&d_zaut[ill], tmp);
	}
#endif
  }
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cudaD_block : end\n");
#endif
}

// ######## ######## ######## ######## ######## ######## ######## ########

#if 1
/*
  <<<ndense,32>>>
  1 GEMV by 1 TB
  1 line by 1/div WARP
*/
// nlf block, 32 thread, all atomic版とwarp shuffle版
template <class T, int div, int atomic>
__global__ void hmvm_cuda_hybrid1
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, T *rowmat,
 int napprox, int *approx, int ndense, int *dense)
{
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid1 : begin\n");
#endif
  int gid   = blockIdx.x;
  int tid   = threadIdx.x;
  int bid   = threadIdx.x/(32/div);
  int blen  = div;
  int xid   = threadIdx.x%(32/div);
  int xlen  = (32/div);
  int ndl, ndt, nstrtl, nstrtt, ltmtx;
  int ip, kt, il, it, itt, itl, ill;
  size_t head;
  T tmp = 0.0;
  extern __shared__ __align__(sizeof(T)) unsigned char my_smem[];
  T *tmp2 = reinterpret_cast<T *>(my_smem);
  cg::thread_block_tile<32/div> g = cg::tiled_partition<32/div>(cg::this_thread_block());

  if(gid<napprox){
#if 1
	// approx
	ip = approx[gid];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	ltmtx = _ltmtx[ip];
	kt = _kt[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	head = a1[ip];
	for(il=bid; il<kt; il+=blen){
	  if(xid==0)tmp2[il] = 0.0;
	  tmp = 0.0;
	  for(it=xid; it<ndt; it+=xlen){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp += rowmat[head+itl]*d_zu[itt];
	  }
	  //for (int offset = warpSize/(2*div); offset > 0; offset /= 2)tmp += __shfl_down_sync(0xffff, tmp, offset, warpSize);
	  for (int offset = g.size()/2; offset > 0; offset /= 2)tmp += g.shfl_down(tmp, offset);
	  if(xid==0)tmp2[il] = tmp;
	}
	head = a2[ip];
	for(il=bid; il<kt; il+=blen){
	  for(it=xid; it<ndl; it+=xlen){
		ill=it+nstrtl-1;
		itl=it+il*ndl;
		myAtomicAdd(&d_zaut[ill], rowmat[head+itl]*tmp2[il]);
	  }
	}
#endif // approx
  }else{
#if 1
	// dense
	ip = dense[gid-napprox];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	ltmtx = _ltmtx[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	head = a1[ip];
	for(il=bid; il<ndl; il+=blen){
	  tmp = 0.0;
	  ill=il+nstrtl-1;
	  for(it=xid; it<ndt; it+=xlen){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp += rowmat[head+itl]*d_zu[itt];
	  }
	  if(atomic==0){
		//for (int offset = warpSize/(2*div); offset > 0; offset /= 2)tmp += __shfl_down_sync(0xffff, tmp, offset, warpSize);
		for (int offset = g.size()/2; offset > 0; offset /= 2)tmp += g.shfl_down(tmp, offset);
		if(xid==0){
		  myAtomicAdd(&d_zaut[ill], tmp);
		}
	  }else{
		myAtomicAdd(&d_zaut[ill], tmp);
	  }
#endif // dense
	}
  }

#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid1 : end\n");
#endif
}
#endif


// ######## ######## ######## ######## ######## ######## ######## ########
// template関数の実体化のための宣言
// ######## ######## ######## ######## ######## ######## ######## ########
// float
#if 1
template __global__ void hmvm_cuda_seq<float>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense);

template __global__ void hmvm_cuda_block<float>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense);

template __global__ void hmvm_cuda_hybrid1<float,1,0>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<float,2,0>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<float,4,0>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<float,8,0>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<float,16,0>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<float,32,0>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense);

template __global__ void hmvm_cuda_hybrid1<float,1,1>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<float,2,1>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<float,4,1>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<float,8,1>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<float,16,1>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<float,32,1>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense);
#endif

// double
#if 1
template __global__ void hmvm_cuda_seq<double>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_block<double>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense);

template __global__ void hmvm_cuda_hybrid1<double,1,0>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<double,2,0>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<double,4,0>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<double,8,0>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<double,16,0>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<double,32,0>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense);

template __global__ void hmvm_cuda_hybrid1<double,1,1>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<double,2,1>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<double,4,1>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<double,8,1>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<double,16,1>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<double,32,1>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense);
#endif
