#include "hip/hip_runtime.h"
// -*- c++ -*-
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <typeinfo>
#include <omp.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_cooperative_groups.h>

#include "hacapk.h"

// できれば重複計算していないかチェックもしたい

#define CHECK_DO(act,msg) {ret=act; if(ret!=hipSuccess){printf("%s failed\n",msg);exit(-1);};}

namespace cg = cooperative_groups;

#if __CUDA_ARCH__ < 600
__device__ double myAtomicAdd(double* address, double val)
{
  unsigned long long int* address_as_ull =
	(unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;

  do {
	assumed = old;
	old = atomicCAS(address_as_ull, assumed,
					__double_as_longlong(val +
										 __longlong_as_double(assumed)));

	// Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
  } while (assumed != old);

  return __longlong_as_double(old);
}
__device__ float myAtomicAdd(float* address, float val)
{
  atomicAdd(address, val);
}
#else
#define myAtomicAdd atomicAdd
#endif

#if 1
// ######## ######## ######## ######## ######## ######## ######## ########
/*
  完全逐次
  バリエーション：なし
*/
template <class T>
__global__ void hmvm_cuda_seq
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, T *rowmat,
 int napprox, int *approx, int ndense, int *dense)
{
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_seq : begin\n");
#endif
  int ndl, ndt, nstrtl, nstrtt, ltmtx;
  int ip, kt, il, it, itt, itl, ill;
  size_t head;
  //extern __shared__ T tmp2[];
  extern __shared__ __align__(sizeof(T)) unsigned char my_smem[];
  T *tmp2 = reinterpret_cast<T *>(my_smem);
  int i;

  // approx
  for(i=0; i<napprox; i++){
#ifndef _SKIP_APPROX
	ip = approx[i];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	ltmtx = _ltmtx[ip];
	kt = _kt[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	head = a1[ip];
	for(il=0; il<kt; il++){
	  tmp2[il] = 0.0;
	  for(it=0; it<ndt; it++){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp2[il] += rowmat[head+itl]*d_zu[itt];
	  }
	}
	head = a2[ip];
	for(il=0; il<kt; il++){
	  for(it=0; it<ndl; it++){
		ill=it+nstrtl-1;
		itl=it+il*ndl;
		myAtomicAdd(&d_zaut[ill], rowmat[head+itl]*tmp2[il]);
	  }
	}
#endif
  }

  // dense
  for(i=0; i<ndense; i++){
#ifndef _SKIP_DENSE
	ip = dense[i];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	ltmtx = _ltmtx[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	head = a1[ip];
	for(il=0; il<ndl; il++){
	  T tmp = (T)0.0;
	  ill=il+nstrtl-1;
	  for(it=0; it<ndt; it++){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp += rowmat[head+itl]*d_zu[itt];
	  }
	  myAtomicAdd(&d_zaut[ill], tmp);
	}
#endif
  }
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_seq : end\n");
#endif
}

template __global__ void hmvm_cuda_seq<float>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_seq<double>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense);
#endif
// ######## ######## ######## ######## ######## ######## ######## ########

// ######## ######## ######## ######## ######## ######## ######## ########
/*
  block並列化
  ThreadBlockごとに1つの部分行列積(mat-mat-vecまたはmat-vec)を行う
  ThreadBlock内部は逐次
  バリエーション：なし
 */
template <class T>
__global__ void hmvm_cuda_block
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, T *rowmat,
 int napprox, int *approx, int ndense, int *dense)
{
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cudaD_block : begin\n");
#endif
  int ndl, ndt, nstrtl, nstrtt, ltmtx;
  int ip, kt, il, it, itt, itl, ill;
  size_t head;
  extern __shared__ __align__(sizeof(T)) unsigned char my_smem[];
  T *tmp2 = reinterpret_cast<T *>(my_smem);
  int i;

  if(blockIdx.x<napprox){
#ifndef _SKIP_APPROX
	// approx
	//for(i=0; i<napprox; i++){
	ip = approx[blockIdx.x];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	ltmtx = _ltmtx[ip];
	kt = _kt[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	head = a1[ip];
	for(il=0; il<kt; il++){
	  tmp2[il] = 0.0;
	  for(it=0; it<ndt; it++){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp2[il] += rowmat[head+itl]*d_zu[itt];
	  }
	}
	head = a2[ip];
	for(il=0; il<kt; il++){
	  for(it=0; it<ndl; it++){
		ill=it+nstrtl-1;
		itl=it+il*ndl;
		myAtomicAdd(&d_zaut[ill], rowmat[head+itl]*tmp2[il]);
	  }
	}
#endif
  }else{
#ifndef _SKIP_DENSE
	// dense
	//for(i=0; i<ndense; i++){
	ip = dense[blockIdx.x - napprox];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	ltmtx = _ltmtx[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	head = a1[ip];
	for(il=0; il<ndl; il++){
	  T tmp = 0.0;
	  ill=il+nstrtl-1;
	  for(it=0; it<ndt; it++){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp += rowmat[head+itl]*d_zu[itt];
	  }
	  myAtomicAdd(&d_zaut[ill], tmp);
	}
#endif
  }
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cudaD_block : end\n");
#endif
}

template __global__ void hmvm_cuda_block<float>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_block<double>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense);
// ######## ######## ######## ######## ######## ######## ######## ########

// ######## ######## ######## ######## ######## ######## ######## ########
#if 1
/*
  hybrid1
  TBあたりスレッド数は32に固定
  1TBが1つのPMVを担当
  PMV内の1行を1/div WARPが担当
  <<<napprox+ndense,32>>>
  1 GEMV by 1 TB(=1WARP)
  1 line by 1/div WARP
  バリエーション
  - div：1行を1/divのWARPで計算する、div=1,2,4,8,16,32
  - a2t：a2を転置版で計算するか否か
  - a2interchange：a2のループを入れ替えるか否か
  - aatomic：approxの計算をatomic優先にするかwarp shuffle併用するか
  - datomic：denseの計算をatomic優先にするかwarp shuffle併用するか
*/
template <class T, int div, int atomic>
__global__ void hmvm_cuda_hybrid1
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, T *rowmat,
 int napprox, int *approx, int ndense, int *dense)
{
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid1 : begin\n");
#endif
  int gid   = blockIdx.x;
  //int tid   = threadIdx.x;
  int bid   = threadIdx.x/(32/div);
  int blen  = div;
  int xid   = threadIdx.x%(32/div);
  int xlen  = (32/div);
  int ndl, ndt, nstrtl, nstrtt, ltmtx;
  int ip, kt, il, it, itt, itl, ill;
  size_t head;
  T tmp = 0.0;
  extern __shared__ __align__(sizeof(T)) unsigned char my_smem[];
  T *tmp2 = reinterpret_cast<T *>(my_smem);
  cg::thread_block_tile<32/div> g = cg::tiled_partition<32/div>(cg::this_thread_block());

  if(gid<napprox){
#ifndef _SKIP_APPROX
	// approx
	ip = approx[gid];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	ltmtx = _ltmtx[ip];
	kt = _kt[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	head = a1[ip];
	for(il=bid; il<kt; il+=blen){
	  if(xid==0)tmp2[il] = 0.0;
	  tmp = 0.0;
	  for(it=xid; it<ndt; it+=xlen){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp += rowmat[head+itl]*d_zu[itt];
	  }
	  //for (int offset = warpSize/(2*div); offset > 0; offset /= 2)tmp += __shfl_down_sync(0xffff, tmp, offset, warpSize);
	  for (int offset = g.size()/2; offset > 0; offset /= 2)tmp += g.shfl_down(tmp, offset);
	  if(xid==0)tmp2[il] = tmp;
	}
	head = a2[ip];
	for(il=bid; il<kt; il+=blen){
	  for(it=xid; it<ndl; it+=xlen){
		ill=it+nstrtl-1;
		itl=it+il*ndl;
		myAtomicAdd(&d_zaut[ill], rowmat[head+itl]*tmp2[il]);
	  }
	}
#endif // approx
  }else{
#ifndef _SKIP_DENSE
	// dense
	ip = dense[gid-napprox];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	ltmtx = _ltmtx[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	head = a1[ip];
	for(il=bid; il<ndl; il+=blen){
	  tmp = 0.0;
	  ill=il+nstrtl-1;
	  for(it=xid; it<ndt; it+=xlen){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp += rowmat[head+itl]*d_zu[itt];
	  }
	  if(atomic==0){
		//for (int offset = warpSize/(2*div); offset > 0; offset /= 2)tmp += __shfl_down_sync(0xffff, tmp, offset, warpSize);
		for (int offset = g.size()/2; offset > 0; offset /= 2)tmp += g.shfl_down(tmp, offset);
		if(xid==0){
		  myAtomicAdd(&d_zaut[ill], tmp);
		}
	  }else{
		myAtomicAdd(&d_zaut[ill], tmp);
	  }
	}
#endif // dense
  }

#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid1 : end\n");
#endif
}

template <class T>
void hmvm_cuda_hybrid1_proxy
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *ltmtx, int *ndt, int *ndl, int *nstrtl, int *nstrtt, int *kt, int *a1, int *a2, T *rowmat,
 int napprox, int *approx, int ndense, int *dense,
 int blocks, int threads, int shms, int div, int atomic, T *v, T *b, int nd, char *fname, int bench)
{
#if 1
  const int L=10, M=5;
  FILE *F;
  int i, l, lmax;
  double d1, d2, dtimes[L], dmin, dmax, davg;
  hipError_t ret;
  if(bench==0){lmax=1;}else{lmax=L;}
  for(l=0;l<lmax;l++){
	for(i=0;i<nd;i++)v[i] = (T)0.0;
	CHECK_DO(hipMemcpy(d_zaut, v, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy v to d_v");
	CHECK_DO(hipMemcpy(d_zu, b, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy b to d_b");
	hipDeviceSynchronize();
	d1 = omp_get_wtime();
	switch(atomic){
	case 0:
	  switch(div){
	  case  1: hmvm_cuda_hybrid1<T, 1,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense); break;
	  case  2: hmvm_cuda_hybrid1<T, 2,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense); break;
	  case  4: hmvm_cuda_hybrid1<T, 4,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense); break;
	  case  8: hmvm_cuda_hybrid1<T, 8,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense); break;
	  case 16: hmvm_cuda_hybrid1<T,16,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense); break;
	  case 32: hmvm_cuda_hybrid1<T,32,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense); break;
	  }
	  break;
	case 1:
	  switch(div){
	  case  1: hmvm_cuda_hybrid1<T, 1,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense); break;
	  case  2: hmvm_cuda_hybrid1<T, 2,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense); break;
	  case  4: hmvm_cuda_hybrid1<T, 4,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense); break;
	  case  8: hmvm_cuda_hybrid1<T, 8,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense); break;
	  case 16: hmvm_cuda_hybrid1<T,16,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense); break;
	  case 32: hmvm_cuda_hybrid1<T,32,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense); break;
	  }
	  break;
	}
	hipDeviceSynchronize();
	d2 = omp_get_wtime();
	dtimes[l] = d2-d1;
  }
  if(bench==0){
	CHECK_DO(hipMemcpy(v, d_zaut, sizeof(T)*nd, hipMemcpyDeviceToHost),"hipMemcpy d_v to v");
	printf("write to %s\n", fname);
	F = fopen(fname, "w");
	for(i=0;i<nd;i++)fprintf(F, "%.3E\n", v[i]);
	fclose(F);
  }else{
	dmin = 9999.99;
	dmax = 0.0;
	davg = 0.0;
	for(i=M;i<L;i++){
	  davg += dtimes[i];
	  if(dmin>dtimes[i])dmin=dtimes[i];
	  if(dmax<dtimes[i])dmax=dtimes[i];
	}
	davg /= (L-M);
	printf("TIME %d hmvm_cuda1_hybrid1b%s min %e max %e avg %e\n", L-M, typeid(T).name(), dmin, dmax, davg);
  }
#endif
}

template
void hmvm_cuda_hybrid1_proxy<float>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense,
 int blocks, int threads, int shms, int div, int atomic, float *v, float *b, int nd, char *fname, int bench);
template
void hmvm_cuda_hybrid1_proxy<double>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense,
 int blocks, int threads, int shms, int div, int atomic, double *v, double *b, int nd, char *fname, int bench);
// ######## ######## ######## ######## ######## ######## ######## ########
#endif

/*
template __global__ void hmvm_cuda_hybrid2<float,DIV,MUL,ATOMIC>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
int napprox, int *approx, int ndense, int *dense);

for a in 0 1
do
for m in `seq 1 16`
do
for d in 1 2 4 8 16 32
do
echo "template __global__ void hmvm_cuda_hybrid2<float,${d},${m},${a}>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
int napprox, int *approx, int ndense, int *dense);"
done
done
done
*/

/*
 template __global__ void hmvm_cuda_hybrid2<double,DIV,MUL,ATOMIC>
(T *d_zaut, T *d_zu, int nlf, int ktmax,
int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, T *rowmat,
int napprox, int *approx, int ndense, int *dense);

for a in 0 1
do
for m in `seq 1 16`
do
for d in 1 2 4 8 16 32
do
echo "template __global__ void hmvm_cuda_hybrid2<double,${d},${m},${a}>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
int napprox, int *approx, int ndense, int *dense);"
done
done
done
*/

#if 1
// ######## ######## ######## ######## ######## ######## ######## ########
/*
  hybrid2
  複数WARP単一GEMV個別行カーネル
  1PMVを担当するのは1TBのまま
  1TBあたりスレッド数を32*mulに増やす
  PMV内の1行を1/div WARPが担当
  <<<napprox+ndense,32*mul>>>
  1 PMV by 1 TB
  1 line by 1/div WARP
  バリエーション
  - div：1行を1/divのWARPで計算する、div=1,2,4,8,16,32
  - mul：立ち上げるスレッド数(mul*32)、1つのmat-mat-vecまたはmat-vecをmul TBで実行、mul=1,2,3,...,
  - a2t：a2を転置版で計算するか否か
  - a2interchange：a2のループを入れ替えるか否か
  - aatomic：approxの計算をatomic優先にするかwarp shuffle併用するか
  - datomic：denseの計算をatomic優先にするかwarp shuffle併用するか
*/
template <class T, int div, int atomic>//int div, int mul, int atomic>
__global__ void hmvm_cuda_hybrid2
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, T *rowmat,
 int napprox, int *approx, int ndense, int *dense, int mul)
{
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid2 : begin\n");
#endif
  int gid   = blockIdx.x;
  //int tid   = threadIdx.x;
  int bid   = threadIdx.x/(32/div);
  int blen  = mul*div;
  int xid   = threadIdx.x%(32/div);
  int xlen  = (32/div);
  int ndl, ndt, nstrtl, nstrtt, ltmtx;
  int ip, kt, il, it, itt, itl, ill;
  size_t head;
  T tmp = 0.0;
  extern __shared__ __align__(sizeof(T)) unsigned char my_smem[];
  T *tmp2 = reinterpret_cast<T *>(my_smem);
  cg::thread_block_tile<32/div> g = cg::tiled_partition<32/div>(cg::this_thread_block());

  if(gid<napprox){
#ifndef _SKIP_APPROX
	// approx
	ip = approx[gid];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	ltmtx = _ltmtx[ip];
	kt = _kt[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	head = a1[ip];
	for(il=bid; il<kt; il+=blen){
	  if(xid==0)tmp2[il] = 0.0;
	  tmp = 0.0;
	  for(it=xid; it<ndt; it+=xlen){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp += rowmat[head+itl]*d_zu[itt];
	  }
	  //for (int offset = warpSize/(2*div); offset > 0; offset /= 2)tmp += __shfl_down_sync(0xffff, tmp, offset, warpSize);
	  for (int offset = g.size()/2; offset > 0; offset /= 2)tmp += g.shfl_down(tmp, offset);
	  if(xid==0)tmp2[il] = tmp;
	}
	head = a2[ip];
	for(il=bid; il<kt; il+=blen){
	  for(it=xid; it<ndl; it+=xlen){
		ill=it+nstrtl-1;
		itl=it+il*ndl;
		myAtomicAdd(&d_zaut[ill], rowmat[head+itl]*tmp2[il]);
	  }
	}
#endif // approx
  }else if(gid-napprox<ndense){
#ifndef _SKIP_DENSE
	// dense
	ip = dense[gid-napprox];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	ltmtx = _ltmtx[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	head = a1[ip];
	for(il=bid; il<ndl; il+=blen){
	  tmp = 0.0;
	  ill=il+nstrtl-1;
	  for(it=xid; it<ndt; it+=xlen){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp += rowmat[head+itl]*d_zu[itt];
	  }
	  if(atomic==0){
		//for (int offset = warpSize/(2*div); offset > 0; offset /= 2)tmp += __shfl_down_sync(tmp, offset, warpSize);
		for (int offset = g.size()/2; offset > 0; offset /= 2)tmp += g.shfl_down(tmp, offset);
		if(xid==0){
		  myAtomicAdd(&d_zaut[ill], tmp);
		}
	  }else{
		myAtomicAdd(&d_zaut[ill], tmp);
	  }
	}
#endif // dense
  }

#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid2 : end\n");
#endif
}

template <class T>
void hmvm_cuda_hybrid2_proxy
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *ltmtx, int *ndt, int *ndl, int *nstrtl, int *nstrtt, int *kt, int *a1, int *a2, T *rowmat,
 int napprox, int *approx, int ndense, int *dense,
 int blocks, int threads, int shms, int div, int mul, int atomic, T *v, T *b, int nd, char *fname, int bench)
{
#if 1
  const int L=10, M=5;
  FILE *F;
  int i, l, lmax;
  double d1, d2, dtimes[L], dmin, dmax, davg;
  hipError_t ret;
  if(bench==0){lmax=L;}else{lmax=1;}
  for(l=0;l<L;l++){
	for(i=0;i<nd;i++)v[i] = (T)0.0;
	CHECK_DO(hipMemcpy(d_zaut, v, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy v to d_v");
	CHECK_DO(hipMemcpy(d_zu, b, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy b to d_b");
	hipDeviceSynchronize();
	switch(atomic){
	case 0:
	  switch(div){
	  case  1: hmvm_cuda_hybrid2<T, 1,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  case  2: hmvm_cuda_hybrid2<T, 2,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  case  4: hmvm_cuda_hybrid2<T, 4,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  case  8: hmvm_cuda_hybrid2<T, 8,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  case 16: hmvm_cuda_hybrid2<T,16,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  case 32: hmvm_cuda_hybrid2<T,32,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  }
	  break;
	case 1:
	  switch(div){
	  case  1: hmvm_cuda_hybrid2<T, 1,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  case  2: hmvm_cuda_hybrid2<T, 2,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  case  4: hmvm_cuda_hybrid2<T, 4,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  case  8: hmvm_cuda_hybrid2<T, 8,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  case 16: hmvm_cuda_hybrid2<T,16,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  case 32: hmvm_cuda_hybrid2<T,32,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  }
	  break;
	}
	hipDeviceSynchronize();
	d2 = omp_get_wtime();
	dtimes[l] = d2-d1;
  }
  if(bench==0){
	CHECK_DO(hipMemcpy(v, d_zaut, sizeof(T)*nd, hipMemcpyDeviceToHost),"hipMemcpy d_v to v");
	printf("write to %s\n", fname);
	F = fopen(fname, "w");
	for(i=0;i<nd;i++)fprintf(F, "%.3E\n", v[i]);
	fclose(F);
  }else{
	dmin = 9999.99;
	dmax = 0.0;
	davg = 0.0;
	for(i=M;i<L;i++){
	  davg += dtimes[i];
	  if(dmin>dtimes[i])dmin=dtimes[i];
	  if(dmax<dtimes[i])dmax=dtimes[i];
	}
	davg /= (L-M);
	printf("TIME %d hmvm_cuda1_hybrid2b%s min %e max %e avg %e\n", L-M, typeid(T).name(), dmin, dmax, davg);
  }
#endif
}

template
void hmvm_cuda_hybrid2_proxy<float>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense,
 int blocks, int threads, int shms, int div, int mul, int atomic, float *v, float *b, int nd, char *fname, int bench);
template
void hmvm_cuda_hybrid2_proxy<double>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense,
 int blocks, int threads, int shms, int div, int mul, int atomic, double *v, double *b, int nd, char *fname, int bench);
// ######## ######## ######## ######## ######## ######## ######## ########
#endif

#if 1
// ######## ######## ######## ######## ######## ######## ######## ########
/*
  hybrid3
  複数WARP個別GEMVカーネル
  1PMVを1WARPが担当
  1TBあたりスレッド数は32*mul(mul WARP)
  PMV内の1行を1/div WARPが担当
  <<<napprox/mul+ndense/mul, 32*mul>>>
  1 PMV by 1 WARP
  1 line by 1/div WARP
  バリエーション
  - div：1行を1/divのWARPで計算する、div=1,2,4,8,16,32
  - mul：立ち上げるブロック数スレッド数の係数、1つのmat-mat-vecまたはmat-vecを1WARPで実行、mul=1,2,3,...,
  - a2t：a2を転置版で計算するか否か
  - a2interchange：a2のループを入れ替えるか否か
  - aatomic：approxの計算をatomic優先にするかwarp shuffle併用するか
  - datomic：denseの計算をatomic優先にするかwarp shuffle併用するか
*/
template <class T, int div, int atomic>
__global__ void hmvm_cuda_hybrid3
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, T *rowmat,
 int napprox, int *approx, int ndense, int *dense, int mul)
{
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid3 : begin\n");
#endif
  int gid   = blockIdx.x*mul+threadIdx.x/32;
  //int tid   = threadIdx.x;
  int bid   = ((threadIdx.x/mul)/32)/div;
  int blen  = div;
  int xid   = threadIdx.x%(32/div);
  int xlen  = (32/div);
  int ndl, ndt, nstrtl, nstrtt, ltmtx;
  int ip, kt, il, it, itt, itl, ill;
  size_t head;
  T tmp = 0.0;
  extern __shared__ __align__(sizeof(T)) unsigned char my_smem[];
  T *tmp2 = reinterpret_cast<T *>(my_smem);
  cg::thread_block_tile<32/div> g = cg::tiled_partition<32/div>(cg::this_thread_block());

#if 1
  if(gid<((napprox+mul-1)/mul)){
#ifndef _SKIP_APPROX
	if(gid<napprox){
	  // approx
	  ip = approx[gid];
	  ndl = _ndl[ip];
	  ndt = _ndt[ip];
	  nstrtl = _nstrtl[ip];
	  nstrtt = _nstrtt[ip];
	  ltmtx = _ltmtx[ip];
	  kt = _kt[ip];
#if _DEBUG_LEVEL >= 3
	  printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	  head = a1[ip];
	  for(il=bid; il<kt; il+=blen){
  	    if(xid==0)tmp2[il] = 0.0;
		tmp = 0.0;
		for(it=xid; it<ndt; it+=xlen){
	      itt=it+nstrtt-1;
		  itl=it+il*ndt;
		  tmp += rowmat[head+itl]*d_zu[itt];
        }
		//for (int offset = warpSize/(2*div); offset > 0; offset /= 2)tmp += __shfl_down_sync(0xffff, tmp, offset, warpSize);
		for (int offset = g.size()/2; offset > 0; offset /= 2)tmp += g.shfl_down(tmp, offset);
		if(xid==0)tmp2[il] = tmp;
      }
	  head = a2[ip];
	  for(il=bid; il<kt; il+=blen){
	    for(it=xid; it<ndl; it+=xlen){
	      ill=it+nstrtl-1;
		  itl=it+il*ndl;
		  myAtomicAdd(&d_zaut[ill], rowmat[head+itl]*tmp2[il]);
        }
      }
    }
#endif // approx
  }else{
#ifndef _SKIP_DENSE
	if(gid-((napprox+mul-1)/mul)<ndense){
	ip = dense[gid-((napprox+mul-1)/mul)];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	ltmtx = _ltmtx[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	head = a1[ip];
	for(il=bid; il<ndl; il+=blen){
      tmp = 0.0;
	  ill=il+nstrtl-1;
	  for(it=xid; it<ndt; it+=xlen){
	    itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp += rowmat[head+itl]*d_zu[itt];
      }
	  if(atomic==0){
	    //for (int offset = warpSize/(2*div); offset > 0; offset /= 2)tmp += __shfl_down(tmp, offset);
	    for (int offset = g.size()/2; offset > 0; offset /= 2)tmp += g.shfl_down(tmp, offset);
		if(xid==0){
	      atomicAdd(&d_zaut[ill], tmp);
        }
      }else{
  	    atomicAdd(&d_zaut[ill], tmp);
      }
    }
    }
#endif // dense
  }
#endif
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid3 : end\n");
#endif
}

template <class T>
void hmvm_cuda_hybrid3_proxy
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *ltmtx, int *ndt, int *ndl, int *nstrtl, int *nstrtt, int *kt, int *a1, int *a2, T *rowmat,
 int napprox, int *approx, int ndense, int *dense,
 int blocks, int threads, int shms, int div, int mul, int atomic, T *v, T *b, int nd, char *fname, int bench)
{
#if 1
  const int L=10, M=5;
  FILE *F;
  int i, l, lmax;
  double d1, d2, dtimes[L], dmin, dmax, davg;
  hipError_t ret;
  if(bench==0){lmax=1;}else{lmax=L;}
  for(l=0;l<lmax;l++){
	for(i=0;i<nd;i++)v[i] = (T)0.0;
	CHECK_DO(hipMemcpy(d_zaut, v, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy v to d_v");
	CHECK_DO(hipMemcpy(d_zu, b, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy b to d_b");
	hipDeviceSynchronize();
	d1 = omp_get_wtime();
	switch(atomic){
	case 0:
	  switch(div){
	  case  1: hmvm_cuda_hybrid3<T, 1,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  case  2: hmvm_cuda_hybrid3<T, 2,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  case  4: hmvm_cuda_hybrid3<T, 4,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  case  8: hmvm_cuda_hybrid3<T, 8,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  case 16: hmvm_cuda_hybrid3<T,16,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  case 32: hmvm_cuda_hybrid3<T,32,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  }
	  break;
	case 1:
	  switch(div){
	  case  1: hmvm_cuda_hybrid3<T, 1,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  case  2: hmvm_cuda_hybrid3<T, 2,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  case  4: hmvm_cuda_hybrid3<T, 4,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  case  8: hmvm_cuda_hybrid3<T, 8,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  case 16: hmvm_cuda_hybrid3<T,16,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  case 32: hmvm_cuda_hybrid3<T,32,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  }
	  break;
	}
	hipDeviceSynchronize();
	d2 = omp_get_wtime();
	dtimes[l] = d2-d1;
  }
  if(bench==0){
	CHECK_DO(hipMemcpy(v, d_zaut, sizeof(T)*nd, hipMemcpyDeviceToHost),"hipMemcpy d_v to v");
	printf("write to %s\n", fname);
	F = fopen(fname, "w");
	for(i=0;i<nd;i++)fprintf(F, "%.3E\n", v[i]);
	fclose(F);
  }else{
	dmin = 9999.99;
	dmax = 0.0;
	davg = 0.0;
	for(i=M;i<L;i++){
	  davg += dtimes[i];
	  if(dmin>dtimes[i])dmin=dtimes[i];
	  if(dmax<dtimes[i])dmax=dtimes[i];
	}
	davg /= (L-M);
	printf("TIME %d hmvm_cuda1_hybrid3b%s min %e max %e avg %e\n", L-M, typeid(T).name(), dmin, dmax, davg);
  }
#endif
}

template
void hmvm_cuda_hybrid3_proxy<float>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense,
 int blocks, int threads, int shms, int div, int mul, int atomic, float *v, float *b, int nd, char *fname, int bench);
template
void hmvm_cuda_hybrid3_proxy<double>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense,
 int blocks, int threads, int shms, int div, int mul, int atomic, double *v, double *b, int nd, char *fname, int bench);
// ######## ######## ######## ######## ######## ######## ######## ########
#endif

#if 0
// ######## ######## ######## ######## ######## ######## ######## ########
/*
  hybrid4
  複数WARP単一GEMV単一行GEMVカーネル
  <<<napprox+ndense, 32*mul>>>
  1 GEMV by mul TB
  1 line by 1/mul WARP
  バリエーション
  - mul：立ち上げるブロック数スレッド数の係数、1つのmat-mat-vecまたはmat-vecを1WARPで実行、mul=1,2,3,...,
  - a2t：a2を転置版で計算するか否か
  - a2interchange：a2のループを入れ替えるか否か
  - aatomic：approxの計算をatomic優先にするかwarp shuffle併用するか
  - datomic：denseの計算をatomic優先にするかwarp shuffle併用するか
*/
template <class T>
__global__ void hmvm_cuda_hybrid4
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, T *rowmat,
 int napprox, int *approx, int ndense, int *dense)
{
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid4 : begin\n");
#endif
  int gid  = blockIdx.x;
  int tid  = threadIdx.x;
  int tlen = blockDim.x;
  int ndl, ndt, nstrtl, nstrtt, ltmtx;
  int ip, kt, il, it, itt, itl, ill;
  size_t head;
  double tmp;

  ip = dense[gid];
  {
#ifndef _SKIP_APPROX
    ndl = _ndl[ip];
    ndt = _ndt[ip];
    nstrtl = _nstrtl[ip];
    nstrtt = _nstrtt[ip];
    ltmtx = _ltmtx[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif

	head = a1[ip];
	for(il=0; il<ndl; il++){
	  tmp = 0.0;
	  ill=il+nstrtl-1;
	  for(it=tid; it<ndt; it+=tlen){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp += rowmat[head+itl]*d_zu[itt];
	  }
	  for (int offset = warpSize/2; offset > 0; offset /= 2)tmp += __shfl_down_sync(tmp, offset, warpSize);
	  if(tid%32==0){
		atomicAdd(&d_zaut[ill], tmp);
	  }
	}
#endif
  }
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid4 : end\n");
#endif
}
// ######## ######## ######## ######## ######## ######## ######## ########
#endif
