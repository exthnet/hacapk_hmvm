// -*- c++ -*-
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <typeinfo>
#include <omp.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_cooperative_groups.h>

#include "hacapk.h"

// できれば重複計算していないかチェックもしたい

#define CHECK_DO(act,msg) {ret=act; if(ret!=hipSuccess){printf("%s failed\n",msg);exit(-1);};}






template
void hmvm_cuda_hybrid4_proxy<float>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat, float *rowmat_t,
 int napprox, int *approx, int ndense, int *dense,
 int blocks, int threads, int shms, int mul, int atomic, float *v, float *b, int nd, char *fname, int bench);
template
void hmvm_cuda_hybrid4_proxy<double>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat, double *rowmat_t,
 int napprox, int *approx, int ndense, int *dense,
 int blocks, int threads, int shms, int mul, int atomic, double *v, double *b, int nd, char *fname, int bench);
// ######## ######## ######## ######## ######## ######## ######## ########
#endif
