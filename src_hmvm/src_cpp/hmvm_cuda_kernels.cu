#include "hip/hip_runtime.h"
// -*- c++ -*-
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <omp.h>
#include <hip/hip_runtime_api.h>

#include "hacapk.h"

#if __CUDA_ARCH__ < 600
__device__ double myAtomicAdd(double* address, double val)
{
  unsigned long long int* address_as_ull =
	(unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;

  do {
	assumed = old;
	old = atomicCAS(address_as_ull, assumed,
					__double_as_longlong(val +
										 __longlong_as_double(assumed)));

	// Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
  } while (assumed != old);

  return __longlong_as_double(old);
}
__device__ float myAtomicAdd(float* address, float val)
{
  atomicAdd(address, val);
}
#else
#define myAtomicAdd atomicAdd
#endif

// ######## ######## ######## ######## ######## ######## ######## ########
// 0: 完全逐次

template <class T>
__global__ void hmvm_cuda_seq
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, T *rowmat,
 int napprox, int *approx, int ndense, int *dense)
{
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cudaD : begin\n");
#endif
  int ndl, ndt, nstrtl, nstrtt, ltmtx;
  int ip, kt, il, it, itt, itl, ill;
  size_t head;
  T tmp;
  //extern __shared__ T tmp2[];
  extern __shared__ __align__(sizeof(T)) unsigned char my_smem[];
  T *tmp2 = reinterpret_cast<T *>(my_smem);
  int i;

  // approx
  for(i=0; i<napprox; i++){
	ip = approx[i];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	ltmtx = _ltmtx[ip];
	kt = _kt[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	head = a1[ip];
	for(il=0; il<kt; il++){
	  tmp2[il] = 0.0;
	  for(it=0; it<ndt; it++){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp2[il] += rowmat[head+itl]*d_zu[itt];
	  }
	}
	head = a2[ip];
	for(il=0; il<kt; il++){
	  for(it=0; it<ndl; it++){
		ill=it+nstrtl-1;
		itl=it+il*ndl;
		myAtomicAdd(&d_zaut[ill], rowmat[head+itl]*tmp2[il]);
	  }
	}
  }
  // dense
  for(i=0; i<ndense; i++){
	ip = dense[i];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	ltmtx = _ltmtx[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	head = a1[ip];
	for(il=0; il<ndl; il++){
	  tmp = 0.0;
	  ill=il+nstrtl-1;
	  for(it=0; it<ndt; it++){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp += rowmat[head+itl]*d_zu[itt];
	  }
	  myAtomicAdd(&d_zaut[ill], tmp);
	}
  }
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cudaD : end\n");
#endif
}

// ######## ######## ######## ######## ######## ######## ######## ########

// block並列化カーネル
template <class T>
__global__ void hmvm_cuda_block
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, T *rowmat,
 int napprox, int *approx, int ndense, int *dense)
{
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cudaD_block : begin\n");
#endif
  int ndl, ndt, nstrtl, nstrtt, ltmtx;
  int ip, kt, il, it, itt, itl, ill;
  size_t head;
  T tmp;
  //extern __shared__ T tmp2[];
  extern __shared__ __align__(sizeof(T)) unsigned char my_smem[];
  T *tmp2 = reinterpret_cast<T *>(my_smem);
  int i;

  if(blockIdx.x<napprox){
	// approx
	//for(i=0; i<napprox; i++){
	ip = approx[blockIdx.x];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	ltmtx = _ltmtx[ip];
	kt = _kt[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	head = a1[ip];
	for(il=0; il<kt; il++){
	  tmp2[il] = 0.0;
	  for(it=0; it<ndt; it++){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp2[il] += rowmat[head+itl]*d_zu[itt];
	  }
	}
	head = a2[ip];
	for(il=0; il<kt; il++){
	  for(it=0; it<ndl; it++){
		ill=it+nstrtl-1;
		itl=it+il*ndl;
		myAtomicAdd(&d_zaut[ill], rowmat[head+itl]*tmp2[il]);
	  }
	}
  }else{
	// dense
	//for(i=0; i<ndense; i++){
	ip = dense[blockIdx.x - napprox];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	ltmtx = _ltmtx[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	head = a1[ip];
	for(il=0; il<ndl; il++){
	  tmp = 0.0;
	  ill=il+nstrtl-1;
	  for(it=0; it<ndt; it++){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp += rowmat[head+itl]*d_zu[itt];
	  }
	  myAtomicAdd(&d_zaut[ill], tmp);
	}
  }
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cudaD_block : end\n");
#endif
}

// ######## ######## ######## ######## ######## ######## ######## ########

// まだデバッグ中

#if 1
/*
  <<<ndense,32>>>
  1 GEMV by 1 TB
  1 line by 1/div WARP
*/

// nlf block, 32 thread
template <class T, int div>
__global__ void hmvm_cuda_hybrid1
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, T *rowmat,
 int napprox, int *approx, int ndense, int *dense)
{
  return;
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid1 : begin\n");
#endif
  int gid   = blockIdx.x;
  int tid   = threadIdx.x;
  int bid   = threadIdx.x/(32/div);
  int blen  = div;
  int xid   = threadIdx.x%(32/div);
  int xlen  = (32/div);
  int ndl, ndt, nstrtl, nstrtt, ltmtx;
  int ip, kt, il, it, itt, itl, ill;
  size_t head;
  T tmp;
  extern __shared__ __align__(sizeof(T)) unsigned char my_smem[];
  T *tmp2 = reinterpret_cast<T *>(my_smem);

  if(gid<napprox){
#if 0
	// approx
	ip = approx[gid];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	ltmtx = _ltmtx[ip];
	kt = _kt[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	head = a1[ip];
	for(il=bid; il<kt; il+=blen){
	  tmp2[il] = 0.0;
	  for(it=xid; it<ndt; it+=xlen){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp2[il] += rowmat[head+itl]*d_zu[itt];
	  }
	}
	head = a2[ip];
	for(il=bid; il<kt; il+=blen){
	  for(it=xid; it<ndl; it+=xlen){
		ill=it+nstrtl-1;
		itl=it+il*ndl;
		myAtomicAdd(&d_zaut[ill], rowmat[head+itl]*tmp2[il]);
	  }
	}
#endif
  }else{
#if 1
	// dense
	ip = dense[gid-napprox];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	ltmtx = _ltmtx[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	head = a1[ip];
	for(il=bid; il<ndl; il+=blen){
	  tmp = 0.0;
	  ill=il+nstrtl-1;
	  for(it=xid; it<ndt; it+=xlen){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp += rowmat[head+itl]*d_zu[itt];
	  }
	  //for (int offset = warpSize/(2*div); offset > 0; offset /= 2)tmp += __shfl_down(tmp, offset);
	  for (int offset = warpSize/(2*div); offset > 0; offset /= 2)tmp += __shfl_down_sync(tmp, offset, warpSize);
	  if(xid==0){
		myAtomicAdd(&d_zaut[ill], tmp);
	  }
	}
#endif
  }

#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid1 : end\n");
#endif
}
#endif


// ######## ######## ######## ######## ######## ######## ######## ########
// template関数の実体化のための宣言
// ######## ######## ######## ######## ######## ######## ######## ########
template __global__ void hmvm_cuda_seq<float>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_seq<double>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense);

template __global__ void hmvm_cuda_block<float>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_block<double>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense);

template __global__ void hmvm_cuda_hybrid1<double,1>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<double,2>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<double,4>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<float,1>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<float,2>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<float,4>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense);
