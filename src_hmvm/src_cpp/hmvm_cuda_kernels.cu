#include "hip/hip_runtime.h"
// -*- c++ -*-
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <typeinfo>
#include <omp.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_cooperative_groups.h>

#include "hacapk.h"

namespace cg = cooperative_groups;

#if __CUDA_ARCH__ < 600
__device__ double myAtomicAdd(double* address, double val)
{
  unsigned long long int* address_as_ull =
	(unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;

  do {
	assumed = old;
	old = atomicCAS(address_as_ull, assumed,
					__double_as_longlong(val +
										 __longlong_as_double(assumed)));

	// Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
  } while (assumed != old);

  return __longlong_as_double(old);
}
__device__ float myAtomicAdd(float* address, float val)
{
  atomicAdd(address, val);
}
#else
#define myAtomicAdd atomicAdd
#endif

// ######## ######## ######## ######## ######## ######## ######## ########
// 0: 完全逐次
#if 1
template <class T>
__global__ void hmvm_cuda_seq
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, T *rowmat,
 int napprox, int *approx, int ndense, int *dense)
{
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_seq : begin\n");
#endif
  int ndl, ndt, nstrtl, nstrtt, ltmtx;
  int ip, kt, il, it, itt, itl, ill;
  size_t head;
  //extern __shared__ T tmp2[];
  extern __shared__ __align__(sizeof(T)) unsigned char my_smem[];
  T *tmp2 = reinterpret_cast<T *>(my_smem);
  int i;

  // approx
  for(i=0; i<napprox; i++){
#if 1
	ip = approx[i];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	ltmtx = _ltmtx[ip];
	kt = _kt[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	head = a1[ip];
	for(il=0; il<kt; il++){
	  tmp2[il] = 0.0;
	  for(it=0; it<ndt; it++){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp2[il] += rowmat[head+itl]*d_zu[itt];
	  }
	}
	head = a2[ip];
	for(il=0; il<kt; il++){
	  for(it=0; it<ndl; it++){
		ill=it+nstrtl-1;
		itl=it+il*ndl;
		myAtomicAdd(&d_zaut[ill], rowmat[head+itl]*tmp2[il]);
	  }
	}
#endif
  }
  // dense
  for(i=0; i<ndense; i++){
#if 1
	ip = dense[i];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	ltmtx = _ltmtx[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	head = a1[ip];
	for(il=0; il<ndl; il++){
	  T tmp = (T)0.0;
	  ill=il+nstrtl-1;
	  for(it=0; it<ndt; it++){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp += rowmat[head+itl]*d_zu[itt];
	  }
	  myAtomicAdd(&d_zaut[ill], tmp);
	}
#endif
  }
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_seq : end\n");
#endif
}
#endif
// ######## ######## ######## ######## ######## ######## ######## ########

// block並列化カーネル
template <class T>
__global__ void hmvm_cuda_block
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, T *rowmat,
 int napprox, int *approx, int ndense, int *dense)
{
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cudaD_block : begin\n");
#endif
  int ndl, ndt, nstrtl, nstrtt, ltmtx;
  int ip, kt, il, it, itt, itl, ill;
  size_t head;
  extern __shared__ __align__(sizeof(T)) unsigned char my_smem[];
  T *tmp2 = reinterpret_cast<T *>(my_smem);
  int i;

  if(blockIdx.x<napprox){
#if 1
	// approx
	//for(i=0; i<napprox; i++){
	ip = approx[blockIdx.x];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	ltmtx = _ltmtx[ip];
	kt = _kt[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	head = a1[ip];
	for(il=0; il<kt; il++){
	  tmp2[il] = 0.0;
	  for(it=0; it<ndt; it++){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp2[il] += rowmat[head+itl]*d_zu[itt];
	  }
	}
	head = a2[ip];
	for(il=0; il<kt; il++){
	  for(it=0; it<ndl; it++){
		ill=it+nstrtl-1;
		itl=it+il*ndl;
		myAtomicAdd(&d_zaut[ill], rowmat[head+itl]*tmp2[il]);
	  }
	}
#endif
  }else{
#if 1
	// dense
	//for(i=0; i<ndense; i++){
	ip = dense[blockIdx.x - napprox];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	ltmtx = _ltmtx[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	head = a1[ip];
	for(il=0; il<ndl; il++){
	  T tmp = 0.0;
	  ill=il+nstrtl-1;
	  for(it=0; it<ndt; it++){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp += rowmat[head+itl]*d_zu[itt];
	  }
	  myAtomicAdd(&d_zaut[ill], tmp);
	}
#endif
  }
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cudaD_block : end\n");
#endif
}

// ######## ######## ######## ######## ######## ######## ######## ########

#if 1
/*
  <<<ndense,32>>>
  1 GEMV by 1 TB
  1 line by 1/div WARP
*/
// nlf block, 32 thread, all atomic版とwarp(cooperative group) shuffle併用版
template <class T, int div, int atomic>
__global__ void hmvm_cuda_hybrid1
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, T *rowmat,
 int napprox, int *approx, int ndense, int *dense)
{
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid1 : begin\n");
#endif
  int gid   = blockIdx.x;
  int tid   = threadIdx.x;
  int bid   = threadIdx.x/(32/div);
  int blen  = div;
  int xid   = threadIdx.x%(32/div);
  int xlen  = (32/div);
  int ndl, ndt, nstrtl, nstrtt, ltmtx;
  int ip, kt, il, it, itt, itl, ill;
  size_t head;
  T tmp = 0.0;
  extern __shared__ __align__(sizeof(T)) unsigned char my_smem[];
  T *tmp2 = reinterpret_cast<T *>(my_smem);
  cg::thread_block_tile<32/div> g = cg::tiled_partition<32/div>(cg::this_thread_block());

  if(gid<napprox){
#if 1
	// approx
	ip = approx[gid];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	ltmtx = _ltmtx[ip];
	kt = _kt[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	head = a1[ip];
	for(il=bid; il<kt; il+=blen){
	  if(xid==0)tmp2[il] = 0.0;
	  tmp = 0.0;
	  for(it=xid; it<ndt; it+=xlen){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp += rowmat[head+itl]*d_zu[itt];
	  }
	  //for (int offset = warpSize/(2*div); offset > 0; offset /= 2)tmp += __shfl_down_sync(0xffff, tmp, offset, warpSize);
	  for (int offset = g.size()/2; offset > 0; offset /= 2)tmp += g.shfl_down(tmp, offset);
	  if(xid==0)tmp2[il] = tmp;
	}
	head = a2[ip];
	for(il=bid; il<kt; il+=blen){
	  for(it=xid; it<ndl; it+=xlen){
		ill=it+nstrtl-1;
		itl=it+il*ndl;
		myAtomicAdd(&d_zaut[ill], rowmat[head+itl]*tmp2[il]);
	  }
	}
#endif // approx
  }else{
#if 1
	// dense
	ip = dense[gid-napprox];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	ltmtx = _ltmtx[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	head = a1[ip];
	for(il=bid; il<ndl; il+=blen){
	  tmp = 0.0;
	  ill=il+nstrtl-1;
	  for(it=xid; it<ndt; it+=xlen){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp += rowmat[head+itl]*d_zu[itt];
	  }
	  if(atomic==0){
		//for (int offset = warpSize/(2*div); offset > 0; offset /= 2)tmp += __shfl_down_sync(0xffff, tmp, offset, warpSize);
		for (int offset = g.size()/2; offset > 0; offset /= 2)tmp += g.shfl_down(tmp, offset);
		if(xid==0){
		  myAtomicAdd(&d_zaut[ill], tmp);
		}
	  }else{
		myAtomicAdd(&d_zaut[ill], tmp);
	  }
	}
#endif // dense
  }

#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid1 : end\n");
#endif
}
#endif


#if 1
/*
  複数WARP単一GEMV個別行カーネル
  <<<ndense,32*mul>>>
  1 GEMV by mul TB
  1 line by 1/div WARP
*/
template <class T, int div, int atomic>//int div, int mul, int atomic>
__global__ void hmvm_cuda_hybrid2
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, T *rowmat,
 int napprox, int *approx, int ndense, int *dense, int mul)
{
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid2 : begin\n");
#endif
  int gid   = blockIdx.x;
  int tid   = threadIdx.x;
  int bid   = threadIdx.x/(32/div);
  int blen  = div*mul;
  int xid   = threadIdx.x%(32/div);
  int xlen  = (32/div);
  int ndl, ndt, nstrtl, nstrtt, ltmtx;
  int ip, kt, il, it, itt, itl, ill;
  size_t head;
  T tmp = 0.0;
  extern __shared__ __align__(sizeof(T)) unsigned char my_smem[];
  T *tmp2 = reinterpret_cast<T *>(my_smem);
  cg::thread_block_tile<32/div> g = cg::tiled_partition<32/div>(cg::this_thread_block());

  if(gid<napprox){
#if 1
	// approx
	ip = approx[gid];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	ltmtx = _ltmtx[ip];
	kt = _kt[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	head = a1[ip];
	for(il=bid; il<kt; il+=blen){
	if(xid==0)tmp2[il] = 0.0;
	tmp = 0.0;
	  for(it=xid; it<ndt; it+=xlen){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp += rowmat[head+itl]*d_zu[itt];
	  }
	  //for (int offset = warpSize/(2*div); offset > 0; offset /= 2)tmp += __shfl_down_sync(0xffff, tmp, offset, warpSize);
	  for (int offset = g.size()/2; offset > 0; offset /= 2)tmp += g.shfl_down(tmp, offset);
	  if(xid==0)tmp2[il] = tmp;
	}
	head = a2[ip];
	for(il=bid; il<kt; il+=blen){
	  for(it=xid; it<ndl; it+=xlen){
		ill=it+nstrtl-1;
		itl=it+il*ndl;
		myAtomicAdd(&d_zaut[ill], rowmat[head+itl]*tmp2[il]);
	  }
	}
#endif // approx
  }else{
#if 1
	// dense
	ip = dense[gid-napprox];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	ltmtx = _ltmtx[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	head = a1[ip];
	for(il=bid; il<ndl; il+=blen){
	  tmp = 0.0;
	  ill=il+nstrtl-1;
	  for(it=xid; it<ndt; it+=xlen){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp += rowmat[head+itl]*d_zu[itt];
	  }
	  if(atomic==0){
		//for (int offset = warpSize/(2*div); offset > 0; offset /= 2)tmp += __shfl_down_sync(tmp, offset, warpSize);
		for (int offset = g.size()/2; offset > 0; offset /= 2)tmp += g.shfl_down(tmp, offset);
		if(xid==0){
		  myAtomicAdd(&d_zaut[ill], tmp);
		}
	  }else{
		myAtomicAdd(&d_zaut[ill], tmp);
	  }
	}
#endif // dense
  }

#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid2 : end\n");
#endif
}
#endif


#if 1
// ######## ######## ######## ######## ######## ######## ######## ########
/*
  複数WARP個別GEMVカーネル
  <<<ndense/mul, 32*mul>>>
  1 GEMV by 1 WARP
  1 line by 1/div WARP
  mul GEMVs in parallel
*/
template <class T, int div, int atomic>
__global__ void hmvm_cuda_hybrid3
(T *d_zaut, T *d_zu, int nlf, int ktmax,
int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, T *rowmat,
int napprox, int *approx, int ndense, int *dense, int mul)
{
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid3 : begin\n");
#endif
  int gid   = blockIdx.x;
  int tid   = threadIdx.x;
  int bid   = threadIdx.x%32/(32/div);
  int blen  = div;
  int xid   = threadIdx.x%(32/div);
  int xlen  = (32/div);
  int ndl, ndt, nstrtl, nstrtt, ltmtx;
  int ip, kt, il, it, itt, itl, ill;
  size_t head;
  T tmp = 0.0;
  extern __shared__ __align__(sizeof(T)) unsigned char my_smem[];
  T *tmp2 = reinterpret_cast<T *>(my_smem);
  cg::thread_block_tile<32/div> g = cg::tiled_partition<32/div>(cg::this_thread_block());
  int id;

  id = gid*mul + tid/32;
  if(id<napprox){
#if 1
	// approx
	ip = approx[gid];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	ltmtx = _ltmtx[ip];
	kt = _kt[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	head = a1[ip];
	for(il=bid; il<kt; il+=blen){
	if(xid==0)tmp2[il] = 0.0;
	tmp = 0.0;
	  for(it=xid; it<ndt; it+=xlen){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp += rowmat[head+itl]*d_zu[itt];
	  }
	  //for (int offset = warpSize/(2*div); offset > 0; offset /= 2)tmp += __shfl_down_sync(0xffff, tmp, offset, warpSize);
	  for (int offset = g.size()/2; offset > 0; offset /= 2)tmp += g.shfl_down(tmp, offset);
	  if(xid==0)tmp2[il] = tmp;
	}
	head = a2[ip];
	for(il=bid; il<kt; il+=blen){
	  for(it=xid; it<ndl; it+=xlen){
		ill=it+nstrtl-1;
		itl=it+il*ndl;
		myAtomicAdd(&d_zaut[ill], rowmat[head+itl]*tmp2[il]);
	  }
	}
#endif // approx
  }else{
#if 1
	ip = dense[id-napprox];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	ltmtx = _ltmtx[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	head = a1[ip];
	for(il=bid; il<ndl; il+=blen){
      tmp = 0.0;
	  ill=il+nstrtl-1;
	  for(it=xid; it<ndt; it+=xlen){
	    itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp += rowmat[head+itl]*d_zu[itt];
      }
	  if(atomic==0){
	    //for (int offset = warpSize/(2*div); offset > 0; offset /= 2)tmp += __shfl_down(tmp, offset);
	    for (int offset = warpSize/(2*div); offset > 0; offset /= 2)tmp += __shfl_down_sync(tmp, offset, warpSize);
		if(xid==0){
	      atomicAdd(&d_zaut[ill], tmp);
        }
      }else{
  	    atomicAdd(&d_zaut[ill], tmp);
      }
    }
#endif // dense
  }
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid3 : end\n");
#endif
}
#endif

// ######## ######## ######## ######## ######## ######## ######## ########
// template関数の実体化のための宣言
// ######## ######## ######## ######## ######## ######## ######## ########
// float
#if 1
template __global__ void hmvm_cuda_seq<float>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense);

template __global__ void hmvm_cuda_block<float>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense);

template __global__ void hmvm_cuda_hybrid1<float,1,0>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<float,2,0>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<float,4,0>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<float,8,0>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<float,16,0>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<float,32,0>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense);

template __global__ void hmvm_cuda_hybrid1<float,1,1>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<float,2,1>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<float,4,1>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<float,8,1>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<float,16,1>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<float,32,1>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense);

/*
template __global__ void hmvm_cuda_hybrid2<float,DIV,MUL,ATOMIC>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
int napprox, int *approx, int ndense, int *dense);

for a in 0 1
do
for m in `seq 1 16`
do
for d in 1 2 4 8 16 32
do
echo "template __global__ void hmvm_cuda_hybrid2<float,${d},${m},${a}>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
int napprox, int *approx, int ndense, int *dense);"
done
done
done
*/
#endif

// double
#if 1
template __global__ void hmvm_cuda_seq<double>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_block<double>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense);

template __global__ void hmvm_cuda_hybrid1<double,1,0>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<double,2,0>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<double,4,0>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<double,8,0>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<double,16,0>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<double,32,0>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense);

template __global__ void hmvm_cuda_hybrid1<double,1,1>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<double,2,1>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<double,4,1>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<double,8,1>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<double,16,1>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense);
template __global__ void hmvm_cuda_hybrid1<double,32,1>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense);

/*
 template __global__ void hmvm_cuda_hybrid2<double,DIV,MUL,ATOMIC>
(T *d_zaut, T *d_zu, int nlf, int ktmax,
int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, T *rowmat,
int napprox, int *approx, int ndense, int *dense);

for a in 0 1
do
for m in `seq 1 16`
do
for d in 1 2 4 8 16 32
do
echo "template __global__ void hmvm_cuda_hybrid2<double,${d},${m},${a}>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
int napprox, int *approx, int ndense, int *dense);"
done
done
done
*/
#endif

#define CHECK_DO(act,msg) {ret=act; if(ret!=hipSuccess){printf("%s failed\n",msg);exit(-1);};}

#if 1
template <class T>
void hmvm_cuda_hybrid2_proxy
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *ltmtx, int *ndt, int *ndl, int *nstrtl, int *nstrtt, int *kt, int *a1, int *a2, T *rowmat,
 int napprox, int *approx, int ndense, int *dense,
 int blocks, int threads, int shms, int div, int mul, int atomic, T *v, T *b, int nd, char *fname, int bench)
{
#if 1
  const int L=10, M=5;
  FILE *F;
  int i, l;
  double d1, d2, dtimes[L], dmin, dmax, davg;
  hipError_t ret;
  if(bench==0){
	// EXEC
	for(i=0;i<nd;i++)v[i] = (T)0.0;
	CHECK_DO(hipMemcpy(d_zaut, v, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy v to d_v");
	CHECK_DO(hipMemcpy(d_zu, b, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy b to d_b");
	hipDeviceSynchronize();
	switch(atomic){
	case 0:
	  switch(div){
	  case  1: hmvm_cuda_hybrid2<T, 1,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  case  2: hmvm_cuda_hybrid2<T, 2,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  case  4: hmvm_cuda_hybrid2<T, 4,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  case  8: hmvm_cuda_hybrid2<T, 8,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  case 16: hmvm_cuda_hybrid2<T,16,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  case 32: hmvm_cuda_hybrid2<T,32,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  }
	  break;
	case 1:
	  switch(div){
	  case  1: hmvm_cuda_hybrid2<T, 1,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  case  2: hmvm_cuda_hybrid2<T, 2,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  case  4: hmvm_cuda_hybrid2<T, 4,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  case  8: hmvm_cuda_hybrid2<T, 8,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  case 16: hmvm_cuda_hybrid2<T,16,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  case 32: hmvm_cuda_hybrid2<T,32,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  }
	  break;
	}
	hipDeviceSynchronize();
	CHECK_DO(hipMemcpy(v, d_zaut, sizeof(T)*nd, hipMemcpyDeviceToHost),"hipMemcpy d_v to v");
	printf("write to %s\n", fname);
	F = fopen(fname, "w");
	for(i=0;i<nd;i++)fprintf(F, "%.3E\n", v[i]);
	fclose(F);
  }else{
	// BENCH
	  for(l=0;l<L;l++){
		for(i=0;i<nd;i++)v[i] = (T)0.0;
		CHECK_DO(hipMemcpy(d_zaut, v, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy v to d_v");
		CHECK_DO(hipMemcpy(d_zu, b, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy b to d_b");
		hipDeviceSynchronize();
		d1 = omp_get_wtime();
		switch(atomic){
		case 0:
		  switch(div){
		  case  1: hmvm_cuda_hybrid2<T, 1,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
		  case  2: hmvm_cuda_hybrid2<T, 2,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
		  case  4: hmvm_cuda_hybrid2<T, 4,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
		  case  8: hmvm_cuda_hybrid2<T, 8,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
		  case 16: hmvm_cuda_hybrid2<T,16,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
		  case 32: hmvm_cuda_hybrid2<T,32,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
		  }
		  break;
		case 1:
		  switch(div){
		  case  1: hmvm_cuda_hybrid2<T, 1,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
		  case  2: hmvm_cuda_hybrid2<T, 2,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
		  case  4: hmvm_cuda_hybrid2<T, 4,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
		  case  8: hmvm_cuda_hybrid2<T, 8,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
		  case 16: hmvm_cuda_hybrid2<T,16,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
		  case 32: hmvm_cuda_hybrid2<T,32,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
		  }
		  break;
		}
		hipDeviceSynchronize();
		d2 = omp_get_wtime();
		dtimes[l] = d2-d1;
	  }
	  dmin = 9999.99;
	  dmax = 0.0;
	  davg = 0.0;
	  for(i=M;i<L;i++){
		davg += dtimes[i];
		if(dmin>dtimes[i])dmin=dtimes[i];
		if(dmax<dtimes[i])dmax=dtimes[i];
	  }
	  davg /= (L-M);
	  printf("TIME %d hmvm_cuda1_hybrid2b%s min %e max %e avg %e\n", L-M, typeid(T).name(), dmin, dmax, davg);
  }
#endif
}
template
void hmvm_cuda_hybrid2_proxy<float>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense,
 int blocks, int threads, int shms, int div, int mul, int atomic, float *v, float *b, int nd, char *fname, int bench);
template
void hmvm_cuda_hybrid2_proxy<double>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense,
 int blocks, int threads, int shms, int div, int mul, int atomic, double *v, double *b, int nd, char *fname, int bench);
#endif


#if 1
template <class T>
void hmvm_cuda_hybrid3_proxy
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *ltmtx, int *ndt, int *ndl, int *nstrtl, int *nstrtt, int *kt, int *a1, int *a2, T *rowmat,
 int napprox, int *approx, int ndense, int *dense,
 int blocks, int threads, int shms, int div, int mul, int atomic, T *v, T *b, int nd, char *fname, int bench)
{
#if 1
  const int L=10, M=5;
  FILE *F;
  int i, l;
  double d1, d2, dtimes[L], dmin, dmax, davg;
  hipError_t ret;
  if(bench==0){
	// EXEC
	for(i=0;i<nd;i++)v[i] = (T)0.0;
	CHECK_DO(hipMemcpy(d_zaut, v, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy v to d_v");
	CHECK_DO(hipMemcpy(d_zu, b, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy b to d_b");
	hipDeviceSynchronize();
	switch(atomic){
	case 0:
	  switch(div){
	  case  1: hmvm_cuda_hybrid3<T, 1,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  case  2: hmvm_cuda_hybrid3<T, 2,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  case  4: hmvm_cuda_hybrid3<T, 4,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  case  8: hmvm_cuda_hybrid3<T, 8,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  case 16: hmvm_cuda_hybrid3<T,16,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  case 32: hmvm_cuda_hybrid3<T,32,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  }
	  break;
	case 1:
	  switch(div){
	  case  1: hmvm_cuda_hybrid3<T, 1,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  case  2: hmvm_cuda_hybrid3<T, 2,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  case  4: hmvm_cuda_hybrid3<T, 4,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  case  8: hmvm_cuda_hybrid3<T, 8,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  case 16: hmvm_cuda_hybrid3<T,16,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  case 32: hmvm_cuda_hybrid3<T,32,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
	  }
	  break;
	}
	hipDeviceSynchronize();
	CHECK_DO(hipMemcpy(v, d_zaut, sizeof(T)*nd, hipMemcpyDeviceToHost),"hipMemcpy d_v to v");
	printf("write to %s\n", fname);
	F = fopen(fname, "w");
	for(i=0;i<nd;i++)fprintf(F, "%.3E\n", v[i]);
	fclose(F);
  }else{
	// BENCH
	  for(l=0;l<L;l++){
		for(i=0;i<nd;i++)v[i] = (T)0.0;
		CHECK_DO(hipMemcpy(d_zaut, v, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy v to d_v");
		CHECK_DO(hipMemcpy(d_zu, b, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy b to d_b");
		hipDeviceSynchronize();
		d1 = omp_get_wtime();
		switch(atomic){
		case 0:
		  switch(div){
		  case  1: hmvm_cuda_hybrid3<T, 1,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
		  case  2: hmvm_cuda_hybrid3<T, 2,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
		  case  4: hmvm_cuda_hybrid3<T, 4,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
		  case  8: hmvm_cuda_hybrid3<T, 8,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
		  case 16: hmvm_cuda_hybrid3<T,16,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
		  case 32: hmvm_cuda_hybrid3<T,32,0><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
		  }
		  break;
		case 1:
		  switch(div){
		  case  1: hmvm_cuda_hybrid3<T, 1,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
		  case  2: hmvm_cuda_hybrid3<T, 2,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
		  case  4: hmvm_cuda_hybrid3<T, 4,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
		  case  8: hmvm_cuda_hybrid3<T, 8,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
		  case 16: hmvm_cuda_hybrid3<T,16,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
		  case 32: hmvm_cuda_hybrid3<T,32,1><<<blocks,threads,shms>>>(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, napprox, approx, ndense, dense, mul); break;
		  }
		  break;
		}
		hipDeviceSynchronize();
		d2 = omp_get_wtime();
		dtimes[l] = d2-d1;
	  }
	  dmin = 9999.99;
	  dmax = 0.0;
	  davg = 0.0;
	  for(i=M;i<L;i++){
		davg += dtimes[i];
		if(dmin>dtimes[i])dmin=dtimes[i];
		if(dmax<dtimes[i])dmax=dtimes[i];
	  }
	  davg /= (L-M);
	  printf("TIME %d hmvm_cuda1_hybrid3b%s min %e max %e avg %e\n", L-M, typeid(T).name(), dmin, dmax, davg);
  }
#endif
}
template
void hmvm_cuda_hybrid3_proxy<float>
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense,
 int blocks, int threads, int shms, int div, int mul, int atomic, float *v, float *b, int nd, char *fname, int bench);
template
void hmvm_cuda_hybrid3_proxy<double>
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense,
 int blocks, int threads, int shms, int div, int mul, int atomic, double *v, double *b, int nd, char *fname, int bench);
#endif
