#include "hip/hip_runtime.h"
// -*- C++ -*-
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <typeinfo>
#include <omp.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_cooperative_groups.h>

#include "hacapk.h"

#define CHECK_DO(act,msg) {ret=act; if(ret!=hipSuccess){printf("%s failed\n",msg);exit(-1);};}

namespace cg = cooperative_groups;

#if __CUDA_ARCH__ < 600
__device__ static double myAtomicAdd(double* address, double val)
{
  unsigned long long int* address_as_ull =
	(unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;

  do {
	assumed = old;
	old = atomicCAS(address_as_ull, assumed,
					__double_as_longlong(val +
										 __longlong_as_double(assumed)));

	// Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
  } while (assumed != old);

  return __longlong_as_double(old);
}
__device__ static inline float myAtomicAdd(float* address, float val)
{
  atomicAdd(address, val);
}
#else
#define myAtomicAdd atomicAdd
#endif

// ######## ######## ######## ######## ######## ######## ######## ########
/*
  hybrid3
  複数WARP個別GEMVカーネル
  1PMVを1WARPが担当
  1TBあたりスレッド数は32*mul(mul WARP)
  PMV内の1行を1/div WARPが担当
  hybrid1->hybrid2では増やしたWARPを同一PMVの計算に割り当てたが、hybrid3ではそれぞれ個別のPMVを計算する
  <<<napprox/mul+ndense/mul, 32*mul>>>
  1 PMV by 1 WARP
  1 line by 1/div WARP
  バリエーション
  - div：1行を1/divのWARPで計算する、div=1,2,4,8,16,32
  - mul：立ち上げるブロック数スレッド数の係数、1つのmat-mat-vecまたはmat-vecを1WARPで実行、mul=1,2,3,...,
  - a2t：a2を転置版で計算するか否か
  - a2interchange：a2のループを入れ替えるか否か
  - aatomic：approxの計算をatomic優先にするかwarp shuffle併用するか
  - datomic：denseの計算をatomic優先にするかwarp shuffle併用するか
*/
template <class T, int div>
__global__ void hmvm_cuda_hybrid3
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt,
 int *a1, int *a2, T *rowmat, T *rowmat_t,
 int napprox, int *approx, int ndense, int *dense,
 int mul, int a2t, int a2i, int aatomic, int datomic)
{
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid3 : begin\n");
#endif
  int gid   = blockIdx.x*mul+threadIdx.x/32;
  int bid   = ((threadIdx.x%32)/(32/div));
  int blen  = (32/(32/div));
  int xid   = (threadIdx.x%(32/div));
  int xlen  = (32/div);
  int ndl, ndt, nstrtl, nstrtt;
  int ip, kt, il, it, itt, itl, ill;
  size_t head;
  T tmp1 = (T)0.0;
  extern __shared__ __align__(sizeof(T)) unsigned char my_smem[];
  T *tmp2 = reinterpret_cast<T *>(my_smem);
  cg::thread_block_tile<32/div> g = cg::tiled_partition<32/div>(cg::this_thread_block());

#if 1
  if(gid<((napprox+mul-1)/mul)*mul){
#ifndef _SKIP_APPROX
	if(gid<napprox){
	  // approx
	  ip = approx[gid];
	  ndl = _ndl[ip];
	  ndt = _ndt[ip];
	  nstrtl = _nstrtl[ip];
	  nstrtt = _nstrtt[ip];
	  kt = _kt[ip];
#if _DEBUG_LEVEL >= 3
	  printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, _ltmtx[ip]);
#endif
	  head = a1[ip];
	  for(il=bid; il<kt; il+=blen){
  	    if(xid==0)tmp2[(threadIdx.x/32)*ktmax+il] = (T)0.0;
		tmp1 = (T)0.0;
		for(it=xid; it<ndt; it+=xlen){
	      itt=it+nstrtt-1;
		  itl=it+il*ndt;
		  if(a2t==0){
			tmp1 += rowmat[head+itl]*d_zu[itt];
		  }else{
			tmp1 += rowmat_t[head+itl]*d_zu[itt];
		  }
        }
		for (int offset = g.size()/2; offset > 0; offset /= 2)tmp1 += g.shfl_down(tmp1, offset);
		if(xid==0)tmp2[(threadIdx.x/32)*ktmax+il] = tmp1;
      }
	  __syncwarp();
	  head = a2[ip];
	  if(a2t==0){ // a2t==0
		if(a2i==0){ // a2i==0
		  for(il=bid; il<kt; il+=blen){
			for(it=xid; it<ndl; it+=xlen){
			  ill=it+nstrtl-1;
			  itl=it+il*ndl;
			  myAtomicAdd(&d_zaut[ill], rowmat[head+itl]*tmp2[(threadIdx.x/32)*ktmax+il]);
			}
		  }
		}else{ // a2i==1
		  if(aatomic==0){ // aatomic==0
			for(it=bid; it<ndl; it+=blen){
			  ill=it+nstrtl-1;
			  tmp1 = (T)0.0;
			  for(il=xid; il<kt; il+=xlen){
				itl=it+il*ndl;
				tmp1 += rowmat[head+itl]*tmp2[(threadIdx.x/32)*ktmax+il];
			  }
			  for (int offset = g.size()/2; offset > 0; offset /= 2)tmp1 += g.shfl_down(tmp1, offset);
			  if(xid==0){
				myAtomicAdd(&d_zaut[ill], tmp1);
			  }
			}
		  }else{ // aatomic==1
			for(it=bid; it<ndl; it+=blen){
			  ill=it+nstrtl-1;
			  tmp1 = (T)0.0;
			  for(il=xid; il<kt; il+=xlen){
				itl=it+il*ndl;
				tmp1 += rowmat[head+itl]*tmp2[(threadIdx.x/32)*ktmax+il];
			  }
			  myAtomicAdd(&d_zaut[ill], tmp1);
			}
		  }
		}
	  }else{ // a2t==1
		if(a2i==0){ // a2i==0
		  for(il=bid; il<kt; il+=blen){
			for(it=xid; it<ndl; it+=xlen){
			  ill=it+nstrtl-1;
			  itl=it*kt+il;
			  myAtomicAdd(&d_zaut[ill], rowmat_t[head+itl]*tmp2[(threadIdx.x/32)*ktmax+il]);
			}
		  }
		}else{ // a2i==1
		  if(aatomic==0){ // aatomic==0
			for(it=bid; it<ndl; it+=blen){
			  ill=it+nstrtl-1;
			  tmp1 = (T)0.0;
			  for(il=xid; il<kt; il+=xlen){
				itl=it*kt+il;
				tmp1 += rowmat_t[head+itl]*tmp2[(threadIdx.x/32)*ktmax+il];
			  }
			  for (int offset = g.size()/2; offset > 0; offset /= 2)tmp1 += g.shfl_down(tmp1, offset);
			  if(xid==0){
				myAtomicAdd(&d_zaut[ill], tmp1);
			  }
			}
		  }else{ // atomic==1
			for(it=bid; it<ndl; it+=blen){
			  ill=it+nstrtl-1;
			  tmp1 = (T)0.0;
			  for(il=xid; il<kt; il+=xlen){
				itl=it*kt+il;
				tmp1 += rowmat_t[head+itl]*tmp2[(threadIdx.x/32)*ktmax+il];
			  }
			  myAtomicAdd(&d_zaut[ill], tmp1);
			}
		  }
		}
	  }
    }
#endif // approx
  }else{
#ifndef _SKIP_DENSE
	ip = gid-((napprox+mul-1)/mul)*mul;
    if(ip<ndense){
	  ip = dense[ip];
	  ndl = _ndl[ip];
	  ndt = _ndt[ip];
	  nstrtl = _nstrtl[ip];
	  nstrtt = _nstrtt[ip];
#if _DEBUG_LEVEL >= 3
	  printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, _ltmtx[ip]);
#endif
	  head = a1[ip];
	  for(il=bid; il<ndl; il+=blen){
		tmp1 = (T)0.0;
		ill=il+nstrtl-1;
		for(it=xid; it<ndt; it+=xlen){
		  itt=it+nstrtt-1;
		  itl=it+il*ndt;
		  if(a2t==0){ // a2t==0
			tmp1 += rowmat[head+itl]*d_zu[itt];
		  }else{ // a2t==1
			tmp1 += rowmat_t[head+itl]*d_zu[itt];
		  }
		}
		if(datomic==0){ // datomic==0
		  for (int offset = g.size()/2; offset > 0; offset /= 2)tmp1 += g.shfl_down(tmp1, offset);
		  if(xid==0){
			atomicAdd(&d_zaut[ill], tmp1);
		  }
		}else{ // datomic==1
		  atomicAdd(&d_zaut[ill], tmp1);
		}
	  }
	}
#endif // dense
  }
#endif

#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid3 : end\n");
#endif
}

template <class T>
void hmvm_cuda_hybrid3_proxy
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *ltmtx, int *ndt, int *ndl, int *nstrtl, int *nstrtt, int *kt,
 int *a1, int *a2, T *rowmat, T *rowmat_t,
 int napprox, int *approx, int ndense, int *dense,
 int blocks, int threads, int shms,
 T *v, T *b, int nd, char *fname, int bench,
 int div, int mul, int a2t, int a2i, int aa, int da)
{
  int M=5, L=M+bench;
  FILE *F;
  int i, l, lmax;
  double d1, d2, *dtimes, dmin, dmax, davg;
  hipError_t ret;
  dtimes = new double[L];
  if(bench==0){lmax=1;}else{lmax=L;}
  for(l=0;l<lmax;l++){
	for(i=0;i<nd;i++)v[i] = (T)0.0;
	CHECK_DO(hipMemcpy(d_zaut, v, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy v to d_v");
	CHECK_DO(hipMemcpy(d_zu, b, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy b to d_b");
	hipDeviceSynchronize();
	d1 = omp_get_wtime();
	switch(div){
	case  1: hmvm_cuda_hybrid3<T, 1><<<blocks,threads,shms>>>
		(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt,
		 a1, a2, rowmat, rowmat_t, napprox, approx, ndense, dense, mul, a2t, a2i, aa, da);
	  break;
	case  2: hmvm_cuda_hybrid3<T, 2><<<blocks,threads,shms>>>
		(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt,
		 a1, a2, rowmat, rowmat_t, napprox, approx, ndense, dense, mul, a2t, a2i, aa, da);
	  break;
	case  4: hmvm_cuda_hybrid3<T, 4><<<blocks,threads,shms>>>
		(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt,
		 a1, a2, rowmat, rowmat_t, napprox, approx, ndense, dense, mul, a2t, a2i, aa, da);
	  break;
	case  8: hmvm_cuda_hybrid3<T, 8><<<blocks,threads,shms>>>
		(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt,
		 a1, a2, rowmat, rowmat_t, napprox, approx, ndense, dense, mul, a2t, a2i, aa, da);
	  break;
	case 16: hmvm_cuda_hybrid3<T,16><<<blocks,threads,shms>>>
		(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt,
		 a1, a2, rowmat, rowmat_t, napprox, approx, ndense, dense, mul, a2t, a2i, aa, da);
	  break;
	case 32: hmvm_cuda_hybrid3<T,32><<<blocks,threads,shms>>>
		(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt,
		 a1, a2, rowmat, rowmat_t, napprox, approx, ndense, dense, mul, a2t, a2i, aa, da);
	  break;
	}
	hipDeviceSynchronize();
	d2 = omp_get_wtime();
	dtimes[l] = d2-d1;
  }
  if(bench==0){
	CHECK_DO(hipMemcpy(v, d_zaut, sizeof(T)*nd, hipMemcpyDeviceToHost),"hipMemcpy d_v to v");
	printf("write to %s\n", fname);
	F = fopen(fname, "w");
	for(i=0;i<nd;i++)fprintf(F, "%.3E\n", v[i]);
	fclose(F);
  }else{
	dmin = 9999.99;
	dmax = 0.0;
	davg = 0.0;
	for(i=M;i<L;i++){
	  davg += dtimes[i];
	  if(dmin>dtimes[i])dmin=dtimes[i];
	  if(dmax<dtimes[i])dmax=dtimes[i];
	}
	davg /= (L-M);
	printf("TIME %d hmvm_cuda3_hybrid3%s min %e max %e avg %e\n", L-M, typeid(T).name(), dmin, dmax, davg);
  }
  delete [] dtimes;
}
// ######## ######## ######## ######## ######## ######## ######## ########

template<class T>
void hmvm_cuda3(matrix2<T> *mat2, T *b, int kernel, int dump_result, int nbench)
{
  matrix2<T> d_sm;
  int i, nd = mat2->nd, ktmax = mat2->ktmax, nlf = mat2->nlf;
  T *v=NULL;
  T *d_b, *d_v;
  int ip;
  int len;
  hipError_t ret;
  printf("hmvm_cuda3_%s: begin\n", typeid(T).name()); fflush(stdout);
  v    = new T[nd];
  for(i=0;i<nd;i++){
	v[i] = (T)0.0;
  }
  CHECK_DO(hipMalloc((void**)&d_b, sizeof(T)*nd),"hipMalloc d_b");
  CHECK_DO(hipMalloc((void**)&d_v, sizeof(T)*nd),"hipMalloc d_v");

  printf("nd = %d\n", nd);												\
  len = mat2->len;
  printf("total length = %d\n", len);
  // host alloc
  // device alloc
  d_sm.nd    = nd;
  d_sm.nlf   = nlf;
  d_sm.ktmax = ktmax;
  hipMalloc((void**)&d_sm.ltmtx, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.ndl, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.ndt, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.nstrtl, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.nstrtt, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.kt, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.a1, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.a2, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.rowmat,sizeof(T)*mat2->len);
  hipMalloc((void**)&d_sm.rowmat_t,sizeof(T)*mat2->len);
  // memcpy
  CHECK_DO(hipMemcpy(d_sm.ltmtx, mat2->ltmtx, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.ltmtx");
  CHECK_DO(hipMemcpy(d_sm.ndt, mat2->ndt, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.ndt");
  CHECK_DO(hipMemcpy(d_sm.ndl, mat2->ndl, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.ndl");
  CHECK_DO(hipMemcpy(d_sm.nstrtl, mat2->nstrtl, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.nstrtl");
  CHECK_DO(hipMemcpy(d_sm.nstrtt, mat2->nstrtt, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.nstrtt");
  CHECK_DO(hipMemcpy(d_sm.kt, mat2->kt, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.kt");
  CHECK_DO(hipMemcpy(d_sm.a1, mat2->a1, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.a1");
  CHECK_DO(hipMemcpy(d_sm.a2, mat2->a2, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.a2");
  CHECK_DO(hipMemcpy(d_sm.rowmat, mat2->rowmat, sizeof(T)*mat2->len, hipMemcpyHostToDevice),"d_sm.rowmat");
  CHECK_DO(hipMemcpy(d_sm.rowmat_t, mat2->rowmat_t, sizeof(T)*mat2->len, hipMemcpyHostToDevice),"d_sm.rowmat_t");

  // 分離
  printf("begin splitting\n");
  mat2->approx = (int*)malloc(sizeof(int)*nlf);
  mat2->dense  = (int*)malloc(sizeof(int)*nlf);
  mat2->napprox = mat2->ndense = 0;
  for(ip=0; ip<nlf; ip++){
	if(mat2->ltmtx[ip]==1){
	  mat2->approx[mat2->napprox++] = ip;
	}else{
	  mat2->dense[mat2->ndense++] = ip;
	}
  }
  hipMalloc((void**)&d_sm.approx,sizeof(int)*mat2->napprox);
  hipMalloc((void**)&d_sm.dense,sizeof(int)*mat2->ndense);
  CHECK_DO(hipMemcpy(d_sm.approx, mat2->approx, sizeof(int)*mat2->napprox, hipMemcpyHostToDevice),"d_sm.approx");
  CHECK_DO(hipMemcpy(d_sm.dense, mat2->dense, sizeof(int)*mat2->ndense, hipMemcpyHostToDevice),"d_sm.dense");
  d_sm.napprox = mat2->napprox;
  d_sm.ndense  = mat2->ndense;
  printf("end splitting (napprox=%d, ndense=%d)\n", mat2->napprox, mat2->ndense);

#if 1
/*
  hybrid3
  複数WARP個別GEMVカーネル
  1PMVを1WARPが担当
  1TBあたりスレッド数は32*mul(mul WARP)
  PMV内の1行を1/div WARPが担当
  <<<napprox/mul+ndense/mul, 32*mul>>>
  1 GEMV by 1 WARP
  1 line by 1/div WARP
  バリエーション
  - div：1行を1/divのWARPで計算する、div=1,2,4,8,16,32
  - mul：立ち上げるブロック数スレッド数の係数、1つのmat-mat-vecまたはmat-vecを1WARPで実行、mul=1,2,3,...,
  - a2t：a2を転置版で計算するか否か
  - a2interchange：a2のループを入れ替えるか否か
  - aatomic：approxの計算をatomic優先にするかwarp shuffle併用するか
  - datomic：denseの計算をatomic優先にするかwarp shuffle併用するか
*/
  if((kernel>=0)&&(kernel<1536)){
	int subkernel = kernel;
	int div, mul, a2t, a2i, aa, da;
	div = subkernel%6;
	mul = (subkernel/6)%16 + 1;
	a2t = ((subkernel/6)/16)%2;
	a2i = (((subkernel/6)/16)/2)%2;
	aa = ((((subkernel/6)/16)/2)/2)%2;
	da = (((((subkernel/6)/16)/2)/2)/2)%2;
	div = pow(2,div);
	char name[0xff], fname[0xff];
	snprintf(name,0xff,"hybrid3_div%d_mul%d_a2t%d_a2i%d_aa%d_da%d_%s", div, mul, a2t, a2i, aa, da, typeid(T).name());
	snprintf(fname,0xff,"result_cuda3_%s.txt", name);
	printf("subkernel=%d = %s\n", subkernel, fname);
	printf("fname = %s\n", fname);
	// EXEC
	if(dump_result)
	  hmvm_cuda_hybrid3_proxy<T>
		(d_v, d_b, d_sm.nlf, d_sm.ktmax,
		 d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt,
		 d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat, d_sm.rowmat_t,
		 d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense,
		 (d_sm.napprox+mul-1)/mul+(d_sm.ndense+mul-1)/mul, 32*mul, d_sm.ktmax*sizeof(T)*mul,
		 v, b, nd, fname, 0,
		 div, mul, a2t, a2i, aa, da);
	// BENCH
	if(nbench>0)
	  hmvm_cuda_hybrid3_proxy<T>
		(d_v, d_b, d_sm.nlf, d_sm.ktmax,
		 d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt,
		 d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat, d_sm.rowmat_t,
		 d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense,
		 (d_sm.napprox+mul-1)/mul+(d_sm.ndense+mul-1)/mul, 32*mul, d_sm.ktmax*sizeof(T)*mul,
		 v, b, nd, fname, nbench,
		 div, mul, a2t, a2i, aa, da);
  }
#endif

  // ######## ######## ######## ######## ######## ######## ######## ########
  hipFree(d_sm.ltmtx);
  hipFree(d_sm.ndl);
  hipFree(d_sm.ndt);
  hipFree(d_sm.nstrtl);
  hipFree(d_sm.nstrtt);
  hipFree(d_sm.kt);
  hipFree(d_sm.a1);
  hipFree(d_sm.a2);
  hipFree(d_sm.rowmat);
  hipFree(d_sm.rowmat_t);
  hipFree(d_b);
  hipFree(d_v);

  delete [] v;
  printf("hmvm_cuda3: end\n");
}


// ######## ######## ######## ######## ######## ######## ######## ########
// template関数の実体化のための宣言
// ######## ######## ######## ######## ######## ######## ######## ########
template void hmvm_cuda3<float>(matrix2<float>  *mat2, float *b, int kernel, int dump_result, int nbench);
template void hmvm_cuda3<double>(matrix2<double> *mat2, double *b, int kernel, int dump_result, int nbench);
