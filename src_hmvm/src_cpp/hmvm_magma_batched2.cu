#include "hip/hip_runtime.h"
// -*- C++ -*-
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <typeinfo>
#include <omp.h>
#include <hip/hip_runtime_api.h>
#include <magma_v2.h>

#include "hacapk.h"

#define CHECK_DO(act,msg) {ret=act; if(ret!=hipSuccess){printf("%s failed\n",msg);exit(-1);};}

// ######## ######## ######## ######## ######## ######## ######## ########
/*
  magma batched
  行列積を2つにわける
  (多すぎるとダメらしい)
 */

void  hmvm_magma_batched2_calc
(
 int n1a,
 double **d_As1a, double **d_Xs1a, double **d_Ys1a,
 int *d_Ms1a, int *d_Ns1a,
 int *d_lddas1a, int *d_incxs1a, int *d_incys1a,
 int n1b,
 double **d_As1b, double **d_Xs1b, double **d_Ys1b,
 int *d_Ms1b, int *d_Ns1b,
 int *d_lddas1b, int *d_incxs1b, int *d_incys1b,
 int n2a,
 double **d_As2a, double **d_Xs2a, double **d_Ys2a,
 int *d_Ms2a, int *d_Ns2a,
 int *d_lddas2a, int *d_incxs2a, int *d_incys2a,
 int n2b,
 double **d_As2b, double **d_Xs2b, double **d_Ys2b,
 int *d_Ms2b, int *d_Ns2b,
 int *d_lddas2b, int *d_incxs2b, int *d_incys2b,
 magma_queue_t queue
 )
{
  double done=1.0, dzero=0.0;

  magmablas_dgemv_vbatched
	(MagmaTrans, d_Ms1a, d_Ns1a,
	 done, d_As1a, d_lddas1a,
	       d_Xs1a, d_incxs1a,
	dzero, d_Ys1a, d_incys1a, n1a, queue);
  magmablas_dgemv_vbatched_atomic
	(MagmaNoTrans, d_Ms1b, d_Ns1b,
	 done, d_As1b, d_lddas1b,
	       d_Xs1b, d_incxs1b,
	       d_Ys1b, d_incys1b, n1b, queue);
  magmablas_dgemv_vbatched_atomic
	(MagmaTrans, d_Ms2a, d_Ns2a,
	 done, d_As2a, d_lddas2a,
	       d_Xs2a, d_incxs2a,
	       d_Ys2a, d_incys2a, n2a, queue);
  magmablas_dgemv_vbatched_atomic
	(MagmaTrans, d_Ms2b, d_Ns2b,
	 done, d_As2b, d_lddas2b,
	       d_Xs2b, d_incxs2b,
	       d_Ys2b, d_incys2b, n2b, queue);
}

void  hmvm_magma_batched2_calc
(
 int n1a,
 float **d_As1a, float **d_Xs1a, float **d_Ys1a,
 int *d_Ms1a, int *d_Ns1a,
 int *d_lddas1a, int *d_incxs1a, int *d_incys1a,
 int n1b,
 float **d_As1b, float **d_Xs1b, float **d_Ys1b,
 int *d_Ms1b, int *d_Ns1b,
 int *d_lddas1b, int *d_incxs1b, int *d_incys1b,
 int n2a,
 float **d_As2a, float **d_Xs2a, float **d_Ys2a,
 int *d_Ms2a, int *d_Ns2a,
 int *d_lddas2a, int *d_incxs2a, int *d_incys2a,
 int n2b,
 float **d_As2b, float **d_Xs2b, float **d_Ys2b,
 int *d_Ms2b, int *d_Ns2b,
 int *d_lddas2b, int *d_incxs2b, int *d_incys2b,
 magma_queue_t queue
 )
{
  float done=1.0f, dzero=0.0f;

  magmablas_sgemv_vbatched
	(MagmaTrans, d_Ms1a, d_Ns1a,
	 done, d_As1a, d_lddas1a,
	       d_Xs1a, d_incxs1a,
	dzero, d_Ys1a, d_incys1a, n1a, queue);
  magmablas_sgemv_vbatched_atomic
	(MagmaNoTrans, d_Ms1b, d_Ns1b,
	 done, d_As1b, d_lddas1b,
	       d_Xs1b, d_incxs1b,
	       d_Ys1b, d_incys1b, n1b, queue);
  magmablas_sgemv_vbatched_atomic
	(MagmaTrans, d_Ms2a, d_Ns2a,
	 done, d_As2a, d_lddas2a,
	       d_Xs2a, d_incxs2a,
	       d_Ys2a, d_incys2a, n2a, queue);
  magmablas_sgemv_vbatched_atomic
	(MagmaTrans, d_Ms2b, d_Ns2b,
	 done, d_As2b, d_lddas2b,
	       d_Xs2b, d_incxs2b,
	       d_Ys2b, d_incys2b, n2b, queue);
}

void  hmvm_magma_batched2_calc_nocheck
(
 int n1a,
 double **d_As1a, double **d_Xs1a, double **d_Ys1a,
 int *d_Ms1a, int *d_Ns1a,
 int *d_lddas1a, int *d_incxs1a, int *d_incys1a,
 int n1b,
 double **d_As1b, double **d_Xs1b, double **d_Ys1b,
 int *d_Ms1b, int *d_Ns1b,
 int *d_lddas1b, int *d_incxs1b, int *d_incys1b,
 int n2a,
 double **d_As2a, double **d_Xs2a, double **d_Ys2a,
 int *d_Ms2a, int *d_Ns2a,
 int *d_lddas2a, int *d_incxs2a, int *d_incys2a,
 int n2b,
 double **d_As2b, double **d_Xs2b, double **d_Ys2b,
 int *d_Ms2b, int *d_Ns2b,
 int *d_lddas2b, int *d_incxs2b, int *d_incys2b,
 magma_queue_t queue
 )
{
  double done=1.0, dzero=0.0;

  magmablas_dgemv_vbatched_nocheck
	(MagmaTrans, d_Ms1a, d_Ns1a,
	 done, d_As1a, d_lddas1a,
	       d_Xs1a, d_incxs1a,
	dzero, d_Ys1a, d_incys1a, n1a, queue);
  magmablas_dgemv_vbatched_nocheck_atomic
	(MagmaNoTrans, d_Ms1b, d_Ns1b,
	 done, d_As1b, d_lddas1b,
	       d_Xs1b, d_incxs1b,
	       d_Ys1b, d_incys1b, n1b, queue);
  magmablas_dgemv_vbatched_nocheck_atomic
	(MagmaTrans, d_Ms2a, d_Ns2a,
	 done, d_As2a, d_lddas2a,
	       d_Xs2a, d_incxs2a,
	       d_Ys2a, d_incys2a, n2a, queue);
  magmablas_dgemv_vbatched_nocheck_atomic
	(MagmaTrans, d_Ms2b, d_Ns2b,
	 done, d_As2b, d_lddas2b,
	       d_Xs2b, d_incxs2b,
	       d_Ys2b, d_incys2b, n2b, queue);
}

void  hmvm_magma_batched2_calc_nocheck
(
 int n1a,
 float **d_As1a, float **d_Xs1a, float **d_Ys1a,
 int *d_Ms1a, int *d_Ns1a,
 int *d_lddas1a, int *d_incxs1a, int *d_incys1a,
 int n1b,
 float **d_As1b, float **d_Xs1b, float **d_Ys1b,
 int *d_Ms1b, int *d_Ns1b,
 int *d_lddas1b, int *d_incxs1b, int *d_incys1b,
 int n2a,
 float **d_As2a, float **d_Xs2a, float **d_Ys2a,
 int *d_Ms2a, int *d_Ns2a,
 int *d_lddas2a, int *d_incxs2a, int *d_incys2a,
 int n2b,
 float **d_As2b, float **d_Xs2b, float **d_Ys2b,
 int *d_Ms2b, int *d_Ns2b,
 int *d_lddas2b, int *d_incxs2b, int *d_incys2b,
 magma_queue_t queue
 )
{
  float done=1.0f, dzero=0.0f;

  magmablas_sgemv_vbatched_nocheck
	(MagmaTrans, d_Ms1a, d_Ns1a,
	 done, d_As1a, d_lddas1a,
	       d_Xs1a, d_incxs1a,
	dzero, d_Ys1a, d_incys1a, n1a, queue);
  magmablas_sgemv_vbatched_nocheck_atomic
	(MagmaNoTrans, d_Ms1b, d_Ns1b,
	 done, d_As1b, d_lddas1b,
	       d_Xs1b, d_incxs1b,
	       d_Ys1b, d_incys1b, n1b, queue);
  magmablas_sgemv_vbatched_nocheck_atomic
	(MagmaTrans, d_Ms2a, d_Ns2a,
	 done, d_As2a, d_lddas2a,
	       d_Xs2a, d_incxs2a,
	       d_Ys2a, d_incys2a, n2a, queue);
  magmablas_sgemv_vbatched_nocheck_atomic
	(MagmaTrans, d_Ms2b, d_Ns2b,
	 done, d_As2b, d_lddas2b,
	       d_Xs2b, d_incxs2b,
	       d_Ys2b, d_incys2b, n2b, queue);
}

template <class T>
void hmvm_magma_batched2_proxy
(
 int nbatch1a,
 T **d_As1a, T **d_Xs1a, T **d_Ys1a,
 int *d_Ms1a, int *d_Ns1a,
 int *d_lddas1a, int *d_incxs1a, int *d_incys1a,
 int nbatch1b,
 T **d_As1b, T **d_Xs1b, T **d_Ys1b,
 int *d_Ms1b, int *d_Ns1b,
 int *d_lddas1b, int *d_incxs1b, int *d_incys1b,
 int nbatch2a,
 T **d_As2a, T **d_Xs2a, T **d_Ys2a,
 int *d_Ms2a, int *d_Ns2a,
 int *d_lddas2a, int *d_incxs2a, int *d_incys2a,
 int nbatch2b,
 T **d_As2b, T **d_Xs2b, T **d_Ys2b,
 int *d_Ms2b, int *d_Ns2b,
 int *d_lddas2b, int *d_incxs2b, int *d_incys2b,
 magma_queue_t queue, int opt,
 T *d_v, int nd, char *fname, int bench
){
#if 1
  int M=5, L=M+bench;
  FILE *F;
  int i, l, lmax;
  double d1, d2, *dtimes, dmin, dmax, davg;
  hipError_t ret;
  T *v;
  v = new T[nd];
  dtimes = new double[L];
  if(bench==0){lmax=1;}else{lmax=L;}
  for(l=0;l<lmax;l++){
	for(i=0;i<nd;i++)v[i] = (T)0.0;
	CHECK_DO(hipMemcpy(d_v, v, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy v to d_v");
	//CHECK_DO(hipMemcpy(d_b, b, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy b to d_b");
	hipDeviceSynchronize();
	d1 = omp_get_wtime();

	if(opt==0){
	  hmvm_magma_batched2_calc
		(
		 nbatch1a, d_As1a, d_Xs1a, d_Ys1a, d_Ms1a, d_Ns1a, d_lddas1a, d_incxs1a, d_incys1a,
		 nbatch1b, d_As1b, d_Xs1b, d_Ys1b, d_Ms1b, d_Ns1b, d_lddas1b, d_incxs1b, d_incys1b,
		 nbatch2a, d_As2a, d_Xs2a, d_Ys2a, d_Ms2a, d_Ns2a, d_lddas2a, d_incxs2a, d_incys2a,
		 nbatch2b, d_As2b, d_Xs2b, d_Ys2b, d_Ms2b, d_Ns2b, d_lddas2b, d_incxs2b, d_incys2b,
		 queue
		 );
	}else{
	  hmvm_magma_batched2_calc_nocheck
		(
		 nbatch1a, d_As1a, d_Xs1a, d_Ys1a, d_Ms1a, d_Ns1a, d_lddas1a, d_incxs1a, d_incys1a,
		 nbatch1b, d_As1b, d_Xs1b, d_Ys1b, d_Ms1b, d_Ns1b, d_lddas1b, d_incxs1b, d_incys1b,
		 nbatch2a, d_As2a, d_Xs2a, d_Ys2a, d_Ms2a, d_Ns2a, d_lddas2a, d_incxs2a, d_incys2a,
		 nbatch2b, d_As2b, d_Xs2b, d_Ys2b, d_Ms2b, d_Ns2b, d_lddas2b, d_incxs2b, d_incys2b,
		 queue
		 );
	}

	hipDeviceSynchronize();
	d2 = omp_get_wtime();
	dtimes[l] = d2-d1;
  }
  if(bench==0){
	CHECK_DO(hipMemcpy(v, d_v, sizeof(T)*nd, hipMemcpyDeviceToHost),"hipMemcpy d_v to v");
	printf("write to %s\n", fname);
	F = fopen(fname, "w");
	for(i=0;i<nd;i++)fprintf(F, "%.3E\n", v[i]);
	fclose(F);
  }else{
	dmin = 9999.99;
	dmax = 0.0;
	davg = 0.0;
	for(i=M;i<L;i++){
	  davg += dtimes[i];
	  if(dmin>dtimes[i])dmin=dtimes[i];
	  if(dmax<dtimes[i])dmax=dtimes[i];
	}
	davg /= (L-M);
	printf("TIME %d hmvm_magma_batched2%s min %e max %e avg %e\n", L-M, typeid(T).name(), dmin, dmax, davg);
  }
  delete [] dtimes;
#endif
}

// ######## ######## ######## ######## ######## ######## ######## ########

// ######## ######## ######## ######## ######## ######## ######## ########

template<class T>
void hmvm_magma_batched2(matrix2<T> *mat2, T *b, int kernel, int dump_result, int nbench)
{
  matrix2<T> d_sm;
  int i, nd = mat2->nd, ktmax = mat2->ktmax, nlf = mat2->nlf;
  T *v=NULL;
  T *d_b, *d_v;
  int ip;
  int len;
  hipError_t ret;
  printf("hmvm_magma_batched2_%s: begin\n", typeid(T).name()); fflush(stdout);
  v    = new T[nd];
  for(i=0;i<nd;i++){
	v[i] = (T)0.0;
  }
  CHECK_DO(hipMalloc((void**)&d_b, sizeof(T)*nd),"hipMalloc d_b");
  CHECK_DO(hipMalloc((void**)&d_v, sizeof(T)*nd),"hipMalloc d_v");
  CHECK_DO(hipMemcpy(d_b, b, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy b to d_b");

  printf("nd = %d\n", nd);												\
  len = mat2->len;
  printf("total length = %d\n", len);
  // host alloc
  // device alloc
  d_sm.nd    = nd;
  d_sm.nlf   = nlf;
  d_sm.ktmax = ktmax;
  hipMalloc((void**)&d_sm.ltmtx, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.ndl, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.ndt, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.nstrtl, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.nstrtt, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.kt, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.a1, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.a2, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.rowmat,sizeof(T)*mat2->len);
  hipMalloc((void**)&d_sm.rowmat_t,sizeof(T)*mat2->len);
  // memcpy
  CHECK_DO(hipMemcpy(d_sm.ltmtx, mat2->ltmtx, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.ltmtx");
  CHECK_DO(hipMemcpy(d_sm.ndt, mat2->ndt, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.ndt");
  CHECK_DO(hipMemcpy(d_sm.ndl, mat2->ndl, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.ndl");
  CHECK_DO(hipMemcpy(d_sm.nstrtl, mat2->nstrtl, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.nstrtl");
  CHECK_DO(hipMemcpy(d_sm.nstrtt, mat2->nstrtt, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.nstrtt");
  CHECK_DO(hipMemcpy(d_sm.kt, mat2->kt, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.kt");
  CHECK_DO(hipMemcpy(d_sm.a1, mat2->a1, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.a1");
  CHECK_DO(hipMemcpy(d_sm.a2, mat2->a2, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.a2");
  CHECK_DO(hipMemcpy(d_sm.rowmat, mat2->rowmat, sizeof(T)*mat2->len, hipMemcpyHostToDevice),"d_sm.rowmat");
  CHECK_DO(hipMemcpy(d_sm.rowmat_t, mat2->rowmat_t, sizeof(T)*mat2->len, hipMemcpyHostToDevice),"d_sm.rowmat_t");

#if 0
  // 分離
  printf("begin splitting\n");
  mat2->approx = (int*)malloc(sizeof(int)*nlf);
  mat2->dense  = (int*)malloc(sizeof(int)*nlf);
  mat2->napprox = mat2->ndense = 0;
  for(ip=0; ip<nlf; ip++){
	if(mat2->ltmtx[ip]==1){
	  mat2->approx[mat2->napprox++] = ip;
	}else{
	  mat2->dense[mat2->ndense++] = ip;
	}
  }
  hipMalloc((void**)&d_sm.approx,sizeof(int)*mat2->napprox);
  hipMalloc((void**)&d_sm.dense,sizeof(int)*mat2->ndense);
  CHECK_DO(hipMemcpy(d_sm.approx, mat2->approx, sizeof(int)*mat2->napprox, hipMemcpyHostToDevice),"d_sm.approx");
  CHECK_DO(hipMemcpy(d_sm.dense, mat2->dense, sizeof(int)*mat2->ndense, hipMemcpyHostToDevice),"d_sm.dense");
  d_sm.napprox = mat2->napprox;
  d_sm.ndense  = mat2->ndense;
  printf("end splitting (napprox=%d, ndense=%d)\n", mat2->napprox, mat2->ndense);
#endif

  int n, kt, ndl, ndt, nstrtl, nstrtt, ltmtx, head;
  magma_device_t dev;
  magma_queue_t queue;
  magma_queue_create(dev, &queue);
  int nbatch1a, nbatch1b, nbatch2a, nbatch2b;
  T **tmpvec;
  tmpvec = (T**)malloc(sizeof(T*)*nlf);
  for(ip=0;ip<nlf;ip++){
	CHECK_DO(hipMalloc((void**)&tmpvec[ip],sizeof(T)*mat2->kt[ip]),"hipMalloc ");
	//hipMemcpy(tmpvec[ip], tmpzero, sizeof(T)*mat2->kt[ip], hipMemcpyHostToDevice);
  }

  // host
  T **h_As1a, **h_Xs1a, **h_Ys1a;
  int *h_Ms1a, *h_Ns1a;
  int *h_lddas1a, *h_incxs1a, *h_incys1a;
  T **h_As1b, **h_Xs1b, **h_Ys1b;
  int *h_Ms1b, *h_Ns1b;
  int *h_lddas1b, *h_incxs1b, *h_incys1b;
  T **h_As2a, **h_Xs2a, **h_Ys2a;
  int *h_Ms2a, *h_Ns2a;
  int *h_lddas2a, *h_incxs2a, *h_incys2a;
  T **h_As2b, **h_Xs2b, **h_Ys2b;
  int *h_Ms2b, *h_Ns2b;
  int *h_lddas2b, *h_incxs2b, *h_incys2b;
  // device
  T **d_As1a, **d_Xs1a, **d_Ys1a;
  int *d_Ms1a, *d_Ns1a;
  int *d_lddas1a, *d_incxs1a, *d_incys1a;
  T **d_As1b, **d_Xs1b, **d_Ys1b;
  int *d_Ms1b, *d_Ns1b;
  int *d_lddas1b, *d_incxs1b, *d_incys1b;
  T **d_As2a, **d_Xs2a, **d_Ys2a;
  int *d_Ms2a, *d_Ns2a;
  int *d_lddas2a, *d_incxs2a, *d_incys2a;
  T **d_As2b, **d_Xs2b, **d_Ys2b;
  int *d_Ms2b, *d_Ns2b;
  int *d_lddas2b, *d_incxs2b, *d_incys2b;
  // allocation
  // 1a approx. 1
  h_As1a = (T**)malloc(sizeof(T*)*(nlf+1));
  h_Xs1a = (T**)malloc(sizeof(T*)*(nlf+1));
  h_Ys1a = (T**)malloc(sizeof(T*)*(nlf+1));
  h_Ms1a = (int*)malloc(sizeof(int)*(nlf+1));
  h_Ns1a = (int*)malloc(sizeof(int)*(nlf+1));
  h_lddas1a = (int*)malloc(sizeof(int)*(nlf+1));
  h_incxs1a = (int*)malloc(sizeof(int)*(nlf+1));
  h_incys1a = (int*)malloc(sizeof(int)*(nlf+1));
  CHECK_DO(hipMalloc((void**)&d_As1a, sizeof(T*)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_Xs1a, sizeof(T*)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_Ys1a, sizeof(T*)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_Ms1a, sizeof(int)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_Ns1a, sizeof(int)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_lddas1a, sizeof(int)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_incxs1a, sizeof(int)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_incys1a, sizeof(int)*(nlf+1)),"hipMalloc ");
  // 1b. approx. 2
  h_As1b = (T**)malloc(sizeof(T*)*(nlf+1));
  h_Xs1b = (T**)malloc(sizeof(T*)*(nlf+1));
  h_Ys1b = (T**)malloc(sizeof(T*)*(nlf+1));
  h_Ms1b = (int*)malloc(sizeof(int)*(nlf+1));
  h_Ns1b = (int*)malloc(sizeof(int)*(nlf+1));
  h_lddas1b = (int*)malloc(sizeof(int)*(nlf+1));
  h_incxs1b = (int*)malloc(sizeof(int)*(nlf+1));
  h_incys1b = (int*)malloc(sizeof(int)*(nlf+1));
  CHECK_DO(hipMalloc((void**)&d_As1b, sizeof(T*)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_Xs1b, sizeof(T*)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_Ys1b, sizeof(T*)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_Ms1b, sizeof(int)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_Ns1b, sizeof(int)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_lddas1b, sizeof(int)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_incxs1b, sizeof(int)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_incys1b, sizeof(int)*(nlf+1)),"hipMalloc ");
  // 2a. dense. 1
  h_As2a = (T**)malloc(sizeof(T*)*(nlf+1));
  h_Xs2a = (T**)malloc(sizeof(T*)*(nlf+1));
  h_Ys2a = (T**)malloc(sizeof(T*)*(nlf+1));
  h_Ms2a = (int*)malloc(sizeof(int)*(nlf+1));
  h_Ns2a = (int*)malloc(sizeof(int)*(nlf+1));
  h_lddas2a = (int*)malloc(sizeof(int)*(nlf+1));
  h_incxs2a = (int*)malloc(sizeof(int)*(nlf+1));
  h_incys2a = (int*)malloc(sizeof(int)*(nlf+1));
  CHECK_DO(hipMalloc((void**)&d_As2a, sizeof(T*)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_Xs2a, sizeof(T*)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_Ys2a, sizeof(T*)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_Ms2a, sizeof(int)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_Ns2a, sizeof(int)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_lddas2a, sizeof(int)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_incxs2a, sizeof(int)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_incys2a, sizeof(int)*(nlf+1)),"hipMalloc ");
  // 2b. dense. 2
  h_As2b = (T**)malloc(sizeof(T*)*(nlf+1));
  h_Xs2b = (T**)malloc(sizeof(T*)*(nlf+1));
  h_Ys2b = (T**)malloc(sizeof(T*)*(nlf+1));
  h_Ms2b = (int*)malloc(sizeof(int)*(nlf+1));
  h_Ns2b = (int*)malloc(sizeof(int)*(nlf+1));
  h_lddas2b = (int*)malloc(sizeof(int)*(nlf+1));
  h_incxs2b = (int*)malloc(sizeof(int)*(nlf+1));
  h_incys2b = (int*)malloc(sizeof(int)*(nlf+1));
  CHECK_DO(hipMalloc((void**)&d_As2b, sizeof(T*)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_Xs2b, sizeof(T*)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_Ys2b, sizeof(T*)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_Ms2b, sizeof(int)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_Ns2b, sizeof(int)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_lddas2b, sizeof(int)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_incxs2b, sizeof(int)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_incys2b, sizeof(int)*(nlf+1)),"hipMalloc ");
  // construction
  // 1a. approx. 1
  n=0;
  for(ip=0; ip<nlf; ip++){
	ndl    = mat2->ndl[ip];
	ndt    = mat2->ndt[ip];
	nstrtl = mat2->nstrtl[ip];
	nstrtt = mat2->nstrtt[ip];
	ltmtx  = mat2->ltmtx[ip];
	if(ltmtx==1){
	  kt     = mat2->kt[ip];
	  head = mat2->a1[ip];
	  h_As1a[n] = &d_sm.rowmat[head];
	  h_Xs1a[n] = &d_b[nstrtt-1];
	  h_Ys1a[n] = tmpvec[ip];
	  h_Ms1a[n] = ndt;
	  h_Ns1a[n] = kt;
	  h_lddas1a[n] = ndt;
	  h_incxs1a[n] = 1;
	  h_incys1a[n] = 1;
	  n++;
	  //hipblasDgemv(handle,HIPBLAS_OP_T, ndt,kt, &done, &d_sm.rowmat[head], ndt,&d_zu[nstrtt-1],1,&dzero,d_zbut,1);
	} else if(ltmtx==2){
	}
  }
  nbatch1a = n;
  hipMemcpy(d_As1a, h_As1a, sizeof(T*)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_Xs1a, h_Xs1a, sizeof(T*)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_Ys1a, h_Ys1a, sizeof(T*)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_Ms1a, h_Ms1a, sizeof(int)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_Ns1a, h_Ns1a, sizeof(int)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_lddas1a, h_lddas1a, sizeof(int)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_incxs1a, h_incxs1a, sizeof(int)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_incys1a, h_incys1a, sizeof(int)*n, hipMemcpyHostToDevice);
  // 1b. approx. 2
  n = 0;
  for(ip=0; ip<nlf; ip++){
	ndl    = mat2->ndl[ip];
	ndt    = mat2->ndt[ip];
	nstrtl = mat2->nstrtl[ip];
	nstrtt = mat2->nstrtt[ip];
	ltmtx  = mat2->ltmtx[ip];
	if(ltmtx==1){
	  kt = mat2->kt[ip];
	  head = mat2->a2[ip];
	  h_As1b[n] = &d_sm.rowmat[head];
	  h_Xs1b[n] = tmpvec[ip];
	  h_Ys1b[n] = &d_v[nstrtl-1];
	  h_Ms1b[n] = ndl;
	  h_Ns1b[n] = kt;
	  h_lddas1b[n] = ndl;
	  h_incxs1b[n] = 1;
	  h_incys1b[n] = 1;
	  n++;
	  //hipblasDgemv(handle,HIPBLAS_OP_N, ndl,kt, &done, &d_sm.rowmat[head], ndl,d_zbut,1,&done,&d_zaut[nstrtl-1],1);
	} else if(ltmtx==2){
	}
  }
  nbatch1b = n;
  hipMemcpy(d_As1b, h_As1b, sizeof(T*)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_Xs1b, h_Xs1b, sizeof(T*)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_Ys1b, h_Ys1b, sizeof(T*)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_Ms1b, h_Ms1b, sizeof(int)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_Ns1b, h_Ns1b, sizeof(int)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_lddas1b, h_lddas1b, sizeof(int)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_incxs1b, h_incxs1b, sizeof(int)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_incys1b, h_incys1b, sizeof(int)*n, hipMemcpyHostToDevice);
  // 2a. dense 1
  n = 0;
  for(ip=0; ip<nlf/2; ip++){
	ndl    = mat2->ndl[ip];
	ndt    = mat2->ndt[ip];
	nstrtl = mat2->nstrtl[ip];
	nstrtt = mat2->nstrtt[ip];
	ltmtx  = mat2->ltmtx[ip];
	if(ltmtx==1){
	} else if(ltmtx==2){
	  head = mat2->a1[ip];
	  h_As2a[n] = &d_sm.rowmat[head];
	  h_Xs2a[n] = &d_b[nstrtt-1];
	  h_Ys2a[n] = &d_v[nstrtl-1];
	  h_Ms2a[n] = ndt;
	  h_Ns2a[n] = ndl;
	  h_lddas2a[n] = ndt;
	  h_incxs2a[n] = 1;
	  h_incys2a[n] = 1;
	  n++;
	  //hipblasDgemv(handle,HIPBLAS_OP_T, ndt,ndl, &done, &d_sm.rowmat[head], ndt,&d_zu[nstrtt-1],1,&done,&d_zaut[nstrtl-1],1);
	}
  }
  nbatch2a = n;
  hipMemcpy(d_As2a, h_As2a, sizeof(T*)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_Xs2a, h_Xs2a, sizeof(T*)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_Ys2a, h_Ys2a, sizeof(T*)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_Ms2a, h_Ms2a, sizeof(int)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_Ns2a, h_Ns2a, sizeof(int)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_lddas2a, h_lddas2a, sizeof(int)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_incxs2a, h_incxs2a, sizeof(int)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_incys2a, h_incys2a, sizeof(int)*n, hipMemcpyHostToDevice);
  // 2b. dense 2
  n = 0;
  for(ip=nlf/2; ip<nlf; ip++){
	ndl    = mat2->ndl[ip];
	ndt    = mat2->ndt[ip];
	nstrtl = mat2->nstrtl[ip];
	nstrtt = mat2->nstrtt[ip];
	ltmtx  = mat2->ltmtx[ip];
	if(ltmtx==1){
	} else if(ltmtx==2){
	  head = mat2->a1[ip];
	  h_As2b[n] = &d_sm.rowmat[head];
	  h_Xs2b[n] = &d_b[nstrtt-1];
	  h_Ys2b[n] = &d_v[nstrtl-1];
	  h_Ms2b[n] = ndt;
	  h_Ns2b[n] = ndl;
	  h_lddas2b[n] = ndt;
	  h_incxs2b[n] = 1;
	  h_incys2b[n] = 1;
	  n++;
	  //hipblasDgemv(handle,HIPBLAS_OP_T, ndt,ndl, &done, &d_sm.rowmat[head], ndt,&d_zu[nstrtt-1],1,&done,&d_zaut[nstrtl-1],1);
	}
  }
  nbatch2b = n;
  hipMemcpy(d_As2b, h_As2b, sizeof(T*)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_Xs2b, h_Xs2b, sizeof(T*)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_Ys2b, h_Ys2b, sizeof(T*)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_Ms2b, h_Ms2b, sizeof(int)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_Ns2b, h_Ns2b, sizeof(int)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_lddas2b, h_lddas2b, sizeof(int)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_incxs2b, h_incxs2b, sizeof(int)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_incys2b, h_incys2b, sizeof(int)*n, hipMemcpyHostToDevice);

  /*
	magma batched blas
	0: default
	1: nocheck
  */
  {
	const char *names[]={"default","nocheck"};
	char name[0xff], fname[0xff];
	snprintf(name,0xff,"magma_batched2_%d_%s_%s", kernel, names[kernel], typeid(T).name());
	snprintf(fname,0xff,"result_%s.txt", name);
	printf("fname = %s\n", fname);

	// EXEC
	if(dump_result)hmvm_magma_batched2_proxy<T>
					 (
					  nbatch1a, d_As1a, d_Xs1a, d_Ys1a, d_Ms1a, d_Ns1a, d_lddas1a, d_incxs1a, d_incys1a,
					  nbatch1b, d_As1b, d_Xs1b, d_Ys1b, d_Ms1b, d_Ns1b, d_lddas1b, d_incxs1b, d_incys1b,
					  nbatch2a, d_As2a, d_Xs2a, d_Ys2a, d_Ms2a, d_Ns2a, d_lddas2a, d_incxs2a, d_incys2a,
					  nbatch2b, d_As2b, d_Xs2b, d_Ys2b, d_Ms2b, d_Ns2b, d_lddas2b, d_incxs2b, d_incys2b,
					  queue, kernel, d_v, nd, fname, 0);
	// BENCH
	if(nbench>0)hmvm_magma_batched2_proxy<T>
				  (
				   nbatch1a, d_As1a, d_Xs1a, d_Ys1a, d_Ms1a, d_Ns1a, d_lddas1a, d_incxs1a, d_incys1a,
				   nbatch1b, d_As1b, d_Xs1b, d_Ys1b, d_Ms1b, d_Ns1b, d_lddas1b, d_incxs1b, d_incys1b,
				   nbatch2a, d_As2a, d_Xs2a, d_Ys2a, d_Ms2a, d_Ns2a, d_lddas2a, d_incxs2a, d_incys2a,
				   nbatch2b, d_As2b, d_Xs2b, d_Ys2b, d_Ms2b, d_Ns2b, d_lddas2b, d_incxs2b, d_incys2b,
				   queue, kernel, d_v, nd, fname, nbench);
  }
  magma_queue_destroy(queue);

  // ######## ######## ######## ######## ######## ######## ######## ########
  hipFree(d_As1a); hipFree(d_Xs1a); hipFree(d_Ys1a); hipFree(d_Ms1a); hipFree(d_Ns1a);
  hipFree(d_lddas1a); hipFree(d_incxs1a); hipFree(d_incys1a);
  hipFree(d_As1b); hipFree(d_Xs1b); hipFree(d_Ys1b); hipFree(d_Ms1b); hipFree(d_Ns1b);
  hipFree(d_lddas1b); hipFree(d_incxs1b); hipFree(d_incys1b);
  hipFree(d_As2a); hipFree(d_Xs2a); hipFree(d_Ys2a); hipFree(d_Ms2a); hipFree(d_Ns2a);
  hipFree(d_lddas2a); hipFree(d_incxs2a); hipFree(d_incys2a);
  hipFree(d_As2b); hipFree(d_Xs2b); hipFree(d_Ys2b); hipFree(d_Ms2b); hipFree(d_Ns2b);
  hipFree(d_lddas2b); hipFree(d_incxs2b); hipFree(d_incys2a);
  for(ip=0;ip<nlf;ip++)hipFree(tmpvec[ip]);
  free(tmpvec);

  hipFree(d_sm.ltmtx);
  hipFree(d_sm.ndl);
  hipFree(d_sm.ndt);
  hipFree(d_sm.nstrtl);
  hipFree(d_sm.nstrtt);
  hipFree(d_sm.kt);
  hipFree(d_sm.a1);
  hipFree(d_sm.a2);
  hipFree(d_sm.rowmat);
  hipFree(d_sm.rowmat_t);
  hipFree(d_b);
  hipFree(d_v);

  delete [] v;
  printf("hmvm_magma_batched2: end\n");
}


// ######## ######## ######## ######## ######## ######## ######## ########
// template関数の実体化のための宣言
// ######## ######## ######## ######## ######## ######## ######## ########
template void hmvm_magma_batched2<float>(matrix2<float>  *mat2, float *b, int kernel, int dump_result, int nbench);
template void hmvm_magma_batched2<double>(matrix2<double> *mat2, double *b, int kernel, int dump_result, int nbench);
