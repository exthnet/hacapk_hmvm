#include "hip/hip_runtime.h"
// -*- C++ -*-
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <typeinfo>
#include <omp.h>
#include <hip/hip_runtime_api.h>
#include <magma_v2.h>

#include "hacapk.h"

#define CHECK_DO(act,msg) {ret=act; if(ret!=hipSuccess){printf("%s failed\n",msg);exit(-1);};}

// ######## ######## ######## ######## ######## ######## ######## ########
/*
  magma batched simple blas
 */

void  hmvm_magma_batched_calc
(
 int n1a,
 double **d_As1a, double **d_Xs1a, double **d_Ys1a,
 int *d_Ms1a, int *d_Ns1a,
 int *d_lddas1a, int *d_incxs1a, int *d_incys1a,
 int n1b,
 double **d_As1b, double **d_Xs1b, double **d_Ys1b,
 int *d_Ms1b, int *d_Ns1b,
 int *d_lddas1b, int *d_incxs1b, int *d_incys1b,
 int n2,
 double **d_As2, double **d_Xs2, double **d_Ys2,
 int *d_Ms2, int *d_Ns2,
 int *d_lddas2, int *d_incxs2, int *d_incys2,
 magma_queue_t queue
 )
{
  double done=1.0, dzero=0.0;

  magmablas_dgemv_vbatched
	(MagmaTrans, d_Ms1a, d_Ns1a,
	 done, d_As1a, d_lddas1a,
	       d_Xs1a, d_incxs1a,
	dzero, d_Ys1a, d_incys1a, n1a, queue);

  magmablas_dgemv_vbatched_atomic
	(MagmaNoTrans, d_Ms1b, d_Ns1b,
	 done, d_As1b, d_lddas1b,
	       d_Xs1b, d_incxs1b,
	       d_Ys1b, d_incys1b, n1b, queue);
  magmablas_dgemv_vbatched_atomic
	(MagmaTrans, d_Ms2, d_Ns2,
	 done, d_As2, d_lddas2,
	       d_Xs2, d_incxs2,
	       d_Ys2, d_incys2, n2, queue);
}

void  hmvm_magma_batched_calc
(
 int n1a,
 float **d_As1a, float **d_Xs1a, float **d_Ys1a,
 int *d_Ms1a, int *d_Ns1a,
 int *d_lddas1a, int *d_incxs1a, int *d_incys1a,
 int n1b,
 float **d_As1b, float **d_Xs1b, float **d_Ys1b,
 int *d_Ms1b, int *d_Ns1b,
 int *d_lddas1b, int *d_incxs1b, int *d_incys1b,
 int n2,
 float **d_As2, float **d_Xs2, float **d_Ys2,
 int *d_Ms2, int *d_Ns2,
 int *d_lddas2, int *d_incxs2, int *d_incys2,
 magma_queue_t queue
 )
{
  float done=1.0f, dzero=0.0f;

  magmablas_sgemv_vbatched
	(MagmaTrans, d_Ms1a, d_Ns1a,
	 done, d_As1a, d_lddas1a,
	       d_Xs1a, d_incxs1a,
	dzero, d_Ys1a, d_incys1a, n1a, queue);

  magmablas_sgemv_vbatched_atomic
	(MagmaNoTrans, d_Ms1b, d_Ns1b,
	 done, d_As1b, d_lddas1b,
	       d_Xs1b, d_incxs1b,
	       d_Ys1b, d_incys1b, n1b, queue);
  magmablas_sgemv_vbatched_atomic
	(MagmaTrans, d_Ms2, d_Ns2,
	 done, d_As2, d_lddas2,
	       d_Xs2, d_incxs2,
	       d_Ys2, d_incys2, n2, queue);
}

void  hmvm_magma_batched_calc_nocheck
(
 int n1a,
 double **d_As1a, double **d_Xs1a, double **d_Ys1a,
 int *d_Ms1a, int *d_Ns1a,
 int *d_lddas1a, int *d_incxs1a, int *d_incys1a,
 int n1b,
 double **d_As1b, double **d_Xs1b, double **d_Ys1b,
 int *d_Ms1b, int *d_Ns1b,
 int *d_lddas1b, int *d_incxs1b, int *d_incys1b,
 int n2,
 double **d_As2, double **d_Xs2, double **d_Ys2,
 int *d_Ms2, int *d_Ns2,
 int *d_lddas2, int *d_incxs2, int *d_incys2,
 magma_queue_t queue
 )
{
  double done=1.0, dzero=0.0;

  magmablas_dgemv_vbatched_nocheck
	(MagmaTrans, d_Ms1a, d_Ns1a,
	 done, d_As1a, d_lddas1a,
	       d_Xs1a, d_incxs1a,
	dzero, d_Ys1a, d_incys1a, n1a, queue);

  magmablas_dgemv_vbatched_nocheck_atomic
	(MagmaNoTrans, d_Ms1b, d_Ns1b,
	 done, d_As1b, d_lddas1b,
	       d_Xs1b, d_incxs1b,
	       d_Ys1b, d_incys1b, n1b, queue);
  magmablas_dgemv_vbatched_nocheck_atomic
	(MagmaTrans, d_Ms2, d_Ns2,
	 done, d_As2, d_lddas2,
	       d_Xs2, d_incxs2,
	       d_Ys2, d_incys2, n2, queue);
}

void  hmvm_magma_batched_calc_nocheck
(
 int n1a,
 float **d_As1a, float **d_Xs1a, float **d_Ys1a,
 int *d_Ms1a, int *d_Ns1a,
 int *d_lddas1a, int *d_incxs1a, int *d_incys1a,
 int n1b,
 float **d_As1b, float **d_Xs1b, float **d_Ys1b,
 int *d_Ms1b, int *d_Ns1b,
 int *d_lddas1b, int *d_incxs1b, int *d_incys1b,
 int n2,
 float **d_As2, float **d_Xs2, float **d_Ys2,
 int *d_Ms2, int *d_Ns2,
 int *d_lddas2, int *d_incxs2, int *d_incys2,
 magma_queue_t queue
 )
{
  float done=1.0f, dzero=0.0f;

  magmablas_sgemv_vbatched_nocheck
	(MagmaTrans, d_Ms1a, d_Ns1a,
	 done, d_As1a, d_lddas1a,
	       d_Xs1a, d_incxs1a,
	dzero, d_Ys1a, d_incys1a, n1a, queue);

  magmablas_sgemv_vbatched_nocheck_atomic
	(MagmaNoTrans, d_Ms1b, d_Ns1b,
	 done, d_As1b, d_lddas1b,
	       d_Xs1b, d_incxs1b,
	       d_Ys1b, d_incys1b, n1b, queue);
  magmablas_sgemv_vbatched_nocheck_atomic
	(MagmaTrans, d_Ms2, d_Ns2,
	 done, d_As2, d_lddas2,
	       d_Xs2, d_incxs2,
	       d_Ys2, d_incys2, n2, queue);
}

template <class T>
void hmvm_magma_batched_proxy
(
 int nbatch1a,
 T **d_As1a, T **d_Xs1a, T **d_Ys1a,
 int *d_Ms1a, int *d_Ns1a,
 int *d_lddas1a, int *d_incxs1a, int *d_incys1a,
 int nbatch1b,
 T **d_As1b, T **d_Xs1b, T **d_Ys1b,
 int *d_Ms1b, int *d_Ns1b,
 int *d_lddas1b, int *d_incxs1b, int *d_incys1b,
 int nbatch2,
 T **d_As2, T **d_Xs2, T **d_Ys2,
 int *d_Ms2, int *d_Ns2,
 int *d_lddas2, int *d_incxs2, int *d_incys2,
 magma_queue_t queue, int opt,
 T *d_v, int nd, char *fname, int bench
){
#if 1
  int M=5, L=M+bench;
  FILE *F;
  int i, l, lmax;
  double d1, d2, *dtimes, dmin, dmax, davg;
  hipError_t ret;
  T *d_tmp, *h_zero;
  T *v;
  v = new T[nd];
  dtimes = new double[L];
  if(bench==0){lmax=1;}else{lmax=L;}
  for(l=0;l<lmax;l++){
	for(i=0;i<nd;i++)v[i] = (T)0.0;
	CHECK_DO(hipMemcpy(d_v, v, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy v to d_v");
	//CHECK_DO(hipMemcpy(d_b, b, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy b to d_b");
	hipDeviceSynchronize();
	d1 = omp_get_wtime();

	if(opt==0){
	  hmvm_magma_batched_calc
		(
		 nbatch1a, d_As1a, d_Xs1a, d_Ys1a, d_Ms1a, d_Ns1a, d_lddas1a, d_incxs1a, d_incys1a,
		 nbatch1b, d_As1b, d_Xs1b, d_Ys1b, d_Ms1b, d_Ns1b, d_lddas1b, d_incxs1b, d_incys1b,
		 nbatch2, d_As2, d_Xs2, d_Ys2, d_Ms2, d_Ns2, d_lddas2, d_incxs2, d_incys2,
		 queue
		 );
	}else{
	  hmvm_magma_batched_calc_nocheck
		(
		 nbatch1a, d_As1a, d_Xs1a, d_Ys1a, d_Ms1a, d_Ns1a, d_lddas1a, d_incxs1a, d_incys1a,
		 nbatch1b, d_As1b, d_Xs1b, d_Ys1b, d_Ms1b, d_Ns1b, d_lddas1b, d_incxs1b, d_incys1b,
		 nbatch2, d_As2, d_Xs2, d_Ys2, d_Ms2, d_Ns2, d_lddas2, d_incxs2, d_incys2,
		 queue
		 );
	}

	hipDeviceSynchronize();
	d2 = omp_get_wtime();
	dtimes[l] = d2-d1;
  }
  if(bench==0){
	CHECK_DO(hipMemcpy(v, d_v, sizeof(T)*nd, hipMemcpyDeviceToHost),"hipMemcpy d_v to v");
	printf("write to %s\n", fname);
	F = fopen(fname, "w");
	for(i=0;i<nd;i++)fprintf(F, "%.3E\n", v[i]);
	fclose(F);
  }else{
	dmin = 9999.99;
	dmax = 0.0;
	davg = 0.0;
	for(i=M;i<L;i++){
	  davg += dtimes[i];
	  if(dmin>dtimes[i])dmin=dtimes[i];
	  if(dmax<dtimes[i])dmax=dtimes[i];
	}
	davg /= (L-M);
	printf("TIME %d hmvm_magma_batched%s min %e max %e avg %e\n", L-M, typeid(T).name(), dmin, dmax, davg);
  }
  delete [] dtimes;
#endif
}

// ######## ######## ######## ######## ######## ######## ######## ########

// ######## ######## ######## ######## ######## ######## ######## ########

template<class T>
void hmvm_magma_batched(matrix2<T> *mat2, T *b, int kernel, int dump_result)
{
  matrix2<T> d_sm;
  int i, l, nd = mat2->nd, ktmax = mat2->ktmax, nlf = mat2->nlf;
  T *v=NULL;
  T *d_b, *d_v;
  int ip;
  int len;
  hipError_t ret;
  printf("hmvm_magma_batched_%s: begin\n", typeid(T).name()); fflush(stdout);
  v    = new T[nd];
  for(i=0;i<nd;i++){
	v[i] = (T)0.0;
  }
  CHECK_DO(hipMalloc((void**)&d_b, sizeof(T)*nd),"hipMalloc d_b");
  CHECK_DO(hipMalloc((void**)&d_v, sizeof(T)*nd),"hipMalloc d_v");
  CHECK_DO(hipMemcpy(d_b, b, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy b to d_b");

  printf("nd = %d\n", nd);												\
  len = mat2->len;
  printf("total length = %d\n", len);
  // host alloc
  // device alloc
  d_sm.nd    = nd;
  d_sm.nlf   = nlf;
  d_sm.ktmax = ktmax;
  hipMalloc((void**)&d_sm.ltmtx, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.ndl, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.ndt, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.nstrtl, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.nstrtt, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.kt, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.a1, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.a2, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.rowmat,sizeof(T)*mat2->len);
  hipMalloc((void**)&d_sm.rowmat_t,sizeof(T)*mat2->len);
  // memcpy
  CHECK_DO(hipMemcpy(d_sm.ltmtx, mat2->ltmtx, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.ltmtx");
  CHECK_DO(hipMemcpy(d_sm.ndt, mat2->ndt, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.ndt");
  CHECK_DO(hipMemcpy(d_sm.ndl, mat2->ndl, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.ndl");
  CHECK_DO(hipMemcpy(d_sm.nstrtl, mat2->nstrtl, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.nstrtl");
  CHECK_DO(hipMemcpy(d_sm.nstrtt, mat2->nstrtt, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.nstrtt");
  CHECK_DO(hipMemcpy(d_sm.kt, mat2->kt, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.kt");
  CHECK_DO(hipMemcpy(d_sm.a1, mat2->a1, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.a1");
  CHECK_DO(hipMemcpy(d_sm.a2, mat2->a2, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.a2");
  CHECK_DO(hipMemcpy(d_sm.rowmat, mat2->rowmat, sizeof(T)*mat2->len, hipMemcpyHostToDevice),"d_sm.rowmat");
  CHECK_DO(hipMemcpy(d_sm.rowmat_t, mat2->rowmat_t, sizeof(T)*mat2->len, hipMemcpyHostToDevice),"d_sm.rowmat_t");

#if 0
  // 分離
  printf("begin splitting\n");
  mat2->approx = (int*)malloc(sizeof(int)*nlf);
  mat2->dense  = (int*)malloc(sizeof(int)*nlf);
  mat2->napprox = mat2->ndense = 0;
  for(ip=0; ip<nlf; ip++){
	if(mat2->ltmtx[ip]==1){
	  mat2->approx[mat2->napprox++] = ip;
	}else{
	  mat2->dense[mat2->ndense++] = ip;
	}
  }
  hipMalloc((void**)&d_sm.approx,sizeof(int)*mat2->napprox);
  hipMalloc((void**)&d_sm.dense,sizeof(int)*mat2->ndense);
  CHECK_DO(hipMemcpy(d_sm.approx, mat2->approx, sizeof(int)*mat2->napprox, hipMemcpyHostToDevice),"d_sm.approx");
  CHECK_DO(hipMemcpy(d_sm.dense, mat2->dense, sizeof(int)*mat2->ndense, hipMemcpyHostToDevice),"d_sm.dense");
  d_sm.napprox = mat2->napprox;
  d_sm.ndense  = mat2->ndense;
  printf("end splitting (napprox=%d, ndense=%d)\n", mat2->napprox, mat2->ndense);
#endif

  int n, kt, ndl, ndt, nstrtl, nstrtt, ltmtx, head;
  magma_device_t dev;
  magma_queue_t queue;
  magma_queue_create(dev, &queue);
  int nbatch1a, nbatch1b, nbatch2;
  T **tmpvec;
  tmpvec = (T**)malloc(sizeof(T*)*nlf);
  for(ip=0;ip<nlf;ip++){
	CHECK_DO(hipMalloc((void**)&tmpvec[ip],sizeof(T)*mat2->kt[ip]),"hipMalloc ");
	//hipMemcpy(tmpvec[ip], tmpzero, sizeof(T)*mat2->kt[ip], hipMemcpyHostToDevice);
  }

  // host
  T **h_As1a, **h_Xs1a, **h_Ys1a;
  int *h_Ms1a, *h_Ns1a;
  int *h_lddas1a, *h_incxs1a, *h_incys1a;
  T **h_As1b, **h_Xs1b, **h_Ys1b;
  int *h_Ms1b, *h_Ns1b;
  int *h_lddas1b, *h_incxs1b, *h_incys1b;
  T **h_As2, **h_Xs2, **h_Ys2;
  int *h_Ms2, *h_Ns2;
  int *h_lddas2, *h_incxs2, *h_incys2;
  // device
  T **d_As1a, **d_Xs1a, **d_Ys1a;
  int *d_Ms1a, *d_Ns1a;
  int *d_lddas1a, *d_incxs1a, *d_incys1a;
  T **d_As1b, **d_Xs1b, **d_Ys1b;
  int *d_Ms1b, *d_Ns1b;
  int *d_lddas1b, *d_incxs1b, *d_incys1b;
  T **d_As2, **d_Xs2, **d_Ys2;
  int *d_Ms2, *d_Ns2;
  int *d_lddas2, *d_incxs2, *d_incys2;
  // allocation
  // 1a approx. 1
  h_As1a = (T**)malloc(sizeof(T*)*(nlf+1));
  h_Xs1a = (T**)malloc(sizeof(T*)*(nlf+1));
  h_Ys1a = (T**)malloc(sizeof(T*)*(nlf+1));
  h_Ms1a = (int*)malloc(sizeof(int)*(nlf+1));
  h_Ns1a = (int*)malloc(sizeof(int)*(nlf+1));
  h_lddas1a = (int*)malloc(sizeof(int)*(nlf+1));
  h_incxs1a = (int*)malloc(sizeof(int)*(nlf+1));
  h_incys1a = (int*)malloc(sizeof(int)*(nlf+1));
  CHECK_DO(hipMalloc((void**)&d_As1a, sizeof(T*)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_Xs1a, sizeof(T*)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_Ys1a, sizeof(T*)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_Ms1a, sizeof(int)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_Ns1a, sizeof(int)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_lddas1a, sizeof(int)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_incxs1a, sizeof(int)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_incys1a, sizeof(int)*(nlf+1)),"hipMalloc ");
  // 1b. approx. 2
  h_As1b = (T**)malloc(sizeof(T*)*(nlf+1));
  h_Xs1b = (T**)malloc(sizeof(T*)*(nlf+1));
  h_Ys1b = (T**)malloc(sizeof(T*)*(nlf+1));
  h_Ms1b = (int*)malloc(sizeof(int)*(nlf+1));
  h_Ns1b = (int*)malloc(sizeof(int)*(nlf+1));
  h_lddas1b = (int*)malloc(sizeof(int)*(nlf+1));
  h_incxs1b = (int*)malloc(sizeof(int)*(nlf+1));
  h_incys1b = (int*)malloc(sizeof(int)*(nlf+1));
  CHECK_DO(hipMalloc((void**)&d_As1b, sizeof(T*)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_Xs1b, sizeof(T*)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_Ys1b, sizeof(T*)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_Ms1b, sizeof(int)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_Ns1b, sizeof(int)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_lddas1b, sizeof(int)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_incxs1b, sizeof(int)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_incys1b, sizeof(int)*(nlf+1)),"hipMalloc ");
  // 2. dense
  h_As2 = (T**)malloc(sizeof(T*)*(nlf+1));
  h_Xs2 = (T**)malloc(sizeof(T*)*(nlf+1));
  h_Ys2 = (T**)malloc(sizeof(T*)*(nlf+1));
  h_Ms2 = (int*)malloc(sizeof(int)*(nlf+1));
  h_Ns2 = (int*)malloc(sizeof(int)*(nlf+1));
  h_lddas2 = (int*)malloc(sizeof(int)*(nlf+1));
  h_incxs2 = (int*)malloc(sizeof(int)*(nlf+1));
  h_incys2 = (int*)malloc(sizeof(int)*(nlf+1));
  CHECK_DO(hipMalloc((void**)&d_As2, sizeof(T*)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_Xs2, sizeof(T*)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_Ys2, sizeof(T*)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_Ms2, sizeof(int)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_Ns2, sizeof(int)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_lddas2, sizeof(int)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_incxs2, sizeof(int)*(nlf+1)),"hipMalloc ");
  CHECK_DO(hipMalloc((void**)&d_incys2, sizeof(int)*(nlf+1)),"hipMalloc ");
  // construction
  // 1a. approx. 1
  n=0;
  for(ip=0; ip<nlf; ip++){
	ndl    = mat2->ndl[ip];
	ndt    = mat2->ndt[ip];
	nstrtl = mat2->nstrtl[ip];
	nstrtt = mat2->nstrtt[ip];
	ltmtx  = mat2->ltmtx[ip];
	if(ltmtx==1){
	  kt     = mat2->kt[ip];
	  head = mat2->a1[ip];
	  h_As1a[n] = &d_sm.rowmat[head];
	  h_Xs1a[n] = &d_b[nstrtt-1];
	  h_Ys1a[n] = tmpvec[ip];
	  h_Ms1a[n] = ndt;
	  h_Ns1a[n] = kt;
	  h_lddas1a[n] = ndt;
	  h_incxs1a[n] = 1;
	  h_incys1a[n] = 1;
	  n++;
	  //hipblasDgemv(handle,HIPBLAS_OP_T, ndt,kt, &done, &d_sm.rowmat[head], ndt,&d_zu[nstrtt-1],1,&dzero,d_zbut,1);
	} else if(ltmtx==2){
	}
  }
  nbatch1a = n;
  hipMemcpy(d_As1a, h_As1a, sizeof(T*)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_Xs1a, h_Xs1a, sizeof(T*)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_Ys1a, h_Ys1a, sizeof(T*)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_Ms1a, h_Ms1a, sizeof(int)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_Ns1a, h_Ns1a, sizeof(int)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_lddas1a, h_lddas1a, sizeof(int)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_incxs1a, h_incxs1a, sizeof(int)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_incys1a, h_incys1a, sizeof(int)*n, hipMemcpyHostToDevice);
  // 1b. approx. 2
  n = 0;
  for(ip=0; ip<nlf; ip++){
	ndl    = mat2->ndl[ip];
	ndt    = mat2->ndt[ip];
	nstrtl = mat2->nstrtl[ip];
	nstrtt = mat2->nstrtt[ip];
	ltmtx  = mat2->ltmtx[ip];
	if(ltmtx==1){
	  kt = mat2->kt[ip];
	  head = mat2->a2[ip];
	  h_As1b[n] = &d_sm.rowmat[head];
	  h_Xs1b[n] = tmpvec[ip];
	  h_Ys1b[n] = &d_v[nstrtl-1];
	  h_Ms1b[n] = ndl;
	  h_Ns1b[n] = kt;
	  h_lddas1b[n] = ndl;
	  h_incxs1b[n] = 1;
	  h_incys1b[n] = 1;
	  n++;
	  //hipblasDgemv(handle,HIPBLAS_OP_N, ndl,kt, &done, &d_sm.rowmat[head], ndl,d_zbut,1,&done,&d_zaut[nstrtl-1],1);
	} else if(ltmtx==2){
	}
  }
  nbatch1b = n;
  hipMemcpy(d_As1b, h_As1b, sizeof(T*)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_Xs1b, h_Xs1b, sizeof(T*)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_Ys1b, h_Ys1b, sizeof(T*)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_Ms1b, h_Ms1b, sizeof(int)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_Ns1b, h_Ns1b, sizeof(int)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_lddas1b, h_lddas1b, sizeof(int)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_incxs1b, h_incxs1b, sizeof(int)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_incys1b, h_incys1b, sizeof(int)*n, hipMemcpyHostToDevice);
  // 2. dense
  n = 0;
  for(ip=0; ip<nlf; ip++){
	ndl    = mat2->ndl[ip];
	ndt    = mat2->ndt[ip];
	nstrtl = mat2->nstrtl[ip];
	nstrtt = mat2->nstrtt[ip];
	ltmtx  = mat2->ltmtx[ip];
	if(ltmtx==1){
	} else if(ltmtx==2){
	  head = mat2->a1[ip];
	  h_As2[n] = &d_sm.rowmat[head];
	  h_Xs2[n] = &d_b[nstrtt-1];
	  h_Ys2[n] = &d_v[nstrtl-1];
	  h_Ms2[n] = ndt;
	  h_Ns2[n] = ndl;
	  h_lddas2[n] = ndt;
	  h_incxs2[n] = 1;
	  h_incys2[n] = 1;
	  n++;
	  //hipblasDgemv(handle,HIPBLAS_OP_T, ndt,ndl, &done, &d_sm.rowmat[head], ndt,&d_zu[nstrtt-1],1,&done,&d_zaut[nstrtl-1],1);
	}
  }
  nbatch2 = n;
  hipMemcpy(d_As2, h_As2, sizeof(T*)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_Xs2, h_Xs2, sizeof(T*)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_Ys2, h_Ys2, sizeof(T*)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_Ms2, h_Ms2, sizeof(int)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_Ns2, h_Ns2, sizeof(int)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_lddas2, h_lddas2, sizeof(int)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_incxs2, h_incxs2, sizeof(int)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_incys2, h_incys2, sizeof(int)*n, hipMemcpyHostToDevice);

  /*
	magma batched blas
	0: default
	1: nocheck
  */
  {
	char *names[]={"default","nocheck"};
	char name[0xff], fname[0xff];
	snprintf(name,0xff,"magma_batched_%d_%s_%s", kernel, names[kernel], typeid(T).name());
	snprintf(fname,0xff,"result_%s.txt", name);
	printf("fname = %s\n", fname);

	// EXEC
	hmvm_magma_batched_proxy<T>
	  (
	   nbatch1a, d_As1a, d_Xs1a, d_Ys1a, d_Ms1a, d_Ns1a, d_lddas1a, d_incxs1a, d_incys1a,
	   nbatch1b, d_As1b, d_Xs1b, d_Ys1b, d_Ms1b, d_Ns1b, d_lddas1b, d_incxs1b, d_incys1b,
	   nbatch2, d_As2, d_Xs2, d_Ys2, d_Ms2, d_Ns2, d_lddas2, d_incxs2, d_incys2,
	   queue, kernel,
	   d_v, nd, fname, 0);
	// BENCH
	hmvm_magma_batched_proxy<T>
	  (
	   nbatch1a, d_As1a, d_Xs1a, d_Ys1a, d_Ms1a, d_Ns1a, d_lddas1a, d_incxs1a, d_incys1a,
	   nbatch1b, d_As1b, d_Xs1b, d_Ys1b, d_Ms1b, d_Ns1b, d_lddas1b, d_incxs1b, d_incys1b,
	   nbatch2, d_As2, d_Xs2, d_Ys2, d_Ms2, d_Ns2, d_lddas2, d_incxs2, d_incys2,
	   queue, kernel,
	   d_v, nd, fname, 5);
  }
  magma_queue_destroy(queue);

  // ######## ######## ######## ######## ######## ######## ######## ########
  hipFree(d_sm.ltmtx);
  hipFree(d_sm.ndl);
  hipFree(d_sm.ndt);
  hipFree(d_sm.nstrtl);
  hipFree(d_sm.nstrtt);
  hipFree(d_sm.kt);
  hipFree(d_sm.a1);
  hipFree(d_sm.a2);
  hipFree(d_sm.rowmat);
  hipFree(d_sm.rowmat_t);
  hipFree(d_b);
  hipFree(d_v);

  delete [] v;
  printf("hmvm_magma_batched: end\n");
}


// ######## ######## ######## ######## ######## ######## ######## ########
// template関数の実体化のための宣言
// ######## ######## ######## ######## ######## ######## ######## ########
template void hmvm_magma_batched<float>(matrix2<float>  *mat2, float *b, int kernel, int dump_result);
template void hmvm_magma_batched<double>(matrix2<double> *mat2, double *b, int kernel, int dump_result);
