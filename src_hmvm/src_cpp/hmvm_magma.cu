#include "hip/hip_runtime.h"
// -*- C++ -*-
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <typeinfo>
#include <omp.h>
#include <hip/hip_runtime_api.h>
#include <magma_v2.h>

#include "hacapk.h"

#define CHECK_DO(act,msg) {ret=act; if(ret!=hipSuccess){printf("%s failed\n",msg);exit(-1);};}

// ######## ######## ######## ######## ######## ######## ######## ########
/*
  magma simple blas
 */

void  hmvm_magma_calc
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt,
 int *a1, int *a2, double *rowmat, double *rowmat_t,
 int napprox, int *approx, int ndense, int *dense,
 double *d_zbut, matrix2<double> *mat)
{
  int ip;
  int ndl,ndt,nstrtl,nstrtt,kt,ltmtx;
  double dzero = 0.0;
  double done = 1.0;
  int head;
  magma_device_t dev;
  magma_queue_t queue;
  magma_queue_create(dev, &queue);

  for(ip=0; ip<nlf; ip++){
    ndl    = _ndl[ip];
    ndt    = _ndt[ip];
    nstrtl = _nstrtl[ip];
    nstrtt = _nstrtt[ip];
    ltmtx  = _ltmtx[ip];
    if(ltmtx==1){
      kt = _kt[ip];
	  //hipMemcpy(&d_zbut, &h_zero, sizeof(double)*kt, hipMemcpyHostToDevice);
	  head = a1[ip];
	  magma_dgemv(MagmaTrans, ndt,kt, done, &rowmat[head], ndt,&d_zu[nstrtt-1],1,dzero,d_zbut,1, queue);
	  head = a2[ip];
	  magma_dgemv(MagmaNoTrans, ndl,kt, done, &rowmat[head], ndl,d_zbut,1,done,&d_zaut[nstrtl-1],1, queue);
    } else if(ltmtx==2){
	  head = a1[ip];
	  magma_dgemv(MagmaTrans, ndt,ndl, done, &rowmat[head], ndt,&d_zu[nstrtt-1],1,done,&d_zaut[nstrtl-1],1, queue);
    }
  }
  //magma_daxpy(nd,done,d_zaut,1,d_zau,1, queue);

  magma_queue_destroy(queue);
}

void  hmvm_magma_calc
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt,
 int *a1, int *a2, float *rowmat, float *rowmat_t,
 int napprox, int *approx, int ndense, int *dense,
 float *d_zbut, matrix2<float> *mat)
{
  int ip;
  int ndl,ndt,nstrtl,nstrtt,kt,ltmtx;
  double dzero = 0.0;
  double done = 1.0;
  int head;
  magma_device_t dev;
  magma_queue_t queue;
  magma_queue_create(dev, &queue);

  for(ip=0; ip<nlf; ip++){
    ndl    = _ndl[ip];
    ndt    = _ndt[ip];
    nstrtl = _nstrtl[ip];
    nstrtt = _nstrtt[ip];
    ltmtx  = _ltmtx[ip];
    if(ltmtx==1){
      kt = _kt[ip];
	  //hipMemcpy(&d_zbut, &h_zero, sizeof(double)*kt, hipMemcpyHostToDevice);
	  head = a1[ip];
	  magma_sgemv(MagmaTrans, ndt,kt, done, &rowmat[head], ndt,&d_zu[nstrtt-1],1,dzero,d_zbut,1, queue);
	  head = a2[ip];
	  magma_sgemv(MagmaNoTrans, ndl,kt, done, &rowmat[head], ndl,d_zbut,1,done,&d_zaut[nstrtl-1],1, queue);
    } else if(ltmtx==2){
	  head = a1[ip];
	  magma_sgemv(MagmaTrans, ndt,ndl, done, &rowmat[head], ndt,&d_zu[nstrtt-1],1,done,&d_zaut[nstrtl-1],1, queue);
    }
  }
  //magma_daxpy(nd,done,d_zaut,1,d_zau,1, queue);

  magma_queue_destroy(queue);
}

template <class T>
void hmvm_magma_proxy
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *ltmtx, int *ndt, int *ndl, int *nstrtl, int *nstrtt, int *kt,
 int *a1, int *a2, T *rowmat, T *rowmat_t,
 int napprox, int *approx, int ndense, int *dense,
 T *v, T *b, int nd, char *fname, int bench,
 matrix2<T> *mat2)
{
  int M=5, L=M+bench;
  FILE *F;
  int i, l, lmax;
  double d1, d2, *dtimes, dmin, dmax, davg;
  hipError_t ret;
  T *d_tmp, *h_zero;
  hipMalloc((void**)&d_tmp, sizeof(T)*mat2->ktmax);
  h_zero = new T[mat2->ktmax];
  for(i=0; i<mat2->ktmax; i++)h_zero[i]=(T)0.0;
  hipMemcpy(d_tmp, h_zero, sizeof(T)*mat2->ktmax, hipMemcpyHostToDevice);
  dtimes = new double[L];
  if(bench==0){lmax=1;}else{lmax=L;}
  for(l=0;l<lmax;l++){
	for(i=0;i<nd;i++)v[i] = (T)0.0;
	CHECK_DO(hipMemcpy(d_zaut, v, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy v to d_v");
	//CHECK_DO(hipMemcpy(d_zu, b, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy b to d_b");
	hipDeviceSynchronize();
	d1 = omp_get_wtime();

	hmvm_magma_calc
	  (d_zaut, d_zu, nlf, ktmax,
	   ltmtx, ndt, ndl, nstrtl, nstrtt, kt,
	   a1, a2, rowmat, rowmat_t,
	   napprox, approx, ndense, dense, d_tmp, mat2);

	hipDeviceSynchronize();
	d2 = omp_get_wtime();
	dtimes[l] = d2-d1;
  }
  if(bench==0){
	CHECK_DO(hipMemcpy(v, d_zaut, sizeof(T)*nd, hipMemcpyDeviceToHost),"hipMemcpy d_v to v");
	printf("write to %s\n", fname);
	F = fopen(fname, "w");
	for(i=0;i<nd;i++)fprintf(F, "%.3E\n", v[i]);
	fclose(F);
  }else{
	dmin = 9999.99;
	dmax = 0.0;
	davg = 0.0;
	for(i=M;i<L;i++){
	  davg += dtimes[i];
	  if(dmin>dtimes[i])dmin=dtimes[i];
	  if(dmax<dtimes[i])dmax=dtimes[i];
	}
	davg /= (L-M);
	printf("TIME %d hmvm_magma_%s min %e max %e avg %e\n", L-M, typeid(T).name(), dmin, dmax, davg);
  }
  delete [] dtimes;
}

// ######## ######## ######## ######## ######## ######## ######## ########

// ######## ######## ######## ######## ######## ######## ######## ########

template<class T>
void hmvm_magma(matrix2<T> *mat2, T *b, int kernel, int dump_result, int nbench)
{
  matrix2<T> d_sm;
  int i, nd = mat2->nd, ktmax = mat2->ktmax, nlf = mat2->nlf;
  T *v=NULL;
  T *d_b, *d_v;
  int len;
  hipError_t ret;
  printf("hmvm_magma_%s: begin\n", typeid(T).name()); fflush(stdout);
  v    = new T[nd];
  for(i=0;i<nd;i++){
	v[i] = (T)0.0;
  }
  CHECK_DO(hipMalloc((void**)&d_b, sizeof(T)*nd),"hipMalloc d_b");
  CHECK_DO(hipMalloc((void**)&d_v, sizeof(T)*nd),"hipMalloc d_v");
  CHECK_DO(hipMemcpy(d_b, b, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy b to d_b");

  printf("nd = %d\n", nd);												\
  len = mat2->len;
  printf("total length = %d\n", len);
  // host alloc
  // device alloc
  d_sm.nd    = nd;
  d_sm.nlf   = nlf;
  d_sm.ktmax = ktmax;
  hipMalloc((void**)&d_sm.ltmtx, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.ndl, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.ndt, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.nstrtl, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.nstrtt, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.kt, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.a1, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.a2, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.rowmat,sizeof(T)*mat2->len);
  hipMalloc((void**)&d_sm.rowmat_t,sizeof(T)*mat2->len);
  // memcpy
  CHECK_DO(hipMemcpy(d_sm.ltmtx, mat2->ltmtx, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.ltmtx");
  CHECK_DO(hipMemcpy(d_sm.ndt, mat2->ndt, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.ndt");
  CHECK_DO(hipMemcpy(d_sm.ndl, mat2->ndl, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.ndl");
  CHECK_DO(hipMemcpy(d_sm.nstrtl, mat2->nstrtl, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.nstrtl");
  CHECK_DO(hipMemcpy(d_sm.nstrtt, mat2->nstrtt, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.nstrtt");
  CHECK_DO(hipMemcpy(d_sm.kt, mat2->kt, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.kt");
  CHECK_DO(hipMemcpy(d_sm.a1, mat2->a1, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.a1");
  CHECK_DO(hipMemcpy(d_sm.a2, mat2->a2, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.a2");
  CHECK_DO(hipMemcpy(d_sm.rowmat, mat2->rowmat, sizeof(T)*mat2->len, hipMemcpyHostToDevice),"d_sm.rowmat");
  CHECK_DO(hipMemcpy(d_sm.rowmat_t, mat2->rowmat_t, sizeof(T)*mat2->len, hipMemcpyHostToDevice),"d_sm.rowmat_t");

#if 0
  // 分離
  printf("begin splitting\n");
  mat2->approx = (int*)malloc(sizeof(int)*nlf);
  mat2->dense  = (int*)malloc(sizeof(int)*nlf);
  mat2->napprox = mat2->ndense = 0;
  for(ip=0; ip<nlf; ip++){
	if(mat2->ltmtx[ip]==1){
	  mat2->approx[mat2->napprox++] = ip;
	}else{
	  mat2->dense[mat2->ndense++] = ip;
	}
  }
  hipMalloc((void**)&d_sm.approx,sizeof(int)*mat2->napprox);
  hipMalloc((void**)&d_sm.dense,sizeof(int)*mat2->ndense);
  CHECK_DO(hipMemcpy(d_sm.approx, mat2->approx, sizeof(int)*mat2->napprox, hipMemcpyHostToDevice),"d_sm.approx");
  CHECK_DO(hipMemcpy(d_sm.dense, mat2->dense, sizeof(int)*mat2->ndense, hipMemcpyHostToDevice),"d_sm.dense");
  d_sm.napprox = mat2->napprox;
  d_sm.ndense  = mat2->ndense;
  printf("end splitting (napprox=%d, ndense=%d)\n", mat2->napprox, mat2->ndense);
#endif

  /*
	magma blas
  */
  {
	char name[0xff], fname[0xff];
	snprintf(name,0xff,"magma_%s", typeid(T).name());
	snprintf(fname,0xff,"result_%s.txt", name);
	printf("fname = %s\n", fname);
	// EXEC
	if(dump_result)
	  hmvm_magma_proxy<T>
		(d_v, d_b, mat2->nlf, mat2->ktmax,
		 mat2->ltmtx, mat2->ndt, mat2->ndl, mat2->nstrtl, mat2->nstrtt, mat2->kt,
		 mat2->a1, mat2->a2, d_sm.rowmat, d_sm.rowmat_t,
		 d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense,
		 v, b, nd, fname, 0,
		 mat2);
	// BENCH
	if(nbench>0)
	  hmvm_magma_proxy<T>
		(d_v, d_b, mat2->nlf, mat2->ktmax,
		 mat2->ltmtx, mat2->ndt, mat2->ndl, mat2->nstrtl, mat2->nstrtt, mat2->kt,
		 mat2->a1, mat2->a2, d_sm.rowmat, d_sm.rowmat_t,
		 d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense,
		 v, b, nd, fname, nbench,
		 mat2);
  }

  // ######## ######## ######## ######## ######## ######## ######## ########
  hipFree(d_sm.ltmtx);
  hipFree(d_sm.ndl);
  hipFree(d_sm.ndt);
  hipFree(d_sm.nstrtl);
  hipFree(d_sm.nstrtt);
  hipFree(d_sm.kt);
  hipFree(d_sm.a1);
  hipFree(d_sm.a2);
  hipFree(d_sm.rowmat);
  hipFree(d_sm.rowmat_t);
  hipFree(d_b);
  hipFree(d_v);

  delete [] v;
  printf("hmvm_magma: end\n");
}


// ######## ######## ######## ######## ######## ######## ######## ########
// template関数の実体化のための宣言
// ######## ######## ######## ######## ######## ######## ######## ########
template void hmvm_magma<float>(matrix2<float>  *mat2, float *b, int kernel, int dump_result, int nbench);
template void hmvm_magma<double>(matrix2<double> *mat2, double *b, int kernel, int dump_result, int nbench);
