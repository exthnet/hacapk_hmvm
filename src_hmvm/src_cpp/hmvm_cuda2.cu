#include "hip/hip_runtime.h"
// -*- C++ -*-
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <typeinfo>
#include <omp.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_cooperative_groups.h>

#include "hacapk.h"

#define CHECK_DO(act,msg) {ret=act; if(ret!=hipSuccess){printf("%s failed\n",msg);exit(-1);};}

namespace cg = cooperative_groups;

#if __CUDA_ARCH__ < 600
__device__ static double myAtomicAdd(double* address, double val)
{
  unsigned long long int* address_as_ull =
	(unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;

  do {
	assumed = old;
	old = atomicCAS(address_as_ull, assumed,
					__double_as_longlong(val +
										 __longlong_as_double(assumed)));

	// Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
  } while (assumed != old);

  return __longlong_as_double(old);
}
__device__ static inline float myAtomicAdd(float* address, float val)
{
  atomicAdd(address, val);
}
#else
#define myAtomicAdd atomicAdd
#endif

// ######## ######## ######## ######## ######## ######## ######## ########
/*
  hybrid2
  複数WARP単一PMV個別行カーネル
  1PMVを1TBが担当
  hybrid1と比べて1TBあたりスレッド数を32*mulに増やす
  PMV内の1行を1/div WARPが担当
  <<<napprox+ndense,32*mul>>>
  1 PMV by 1 TB (1 TB = mul WARP)
  1 line by 1/div WARP
  バリエーション
  - div：1行を1/divのWARPで計算する、div=1,2,4,8,16,32
  - mul：立ち上げるスレッド数(mul*32)、1つのmat-mat-vecまたはmat-vecをmul TBで実行、mul=1,2,3,...,16
  - a2t：a2を転置版で計算するか否か(0,1)
  - a2interchange：a2のループを入れ替えるか否か(0,1)
  - aatomic：approxの計算をatomic優先にするかwarp shuffle併用するか(0,1)
  - datomic：denseの計算をatomic優先にするかwarp shuffle併用するか(0,1)
  6x16*2x2x2x2=1536通り
*/
template <class T, int div, int mul, int a2t, int a2i, int aatomic, int datomic>
__global__ void hmvm_cuda_hybrid2
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt,
 int *a1, int *a2, T *rowmat, T *rowmat_t,
 int napprox, int *approx, int ndense, int *dense)
{
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid2 : begin\n");
#endif
  int gid   = blockIdx.x;
  //int tid   = threadIdx.x;
  int bid   = threadIdx.x/(32/div);
  int blen  = mul*div;
  int xid   = threadIdx.x%(32/div);
  int xlen  = (32/div);
  int ndl, ndt, nstrtl, nstrtt;
  int ip, kt, il, it, itt, itl, ill;
  size_t head;
  T tmp = 0.0;
  extern __shared__ __align__(sizeof(T)) unsigned char my_smem[];
  T *tmp2 = reinterpret_cast<T *>(my_smem);
  cg::thread_block_tile<32/div> g = cg::tiled_partition<32/div>(cg::this_thread_block());

  if(gid<napprox){
#ifndef _SKIP_APPROX
	// approx
	ip = approx[gid];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	kt = _kt[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt);
#endif
	head = a1[ip];
	for(il=bid; il<kt; il+=blen){
	  if(xid==0)tmp2[il] = 0.0;
	  tmp = 0.0;
	  for(it=xid; it<ndt; it+=xlen){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp += rowmat[head+itl]*d_zu[itt];
	  }
	  //for (int offset = warpSize/(2*div); offset > 0; offset /= 2)tmp += __shfl_down_sync(0xffff, tmp, offset, warpSize);
	  for (int offset = g.size()/2; offset > 0; offset /= 2)tmp += g.shfl_down(tmp, offset);
	  if(xid==0)tmp2[il] = tmp;
	}
	head = a2[ip];
	for(il=bid; il<kt; il+=blen){
	  for(it=xid; it<ndl; it+=xlen){
		ill=it+nstrtl-1;
		itl=it+il*ndl;
		myAtomicAdd(&d_zaut[ill], rowmat[head+itl]*tmp2[il]);
	  }
	}
#endif // approx
  }else if(gid-napprox<ndense){
#ifndef _SKIP_DENSE
	// dense
	ip = dense[gid-napprox];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt);
#endif
	head = a1[ip];
	for(il=bid; il<ndl; il+=blen){
	  tmp = 0.0;
	  ill=il+nstrtl-1;
	  for(it=xid; it<ndt; it+=xlen){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp += rowmat[head+itl]*d_zu[itt];
	  }
	  if(datomic==0){
		//for (int offset = warpSize/(2*div); offset > 0; offset /= 2)tmp += __shfl_down_sync(tmp, offset, warpSize);
		for (int offset = g.size()/2; offset > 0; offset /= 2)tmp += g.shfl_down(tmp, offset);
		if(xid==0){
		  myAtomicAdd(&d_zaut[ill], tmp);
		}
	  }else{
		myAtomicAdd(&d_zaut[ill], tmp);
	  }
	}
#endif // dense
  }

#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid2 : end\n");
#endif
}

#if 0
template <class T, int div>
__global__ void hmvm_cuda_hybrid2
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt,
 int *a1, int *a2, T *rowmat, T *rowmat_t,
 int napprox, int *approx, int ndense, int *dense,
 int mul, int a2t, int a2i, int aatomic, int datomic)
{
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid2 : begin\n");
#endif
  int gid   = blockIdx.x;
  //int tid   = threadIdx.x;
  int bid   = threadIdx.x/(32/div);
  int blen  = mul*div;
  int xid   = threadIdx.x%(32/div);
  int xlen  = (32/div);
  int ndl, ndt, nstrtl, nstrtt;
  int ip, kt, il, it, itt, itl, ill;
  size_t head;
  T tmp = 0.0;
  extern __shared__ __align__(sizeof(T)) unsigned char my_smem[];
  T *tmp2 = reinterpret_cast<T *>(my_smem);
  cg::thread_block_tile<32/div> g = cg::tiled_partition<32/div>(cg::this_thread_block());

  if(gid<napprox){
#ifndef _SKIP_APPROX
	// approx
	ip = approx[gid];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	kt = _kt[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt);
#endif
	head = a1[ip];
	for(il=bid; il<kt; il+=blen){
	  if(xid==0)tmp2[il] = 0.0;
	  tmp = 0.0;
	  for(it=xid; it<ndt; it+=xlen){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp += rowmat[head+itl]*d_zu[itt];
	  }
	  //for (int offset = warpSize/(2*div); offset > 0; offset /= 2)tmp += __shfl_down_sync(0xffff, tmp, offset, warpSize);
	  for (int offset = g.size()/2; offset > 0; offset /= 2)tmp += g.shfl_down(tmp, offset);
	  if(xid==0)tmp2[il] = tmp;
	}
	head = a2[ip];
	for(il=bid; il<kt; il+=blen){
	  for(it=xid; it<ndl; it+=xlen){
		ill=it+nstrtl-1;
		itl=it+il*ndl;
		myAtomicAdd(&d_zaut[ill], rowmat[head+itl]*tmp2[il]);
	  }
	}
#endif // approx
  }else if(gid-napprox<ndense){
#ifndef _SKIP_DENSE
	// dense
	ip = dense[gid-napprox];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt);
#endif
	head = a1[ip];
	for(il=bid; il<ndl; il+=blen){
	  tmp = 0.0;
	  ill=il+nstrtl-1;
	  for(it=xid; it<ndt; it+=xlen){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp += rowmat[head+itl]*d_zu[itt];
	  }
	  if(datomic==0){
		//for (int offset = warpSize/(2*div); offset > 0; offset /= 2)tmp += __shfl_down_sync(tmp, offset, warpSize);
		for (int offset = g.size()/2; offset > 0; offset /= 2)tmp += g.shfl_down(tmp, offset);
		if(xid==0){
		  myAtomicAdd(&d_zaut[ill], tmp);
		}
	  }else{
		myAtomicAdd(&d_zaut[ill], tmp);
	  }
	}
#endif // dense
  }

#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid2 : end\n");
#endif
}

template <class T>
void hmvm_cuda_hybrid2_proxy
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *ltmtx, int *ndt, int *ndl, int *nstrtl, int *nstrtt, int *kt,
 int *a1, int *a2, T *rowmat, T *rowmat_t,
 int napprox, int *approx, int ndense, int *dense,
 int blocks, int threads, int shms,
 T *v, T *b, int nd, char *fname, int bench,
 int div, int mul, int a2t, int a2i, int aa, int da)
{
#if 1
  int M=5, L=M+bench;
  FILE *F;
  int i, l, lmax;
  double d1, d2, *dtimes, dmin, dmax, davg;
  hipError_t ret;
  dtimes = new double[L];
  if(bench==0){lmax=1;}else{lmax=L;}
  for(l=0;l<lmax;l++){
	for(i=0;i<nd;i++)v[i] = (T)0.0;
	CHECK_DO(hipMemcpy(d_zaut, v, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy v to d_v");
	CHECK_DO(hipMemcpy(d_zu, b, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy b to d_b");
	hipDeviceSynchronize();
	d1 = omp_get_wtime();
	switch(div){
	case 1:
	  hmvm_cuda_hybrid2<T,1><<<blocks,threads,shms>>>
		(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt,
		 a1, a2, rowmat, rowmat_t, napprox, approx, ndense, dense , mul, a2t, a2i, aa, da);
	  break;
	case 2:
	  hmvm_cuda_hybrid2<T,2><<<blocks,threads,shms>>>
		(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt,
		 a1, a2, rowmat, rowmat_t, napprox, approx, ndense, dense , mul, a2t, a2i, aa, da);
	  break;
	case 4:
	  hmvm_cuda_hybrid2<T,4><<<blocks,threads,shms>>>
		(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt,
		 a1, a2, rowmat, rowmat_t, napprox, approx, ndense, dense , mul, a2t, a2i, aa, da);
	  break;
	case 8:
	  hmvm_cuda_hybrid2<T,8><<<blocks,threads,shms>>>
		(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt,
		 a1, a2, rowmat, rowmat_t, napprox, approx, ndense, dense , mul, a2t, a2i, aa, da);
	  break;
	case 16:
	  hmvm_cuda_hybrid2<T,16><<<blocks,threads,shms>>>
		(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt,
		 a1, a2, rowmat, rowmat_t, napprox, approx, ndense, dense , mul, a2t, a2i, aa, da);
	  break;
	case 32:
	  hmvm_cuda_hybrid2<T,32><<<blocks,threads,shms>>>
		(d_zaut, d_zu, nlf, ktmax, ltmtx, ndt, ndl, nstrtl, nstrtt, kt,
		 a1, a2, rowmat, rowmat_t, napprox, approx, ndense, dense , mul, a2t, a2i, aa, da);
	  break;
	}
	hipDeviceSynchronize();
	d2 = omp_get_wtime();
	dtimes[l] = d2-d1;
  }
  if(bench==0){
	CHECK_DO(hipMemcpy(v, d_zaut, sizeof(T)*nd, hipMemcpyDeviceToHost),"hipMemcpy d_v to v");
	printf("write to %s\n", fname);
	F = fopen(fname, "w");
	for(i=0;i<nd;i++)fprintf(F, "%.3E\n", v[i]);
	fclose(F);
  }else{
	dmin = 9999.99;
	dmax = 0.0;
	davg = 0.0;
	for(i=M;i<L;i++){
	  davg += dtimes[i];
	  if(dmin>dtimes[i])dmin=dtimes[i];
	  if(dmax<dtimes[i])dmax=dtimes[i];
	}
	davg /= (L-M);
	printf("TIME %d hmvm_cuda1_hybrid2%s min %e max %e avg %e\n", L-M, typeid(T).name(), dmin, dmax, davg);
  }
  delete [] dtimes;
#endif
}
#else
template <class T, int div, int mul, int a2t, int a2i, int aa, int da>
void hmvm_cuda_hybrid2_proxy
(T *d_zaut, T *d_zu, int nlf, int ktmax,
 int *ltmtx, int *ndt, int *ndl, int *nstrtl, int *nstrtt, int *kt,
 int *a1, int *a2, T *rowmat, T *rowmat_t,
 int napprox, int *approx, int ndense, int *dense,
 int blocks, int threads, int shms,
 T *v, T *b, int nd, char *fname, int bench)
{
#if 1
  int M=5, L=M+bench;
  FILE *F;
  int i, l, lmax;
  double d1, d2, *dtimes, dmin, dmax, davg;
  hipError_t ret;
  dtimes = new double[L];
  if(bench==0){lmax=1;}else{lmax=L;}
  for(l=0;l<lmax;l++){
	for(i=0;i<nd;i++)v[i] = (T)0.0;
	CHECK_DO(hipMemcpy(d_zaut, v, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy v to d_v");
	CHECK_DO(hipMemcpy(d_zu, b, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy b to d_b");
	hipDeviceSynchronize();
	d1 = omp_get_wtime();
	hmvm_cuda_hybrid2<T,div,mul,a2t,a2i,aa,da><<<blocks,threads,shms>>>
	  (d_zaut, d_zu, nlf, ktmax, ltmtx,
	   ndt, ndl, nstrtl, nstrtt, kt, a1, a2, rowmat, rowmat_t,
	   napprox, approx, ndense, dense);
	hipDeviceSynchronize();
	d2 = omp_get_wtime();
	dtimes[l] = d2-d1;
  }
  if(bench==0){
	CHECK_DO(hipMemcpy(v, d_zaut, sizeof(T)*nd, hipMemcpyDeviceToHost),"hipMemcpy d_v to v");
	printf("write to %s\n", fname);
	F = fopen(fname, "w");
	for(i=0;i<nd;i++)fprintf(F, "%.3E\n", v[i]);
	fclose(F);
  }else{
	dmin = 9999.99;
	dmax = 0.0;
	davg = 0.0;
	for(i=M;i<L;i++){
	  davg += dtimes[i];
	  if(dmin>dtimes[i])dmin=dtimes[i];
	  if(dmax<dtimes[i])dmax=dtimes[i];
	}
	davg /= (L-M);
	printf("TIME %d hmvm_cuda1_hybrid2%s min %e max %e avg %e\n", L-M, typeid(T).name(), dmin, dmax, davg);
  }
  delete [] dtimes;
#endif
}
#include "template_hybrid2.hpp"
#endif
// ######## ######## ######## ######## ######## ######## ######## ########

template<class T>
void hmvm_cuda2(matrix2<T> *mat2, T *b, int kernel, int dump_result)
{
  const int L=10, M=5;
  FILE *F;
  matrix2<T> d_sm;
  int i, l, nd = mat2->nd, ktmax = mat2->ktmax, nlf = mat2->nlf;
  double d1, d2, dtimes[L], dmin, dmax, davg;
  T *v=NULL, *tmp=NULL, *zero;
  T *d_b, *d_v;//, *d_zaut, *d_zbut;
  int ip;
  int len;
  hipError_t ret;
  printf("hmvm_cuda2_%s: begin\n", typeid(T).name()); fflush(stdout);
  v    = new T[nd];    //(double*)malloc(sizeof(double)*mat2->nd);
  tmp  = new T[ktmax]; //(double*)malloc(sizeof(double)*mat2->ktmax);
  zero = new T[ktmax]; //(double*)malloc(sizeof(double)*mat2->ktmax);
  for(i=0;i<nd;i++){
	v[i] = (T)0.0;
  }
  for(i=0;i<ktmax;i++){
	zero[i] = (T)0.0;
  }
  //CHECK_DO(hipMalloc((void**)&d_zaut, sizeof(T)*mat2->nd),"hipMalloc z_aut");
  //CHECK_DO(hipMalloc((void**)&d_zbut, sizeof(T)*mat2->ktmax),"hipMalloc zbut");
  CHECK_DO(hipMalloc((void**)&d_b, sizeof(T)*nd),"hipMalloc d_b");
  CHECK_DO(hipMalloc((void**)&d_v, sizeof(T)*nd),"hipMalloc d_v");
  //for(i=0;i<mat2->nd;i++){d_b[i]=NULL;d_v[i]=NULL;}

  printf("nd = %d\n", nd);												\
  len = mat2->len;
  printf("total length = %d\n", len);
  // host alloc
  // device alloc
  d_sm.nd    = nd;
  d_sm.nlf   = nlf;
  d_sm.ktmax = ktmax;
  hipMalloc((void**)&d_sm.ltmtx, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.ndl, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.ndt, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.nstrtl, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.nstrtt, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.kt, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.a1, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.a2, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.rowmat,sizeof(T)*mat2->len);
  hipMalloc((void**)&d_sm.rowmat_t,sizeof(T)*mat2->len);
  // memcpy
  CHECK_DO(hipMemcpy(d_sm.ltmtx, mat2->ltmtx, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.ltmtx");
  CHECK_DO(hipMemcpy(d_sm.ndt, mat2->ndt, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.ndt");
  CHECK_DO(hipMemcpy(d_sm.ndl, mat2->ndl, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.ndl");
  CHECK_DO(hipMemcpy(d_sm.nstrtl, mat2->nstrtl, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.nstrtl");
  CHECK_DO(hipMemcpy(d_sm.nstrtt, mat2->nstrtt, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.nstrtt");
  CHECK_DO(hipMemcpy(d_sm.kt, mat2->kt, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.kt");
  CHECK_DO(hipMemcpy(d_sm.a1, mat2->a1, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.a1");
  CHECK_DO(hipMemcpy(d_sm.a2, mat2->a2, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.a2");
  CHECK_DO(hipMemcpy(d_sm.rowmat, mat2->rowmat, sizeof(T)*mat2->len, hipMemcpyHostToDevice),"d_sm.rowmat");
  CHECK_DO(hipMemcpy(d_sm.rowmat_t, mat2->rowmat_t, sizeof(T)*mat2->len, hipMemcpyHostToDevice),"d_sm.rowmat_t");

  // 分離
  printf("begin splitting\n");
  mat2->approx = (int*)malloc(sizeof(int)*nlf);
  mat2->dense  = (int*)malloc(sizeof(int)*nlf);
  mat2->napprox = mat2->ndense = 0;
  for(ip=0; ip<nlf; ip++){
	if(mat2->ltmtx[ip]==1){
	  mat2->approx[mat2->napprox++] = ip;
	}else{
	  mat2->dense[mat2->ndense++] = ip;
	}
  }
  hipMalloc((void**)&d_sm.approx,sizeof(int)*mat2->napprox);
  hipMalloc((void**)&d_sm.dense,sizeof(int)*mat2->ndense);
  CHECK_DO(hipMemcpy(d_sm.approx, mat2->approx, sizeof(int)*mat2->napprox, hipMemcpyHostToDevice),"d_sm.approx");
  CHECK_DO(hipMemcpy(d_sm.dense, mat2->dense, sizeof(int)*mat2->ndense, hipMemcpyHostToDevice),"d_sm.dense");
  d_sm.napprox = mat2->napprox;
  d_sm.ndense  = mat2->ndense;
  printf("end splitting (napprox=%d, ndense=%d)\n", mat2->napprox, mat2->ndense);

#if 1
/*
  hybrid2
  複数WARP単一GEMV個別行カーネル
  <<<napprox+ndense,32*mul>>>
  1 GEMV by 1 WARP
  1 line by 1/div WARP
  バリエーション
  - div：1行を1/divのWARPで計算する、div=1,2,4,8,16,32
  - mul：立ち上げるスレッド数(mul*32)、1つのmat-mat-vecまたはmat-vecをmul TBで実行、mul=1,2,3,...,16
  - a2t：a2を転置版で計算するか否か(0,1)
  - a2interchange：a2のループを入れ替えるか否か(0,1)
  - aatomic：approxの計算をatomic優先にするかwarp shuffle併用するか(0,1)
  - datomic：denseの計算をatomic優先にするかwarp shuffle併用するか(0,1)
  6x16x2x2x2x2=1536通り
  全部大丈夫そう
*/
  if(kernel>=10000&&kernel<11536){
	int subkernel = kernel-10000;
	int div, mul, a2t, a2i, aa, da;
	div = subkernel%6;
	mul = (subkernel/6)%16 + 1;
	a2t = ((subkernel/6)/16)%2;
	a2i = (((subkernel/6)/16)/2)%2;
	aa = ((((subkernel/6)/16)/2)/2)%2;
	da = (((((subkernel/6)/16)/2)/2)/2)%2;
	div = pow(2,div);
	if((32*mul)%div)printf("invalid parameters: 32*%d %% %d\n", mul, div);
	char name[0xff], fname[0xff];
	snprintf(name,0xff,"hybrid2_div%d_mul%d_a2t%d_a2i%d_aa%d_da%d_%s", div, mul, a2t, a2i, aa, da, typeid(T).name());
	snprintf(fname,0xff,"result_cuda1_%s.txt", name);
	printf("fname = %s\n", fname);
	//printf("DIV = %d, MUL = %d, ATOMIC = %d\n", DIV, MUL, ATOMIC);
	// EXEC
	hmvm_cuda_hybrid2_proxy<T>
	  (d_v, d_b, d_sm.nlf, d_sm.ktmax,
	   d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt, d_sm.kt,
	   d_sm.a1, d_sm.a2, d_sm.rowmat, d_sm.rowmat_t,
	   d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense,
	   d_sm.napprox+d_sm.ndense, 32*mul, d_sm.ktmax*sizeof(T),
	   v, b, nd, fname, 0,
	   div, mul, a2t, a2i, aa, da);
	// BENCH
	/*
	if(0)hmvm_cuda_hybrid2_proxy
	  (d_v, d_b, d_sm.nlf, d_sm.ktmax,
	   d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt, d_sm.kt,
	   d_sm.a1, d_sm.a2, d_sm.rowmat, d_sm.rowmat_t,
	   d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense,
	   d_sm.napprox+d_sm.ndense,32*mul,d_sm.ktmax*sizeof(T),
	   v, b, nd, fname, 5,
	   div, mul, a2t, a2i, aa, da);
	*/
  }
#endif

  // ######## ######## ######## ######## ######## ######## ######## ########
  hipFree(d_sm.ltmtx);
  hipFree(d_sm.ndl);
  hipFree(d_sm.ndt);
  hipFree(d_sm.nstrtl);
  hipFree(d_sm.nstrtt);
  hipFree(d_sm.kt);
  hipFree(d_sm.a1);
  hipFree(d_sm.a2);
  hipFree(d_sm.rowmat);
  hipFree(d_sm.rowmat_t);
  //hipFree(d_zaut);
  //hipFree(d_zbut);
  hipFree(d_b);
  hipFree(d_v);

  delete [] v; delete [] tmp; delete [] zero;
  //free(v); free(tmp); free(zero);
  printf("hmvm_cuda1: end\n");
}


// ######## ######## ######## ######## ######## ######## ######## ########
// template関数の実体化のための宣言
// ######## ######## ######## ######## ######## ######## ######## ########
template void hmvm_cuda2<float>(matrix2<float>  *mat2, float *b, int kernel, int dump_result);
template void hmvm_cuda2<double>(matrix2<double> *mat2, double *b, int kernel, int dump_result);
