#include "hip/hip_runtime.h"
// -*- C++ -*-
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <typeinfo>
#include <omp.h>
#include <hip/hip_runtime_api.h>

#include "hacapk.h"
#include "hmvm_cuda_kernels.h"

#define CHECK_DO(act,msg) {ret=act; if(ret!=hipSuccess){printf("%s failed\n",msg);exit(-1);};}

template<class T>
void hmvm_cuda1(matrix2<T> mat2, T *b, int kernel, int dump_result)
{
  const int L=10, M=5;
  FILE *F;
  matrix2<T> d_sm;
  int i, l, nd = mat2.nd;
  double d1, d2, dtimes[L], dmin, dmax, davg;
  T *v=NULL, *tmp=NULL, *zero;
  T *d_b, *d_v, *d_zaut, *d_zbut;
  int ip;
  int len;
  hipError_t ret;
  printf("hmvm_cuda1_%s: begin\n", typeid(T).name());
  v    = new T[mat2.nd];    //(double*)malloc(sizeof(double)*mat2.nd);
  tmp  = new T[mat2.ktmax]; //(double*)malloc(sizeof(double)*mat2.ktmax);
  zero = new T[mat2.ktmax]; //(double*)malloc(sizeof(double)*mat2.ktmax);
  for(i=0;i<nd;i++){
	v[i] = (T)0.0;
  }
  for(i=0;i<mat2.ktmax;i++){
	zero[i] = (T)0.0;
  }
  CHECK_DO(hipMalloc((void**)&d_zaut, sizeof(T)*mat2.nd),"hipMalloc z_aut");
  CHECK_DO(hipMalloc((void**)&d_zbut, sizeof(T)*mat2.ktmax),"hipMalloc zbut");
  CHECK_DO(hipMalloc((void**)&d_b, sizeof(T)*mat2.nd),"hipMalloc d_b");
  CHECK_DO(hipMalloc((void**)&d_v, sizeof(T)*mat2.nd),"cudaMallod d_v");
  //for(i=0;i<mat2.nd;i++){d_b[i]=NULL;d_v[i]=NULL;}

  len = mat2.len;
  printf("total length = %d\n", len);
  // host alloc
  // device alloc
  d_sm.nd    = mat2.nd;
  d_sm.nlf   = mat2.nlf;
  d_sm.ktmax = mat2.ktmax;
  hipMalloc((void**)&d_sm.ltmtx, sizeof(int)*mat2.nlf);
  hipMalloc((void**)&d_sm.ndl, sizeof(int)*mat2.nlf);
  hipMalloc((void**)&d_sm.ndt, sizeof(int)*mat2.nlf);
  hipMalloc((void**)&d_sm.nstrtl, sizeof(int)*mat2.nlf);
  hipMalloc((void**)&d_sm.nstrtt, sizeof(int)*mat2.nlf);
  hipMalloc((void**)&d_sm.kt, sizeof(int)*mat2.nlf);
  hipMalloc((void**)&d_sm.a1, sizeof(int)*mat2.nlf);
  hipMalloc((void**)&d_sm.a2, sizeof(int)*mat2.nlf);
  hipMalloc((void**)&d_sm.rowmat,sizeof(T)*mat2.len);
  hipMalloc((void**)&d_sm.rowmat_t,sizeof(T)*mat2.len);
  // memcpy
  CHECK_DO(hipMemcpy(d_sm.ltmtx, mat2.ltmtx, sizeof(int)*mat2.nlf, hipMemcpyHostToDevice),"d_sm.ltmtx");
  CHECK_DO(hipMemcpy(d_sm.ndt, mat2.ndt, sizeof(int)*mat2.nlf, hipMemcpyHostToDevice),"d_sm.ndt");
  CHECK_DO(hipMemcpy(d_sm.ndl, mat2.ndl, sizeof(int)*mat2.nlf, hipMemcpyHostToDevice),"d_sm.ndl");
  CHECK_DO(hipMemcpy(d_sm.nstrtl, mat2.nstrtl, sizeof(int)*mat2.nlf, hipMemcpyHostToDevice),"d_sm.nstrtl");
  CHECK_DO(hipMemcpy(d_sm.nstrtt, mat2.nstrtt, sizeof(int)*mat2.nlf, hipMemcpyHostToDevice),"d_sm.nstrtt");
  CHECK_DO(hipMemcpy(d_sm.kt, mat2.kt, sizeof(int)*mat2.nlf, hipMemcpyHostToDevice),"d_sm.kt");
  CHECK_DO(hipMemcpy(d_sm.a1, mat2.a1, sizeof(int)*mat2.nlf, hipMemcpyHostToDevice),"d_sm.a1");
  CHECK_DO(hipMemcpy(d_sm.a2, mat2.a2, sizeof(int)*mat2.nlf, hipMemcpyHostToDevice),"d_sm.a2");
  CHECK_DO(hipMemcpy(d_sm.rowmat, mat2.rowmat, sizeof(T)*mat2.len, hipMemcpyHostToDevice),"d_sm.rowmat");
  CHECK_DO(hipMemcpy(d_sm.rowmat_t, mat2.rowmat_t, sizeof(T)*mat2.len, hipMemcpyHostToDevice),"d_sm.rowmat_t");

  // 分離
  printf("begin splitting\n");
  mat2.approx = (int*)malloc(sizeof(int)*mat2.nlf);
  mat2.dense  = (int*)malloc(sizeof(int)*mat2.nlf);
  mat2.napprox = mat2.ndense = 0;
  for(ip=0; ip<mat2.nlf; ip++){
	if(mat2.ltmtx[ip]==1){
	  mat2.approx[mat2.napprox++] = ip;
	}else{
	  mat2.dense[mat2.ndense++] = ip;
	}
  }
  hipMalloc((void**)&d_sm.approx,sizeof(int)*mat2.napprox);
  hipMalloc((void**)&d_sm.dense,sizeof(int)*mat2.ndense);
  CHECK_DO(hipMemcpy(d_sm.approx, mat2.approx, sizeof(int)*mat2.napprox, hipMemcpyHostToDevice),"d_sm.approx");
  CHECK_DO(hipMemcpy(d_sm.dense, mat2.dense, sizeof(int)*mat2.ndense, hipMemcpyHostToDevice),"d_sm.dense");
  d_sm.napprox = mat2.napprox;
  d_sm.ndense  = mat2.ndense;
  printf("end splitting\n");

#define EXEC(FUNCNAME,B,T,S)											\
  printf("nd = %d\n", nd);												\
  hipMemcpy(d_v, v, sizeof(T)*nd, hipMemcpyHostToDevice);				\
  hipMemcpy(d_b, b, sizeof(T)*nd, hipMemcpyHostToDevice);				\
  FUNCNAME<<<B,T,S>>>													\
	(d_v, d_b, d_sm.nlf, d_sm.ktmax,									\
	 d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt,			\
	 d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat,							\
	 d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense);				\
  hipDeviceSynchronize();												\
  hipMemcpy(v, d_v, sizeof(T)*nd, hipMemcpyDeviceToHost);				\
  printf("write to %s\n", fname);										\
  F = fopen(fname, "w");												\
  for(i=0;i<nd;i++)fprintf(F, "%.3E\n", v[i]);							\
  fclose(F);

#define BENCH(FUNCNAME,B,T,S)											\
  printf("nd = %d\n", nd);												\
  for(l=0;l<L;l++){														\
	for(i=0;i<nd;i++)v[i] = 0.0;										\
	hipMemcpy(d_v, v, sizeof(T)*nd, hipMemcpyHostToDevice);			\
	hipMemcpy(d_b, b, sizeof(T)*nd, hipMemcpyHostToDevice);			\
	hipDeviceSynchronize();											\
	d1 = omp_get_wtime();												\
	FUNCNAME<<<B,T,S>>>													\
	  (d_v, d_b, d_sm.nlf, d_sm.ktmax,									\
	   d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt,		\
	   d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat,							\
	   d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense);				\
	hipDeviceSynchronize();											\
	d2 = omp_get_wtime();												\
	dtimes[l] = d2-d1;													\
  }																		\
  dmin = 9999.99;														\
  dmax = 0.0;															\
  davg = 0.0;															\
  for(i=M;i<L;i++){														\
	davg += dtimes[i];													\
	if(dmin>dtimes[i])dmin=dtimes[i];									\
	if(dmax<dtimes[i])dmax=dtimes[i];									\
  }																		\
  davg /= (L-5);

  // sequential
  if(kernel==0)
  {
	int a1, a2;
	char name[8], fname[0xff];
	a1=a2=0;
	snprintf(name,8,"_%d_%d",a1,a2);
	snprintf(fname,32,"result_cuda1%s_%s.txt", name, typeid(T).name());
	printf("fname = %s\n", fname);
	EXEC(hmvm_cuda_seq<T>,1,1,d_sm.ktmax*sizeof(T));
	BENCH(hmvm_cuda_seq<T>,1,1,d_sm.ktmax*sizeof(T));
	printf("TIME %d hmvm_cuda1%s min %e max %e avg %e\n", L-M, typeid(T).name(), dmin, dmax, davg);
  }

  // block parallel
  // whole hmvm calculation in 1 GPU kernel
  if(kernel==1)
  {
	int a1, a2;
	char name[8], fname[32];
	a1=a2=0;
	snprintf(name,8,"_%d_%d",a1,a2);
	snprintf(fname,32,"result_cuda1blk%s_%s.txt", name, typeid(T).name());
	printf("fname = %s\n", fname);
	EXEC(hmvm_cuda_block<T>,d_sm.napprox+d_sm.ndense,1,d_sm.ktmax*sizeof(T));
	BENCH(hmvm_cuda_block<T>,d_sm.napprox+d_sm.ndense,1,d_sm.ktmax*sizeof(T));
	printf("TIME %d hmvm_cuda1blk%s min %e max %e avg %e\n", L-M, typeid(T).name(), dmin, dmax, davg);
  }

  // block & thread parallel
  // whole hmvm calculation in 1 GPU kernel
  // under development
  if(kernel==2)
  {
	int a1, a2;
	char name[8], fname[32];
	a1=a2=0;
	snprintf(name,8,"_%d_%d",a1,a2);
	snprintf(fname,32,"result_cuda1hyb%s_%s.txt", name, typeid(T).name());
	printf("fname = %s\n", fname);
	//EXEC(hmvm_cuda_hybrid<T>,d_sm.napprox+d_sm.ndense,1,d_sm.ktmax*sizeof(T));
	//BENCH(hmvm_cuda_hybrid<T>,d_sm.napprox+d_sm.ndense,1,d_sm.ktmax*sizeof(T));
	printf("TIME %d hmvm_cuda1hyb%s min %e max %e avg %e\n", L-M, typeid(T).name(), dmin, dmax, davg);
  }

  // ######## ######## ######## ######## ######## ######## ######## ########
  hipFree(d_sm.ltmtx);
  hipFree(d_sm.ndl);
  hipFree(d_sm.ndt);
  hipFree(d_sm.nstrtl);
  hipFree(d_sm.nstrtt);
  hipFree(d_sm.kt);
  hipFree(d_sm.a1);
  hipFree(d_sm.a2);
  hipFree(d_sm.rowmat);
  hipFree(d_sm.rowmat_t);
  hipFree(d_zaut);
  hipFree(d_zbut);
  hipFree(d_b);
  hipFree(d_v);

  delete [] v; delete [] tmp; delete [] zero;
  //free(v); free(tmp); free(zero);
  printf("hmvm_cuda1: end\n");
}

template void hmvm_cuda1<float>(matrix2<float> mat2, float *b, int kernel, int dump_result);
template void hmvm_cuda1<double>(matrix2<double> mat2, double *b, int kernel, int dump_result);
