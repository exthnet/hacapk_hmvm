#include "hip/hip_runtime.h"
// -*- C++ -*-
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <typeinfo>
#include <omp.h>
#include <hip/hip_runtime_api.h>

#include "hacapk.h"
#include "hmvm_cuda_kernels.h"

#define CHECK_DO(act,msg) {ret=act; if(ret!=hipSuccess){printf("%s failed\n",msg);exit(-1);};}




#define myAtomicAdd atomicAdd



/*
  <<<ndense,32>>>
  1 GEMV by 1 TB
  1 line by 1/div WARP
*/

__global__ void hmvm_cuda_hybrid0
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense, int div)
{
}
// nlf block, 32 thread
__global__ void hmvm_cuda_hybrid0
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense, int div)
{
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid1 : begin\n");
#endif
  int gid   = blockIdx.x;
  int tid   = threadIdx.x;
  int bid   = threadIdx.x/(32/div);
  int blen  = div;
  int xid   = threadIdx.x%(32/div);
  int xlen  = (32/div);
  int ndl, ndt, nstrtl, nstrtt, ltmtx;
  int ip, kt, il, it, itt, itl, ill;
  size_t head;
  double tmp = 0.0;
  extern __shared__ __align__(sizeof(double)) unsigned char my_smem[];
  double *tmp2 = reinterpret_cast<double *>(my_smem);

  if(gid<napprox){
#if 0
	// approx
	ip = approx[gid];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	ltmtx = _ltmtx[ip];
	kt = _kt[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	head = a1[ip];
	for(il=bid; il<kt; il+=blen){
	  tmp2[il] = 0.0;
	  for(it=xid; it<ndt; it+=xlen){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp2[il] += rowmat[head+itl]*d_zu[itt];
	  }
	}
	head = a2[ip];
	for(il=bid; il<kt; il+=blen){
	  for(it=xid; it<ndl; it+=xlen){
		ill=it+nstrtl-1;
		itl=it+il*ndl;
		myAtomicAdd(&d_zaut[ill], rowmat[head+itl]*tmp2[il]);
	  }
	}
#endif
  }else{
#if 1
	// dense
	ip = dense[gid-napprox];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	ltmtx = _ltmtx[ip];
#if 1//_DEBUG_LEVEL >= 3
	if(ip==dense[0])printf("dense %d %d: %d %d %d %d %d\n", ip, gid, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	head = a1[ip];
	__syncthreads();
	//if(ip==0&&xid==0)printf("x %d %f + %f\n", ill, d_zaut[ill], tmp);
	__syncthreads();
	//	if(blockIdx.x==0&&threadIdx.x==0)printf("! %d %e + %f\n", 0, d_zaut[0], tmp);
	if(ip==dense[0])printf("! %d %e + %f\n", 0, d_zaut[0], tmp);
	//if(ip<ndense) // 判定いるのか？
	for(il=bid; il<ndl; il+=blen){
	  // 何故かだめ
#if 1
	  tmp = 0.0;
	  ill=il+nstrtl-1;
	  if(ip==dense[0])printf("x %d %e + %f\n", ill, d_zaut[ill], tmp);
	  __syncthreads();
	  for(it=xid; it<ndt; it+=xlen){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp += rowmat[head+itl]*d_zu[itt];
	  }
	  if(ip==dense[0]&&il==bid)printf("tmp %d %f\n", threadIdx.x, tmp);
	  //for (int offset = warpSize/(2*div); offset > 0; offset /= 2)tmp += __shfl_down(tmp, offset);
	  __syncthreads();
	  __syncthreads();
	  for (int offset = warpSize/(2*div); offset > 0; offset /= 2)tmp += __shfl_down_sync(0xffff, tmp, offset, warpSize);
	  __syncthreads();
	  if(xid==0){
		if(ip==dense[0]&&il==bid)printf("1->1 %d %f + %f\n", ill, d_zaut[ill], tmp);
		atomicAdd(&d_zaut[ill], tmp);
		if(ip==dense[0]&&il==bid)printf("1->2 %d %f + %f\n", ill, d_zaut[ill], tmp);
	  }
	  __syncthreads();
#endif
	  // ただしくうごく
#if 0
	  tmp = 0.0;
	  ill=il+nstrtl-1;
	  for(it=xid; it<ndt; it+=xlen){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp += rowmat[head+itl]*d_zu[itt];
	  }
	  //for (int offset = warpSize/(2*div); offset > 0; offset /= 2)tmp += __shfl_down(tmp, offset);
	  __syncthreads();
	  if(ip==dense[0]&&il==bid)printf("2->1 %d %e + %e\n", ill, d_zaut[ill], tmp);
	  __syncthreads();
	  atomicAdd(&d_zaut[ill], tmp);
	  __syncthreads();
	  if(ip==dense[0]&&il==bid)printf("2->2 %d %e + %e\n", ill, d_zaut[ill], tmp);
#endif
	}
#endif
  }

#if _DEBUG_LEVEL >= 2
  printf("hmvm_cuda_hybrid1 : end\n");
#endif
}

__global__ void hmvm_cuda_test
(float *d_zaut, float *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, float *rowmat,
 int napprox, int *approx, int ndense, int *dense)
{
}
__global__ void hmvm_cuda_test
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense)
{
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cudaD : begin\n");
#endif
  int ndl, ndt, nstrtl, nstrtt, ltmtx;
  int ip, kt, il, it, itt, itl, ill;
  size_t head;
  //extern __shared__ double tmp2[];
  extern __shared__ __align__(sizeof(double)) unsigned char my_smem[];
  double *tmp2 = reinterpret_cast<double *>(my_smem);
  int i;

  // approx
  for(i=0; i<napprox; i++){
#if 1
	ip = approx[i];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	ltmtx = _ltmtx[ip];
	kt = _kt[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	head = a1[ip];
	for(il=0; il<kt; il++){
	  tmp2[il] = 0.0;
	  for(it=0; it<ndt; it++){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp2[il] += rowmat[head+itl]*d_zu[itt];
	  }
	}
	head = a2[ip];
	for(il=0; il<kt; il++){
	  for(it=0; it<ndl; it++){
		ill=it+nstrtl-1;
		itl=it+il*ndl;
		myAtomicAdd(&d_zaut[ill], rowmat[head+itl]*tmp2[il]);
	  }
	}
#endif
  }
  // dense
  for(i=0; i<ndense; i++){
#if 1
	ip = dense[i];
	ndl = _ndl[ip];
	ndt = _ndt[ip];
	nstrtl = _nstrtl[ip];
	nstrtt = _nstrtt[ip];
	ltmtx = _ltmtx[ip];
#if _DEBUG_LEVEL >= 3
	printf("%d: %d %d %d %d %d\n", ip, ndl, ndt, nstrtl, nstrtt, ltmtx);
#endif
	head = a1[ip];
	for(il=0; il<ndl; il++){
	  double tmp = 0.0;
	  ill=il+nstrtl-1;
	  for(it=0; it<ndt; it++){
		itt=it+nstrtt-1;
		itl=it+il*ndt;
		tmp += rowmat[head+itl]*d_zu[itt];
	  }
	  myAtomicAdd(&d_zaut[ill], tmp);
	  printf("myAtomicAdd %d %e\n", ill, tmp);
	}
#endif
  }
#if _DEBUG_LEVEL >= 2
  printf("hmvm_cudaD : end\n");
#endif
}


template<class T>
void hmvm_cuda1(matrix2<T> *mat2, T *b, int kernel, int dump_result)
{
  const int L=10, M=5;
  FILE *F;
  matrix2<T> d_sm;
  int i, l, nd = mat2->nd;
  double d1, d2, dtimes[L], dmin, dmax, davg;
  T *v=NULL, *tmp=NULL, *zero;
  T *d_b, *d_v;//, *d_zaut, *d_zbut;
  int ip;
  int len;
  hipError_t ret;
  printf("hmvm_cuda1_%s: begin\n", typeid(T).name());
  v    = new T[mat2->nd];    //(double*)malloc(sizeof(double)*mat2->nd);
  tmp  = new T[mat2->ktmax]; //(double*)malloc(sizeof(double)*mat2->ktmax);
  zero = new T[mat2->ktmax]; //(double*)malloc(sizeof(double)*mat2->ktmax);
  for(i=0;i<nd;i++){
	v[i] = (T)0.0;
  }
  for(i=0;i<mat2->ktmax;i++){
	zero[i] = (T)0.0;
  }
  //CHECK_DO(hipMalloc((void**)&d_zaut, sizeof(T)*mat2->nd),"hipMalloc z_aut");
  //CHECK_DO(hipMalloc((void**)&d_zbut, sizeof(T)*mat2->ktmax),"hipMalloc zbut");
  CHECK_DO(hipMalloc((void**)&d_b, sizeof(T)*mat2->nd),"hipMalloc d_b");
  CHECK_DO(hipMalloc((void**)&d_v, sizeof(T)*mat2->nd),"cudaMallod d_v");
  //for(i=0;i<mat2->nd;i++){d_b[i]=NULL;d_v[i]=NULL;}

  len = mat2->len;
  printf("total length = %d\n", len);
  // host alloc
  // device alloc
  d_sm.nd    = mat2->nd;
  d_sm.nlf   = mat2->nlf;
  d_sm.ktmax = mat2->ktmax;
  hipMalloc((void**)&d_sm.ltmtx, sizeof(int)*mat2->nlf);
  hipMalloc((void**)&d_sm.ndl, sizeof(int)*mat2->nlf);
  hipMalloc((void**)&d_sm.ndt, sizeof(int)*mat2->nlf);
  hipMalloc((void**)&d_sm.nstrtl, sizeof(int)*mat2->nlf);
  hipMalloc((void**)&d_sm.nstrtt, sizeof(int)*mat2->nlf);
  hipMalloc((void**)&d_sm.kt, sizeof(int)*mat2->nlf);
  hipMalloc((void**)&d_sm.a1, sizeof(int)*mat2->nlf);
  hipMalloc((void**)&d_sm.a2, sizeof(int)*mat2->nlf);
  hipMalloc((void**)&d_sm.rowmat,sizeof(T)*mat2->len);
  hipMalloc((void**)&d_sm.rowmat_t,sizeof(T)*mat2->len);
  // memcpy
  CHECK_DO(hipMemcpy(d_sm.ltmtx, mat2->ltmtx, sizeof(int)*mat2->nlf, hipMemcpyHostToDevice),"d_sm.ltmtx");
  CHECK_DO(hipMemcpy(d_sm.ndt, mat2->ndt, sizeof(int)*mat2->nlf, hipMemcpyHostToDevice),"d_sm.ndt");
  CHECK_DO(hipMemcpy(d_sm.ndl, mat2->ndl, sizeof(int)*mat2->nlf, hipMemcpyHostToDevice),"d_sm.ndl");
  CHECK_DO(hipMemcpy(d_sm.nstrtl, mat2->nstrtl, sizeof(int)*mat2->nlf, hipMemcpyHostToDevice),"d_sm.nstrtl");
  CHECK_DO(hipMemcpy(d_sm.nstrtt, mat2->nstrtt, sizeof(int)*mat2->nlf, hipMemcpyHostToDevice),"d_sm.nstrtt");
  CHECK_DO(hipMemcpy(d_sm.kt, mat2->kt, sizeof(int)*mat2->nlf, hipMemcpyHostToDevice),"d_sm.kt");
  CHECK_DO(hipMemcpy(d_sm.a1, mat2->a1, sizeof(int)*mat2->nlf, hipMemcpyHostToDevice),"d_sm.a1");
  CHECK_DO(hipMemcpy(d_sm.a2, mat2->a2, sizeof(int)*mat2->nlf, hipMemcpyHostToDevice),"d_sm.a2");
  CHECK_DO(hipMemcpy(d_sm.rowmat, mat2->rowmat, sizeof(T)*mat2->len, hipMemcpyHostToDevice),"d_sm.rowmat");
  CHECK_DO(hipMemcpy(d_sm.rowmat_t, mat2->rowmat_t, sizeof(T)*mat2->len, hipMemcpyHostToDevice),"d_sm.rowmat_t");

  // 分離
  printf("begin splitting\n");
  mat2->approx = (int*)malloc(sizeof(int)*mat2->nlf);
  mat2->dense  = (int*)malloc(sizeof(int)*mat2->nlf);
  mat2->napprox = mat2->ndense = 0;
  for(ip=0; ip<mat2->nlf; ip++){
	if(mat2->ltmtx[ip]==1){
	  mat2->approx[mat2->napprox++] = ip;
	}else{
	  mat2->dense[mat2->ndense++] = ip;
	}
  }
  hipMalloc((void**)&d_sm.approx,sizeof(int)*mat2->napprox);
  hipMalloc((void**)&d_sm.dense,sizeof(int)*mat2->ndense);
  CHECK_DO(hipMemcpy(d_sm.approx, mat2->approx, sizeof(int)*mat2->napprox, hipMemcpyHostToDevice),"d_sm.approx");
  CHECK_DO(hipMemcpy(d_sm.dense, mat2->dense, sizeof(int)*mat2->ndense, hipMemcpyHostToDevice),"d_sm.dense");
  d_sm.napprox = mat2->napprox;
  d_sm.ndense  = mat2->ndense;
  printf("end splitting (napprox=%d, ndense=%d)\n", mat2->napprox, mat2->ndense);

#define EXEC(FUNCNAME,BLOCKS,THREADS,S)											\
  printf("nd = %d\n", nd);												\
  for(i=0;i<nd;i++)v[i] = (T)0.0;										\
  hipMemcpy(d_v, v, sizeof(T)*nd, hipMemcpyHostToDevice);				\
  hipMemcpy(d_b, b, sizeof(T)*nd, hipMemcpyHostToDevice);				\
  FUNCNAME<<<BLOCKS,THREADS,S>>>													\
	(d_v, d_b, d_sm.nlf, d_sm.ktmax,									\
	 d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt,			\
	 d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat,							\
	 d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense);				\
  hipDeviceSynchronize();												\
  hipMemcpy(v, d_v, sizeof(T)*nd, hipMemcpyDeviceToHost);				\
  printf("write to %s\n", fname);										\
  F = fopen(fname, "w");												\
  for(i=0;i<nd;i++)fprintf(F, "%.3E\n", v[i]);							\
  fclose(F);

#define BENCH(FUNCNAME,B,T,S)											\
  printf("nd = %d\n", nd);												\
  for(l=0;l<L;l++){														\
	for(i=0;i<nd;i++)v[i] = (T)0.0;										\
	hipMemcpy(d_v, v, sizeof(T)*nd, hipMemcpyHostToDevice);			\
	hipMemcpy(d_b, b, sizeof(T)*nd, hipMemcpyHostToDevice);			\
	hipDeviceSynchronize();											\
	d1 = omp_get_wtime();												\
	FUNCNAME<<<B,T,S>>>													\
	  (d_v, d_b, d_sm.nlf, d_sm.ktmax,									\
	   d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt,		\
	   d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat,							\
	   d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense);				\
	hipDeviceSynchronize();											\
	d2 = omp_get_wtime();												\
	dtimes[l] = d2-d1;													\
  }																		\
  dmin = 9999.99;														\
  dmax = 0.0;															\
  davg = 0.0;															\
  for(i=M;i<L;i++){														\
	davg += dtimes[i];													\
	if(dmin>dtimes[i])dmin=dtimes[i];									\
	if(dmax<dtimes[i])dmax=dtimes[i];									\
  }																		\
  davg /= (L-5);

  // sequential
  if(kernel==0)
  {
	int a1, a2;
	char name[8], fname[0xff];
	a1=a2=0;
	snprintf(name,8,"_%d_%d",a1,a2);
	snprintf(fname,32,"result_cuda1%s_%s.txt", name, typeid(T).name());
	printf("fname = %s\n", fname);
	/*
	printf("nd = %d\n", nd);
	for(i=0;i<nd;i++)v[i] = (T)0.0;
	hipMemcpy(d_v, v, sizeof(T)*nd, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, sizeof(T)*nd, hipMemcpyHostToDevice);
	hmvm_cuda_seq<<<1,1,d_sm.ktmax*sizeof(T)>>>
	  //hmvm_cuda_test<<<1,1,d_sm.ktmax*sizeof(T)>>>
	  //hmvm_cuda_test<<<1,1,d_sm.ktmax*sizeof(double)>>>
	  (d_v, d_b, d_sm.nlf, d_sm.ktmax,
	   d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt,
	   d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat,
	   d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense);
	hipDeviceSynchronize();
	hipMemcpy(v, d_v, sizeof(T)*nd, hipMemcpyDeviceToHost);
	printf("write to %s\n", fname);
	F = fopen(fname, "w");
	for(i=0;i<nd;i++)fprintf(F, "%.3E\n", v[i]);
	fclose(F);
	*/
	EXEC(hmvm_cuda_seq,1,1,d_sm.ktmax*sizeof(T));
	//BENCH(hmvm_cuda_seq<T>,1,1,d_sm.ktmax*sizeof(T));
	printf("TIME %d hmvm_cuda1%s min %e max %e avg %e\n", L-M, typeid(T).name(), dmin, dmax, davg);
  }

  // block parallel
  // whole hmvm calculation in 1 GPU kernel
  if(kernel==1)
  {
	int a1, a2;
	char name[8], fname[32];
	a1=a2=0;
	snprintf(name,8,"_%d_%d",a1,a2);
	snprintf(fname,32,"result_cuda1blk%s_%s.txt", name, typeid(T).name());
	printf("fname = %s\n", fname);
	/*
	printf("nd = %d\n", nd);
	for(i=0;i<nd;i++)v[i] = (T)0.0;
	hipMemcpy(d_v, v, sizeof(T)*nd, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, sizeof(T)*nd, hipMemcpyHostToDevice);
	hmvm_cuda_block<<<d_sm.napprox+d_sm.ndense,1,d_sm.ktmax*sizeof(T)>>>
	  (d_v, d_b, d_sm.nlf, d_sm.ktmax,
	   d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt,
	   d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat,
	   d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense);
	hipDeviceSynchronize();
	hipMemcpy(v, d_v, sizeof(T)*nd, hipMemcpyDeviceToHost);
	printf("write to %s\n", fname);
	F = fopen(fname, "w");
	for(i=0;i<nd;i++)fprintf(F, "%.3E\n", v[i]);
	fclose(F);
	*/
	EXEC(hmvm_cuda_block,d_sm.napprox+d_sm.ndense,1,d_sm.ktmax*sizeof(T));
	//BENCH(hmvm_cuda_block<T>,d_sm.napprox+d_sm.ndense,1,d_sm.ktmax*sizeof(T));
	printf("TIME %d hmvm_cuda1blk%s min %e max %e avg %e\n", L-M, typeid(T).name(), dmin, dmax, davg);
  }

#define EXEC_1(FUNCNAME,BK,TH,S)											\
  printf("nd = %d\n", nd);												\
  for(i=0;i<nd;i++)v[i] = (T)0.0;										\
  hipMemcpy(d_v, v, sizeof(T)*nd, hipMemcpyHostToDevice);				\
  hipMemcpy(d_b, b, sizeof(T)*nd, hipMemcpyHostToDevice);				\
  FUNCNAME<T,1><<<BK,TH,S>>>												\
	(d_v, d_b, d_sm.nlf, d_sm.ktmax,									\
	 d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt,			\
	 d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat,							\
	 d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense);				\
  hipDeviceSynchronize();												\
  hipMemcpy(v, d_v, sizeof(T)*nd, hipMemcpyDeviceToHost);				\
  printf("write to %s\n", fname);										\
  F = fopen(fname, "w");												\
  for(i=0;i<nd;i++)fprintf(F, "%.3E\n", v[i]);							\
  fclose(F);

  // block & thread parallel
  // whole hmvm calculation in 1 GPU kernel
  // under development
  if(kernel==2)
  {
	int a1, a2;
	char name[8], fname[32];
	a1=a2=0;
	snprintf(name,8,"_%d_%d",a1,a2);
	snprintf(fname,32,"result_cuda1hyb%s_%s.txt", name, typeid(T).name());
	printf("fname = %s\n", fname);

	printf("nd = %d\n", nd);
	for(i=0;i<nd;i++)v[i] = (T)0.0;
	hipMemcpy(d_v, v, sizeof(T)*nd, hipMemcpyHostToDevice); // 初期化したはずなのに参照するとおかしくない？
	hipMemcpy(d_b, b, sizeof(T)*nd, hipMemcpyHostToDevice);
	printf("launch %d blocks\n", d_sm.napprox+d_sm.ndense);
	//hmvm_cuda_hybrid0<<<d_sm.napprox+1,32,d_sm.ktmax*sizeof(T)>>>
	hmvm_cuda_hybrid0<<<d_sm.napprox+d_sm.ndense,32,d_sm.ktmax*sizeof(T)>>>
	  (d_v, d_b, d_sm.nlf, d_sm.ktmax,
	   d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt,
	   d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat,
	   d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense, 1);
	/*
	hmvm_cuda_hybrid1<T,1><<<d_sm.napprox+d_sm.ndense,32,d_sm.ktmax*sizeof(T)>>>
	  (d_v, d_b, d_sm.nlf, d_sm.ktmax,
	   d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt,
	   d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat,
	   d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense);
	*/
	hipDeviceSynchronize();
	hipMemcpy(v, d_v, sizeof(T)*nd, hipMemcpyDeviceToHost);
	printf("write to %s\n", fname);
	F = fopen(fname, "w");
	for(i=0;i<nd;i++)fprintf(F, "%.3E\n", v[i]);
	fclose(F);
	//EXEC_1(hmvm_cuda_hybrid1,d_sm.napprox+d_sm.ndense,32,d_sm.ktmax*sizeof(T));
	/*
	proxy_hmvm_cuda_hybrid1<T,1><<<d_sm.napprox+d_sm.ndense,32,d_sm.ktmax*sizeof(T)>>>
	  (d_v, d_b, d_sm.nlf, d_sm.ktmax,
	   d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt,
	   d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat,
	   d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense);
	*/
	//BENCH(hmvm_cuda_hybrid<T>,d_sm.napprox+d_sm.ndense,1,d_sm.ktmax*sizeof(T));
	printf("TIME %d hmvm_cuda1hyb%s min %e max %e avg %e\n", L-M, typeid(T).name(), dmin, dmax, davg);
  }

  // ######## ######## ######## ######## ######## ######## ######## ########
  hipFree(d_sm.ltmtx);
  hipFree(d_sm.ndl);
  hipFree(d_sm.ndt);
  hipFree(d_sm.nstrtl);
  hipFree(d_sm.nstrtt);
  hipFree(d_sm.kt);
  hipFree(d_sm.a1);
  hipFree(d_sm.a2);
  hipFree(d_sm.rowmat);
  hipFree(d_sm.rowmat_t);
  //hipFree(d_zaut);
  //hipFree(d_zbut);
  hipFree(d_b);
  hipFree(d_v);

  delete [] v; delete [] tmp; delete [] zero;
  //free(v); free(tmp); free(zero);
  printf("hmvm_cuda1: end\n");
}


// ######## ######## ######## ######## ######## ######## ######## ########
// template関数の実体化のための宣言
// ######## ######## ######## ######## ######## ######## ######## ########
template void hmvm_cuda1<float>(matrix2<float>  *mat2, float *b, int kernel, int dump_result);
template void hmvm_cuda1<double>(matrix2<double> *mat2, double *b, int kernel, int dump_result);
