#include "hip/hip_runtime.h"
// -*- C++ -*-
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <typeinfo>
#include <omp.h>
#include <hip/hip_runtime_api.h>

#include "hacapk.h"
#include "hmvm_cuda_kernels.h"

#define CHECK_DO(act,msg) {ret=act; if(ret!=hipSuccess){printf("%s failed\n",msg);exit(-1);};}

template<class T>
void hmvm_cuda1(matrix2<T> mat2, T *b, int kernel, int dump_result)
{
  const int L=5, M=5;
  FILE *F;
  matrix2<T> d_sm;
  int i, l, nd = mat2.nd;
  double d1, d2, dtimes[L+M], dmin, dmax, davg1, davg2;
  T *v=NULL, *tmp=NULL, *zero;
  T *d_b, *d_v, *d_zaut, *d_zbut;
  int ip;
  int len, offset=0;
  hipError_t ret;
  printf("hmvm_cuda1_%s: begin\n", typeid(T).name());
  v    = new T[mat2.nd];    //(double*)malloc(sizeof(double)*mat2.nd);
  tmp  = new T[mat2.ktmax]; //(double*)malloc(sizeof(double)*mat2.ktmax);
  zero = new T[mat2.ktmax]; //(double*)malloc(sizeof(double)*mat2.ktmax);
  for(i=0;i<nd;i++){
	v[i] = (T)0.0;
  }
  for(i=0;i<mat2.ktmax;i++){
	zero[i] = (T)0.0;
  }
  CHECK_DO(hipMalloc((void**)&d_zaut, sizeof(T)*mat2.nd),"hipMalloc z_aut");
  CHECK_DO(hipMalloc((void**)&d_zbut, sizeof(T)*mat2.ktmax),"hipMalloc zbut");
  CHECK_DO(hipMalloc((void**)&d_b, sizeof(T)*mat2.nd),"hipMalloc d_b");
  CHECK_DO(hipMalloc((void**)&d_v, sizeof(T)*mat2.nd),"cudaMallod d_v");
  //for(i=0;i<mat2.nd;i++){d_b[i]=NULL;d_v[i]=NULL;}

  len = mat2.len;
  printf("total length = %d\n", len);
  // host alloc
  // device alloc
  d_sm.nd    = mat2.nd;
  d_sm.nlf   = mat2.nlf;
  d_sm.ktmax = mat2.ktmax;
  hipMalloc((void**)&d_sm.ltmtx, sizeof(int)*mat2.nlf);
  hipMalloc((void**)&d_sm.ndl, sizeof(int)*mat2.nlf);
  hipMalloc((void**)&d_sm.ndt, sizeof(int)*mat2.nlf);
  hipMalloc((void**)&d_sm.nstrtl, sizeof(int)*mat2.nlf);
  hipMalloc((void**)&d_sm.nstrtt, sizeof(int)*mat2.nlf);
  hipMalloc((void**)&d_sm.kt, sizeof(int)*mat2.nlf);
  hipMalloc((void**)&d_sm.a1, sizeof(int)*mat2.nlf);
  hipMalloc((void**)&d_sm.a2, sizeof(int)*mat2.nlf);
  hipMalloc((void**)&d_sm.rowmat,sizeof(T)*mat2.len);
  hipMalloc((void**)&d_sm.rowmat_t,sizeof(T)*mat2.len);
  // memcpy
  CHECK_DO(hipMemcpy(d_sm.ltmtx, mat2.ltmtx, sizeof(int)*mat2.nlf, hipMemcpyHostToDevice),"d_sm.ltmtx");
  CHECK_DO(hipMemcpy(d_sm.ndt, mat2.ndt, sizeof(int)*mat2.nlf, hipMemcpyHostToDevice),"d_sm.ndt");
  CHECK_DO(hipMemcpy(d_sm.ndl, mat2.ndl, sizeof(int)*mat2.nlf, hipMemcpyHostToDevice),"d_sm.ndl");
  CHECK_DO(hipMemcpy(d_sm.nstrtl, mat2.nstrtl, sizeof(int)*mat2.nlf, hipMemcpyHostToDevice),"d_sm.nstrtl");
  CHECK_DO(hipMemcpy(d_sm.nstrtt, mat2.nstrtt, sizeof(int)*mat2.nlf, hipMemcpyHostToDevice),"d_sm.nstrtt");
  CHECK_DO(hipMemcpy(d_sm.kt, mat2.kt, sizeof(int)*mat2.nlf, hipMemcpyHostToDevice),"d_sm.kt");
  CHECK_DO(hipMemcpy(d_sm.a1, mat2.a1, sizeof(int)*mat2.nlf, hipMemcpyHostToDevice),"d_sm.a1");
  CHECK_DO(hipMemcpy(d_sm.a2, mat2.a2, sizeof(int)*mat2.nlf, hipMemcpyHostToDevice),"d_sm.a2");
  CHECK_DO(hipMemcpy(d_sm.rowmat, mat2.rowmat, sizeof(T)*mat2.len, hipMemcpyHostToDevice),"d_sm.rowmat");
  CHECK_DO(hipMemcpy(d_sm.rowmat_t, mat2.rowmat_t, sizeof(T)*mat2.len, hipMemcpyHostToDevice),"d_sm.rowmat_t");

  // 分離
  printf("begin splitting\n");
  mat2.approx = (int*)malloc(sizeof(int)*mat2.nlf);
  mat2.dense  = (int*)malloc(sizeof(int)*mat2.nlf);
  mat2.napprox = mat2.ndense = 0;
  for(ip=0; ip<mat2.nlf; ip++){
	if(mat2.ltmtx[ip]==1){
	  mat2.approx[mat2.napprox++] = ip;
	}else{
	  mat2.dense[mat2.ndense++] = ip;
	}
  }
  hipMalloc((void**)&d_sm.approx,sizeof(int)*mat2.napprox);
  hipMalloc((void**)&d_sm.dense,sizeof(int)*mat2.ndense);
  CHECK_DO(hipMemcpy(d_sm.approx, mat2.approx, sizeof(int)*mat2.napprox, hipMemcpyHostToDevice),"d_sm.approx");
  CHECK_DO(hipMemcpy(d_sm.dense, mat2.dense, sizeof(int)*mat2.ndense, hipMemcpyHostToDevice),"d_sm.dense");
  d_sm.napprox = mat2.napprox;
  d_sm.ndense  = mat2.ndense;
  printf("end splitting\n");

#if 0
  hmvm_cudaD_kernel00dd00<16><<<mat2.ndense,32>>>						\
  (d_v, d_b, mat2.nlf, mat2.ktmax, mat2.ltmtx, mat2.ndt, mat2.ndl, mat2.nstrtl, mat2.nstrtt, mat2.kt, mat2.a1, mat2.a2, mat2.rowmat, mat2.ndense, mat2.dense);\


	hmvm_cudaD_kernel000000<<<mat2.ndense,32>>>						\
	  (d_v, d_b, mat2.nlf, mat2.ktmax, mat2.ltmtx, mat2.ndt, mat2.ndl, mat2.nstrtl, mat2.nstrtt, mat2.kt, mat2.a1, mat2.a2, mat2.rowmat, mat2.ndense, mat2.dense); \

  //FUNCNAME(d_v, d_b, d_sm, a1, a2);									\

#endif

#define BENCH(FUNCNAME,B,T,S)											\
  printf("nd = %d\n", nd);												\
  hipMemcpy(d_v, v, sizeof(T)*nd, hipMemcpyHostToDevice);				\
  hipMemcpy(d_b, b, sizeof(T)*nd, hipMemcpyHostToDevice);				\
  FUNCNAME<<<B,T,S>>>													\
	(d_v, d_b, d_sm.nlf, d_sm.ktmax,									\
	 d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt,			\
	 d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat,							\
	 d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense);				\
  hipDeviceSynchronize();												\
  hipMemcpy(v, d_v, sizeof(T)*nd, hipMemcpyDeviceToHost);				\
  printf("write to %s\n", fname);										\
  F = fopen(fname, "w");												\
  for(i=0;i<nd;i++)fprintf(F, "%.3E\n", v[i]);							\
  fclose(F);

#if 0
	hmvm_cudaD<<<1,1,d_sm.ktmax*sizeof(T)>>>							\
	  (d_v, d_b, d_sm.nlf, d_sm.ktmax, d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt, d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat, \
	   d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense);				\

#endif

#if 0
  for(l=0;l<M+L;l++){													\
	for(i=0;i<nd;i++)v[i] = 0.0;										\
	d1 = omp_get_wtime();												\
	hipDeviceSynchronize();											\
	d2 = omp_get_wtime();												\
	dtimes[l] = d2-d1;													\
  }																		\
  dmin = 9999.99;         dmax = 0.0;									\
  davg1 = 0.0;            davg2 = 0.0;									\
  for(i=0;i<M+L;i++)davg1 += dtimes[i];									\
  for(i=M;i<M+L;i++){													\
	if(dmin>dtimes[i])dmin=dtimes[i];									\
	if(dmax<dtimes[i])dmax=dtimes[i];									\
	davg2 += dtimes[i];													\
  }																		\
  davg1 /= (M+L);         davg2 /= L;
#endif

  // sequential
  if(kernel==0)
  {
	int a1, a2;
	char name[8], fname[0xff];
	a1=a2=0;
	snprintf(name,8,"_%d_%d",a1,a2);
	snprintf(fname,32,"result_cuda1%s_%s.txt", name, typeid(T).name());
	printf("fname = %s\n", fname);
	BENCH(hmvm_cuda_seq<T>,1,1,d_sm.ktmax*sizeof(T));
	printf("TIME %d hmvm_cuda1%s min %e max %e avg1 %e avg2 %e |", M+L, name, dmin, dmax, davg1, davg2);
	for(i=0;i<M+L;i++)printf(" %.3E", dtimes[i]);
	printf("\n");
  }
  // block parallel
  if(kernel==1)
  {
	int a1, a2;
	char name[8], fname[32];
	a1=a2=0;
	snprintf(name,8,"_%d_%d",a1,a2);
	snprintf(fname,32,"result_cuda1blk%s_%s.txt", name, typeid(T).name());
	printf("fname = %s\n", fname);
	BENCH(hmvm_cuda_block<T>,d_sm.napprox+d_sm.ndense,1,d_sm.ktmax*sizeof(T));
	printf("TIME %d hmvm_cuda1%s min %e max %e avg1 %e avg2 %e |", M+L, name, dmin, dmax, davg1, davg2);
	for(i=0;i<M+L;i++)printf(" %.3E", dtimes[i]);
	printf("\n");
  }

  // ######## ######## ######## ######## ######## ######## ######## ########
  hipFree(d_sm.ltmtx);
  hipFree(d_sm.ndl);
  hipFree(d_sm.ndt);
  hipFree(d_sm.nstrtl);
  hipFree(d_sm.nstrtt);
  hipFree(d_sm.kt);
  hipFree(d_sm.a1);
  hipFree(d_sm.a2);
  hipFree(d_sm.rowmat);
  hipFree(d_sm.rowmat_t);
  hipFree(d_zaut);
  hipFree(d_zbut);
  hipFree(d_b);
  hipFree(d_v);

  delete [] v; delete [] tmp; delete [] zero;
  //free(v); free(tmp); free(zero);
  printf("hmvm_cuda1: end\n");
}

template void hmvm_cuda1<float>(matrix2<float> mat2, float *b, int kernel, int dump_result);
template void hmvm_cuda1<double>(matrix2<double> mat2, double *b, int kernel, int dump_result);
