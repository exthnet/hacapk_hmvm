#include "hip/hip_runtime.h"
// -*- C++ -*-
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <typeinfo>
#include <omp.h>
#include <hip/hip_runtime_api.h>

#include "hacapk.h"
#include "hmvm_cuda_kernels.cu"

#define CHECK_DO(act,msg) {ret=act; if(ret!=hipSuccess){printf("%s failed\n",msg);exit(-1);};}


template<class T>
void hmvm_cuda1(matrix2<T> *mat2, T *b, int kernel, int dump_result)
{
  const int L=10, M=5;
  FILE *F;
  matrix2<T> d_sm;
  int i, l, nd = mat2->nd, ktmax = mat2->ktmax, nlf = mat2->nlf;
  double d1, d2, dtimes[L], dmin, dmax, davg;
  T *v=NULL, *tmp=NULL, *zero;
  T *d_b, *d_v;//, *d_zaut, *d_zbut;
  int ip;
  int len;
  hipError_t ret;
  printf("hmvm_cuda1_%s: begin\n", typeid(T).name()); fflush(stdout);
  v    = new T[nd];    //(double*)malloc(sizeof(double)*mat2->nd);
  tmp  = new T[ktmax]; //(double*)malloc(sizeof(double)*mat2->ktmax);
  zero = new T[ktmax]; //(double*)malloc(sizeof(double)*mat2->ktmax);
  for(i=0;i<nd;i++){
	v[i] = (T)0.0;
  }
  for(i=0;i<ktmax;i++){
	zero[i] = (T)0.0;
  }
  //CHECK_DO(hipMalloc((void**)&d_zaut, sizeof(T)*mat2->nd),"hipMalloc z_aut");
  //CHECK_DO(hipMalloc((void**)&d_zbut, sizeof(T)*mat2->ktmax),"hipMalloc zbut");
  CHECK_DO(hipMalloc((void**)&d_b, sizeof(T)*nd),"hipMalloc d_b");
  CHECK_DO(hipMalloc((void**)&d_v, sizeof(T)*nd),"hipMalloc d_v");
  //for(i=0;i<mat2->nd;i++){d_b[i]=NULL;d_v[i]=NULL;}

  printf("nd = %d\n", nd);												\
  len = mat2->len;
  printf("total length = %d\n", len);
  // host alloc
  // device alloc
  d_sm.nd    = nd;
  d_sm.nlf   = nlf;
  d_sm.ktmax = ktmax;
  hipMalloc((void**)&d_sm.ltmtx, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.ndl, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.ndt, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.nstrtl, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.nstrtt, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.kt, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.a1, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.a2, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.rowmat,sizeof(T)*mat2->len);
  hipMalloc((void**)&d_sm.rowmat_t,sizeof(T)*mat2->len);
  // memcpy
  CHECK_DO(hipMemcpy(d_sm.ltmtx, mat2->ltmtx, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.ltmtx");
  CHECK_DO(hipMemcpy(d_sm.ndt, mat2->ndt, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.ndt");
  CHECK_DO(hipMemcpy(d_sm.ndl, mat2->ndl, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.ndl");
  CHECK_DO(hipMemcpy(d_sm.nstrtl, mat2->nstrtl, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.nstrtl");
  CHECK_DO(hipMemcpy(d_sm.nstrtt, mat2->nstrtt, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.nstrtt");
  CHECK_DO(hipMemcpy(d_sm.kt, mat2->kt, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.kt");
  CHECK_DO(hipMemcpy(d_sm.a1, mat2->a1, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.a1");
  CHECK_DO(hipMemcpy(d_sm.a2, mat2->a2, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.a2");
  CHECK_DO(hipMemcpy(d_sm.rowmat, mat2->rowmat, sizeof(T)*mat2->len, hipMemcpyHostToDevice),"d_sm.rowmat");
  CHECK_DO(hipMemcpy(d_sm.rowmat_t, mat2->rowmat_t, sizeof(T)*mat2->len, hipMemcpyHostToDevice),"d_sm.rowmat_t");

  // 分離
  printf("begin splitting\n");
  mat2->approx = (int*)malloc(sizeof(int)*nlf);
  mat2->dense  = (int*)malloc(sizeof(int)*nlf);
  mat2->napprox = mat2->ndense = 0;
  for(ip=0; ip<nlf; ip++){
	if(mat2->ltmtx[ip]==1){
	  mat2->approx[mat2->napprox++] = ip;
	}else{
	  mat2->dense[mat2->ndense++] = ip;
	}
  }
  hipMalloc((void**)&d_sm.approx,sizeof(int)*mat2->napprox);
  hipMalloc((void**)&d_sm.dense,sizeof(int)*mat2->ndense);
  CHECK_DO(hipMemcpy(d_sm.approx, mat2->approx, sizeof(int)*mat2->napprox, hipMemcpyHostToDevice),"d_sm.approx");
  CHECK_DO(hipMemcpy(d_sm.dense, mat2->dense, sizeof(int)*mat2->ndense, hipMemcpyHostToDevice),"d_sm.dense");
  d_sm.napprox = mat2->napprox;
  d_sm.ndense  = mat2->ndense;
  printf("end splitting (napprox=%d, ndense=%d)\n", mat2->napprox, mat2->ndense);

#if 0
#define EXEC(FUNCNAME,BLOCKS,THREADS,S)									\
  for(i=0;i<nd;i++)v[i] = (T)0.0;										\
  CHECK_DO(hipMemcpy(d_v, v, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy v to d_v"); \
  CHECK_DO(hipMemcpy(d_b, b, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy b to d_b"); \
  hipDeviceSynchronize();												\
  FUNCNAME<<<BLOCKS,THREADS,S>>>										\
	(d_v, d_b, d_sm.nlf, d_sm.ktmax,									\
	 d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt,			\
	 d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat, d_sm.rowmat_t,				\
	 d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense);				\
  hipDeviceSynchronize();												\
  CHECK_DO(hipMemcpy(v, d_v, sizeof(T)*nd, hipMemcpyDeviceToHost),"hipMemcpy d_v to v"); \
  printf("write to %s\n", fname);										\
  F = fopen(fname, "w");												\
  for(i=0;i<nd;i++)fprintf(F, "%.3E\n", v[i]);							\
  fclose(F);

#define BENCH(FUNCNAME,BLOCKS,THREADS,S)

#define EXEC2(FUNCNAME,BLOCKS,THREADS,S,DIV,OPT)							\
  for(i=0;i<nd;i++)v[i] = (T)0.0;										\
  CHECK_DO(hipMemcpy(d_v, v, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy v to d_v"); \
  CHECK_DO(hipMemcpy(d_b, b, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy b to d_b"); \
  hipDeviceSynchronize();												\
  FUNCNAME<T,DIV,OPT><<<BLOCKS,THREADS,S>>>							\
	(d_v, d_b, d_sm.nlf, d_sm.ktmax,									\
	 d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt,			\
	 d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat, d_sm.rowmat_t,	\
	 d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense);				\
  hipDeviceSynchronize();												\
  CHECK_DO(hipMemcpy(v, d_v, sizeof(T)*nd, hipMemcpyDeviceToHost),"hipMemcpy d_v to v"); \
  printf("write to %s\n", fname);										\
  F = fopen(fname, "w");												\
  for(i=0;i<nd;i++)fprintf(F, "%.3E\n", v[i]);							\
  fclose(F);

#define BENCH2(FUNCNAME,BLOCKS,THREADS,S,DIV,OPT)
#endif

  /*
	完全逐次
	バリエーション：
	- a2trans：approxy2を転置版で計算するか否か(0,1)
	- a2interchange：approxy2のループを入れ替えるか否か(0,1)
  */
  if(kernel>=0 && kernel<4)
  {
	int subkernel = kernel;
	int a2t, a2i;
	a2t = subkernel%2;
	a2i = (subkernel/2)%2;
	char name[64], fname[128];
	snprintf(name,64,"seq_a2t%d_a2i%d%s", a2t, a2i, typeid(T).name());
	snprintf(fname,128,"result_cuda1_%s.txt", name);
	printf("fname = %s\n", fname);
	// EXEC
	//EXEC((hmvm_cuda_seq<T,1,1>),1,1,d_sm.ktmax*sizeof(T));
	hmvm_cuda_seq_proxy
	  (d_v, d_b, d_sm.nlf, d_sm.ktmax,
	   d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt, d_sm.kt,
	   d_sm.a1, d_sm.a2, d_sm.rowmat, d_sm.rowmat_t,
	   d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense,
	   v, b, nd, fname, 0,
	   a2t, a2i);
	if(0)hmvm_cuda_seq_proxy
	  (d_v, d_b, d_sm.nlf, d_sm.ktmax,
	   d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt, d_sm.kt,
	   d_sm.a1, d_sm.a2, d_sm.rowmat, d_sm.rowmat_t,
	   d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense,
	   v, b, nd, fname, 5,
	   a2t, a2i);
	printf("TIME %d hmvm_cuda1_seq%s min %e max %e avg %e\n", L-M, typeid(T).name(), dmin, dmax, davg);
  }

  /*
	block並列化
	ThreadBlockごとに1つの部分行列積(mat-mat-vecまたはmat-vec)を行う
	ThreadBlock内部は逐次
	- a2trans：approxy2を転置版で計算するか否か(0,1)
	- a2interchange：approxy2のループを入れ替えるか否か(0,1)
  */
  if(kernel>=10 && kernel<14)
  {
	int subkernel = kernel-10;
	int a2t, a2i;
	a2t = subkernel%2;
	a2i = (subkernel/2)%2;
	char name[64], fname[128];
	snprintf(name,64,"block_a2t%d_a2i%d%s", a2t, a2i, typeid(T).name());
	snprintf(fname,128,"result_cuda1_%s.txt", name);
	printf("fname = %s\n", fname);
	// EXEC
	hmvm_cuda_block_proxy
	  (d_v, d_b, d_sm.nlf, d_sm.ktmax,
	   d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt,
	   d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat, d_sm.rowmat_t,
	   d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense,
	   v, b, nd, fname, 0,
	   a2t, a2i);
	if(0)hmvm_cuda_block_proxy
	  (d_v, d_b, d_sm.nlf, d_sm.ktmax,
	   d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt,
	   d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat, d_sm.rowmat_t,
	   d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense,
	   v, b, nd, fname, 5,
	   a2t, a2i);
	printf("TIME %d hmvm_cuda1_block%s min %e max %e avg %e\n", L-M, typeid(T).name(), dmin, dmax, davg);
  }

/*
  hybrid1
  基本スレッド並列化カーネル
  1PMVを1TB(32Thread=1WARP)で計算する
  1行を1/div WARPで計算する
  <<<napprox+ndense,32>>>
  1 GEMV by 1 TB
  1 line by 1/div WARP
  バリエーション
  - div：1行を1/divのWARPで計算する、div=1,2,4,8,16,32
  - a2t：a2を転置版で計算するか否か(0,1)
  - a2interchange：a2のループを入れ替えるか否か(0,1)
  - aatomic：approxの計算をatomic優先にするかwarp shuffle併用するか(0,1)
  - datomic：denseの計算をatomic優先にするかwarp shuffle併用するか(0,1)
  6x2x2x2x2=96通り
  何故かdivが8or16の時にa2i=1,aa=1の結果がおかしい
  div8は誤差の範囲かも知れない、div16はよりおかしい
*/
  if(kernel>=1000&&kernel<1096){
	int subkernel = kernel-1000;
	int div, a2t, a2i, aa, da;
	div = subkernel%6;
	a2t = (subkernel/6)%2;
	a2i = ((subkernel/6)/2)%2;
	aa = (((subkernel/6)/2)/2)%2;
	da = ((((subkernel/6)/2)/2)/2)%2;
	div = pow(2,div);
	char name[0xff], fname[0xff];
	snprintf(name,0xff,"hybrid1_div%d_a2t%d_a2i%d_aa%d_da%d_%s", div, a2t, a2i, aa, da, typeid(T).name());
	snprintf(fname,0xff,"result_cuda1_%s.txt", name);
	printf("fname = %s\n", fname);
	// EXEC
	hmvm_cuda_hybrid1_proxy<T>
	  (d_v, d_b, d_sm.nlf, d_sm.ktmax,
	   d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt, d_sm.kt,
	   d_sm.a1, d_sm.a2, d_sm.rowmat, d_sm.rowmat_t,
	   d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense,
	   d_sm.napprox+d_sm.ndense, 32, d_sm.ktmax*sizeof(T),
	   v, b, nd, fname, 0,
	   div, a2t, a2i, aa, da);
	/*
	hmvm_cuda_hybrid1_proxy
	  (d_v, d_b, d_sm.nlf, d_sm.ktmax,
	   d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt, d_sm.kt,
	   d_sm.a1, d_sm.a2, d_sm.rowmat, d_sm.rowmat_t,
	   d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense,
	   d_sm.napprox+d_sm.ndense,32,d_sm.ktmax*sizeof(T),
	   v, b, nd, fname, 0,
	   div, a2t, a2i, aa, da);
	// BENCH
	if(0)hmvm_cuda_hybrid1_proxy
	  (d_v, d_b, d_sm.nlf, d_sm.ktmax,
	   d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt, d_sm.kt,
	   d_sm.a1, d_sm.a2, d_sm.rowmat, d_sm.rowmat_t,
	   d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense,
	   d_sm.napprox+d_sm.ndense,32,d_sm.ktmax*sizeof(T),
	   v, b, nd, fname, 5,
	   div, a2t, a2i, aa, da);
	*/
  }

#if 1
/*
  hybrid2
  複数WARP単一GEMV個別行カーネル
  <<<napprox+ndense,32*mul>>>
  1 GEMV by 1 WARP
  1 line by 1/div WARP
  バリエーション
  - div：1行を1/divのWARPで計算する、div=1,2,4,8,16,32
  - mul：立ち上げるスレッド数(mul*32)、1つのmat-mat-vecまたはmat-vecをmul TBで実行、mul=1,2,3,...,16
  - a2t：a2を転置版で計算するか否か(0,1)
  - a2interchange：a2のループを入れ替えるか否か(0,1)
  - aatomic：approxの計算をatomic優先にするかwarp shuffle併用するか(0,1)
  - datomic：denseの計算をatomic優先にするかwarp shuffle併用するか(0,1)
  6x16x2x2x2x2=1536通り
  全部大丈夫そう
*/
  if(kernel>=10000&&kernel<11536){
	int subkernel = kernel-10000;
	int div, mul, a2t, a2i, aa, da;
	div = subkernel%6;
	mul = (subkernel/6)%16 + 1;
	a2t = ((subkernel/6)/16)%2;
	a2i = (((subkernel/6)/16)/2)%2;
	aa = ((((subkernel/6)/16)/2)/2)%2;
	da = (((((subkernel/6)/16)/2)/2)/2)%2;
	div = pow(2,div);
	if((32*mul)%div)printf("invalid parameters: 32*%d %% %d\n", mul, div);
	char name[0xff], fname[0xff];
	snprintf(name,0xff,"hybrid2_div%d_mul%d_a2t%d_a2i%d_aa%d_da%d_%s", div, mul, a2t, a2i, aa, da, typeid(T).name());
	snprintf(fname,0xff,"result_cuda1_%s.txt", name);
	printf("fname = %s\n", fname);
	//printf("DIV = %d, MUL = %d, ATOMIC = %d\n", DIV, MUL, ATOMIC);
	// EXEC
	hmvm_cuda_hybrid2_proxy<T>
	  (d_v, d_b, d_sm.nlf, d_sm.ktmax,
	   d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt, d_sm.kt,
	   d_sm.a1, d_sm.a2, d_sm.rowmat, d_sm.rowmat_t,
	   d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense,
	   d_sm.napprox+d_sm.ndense, 32*mul, d_sm.ktmax*sizeof(T),
	   v, b, nd, fname, 0,
	   div, mul, a2t, a2i, aa, da);
	// BENCH
	/*
	if(0)hmvm_cuda_hybrid2_proxy
	  (d_v, d_b, d_sm.nlf, d_sm.ktmax,
	   d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt, d_sm.kt,
	   d_sm.a1, d_sm.a2, d_sm.rowmat, d_sm.rowmat_t,
	   d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense,
	   d_sm.napprox+d_sm.ndense,32*mul,d_sm.ktmax*sizeof(T),
	   v, b, nd, fname, 5,
	   div, mul, a2t, a2i, aa, da);
	*/
  }
#endif

#if 1
/*
  hybrid3
  複数WARP個別GEMVカーネル
  1PMVを1WARPが担当
  1TBあたりスレッド数は32*mul(mul WARP)
  PMV内の1行を1/div WARPが担当
  <<<napprox/mul+ndense/mul, 32*mul>>>
  1 GEMV by 1 WARP
  1 line by 1/div WARP
  バリエーション
  - div：1行を1/divのWARPで計算する、div=1,2,4,8,16,32
  - mul：立ち上げるブロック数スレッド数の係数、1つのmat-mat-vecまたはmat-vecを1WARPで実行、mul=1,2,3,...,
  - a2t：a2を転置版で計算するか否か
  - a2interchange：a2のループを入れ替えるか否か
  - aatomic：approxの計算をatomic優先にするかwarp shuffle併用するか
  - datomic：denseの計算をatomic優先にするかwarp shuffle併用するか
*/
  if((kernel>=20000)&&(kernel<21536)){
	int subkernel = kernel-20000;
	int div, mul, a2t, a2i, aa, da;
	div = subkernel%6;
	mul = (subkernel/6)%16 + 1;
	a2t = ((subkernel/6)/16)%2;
	a2i = (((subkernel/6)/16)/2)%2;
	aa = ((((subkernel/6)/16)/2)/2)%2;
	da = (((((subkernel/6)/16)/2)/2)/2)%2;
	div = pow(2,div);
	char name[0xff], fname[0xff];
	snprintf(name,0xff,"hybrid3_div%d_mul%d_a2t%d_a2i%d_aa%d_da%d_%s", div, mul, a2t, a2i, aa, da, typeid(T).name());
	snprintf(fname,0xff,"result_cuda1_%s.txt", name);
	printf("fname = %s\n", fname);
	//printf("DIV = %d, MUL = %d, ATOMIC = %d\n", DIV, MUL, ATOMIC);
	// EXEC
#if 1
	hmvm_cuda_hybrid3_proxy<T>
	  (d_v, d_b, d_sm.nlf, d_sm.ktmax,
	   d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt,
	   d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat, d_sm.rowmat_t,
	   d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense,
	   (d_sm.napprox+mul-1)/mul+(d_sm.ndense+mul-1)/mul, 32*mul, d_sm.ktmax*sizeof(T)*mul,
	   v, b, nd, fname, 0,
	   div, mul, a2t, a2i, aa, da);
	  //(d_sm.napprox+MUL-1)/MUL,32*MUL,d_sm.ktmax*sizeof(T)*MUL,DIV,MUL,ATOMIC, v, b, nd, fname, 0);
	  //(d_sm.ndense+MUL-1)/MUL,32*MUL,d_sm.ktmax*sizeof(T),DIV,MUL,ATOMIC, v, b, nd, fname, 0);
#endif
	// BENCH
	/*
	if(0){
	  hmvm_cuda_hybrid3_proxy<T>(d_v, d_b, d_sm.nlf, d_sm.ktmax,
								 d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt,
								 d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat, d_sm.rowmat_t,
								 d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense,
								 (d_sm.napprox+MUL-1)/MUL+(d_sm.ndense+MUL-1)/MUL,32*MUL,d_sm.ktmax*sizeof(T),DIV,MUL,ATOMIC, v, b, nd, fname, 1);
	}
	*/
  }
#endif


#if 1
/*
  hybrid4
  複数WARP単一GEMV単一行GEMVカーネル
  <<<napprox+ndense, 32*mul>>>
  1 GEMV by mul TB
  1 line by 1/mul WARP
  バリエーション
  - mul：
  - a2t：a2を転置版で計算するか否か
  - a2interchange：a2のループを入れ替えるか否か
  - aatomic：approxの計算をatomic優先にするかwarp shuffle併用するか
  - datomic：denseの計算をatomic優先にするかwarp shuffle併用するか
*/
  if((kernel>=4000)&&(kernel<4192)){
	int subkernel = kernel-4000;
	int MUL, ATOMIC;
	ATOMIC = subkernel/16;
	MUL = subkernel%16;
	ATOMIC = 1; MUL = 1; // test
	char name[32], fname[64];
	snprintf(name,32,"hybrid4_mul%d_atomic%d_%s", MUL, ATOMIC, typeid(T).name());
	snprintf(fname,64,"result_cuda1_%s.txt", name);
	printf("fname = %s\n", fname);
	printf("MUL = %d, ATOMIC = %d\n", MUL, ATOMIC);
	// EXEC
	{
    hmvm_cuda_hybrid4_proxy<T>(d_v, d_b, d_sm.nlf, d_sm.ktmax,
	d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt,
	d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat, d_sm.rowmat_t,
	d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense,
	d_sm.napprox+d_sm.ndense,32*MUL,d_sm.ktmax*sizeof(T)*MUL,MUL,ATOMIC, v, b, nd, fname, 0);
	//(d_sm.napprox+MUL-1)/MUL,32*MUL,d_sm.ktmax*sizeof(T)*MUL,DIV,MUL,ATOMIC, v, b, nd, fname, 0);
	//(d_sm.ndense+MUL-1)/MUL,32*MUL,d_sm.ktmax*sizeof(T),DIV,MUL,ATOMIC, v, b, nd, fname, 0);
	}
	// BENCH
	{
	}
  }
#endif

  // ######## ######## ######## ######## ######## ######## ######## ########
  hipFree(d_sm.ltmtx);
  hipFree(d_sm.ndl);
  hipFree(d_sm.ndt);
  hipFree(d_sm.nstrtl);
  hipFree(d_sm.nstrtt);
  hipFree(d_sm.kt);
  hipFree(d_sm.a1);
  hipFree(d_sm.a2);
  hipFree(d_sm.rowmat);
  hipFree(d_sm.rowmat_t);
  //hipFree(d_zaut);
  //hipFree(d_zbut);
  hipFree(d_b);
  hipFree(d_v);

  delete [] v; delete [] tmp; delete [] zero;
  //free(v); free(tmp); free(zero);
  printf("hmvm_cuda1: end\n");
}


// ######## ######## ######## ######## ######## ######## ######## ########
// template関数の実体化のための宣言
// ######## ######## ######## ######## ######## ######## ######## ########
template void hmvm_cuda1<float>(matrix2<float>  *mat2, float *b, int kernel, int dump_result);
template void hmvm_cuda1<double>(matrix2<double> *mat2, double *b, int kernel, int dump_result);
