#include "hip/hip_runtime.h"
// -*- C++ -*-
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <typeinfo>
#include <omp.h>
#include <hip/hip_runtime_api.h>

#include "hacapk.h"
#include "hmvm_cuda_kernels.h"

#define CHECK_DO(act,msg) {ret=act; if(ret!=hipSuccess){printf("%s failed\n",msg);exit(-1);};}


template<class T>
void hmvm_cuda1(matrix2<T> *mat2, T *b, int kernel, int dump_result)
{
  const int L=10, M=5;
  FILE *F;
  matrix2<T> d_sm;
  int i, l, nd = mat2->nd, ktmax = mat2->ktmax, nlf = mat2->nlf;
  double d1, d2, dtimes[L], dmin, dmax, davg;
  T *v=NULL, *tmp=NULL, *zero;
  T *d_b, *d_v;//, *d_zaut, *d_zbut;
  int ip;
  int len;
  hipError_t ret;
  printf("hmvm_cuda1_%s: begin\n", typeid(T).name());
  v    = new T[nd];    //(double*)malloc(sizeof(double)*mat2->nd);
  tmp  = new T[ktmax]; //(double*)malloc(sizeof(double)*mat2->ktmax);
  zero = new T[ktmax]; //(double*)malloc(sizeof(double)*mat2->ktmax);
  for(i=0;i<nd;i++){
	v[i] = (T)0.0;
  }
  for(i=0;i<ktmax;i++){
	zero[i] = (T)0.0;
  }
  //CHECK_DO(hipMalloc((void**)&d_zaut, sizeof(T)*mat2->nd),"hipMalloc z_aut");
  //CHECK_DO(hipMalloc((void**)&d_zbut, sizeof(T)*mat2->ktmax),"hipMalloc zbut");
  CHECK_DO(hipMalloc((void**)&d_b, sizeof(T)*nd),"hipMalloc d_b");
  CHECK_DO(hipMalloc((void**)&d_v, sizeof(T)*nd),"hipMalloc d_v");
  //for(i=0;i<mat2->nd;i++){d_b[i]=NULL;d_v[i]=NULL;}

  printf("nd = %d\n", nd);												\
  len = mat2->len;
  printf("total length = %d\n", len);
  // host alloc
  // device alloc
  d_sm.nd    = nd;
  d_sm.nlf   = nlf;
  d_sm.ktmax = ktmax;
  hipMalloc((void**)&d_sm.ltmtx, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.ndl, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.ndt, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.nstrtl, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.nstrtt, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.kt, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.a1, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.a2, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.rowmat,sizeof(T)*mat2->len);
  hipMalloc((void**)&d_sm.rowmat_t,sizeof(T)*mat2->len);
  // memcpy
  CHECK_DO(hipMemcpy(d_sm.ltmtx, mat2->ltmtx, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.ltmtx");
  CHECK_DO(hipMemcpy(d_sm.ndt, mat2->ndt, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.ndt");
  CHECK_DO(hipMemcpy(d_sm.ndl, mat2->ndl, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.ndl");
  CHECK_DO(hipMemcpy(d_sm.nstrtl, mat2->nstrtl, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.nstrtl");
  CHECK_DO(hipMemcpy(d_sm.nstrtt, mat2->nstrtt, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.nstrtt");
  CHECK_DO(hipMemcpy(d_sm.kt, mat2->kt, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.kt");
  CHECK_DO(hipMemcpy(d_sm.a1, mat2->a1, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.a1");
  CHECK_DO(hipMemcpy(d_sm.a2, mat2->a2, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.a2");
  CHECK_DO(hipMemcpy(d_sm.rowmat, mat2->rowmat, sizeof(T)*mat2->len, hipMemcpyHostToDevice),"d_sm.rowmat");
  CHECK_DO(hipMemcpy(d_sm.rowmat_t, mat2->rowmat_t, sizeof(T)*mat2->len, hipMemcpyHostToDevice),"d_sm.rowmat_t");

  // 分離
  printf("begin splitting\n");
  mat2->approx = (int*)malloc(sizeof(int)*nlf);
  mat2->dense  = (int*)malloc(sizeof(int)*nlf);
  mat2->napprox = mat2->ndense = 0;
  for(ip=0; ip<nlf; ip++){
	if(mat2->ltmtx[ip]==1){
	  mat2->approx[mat2->napprox++] = ip;
	}else{
	  mat2->dense[mat2->ndense++] = ip;
	}
  }
  hipMalloc((void**)&d_sm.approx,sizeof(int)*mat2->napprox);
  hipMalloc((void**)&d_sm.dense,sizeof(int)*mat2->ndense);
  CHECK_DO(hipMemcpy(d_sm.approx, mat2->approx, sizeof(int)*mat2->napprox, hipMemcpyHostToDevice),"d_sm.approx");
  CHECK_DO(hipMemcpy(d_sm.dense, mat2->dense, sizeof(int)*mat2->ndense, hipMemcpyHostToDevice),"d_sm.dense");
  d_sm.napprox = mat2->napprox;
  d_sm.ndense  = mat2->ndense;
  printf("end splitting (napprox=%d, ndense=%d)\n", mat2->napprox, mat2->ndense);

#define EXEC(FUNCNAME,BLOCKS,THREADS,S)									\
  for(i=0;i<nd;i++)v[i] = (T)0.0;										\
  CHECK_DO(hipMemcpy(d_v, v, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy v to d_v"); \
  CHECK_DO(hipMemcpy(d_b, b, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy b to d_b"); \
  hipDeviceSynchronize();												\
  FUNCNAME<<<BLOCKS,THREADS,S>>>										\
	(d_v, d_b, d_sm.nlf, d_sm.ktmax,									\
	 d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt,			\
	 d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat,							\
	 d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense);				\
  hipDeviceSynchronize();												\
  CHECK_DO(hipMemcpy(v, d_v, sizeof(T)*nd, hipMemcpyDeviceToHost),"hipMemcpy d_v to v"); \
  printf("write to %s\n", fname);										\
  F = fopen(fname, "w");												\
  for(i=0;i<nd;i++)fprintf(F, "%.3E\n", v[i]);							\
  fclose(F);

#define BENCH(FUNCNAME,BLOCKS,THREADS,S)
#if 0
#define BENCH(FUNCNAME,BLOCKS,THREADS,S)								\
  for(l=0;l<L;l++){														\
	for(i=0;i<nd;i++)v[i] = (T)0.0;										\
	CHECK_DO(hipMemcpy(d_v, v, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy v to d_v"); \
	CHECK_DO(hipMemcpy(d_b, b, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy b to d_b"); \
	hipDeviceSynchronize();											\
	d1 = omp_get_wtime();												\
	FUNCNAME<<<BLOCKS,THREADS,S>>>										\
	  (d_v, d_b, d_sm.nlf, d_sm.ktmax,									\
	   d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt,		\
	   d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat,							\
	   d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense);				\
	hipDeviceSynchronize();											\
	d2 = omp_get_wtime();												\
	dtimes[l] = d2-d1;													\
  }																		\
  dmin = 9999.99;														\
  dmax = 0.0;															\
  davg = 0.0;															\
  for(i=M;i<L;i++){														\
	davg += dtimes[i];													\
	if(dmin>dtimes[i])dmin=dtimes[i];									\
	if(dmax<dtimes[i])dmax=dtimes[i];									\
  }																		\
  davg /= (L-M);
#endif

#define EXEC2(FUNCNAME,BLOCKS,THREADS,S,DIV,OPT)							\
  for(i=0;i<nd;i++)v[i] = (T)0.0;										\
  CHECK_DO(hipMemcpy(d_v, v, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy v to d_v"); \
  CHECK_DO(hipMemcpy(d_b, b, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy b to d_b"); \
  hipDeviceSynchronize();												\
  FUNCNAME<T,DIV,OPT><<<BLOCKS,THREADS,S>>>							\
	(d_v, d_b, d_sm.nlf, d_sm.ktmax,									\
	 d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt,			\
	 d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat,							\
	 d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense);				\
  hipDeviceSynchronize();												\
  CHECK_DO(hipMemcpy(v, d_v, sizeof(T)*nd, hipMemcpyDeviceToHost),"hipMemcpy d_v to v"); \
  printf("write to %s\n", fname);										\
  F = fopen(fname, "w");												\
  for(i=0;i<nd;i++)fprintf(F, "%.3E\n", v[i]);							\
  fclose(F);

#define BENCH2(FUNCNAME,BLOCKS,THREADS,S,DIV,OPT)
#if 0
#define BENCH2(FUNCNAME,BLOCKS,THREADS,S,DIV,OPT)							\
  for(l=0;l<L;l++){														\
	for(i=0;i<nd;i++)v[i] = (T)0.0;										\
	CHECK_DO(hipMemcpy(d_v, v, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy v to d_v"); \
	CHECK_DO(hipMemcpy(d_b, b, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy b to d_b"); \
	hipDeviceSynchronize();											\
	d1 = omp_get_wtime();												\
	FUNCNAME<T,DIV,OPT><<<BLOCKS,THREADS,S>>>						\
	  (d_v, d_b, d_sm.nlf, d_sm.ktmax,									\
	   d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt,		\
	   d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat,							\
	   d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense);				\
	hipDeviceSynchronize();											\
	d2 = omp_get_wtime();												\
	dtimes[l] = d2-d1;													\
  }																		\
  dmin = 9999.99;														\
  dmax = 0.0;															\
  davg = 0.0;															\
  for(i=M;i<L;i++){														\
	davg += dtimes[i];													\
	if(dmin>dtimes[i])dmin=dtimes[i];									\
	if(dmax<dtimes[i])dmax=dtimes[i];									\
  }																		\
  davg /= (L-M);
#endif

  // sequential
  if(kernel==0)
  {
	int a1, a2;
	char name[8], fname[0xff];
	a1=a2=0;
	snprintf(name,8,"_%d_%d",a1,a2);
	snprintf(fname,32,"result_cuda1_seq%s_%s.txt", name, typeid(T).name());
	printf("fname = %s\n", fname);
	EXEC(hmvm_cuda_seq,1,1,d_sm.ktmax*sizeof(T));
	BENCH(hmvm_cuda_seq,1,1,d_sm.ktmax*sizeof(T));
	printf("TIME %d hmvm_cuda1_seq%s min %e max %e avg %e\n", L-M, typeid(T).name(), dmin, dmax, davg);
  }

  // block parallel
  // whole hmvm calculation in 1 GPU kernel
  if(kernel==1)
  {
	int a1, a2;
	char name[8], fname[32];
	a1=a2=0;
	snprintf(name,8,"_%d_%d",a1,a2);
	snprintf(fname,32,"result_cuda1_block%s_%s.txt", name, typeid(T).name());
	printf("fname = %s\n", fname);
	EXEC(hmvm_cuda_block,d_sm.napprox+d_sm.ndense,1,d_sm.ktmax*sizeof(T));
	BENCH(hmvm_cuda_block,d_sm.napprox+d_sm.ndense,1,d_sm.ktmax*sizeof(T));
	printf("TIME %d hmvm_cuda1_block%s min %e max %e avg %e\n", L-M, typeid(T).name(), dmin, dmax, davg);
  }

  // block & thread parallel
  // whole hmvm calculation in 1 GPU kernel
  // under development
  if(kernel==10)
  {
	char name[32], fname[64];
	snprintf(name,32,"hybrid1_div1_shuffl_%d_%s", kernel, typeid(T).name());
	snprintf(fname,64,"result_cuda1_%s.txt", name);
	printf("fname = %s\n", fname);
	EXEC((hmvm_cuda_hybrid1<T,1,0>),d_sm.napprox+d_sm.ndense,32,d_sm.ktmax*sizeof(T));
	BENCH((hmvm_cuda_hybrid1<T,1,0>),d_sm.napprox+d_sm.ndense,32,d_sm.ktmax*sizeof(T));
	printf("TIME %d hmvm_cuda1_%s min %e max %e avg %e\n", L-M, name, dmin, dmax, davg);
  }
  if(kernel==11)
  {
	char name[32], fname[64];
	snprintf(name,32,"hybrid1_div2_shuffl_%d_%s", kernel, typeid(T).name());
	snprintf(fname,64,"result_cuda1_%s.txt", name);
	printf("fname = %s\n", fname);
	EXEC((hmvm_cuda_hybrid1<T,2,0>),d_sm.napprox+d_sm.ndense,32,d_sm.ktmax*sizeof(T));
	BENCH((hmvm_cuda_hybrid1<T,2,0>),d_sm.napprox+d_sm.ndense,32,d_sm.ktmax*sizeof(T));
	printf("TIME %d hmvm_cuda1_%s min %e max %e avg %e\n", L-M, name, dmin, dmax, davg);
  }
  if(kernel==12)
  {
	char name[32], fname[64];
	snprintf(name,32,"hybrid1_div4_shuffl_%d_%s", kernel, typeid(T).name());
	snprintf(fname,64,"result_cuda1_%s.txt", name);
	printf("fname = %s\n", fname);
	EXEC((hmvm_cuda_hybrid1<T,4,0>),d_sm.napprox+d_sm.ndense,32,d_sm.ktmax*sizeof(T));
	BENCH((hmvm_cuda_hybrid1<T,4,0>),d_sm.napprox+d_sm.ndense,32,d_sm.ktmax*sizeof(T));
	printf("TIME %d hmvm_cuda1_%s min %e max %e avg %e\n", L-M, name, dmin, dmax, davg);
  }
  if(kernel==13)
  {
	char name[32], fname[64];
	snprintf(name,32,"hybrid1_div8_shuffl_%d_%s", kernel, typeid(T).name());
	snprintf(fname,64,"result_cuda1_%s.txt", name);
	printf("fname = %s\n", fname);
	EXEC((hmvm_cuda_hybrid1<T,8,0>),d_sm.napprox+d_sm.ndense,32,d_sm.ktmax*sizeof(T));
	BENCH((hmvm_cuda_hybrid1<T,8,0>),d_sm.napprox+d_sm.ndense,32,d_sm.ktmax*sizeof(T));
	printf("TIME %d hmvm_cuda1_%s min %e max %e avg %e\n", L-M, name, dmin, dmax, davg);
  }
  if(kernel==14)
  {
	char name[32], fname[64];
	snprintf(name,32,"hybrid1_div16_shuffl_%d_%s", kernel, typeid(T).name());
	snprintf(fname,64,"result_cuda1_%s.txt", name);
	printf("fname = %s\n", fname);
	EXEC((hmvm_cuda_hybrid1<T,16,0>),d_sm.napprox+d_sm.ndense,32,d_sm.ktmax*sizeof(T));
	BENCH((hmvm_cuda_hybrid1<T,16,0>),d_sm.napprox+d_sm.ndense,32,d_sm.ktmax*sizeof(T));
	printf("TIME %d hmvm_cuda1_%s min %e max %e avg %e\n", L-M, name, dmin, dmax, davg);
  }
  if(kernel==15)
  {
	char name[32], fname[64];
	snprintf(name,32,"hybrid1_div32_shuffl_%d_%s", kernel, typeid(T).name());
	snprintf(fname,64,"result_cuda1_%s.txt", name);
	printf("fname = %s\n", fname);
	EXEC((hmvm_cuda_hybrid1<T,32,0>),d_sm.napprox+d_sm.ndense,32,d_sm.ktmax*sizeof(T));
	BENCH((hmvm_cuda_hybrid1<T,32,0>),d_sm.napprox+d_sm.ndense,32,d_sm.ktmax*sizeof(T));
	printf("TIME %d hmvm_cuda1_%s min %e max %e avg %e\n", L-M, name, dmin, dmax, davg);
  }

  if(kernel==20)
  {
	char name[32], fname[64];
	snprintf(name,32,"hybrid1_div1_atomic_%d_%s", kernel, typeid(T).name());
	snprintf(fname,64,"result_cuda1_%s.txt", name);
	printf("fname = %s\n", fname);
	EXEC((hmvm_cuda_hybrid1<T,1,1>),d_sm.napprox+d_sm.ndense,32,d_sm.ktmax*sizeof(T));
	BENCH((hmvm_cuda_hybrid1<T,1,1>),d_sm.napprox+d_sm.ndense,32,d_sm.ktmax*sizeof(T));
	printf("TIME %d hmvm_cuda1_hybrid1b%s min %e max %e avg %e\n", L-M, typeid(T).name(), dmin, dmax, davg);
  }
  if(kernel==21)
  {
	char name[32], fname[64];
	snprintf(name,32,"hybrid1_div2_atomic_%d_%s", kernel, typeid(T).name());
	snprintf(fname,64,"result_cuda1_%s.txt", name);
	printf("fname = %s\n", fname);
	EXEC((hmvm_cuda_hybrid1<T,2,1>),d_sm.napprox+d_sm.ndense,32,d_sm.ktmax*sizeof(T));
	BENCH((hmvm_cuda_hybrid1<T,2,1>),d_sm.napprox+d_sm.ndense,32,d_sm.ktmax*sizeof(T));
	printf("TIME %d hmvm_cuda1_hybrid1b%s min %e max %e avg %e\n", L-M, typeid(T).name(), dmin, dmax, davg);
  }
  if(kernel==22)
  {
	char name[32], fname[64];
	snprintf(name,32,"hybrid1_div4_atomic_%d_%s", kernel, typeid(T).name());
	snprintf(fname,64,"result_cuda1_%s.txt", name);
	printf("fname = %s\n", fname);
	EXEC((hmvm_cuda_hybrid1<T,4,1>),d_sm.napprox+d_sm.ndense,32,d_sm.ktmax*sizeof(T));
	BENCH((hmvm_cuda_hybrid1<T,4,1>),d_sm.napprox+d_sm.ndense,32,d_sm.ktmax*sizeof(T));
	printf("TIME %d hmvm_cuda1_hybrid1b%s min %e max %e avg %e\n", L-M, typeid(T).name(), dmin, dmax, davg);
  }
  if(kernel==23)
  {
	char name[32], fname[64];
	snprintf(name,32,"hybrid1_div8_atomic_%d_%s", kernel, typeid(T).name());
	snprintf(fname,64,"result_cuda1_%s.txt", name);
	printf("fname = %s\n", fname);
	EXEC((hmvm_cuda_hybrid1<T,8,1>),d_sm.napprox+d_sm.ndense,32,d_sm.ktmax*sizeof(T));
	BENCH((hmvm_cuda_hybrid1<T,8,1>),d_sm.napprox+d_sm.ndense,32,d_sm.ktmax*sizeof(T));
	printf("TIME %d hmvm_cuda1_hybrid1b%s min %e max %e avg %e\n", L-M, typeid(T).name(), dmin, dmax, davg);
  }
  if(kernel==24)
  {
	char name[32], fname[64];
	snprintf(name,32,"hybrid1_div16_atomic_%d_%s", kernel, typeid(T).name());
	snprintf(fname,64,"result_cuda1_%s.txt", name);
	printf("fname = %s\n", fname);
	EXEC((hmvm_cuda_hybrid1<T,16,1>),d_sm.napprox+d_sm.ndense,32,d_sm.ktmax*sizeof(T));
	BENCH((hmvm_cuda_hybrid1<T,16,1>),d_sm.napprox+d_sm.ndense,32,d_sm.ktmax*sizeof(T));
	printf("TIME %d hmvm_cuda1_hybrid1b%s min %e max %e avg %e\n", L-M, typeid(T).name(), dmin, dmax, davg);
  }
  if(kernel==25)
  {
	char name[32], fname[64];
	snprintf(name,32,"hybrid1_div32_atomic_%d_%s", kernel, typeid(T).name());
	snprintf(fname,64,"result_cuda1_%s.txt", name);
	printf("fname = %s\n", fname);
	EXEC((hmvm_cuda_hybrid1<T,32,1>),d_sm.napprox+d_sm.ndense,32,d_sm.ktmax*sizeof(T));
	BENCH((hmvm_cuda_hybrid1<T,32,1>),d_sm.napprox+d_sm.ndense,32,d_sm.ktmax*sizeof(T));
	printf("TIME %d hmvm_cuda1_hybrid1b%s min %e max %e avg %e\n", L-M, typeid(T).name(), dmin, dmax, davg);
  }

#if 1
  if(kernel>=1000&&kernel<1192){
	int subkernel = kernel-1000;
	int DIV, MUL, ATOMIC;
	ATOMIC = subkernel/96;
	MUL = (subkernel%96)/6 + 1;
	DIV = pow(2,subkernel%6);
	char name[32], fname[64];
	snprintf(name,32,"hybrid2_div%d_mul%d_atomic%d_%s", DIV, MUL, ATOMIC, typeid(T).name());
	snprintf(fname,64,"result_cuda1_%s.txt", name);
	printf("fname = %s\n", fname);
	printf("DIV = %d, MUL = %d, ATOMIC = %d\n", DIV, MUL, ATOMIC);
	// EXEC
	{
	  hmvm_cuda_hybrid2_proxy<T>(d_v, d_b, d_sm.nlf, d_sm.ktmax,
								 d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt,
								 d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat,
								 d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense,
								 d_sm.napprox+d_sm.ndense,32*MUL,d_sm.ktmax*sizeof(T),DIV,MUL,ATOMIC, v, b, nd, fname, 0);
	}
	// BENCH
	{
	  hmvm_cuda_hybrid2_proxy<T>(d_v, d_b, d_sm.nlf, d_sm.ktmax,
								 d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt,
								 d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat,
								 d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense,
								 d_sm.napprox+d_sm.ndense,32*MUL,d_sm.ktmax*sizeof(T),DIV,MUL,ATOMIC, v, b, nd, fname, 1);
	}
  }
#endif

#if 1
  if(kernel>=2000&&kernel<2192){
	int subkernel = kernel-2000;
	int DIV, MUL, ATOMIC;
	ATOMIC = subkernel/96;
	MUL = (subkernel%96)/6 + 1;
	DIV = pow(2,subkernel%6);
	ATOMIC = 1; MUL = 1; DIV = 1; // test
	char name[32], fname[64];
	snprintf(name,32,"hybrid3_div%d_mul%d_atomic%d_%s", DIV, MUL, ATOMIC, typeid(T).name());
	snprintf(fname,64,"result_cuda1_%s.txt", name);
	printf("fname = %s\n", fname);
	printf("DIV = %d, MUL = %d, ATOMIC = %d\n", DIV, MUL, ATOMIC);
	// EXEC
	{
	  hmvm_cuda_hybrid3_proxy<T>(d_v, d_b, d_sm.nlf, d_sm.ktmax,
								 d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt,
								 d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat,
								 d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense,
								 (d_sm.napprox+MUL-1)/MUL+(d_sm.ndense+MUL-1)/MUL,32*MUL,d_sm.ktmax*sizeof(T),DIV,MUL,ATOMIC, v, b, nd, fname, 0);
	}
	// BENCH
	if(0){
	  hmvm_cuda_hybrid3_proxy<T>(d_v, d_b, d_sm.nlf, d_sm.ktmax,
								 d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt,
								 d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat,
								 d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense,
								 (d_sm.napprox+MUL-1)/MUL+(d_sm.ndense+MUL-1)/MUL,32*MUL,d_sm.ktmax*sizeof(T),DIV,MUL,ATOMIC, v, b, nd, fname, 1);
	}
  }
#endif
  // ######## ######## ######## ######## ######## ######## ######## ########
  hipFree(d_sm.ltmtx);
  hipFree(d_sm.ndl);
  hipFree(d_sm.ndt);
  hipFree(d_sm.nstrtl);
  hipFree(d_sm.nstrtt);
  hipFree(d_sm.kt);
  hipFree(d_sm.a1);
  hipFree(d_sm.a2);
  hipFree(d_sm.rowmat);
  hipFree(d_sm.rowmat_t);
  //hipFree(d_zaut);
  //hipFree(d_zbut);
  hipFree(d_b);
  hipFree(d_v);

  delete [] v; delete [] tmp; delete [] zero;
  //free(v); free(tmp); free(zero);
  printf("hmvm_cuda1: end\n");
}


// ######## ######## ######## ######## ######## ######## ######## ########
// template関数の実体化のための宣言
// ######## ######## ######## ######## ######## ######## ######## ########
template void hmvm_cuda1<float>(matrix2<float>  *mat2, float *b, int kernel, int dump_result);
template void hmvm_cuda1<double>(matrix2<double> *mat2, double *b, int kernel, int dump_result);
