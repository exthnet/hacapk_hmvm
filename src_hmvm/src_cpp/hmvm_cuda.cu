#include "hip/hip_runtime.h"
// -*- C++ -*-
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <typeinfo>
#include <omp.h>
#include <hip/hip_runtime_api.h>

#include "hacapk.h"
#include "hmvm_cuda_kernels.h"

#define CHECK_DO(act,msg) {ret=act; if(ret!=hipSuccess){printf("%s failed\n",msg);exit(-1);};}


template<class T>
void hmvm_cuda1(matrix2<T> *mat2, T *b, int kernel, int dump_result)
{
  const int L=10, M=5;
  FILE *F;
  matrix2<T> d_sm;
  int i, l, nd = mat2->nd, ktmax = mat2->ktmax, nlf = mat2->nlf;
  double d1, d2, dtimes[L], dmin, dmax, davg;
  T *v=NULL, *tmp=NULL, *zero;
  T *d_b, *d_v;//, *d_zaut, *d_zbut;
  int ip;
  int len;
  hipError_t ret;
  printf("hmvm_cuda1_%s: begin\n", typeid(T).name());
  v    = new T[nd];    //(double*)malloc(sizeof(double)*mat2->nd);
  tmp  = new T[ktmax]; //(double*)malloc(sizeof(double)*mat2->ktmax);
  zero = new T[ktmax]; //(double*)malloc(sizeof(double)*mat2->ktmax);
  for(i=0;i<nd;i++){
	v[i] = (T)0.0;
  }
  for(i=0;i<ktmax;i++){
	zero[i] = (T)0.0;
  }
  //CHECK_DO(hipMalloc((void**)&d_zaut, sizeof(T)*mat2->nd),"hipMalloc z_aut");
  //CHECK_DO(hipMalloc((void**)&d_zbut, sizeof(T)*mat2->ktmax),"hipMalloc zbut");
  CHECK_DO(hipMalloc((void**)&d_b, sizeof(T)*nd),"hipMalloc d_b");
  CHECK_DO(hipMalloc((void**)&d_v, sizeof(T)*nd),"hipMalloc d_v");
  //for(i=0;i<mat2->nd;i++){d_b[i]=NULL;d_v[i]=NULL;}

  printf("nd = %d\n", nd);												\
  len = mat2->len;
  printf("total length = %d\n", len);
  // host alloc
  // device alloc
  d_sm.nd    = nd;
  d_sm.nlf   = nlf;
  d_sm.ktmax = ktmax;
  hipMalloc((void**)&d_sm.ltmtx, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.ndl, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.ndt, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.nstrtl, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.nstrtt, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.kt, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.a1, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.a2, sizeof(int)*nlf);
  hipMalloc((void**)&d_sm.rowmat,sizeof(T)*mat2->len);
  hipMalloc((void**)&d_sm.rowmat_t,sizeof(T)*mat2->len);
  // memcpy
  CHECK_DO(hipMemcpy(d_sm.ltmtx, mat2->ltmtx, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.ltmtx");
  CHECK_DO(hipMemcpy(d_sm.ndt, mat2->ndt, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.ndt");
  CHECK_DO(hipMemcpy(d_sm.ndl, mat2->ndl, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.ndl");
  CHECK_DO(hipMemcpy(d_sm.nstrtl, mat2->nstrtl, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.nstrtl");
  CHECK_DO(hipMemcpy(d_sm.nstrtt, mat2->nstrtt, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.nstrtt");
  CHECK_DO(hipMemcpy(d_sm.kt, mat2->kt, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.kt");
  CHECK_DO(hipMemcpy(d_sm.a1, mat2->a1, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.a1");
  CHECK_DO(hipMemcpy(d_sm.a2, mat2->a2, sizeof(int)*nlf, hipMemcpyHostToDevice),"d_sm.a2");
  CHECK_DO(hipMemcpy(d_sm.rowmat, mat2->rowmat, sizeof(T)*mat2->len, hipMemcpyHostToDevice),"d_sm.rowmat");
  CHECK_DO(hipMemcpy(d_sm.rowmat_t, mat2->rowmat_t, sizeof(T)*mat2->len, hipMemcpyHostToDevice),"d_sm.rowmat_t");

  // 分離
  printf("begin splitting\n");
  mat2->approx = (int*)malloc(sizeof(int)*nlf);
  mat2->dense  = (int*)malloc(sizeof(int)*nlf);
  mat2->napprox = mat2->ndense = 0;
  for(ip=0; ip<nlf; ip++){
	if(mat2->ltmtx[ip]==1){
	  mat2->approx[mat2->napprox++] = ip;
	}else{
	  mat2->dense[mat2->ndense++] = ip;
	}
  }
  hipMalloc((void**)&d_sm.approx,sizeof(int)*mat2->napprox);
  hipMalloc((void**)&d_sm.dense,sizeof(int)*mat2->ndense);
  CHECK_DO(hipMemcpy(d_sm.approx, mat2->approx, sizeof(int)*mat2->napprox, hipMemcpyHostToDevice),"d_sm.approx");
  CHECK_DO(hipMemcpy(d_sm.dense, mat2->dense, sizeof(int)*mat2->ndense, hipMemcpyHostToDevice),"d_sm.dense");
  d_sm.napprox = mat2->napprox;
  d_sm.ndense  = mat2->ndense;
  printf("end splitting (napprox=%d, ndense=%d)\n", mat2->napprox, mat2->ndense);

#define EXEC(FUNCNAME,BLOCKS,THREADS,S)									\
  for(i=0;i<nd;i++)v[i] = (T)0.0;										\
  CHECK_DO(hipMemcpy(d_v, v, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy v to d_v"); \
  CHECK_DO(hipMemcpy(d_b, b, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy b to d_b"); \
  hipDeviceSynchronize();												\
  FUNCNAME<<<BLOCKS,THREADS,S>>>										\
	(d_v, d_b, d_sm.nlf, d_sm.ktmax,									\
	 d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt,			\
	 d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat,							\
	 d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense);				\
  hipDeviceSynchronize();												\
  CHECK_DO(hipMemcpy(v, d_v, sizeof(T)*nd, hipMemcpyDeviceToHost),"hipMemcpy d_v to v"); \
  printf("write to %s\n", fname);										\
  F = fopen(fname, "w");												\
  for(i=0;i<nd;i++)fprintf(F, "%.3E\n", v[i]);							\
  fclose(F);

#define BENCH(FUNCNAME,BLOCKS,THREADS,S)
#if 0
#define BENCH(FUNCNAME,BLOCKS,THREADS,S)								\
  for(l=0;l<L;l++){														\
	for(i=0;i<nd;i++)v[i] = (T)0.0;										\
	CHECK_DO(hipMemcpy(d_v, v, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy v to d_v"); \
	CHECK_DO(hipMemcpy(d_b, b, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy b to d_b"); \
	hipDeviceSynchronize();											\
	d1 = omp_get_wtime();												\
	FUNCNAME<<<BLOCKS,THREADS,S>>>										\
	  (d_v, d_b, d_sm.nlf, d_sm.ktmax,									\
	   d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt,		\
	   d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat,							\
	   d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense);				\
	hipDeviceSynchronize();											\
	d2 = omp_get_wtime();												\
	dtimes[l] = d2-d1;													\
  }																		\
  dmin = 9999.99;														\
  dmax = 0.0;															\
  davg = 0.0;															\
  for(i=M;i<L;i++){														\
	davg += dtimes[i];													\
	if(dmin>dtimes[i])dmin=dtimes[i];									\
	if(dmax<dtimes[i])dmax=dtimes[i];									\
  }																		\
  davg /= (L-M);
#endif

#define EXEC2(FUNCNAME,BLOCKS,THREADS,S,DIV,OPT)							\
  for(i=0;i<nd;i++)v[i] = (T)0.0;										\
  CHECK_DO(hipMemcpy(d_v, v, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy v to d_v"); \
  CHECK_DO(hipMemcpy(d_b, b, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy b to d_b"); \
  hipDeviceSynchronize();												\
  FUNCNAME<T,DIV,OPT><<<BLOCKS,THREADS,S>>>							\
	(d_v, d_b, d_sm.nlf, d_sm.ktmax,									\
	 d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt,			\
	 d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat,							\
	 d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense);				\
  hipDeviceSynchronize();												\
  CHECK_DO(hipMemcpy(v, d_v, sizeof(T)*nd, hipMemcpyDeviceToHost),"hipMemcpy d_v to v"); \
  printf("write to %s\n", fname);										\
  F = fopen(fname, "w");												\
  for(i=0;i<nd;i++)fprintf(F, "%.3E\n", v[i]);							\
  fclose(F);

#define BENCH2(FUNCNAME,BLOCKS,THREADS,S,DIV,OPT)
#if 0
#define BENCH2(FUNCNAME,BLOCKS,THREADS,S,DIV,OPT)							\
  for(l=0;l<L;l++){														\
	for(i=0;i<nd;i++)v[i] = (T)0.0;										\
	CHECK_DO(hipMemcpy(d_v, v, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy v to d_v"); \
	CHECK_DO(hipMemcpy(d_b, b, sizeof(T)*nd, hipMemcpyHostToDevice),"hipMemcpy b to d_b"); \
	hipDeviceSynchronize();											\
	d1 = omp_get_wtime();												\
	FUNCNAME<T,DIV,OPT><<<BLOCKS,THREADS,S>>>						\
	  (d_v, d_b, d_sm.nlf, d_sm.ktmax,									\
	   d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt,		\
	   d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat,							\
	   d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense);				\
	hipDeviceSynchronize();											\
	d2 = omp_get_wtime();												\
	dtimes[l] = d2-d1;													\
  }																		\
  dmin = 9999.99;														\
  dmax = 0.0;															\
  davg = 0.0;															\
  for(i=M;i<L;i++){														\
	davg += dtimes[i];													\
	if(dmin>dtimes[i])dmin=dtimes[i];									\
	if(dmax<dtimes[i])dmax=dtimes[i];									\
  }																		\
  davg /= (L-M);
#endif

  /*
	完全逐次
	バリエーション：なし
  */
  if(kernel==0)
  {
	int a1, a2;
	char name[8], fname[0xff];
	a1=a2=0;
	snprintf(name,8,"_%d_%d",a1,a2);
	snprintf(fname,32,"result_cuda1_seq%s_%s.txt", name, typeid(T).name());
	printf("fname = %s\n", fname);
	EXEC(hmvm_cuda_seq,1,1,d_sm.ktmax*sizeof(T));
	BENCH(hmvm_cuda_seq,1,1,d_sm.ktmax*sizeof(T));
	printf("TIME %d hmvm_cuda1_seq%s min %e max %e avg %e\n", L-M, typeid(T).name(), dmin, dmax, davg);
  }

  /*
	block並列化
	ThreadBlockごとに1つの部分行列積(mat-mat-vecまたはmat-vec)を行う
	ThreadBlock内部は逐次
	バリエーション：なし
  */
  if(kernel==1)
  {
	int a1, a2;
	char name[8], fname[32];
	a1=a2=0;
	snprintf(name,8,"_%d_%d",a1,a2);
	snprintf(fname,32,"result_cuda1_block%s_%s.txt", name, typeid(T).name());
	printf("fname = %s\n", fname);
	EXEC(hmvm_cuda_block,d_sm.napprox+d_sm.ndense,1,d_sm.ktmax*sizeof(T));
	BENCH(hmvm_cuda_block,d_sm.napprox+d_sm.ndense,1,d_sm.ktmax*sizeof(T));
	printf("TIME %d hmvm_cuda1_block%s min %e max %e avg %e\n", L-M, typeid(T).name(), dmin, dmax, davg);
  }

/*
  hybrid1
  基本スレッド並列化カーネル
  <<<napprox+ndense,32>>>
  1 GEMV by 1 TB
  1 line by 1/div WARP
  バリエーション
  - div：1行を1/divのWARPで計算する、div=1,2,4,8,16,32
  - a2t：a2を転置版で計算するか否か
  - a2interchange：a2のループを入れ替えるか否か
  - aatomic：approxの計算をatomic優先にするかwarp shuffle併用するか
  - datomic：denseの計算をatomic優先にするかwarp shuffle併用するか
*/
  if(kernel>=1000&&kernel<1012){
	int subkernel = kernel-1000;
	int DIV, MUL, ATOMIC;
	ATOMIC = subkernel/6;
	DIV = pow(2,subkernel%6);
	char name[32], fname[64];
	snprintf(name,32,"hybrid1_div%d_atomic%d_%s", DIV, ATOMIC, typeid(T).name());
	snprintf(fname,64,"result_cuda1_%s.txt", name);
	printf("fname = %s\n", fname);
	printf("DIV = %d, ATOMIC = %d\n", DIV, ATOMIC);
	// EXEC
	{
	  hmvm_cuda_hybrid1_proxy<T>(d_v, d_b, d_sm.nlf, d_sm.ktmax,
								 d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt,
								 d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat,
								 d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense,
								 d_sm.napprox+d_sm.ndense,32,d_sm.ktmax*sizeof(T),DIV,ATOMIC, v, b, nd, fname, 0);
	}
	// BENCH
	{
	  hmvm_cuda_hybrid1_proxy<T>(d_v, d_b, d_sm.nlf, d_sm.ktmax,
								 d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt,
								 d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat,
								 d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense,
								 d_sm.napprox+d_sm.ndense,32,d_sm.ktmax*sizeof(T),DIV,ATOMIC, v, b, nd, fname, 1);
	}
  }

#if 1
/*
  hybrid2
  複数WARP単一GEMV個別行カーネル
  <<<napprox+ndense,32*mul>>>
  1 GEMV by 1 WARP
  1 line by 1/div WARP
  バリエーション
  - div：1行を1/divのWARPで計算する、div=1,2,4,8,16,32
  - mul：立ち上げるスレッド数(mul*32)、1つのmat-mat-vecまたはmat-vecをmul TBで実行、mul=1,2,3,...,
  - a2t：a2を転置版で計算するか否か
  - a2interchange：a2のループを入れ替えるか否か
  - aatomic：approxの計算をatomic優先にするかwarp shuffle併用するか
  - datomic：denseの計算をatomic優先にするかwarp shuffle併用するか
*/
  if(kernel>=2000&&kernel<2192){
	int subkernel = kernel-2000;
	int DIV, MUL, ATOMIC;
	ATOMIC = subkernel/96;
	MUL = (subkernel%96)/6 + 1;
	DIV = pow(2,subkernel%6);
	if((32*MUL)%DIV)printf("invalid parameters: 32*%d %% %d\n", MUL, DIV);
	char name[32], fname[64];
	snprintf(name,32,"hybrid2_div%d_mul%d_atomic%d_%s", DIV, MUL, ATOMIC, typeid(T).name());
	snprintf(fname,64,"result_cuda1_%s.txt", name);
	printf("fname = %s\n", fname);
	printf("DIV = %d, MUL = %d, ATOMIC = %d\n", DIV, MUL, ATOMIC);
	// EXEC
	{
	  hmvm_cuda_hybrid2_proxy<T>(d_v, d_b, d_sm.nlf, d_sm.ktmax,
								 d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt,
								 d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat,
								 d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense,
								 d_sm.napprox+d_sm.ndense,32*MUL,d_sm.ktmax*sizeof(T),DIV,MUL,ATOMIC, v, b, nd, fname, 0);
	}
	// BENCH
	{
	  hmvm_cuda_hybrid2_proxy<T>(d_v, d_b, d_sm.nlf, d_sm.ktmax,
								 d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt,
								 d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat,
								 d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense,
								 d_sm.napprox+d_sm.ndense,32*MUL,d_sm.ktmax*sizeof(T),DIV,MUL,ATOMIC, v, b, nd, fname, 1);
	}
  }
#endif

#if 1
/*
  hybrid3
  複数WARP個別GEMVカーネル
  <<<napprox/mul+ndense/mul, 32*mul>>>
  1 GEMV by 1 WARP
  1 line by 1/div WARP
  バリエーション
  - div：1行を1/divのWARPで計算する、div=1,2,4,8,16,32
  - mul：立ち上げるブロック数スレッド数の係数、1つのmat-mat-vecまたはmat-vecを1WARPで実行、mul=1,2,3,...,
  - a2t：a2を転置版で計算するか否か
  - a2interchange：a2のループを入れ替えるか否か
  - aatomic：approxの計算をatomic優先にするかwarp shuffle併用するか
  - datomic：denseの計算をatomic優先にするかwarp shuffle併用するか
*/
  if((kernel>=3000)&&(kernel<3192)){
	int subkernel = kernel-3000;
	int DIV, MUL, ATOMIC;
	ATOMIC = subkernel/96;
	MUL = (subkernel%96)/6 + 1;
	DIV = pow(2,subkernel%6);
	//ATOMIC = 1; MUL = 1; DIV = 1; // test
	char name[32], fname[64];
	snprintf(name,32,"hybrid3_div%d_mul%d_atomic%d_%s", DIV, MUL, ATOMIC, typeid(T).name());
	snprintf(fname,64,"result_cuda1_%s.txt", name);
	printf("fname = %s\n", fname);
	printf("DIV = %d, MUL = %d, ATOMIC = %d\n", DIV, MUL, ATOMIC);
	// EXEC
	{
    hmvm_cuda_hybrid3_proxy<T>(d_v, d_b, d_sm.nlf, d_sm.ktmax,
	d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt,
	d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat,
	d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense,
	(d_sm.napprox+MUL-1)/MUL+(d_sm.ndense+MUL-1)/MUL,32*MUL,d_sm.ktmax*sizeof(T),DIV,MUL,ATOMIC, v, b, nd, fname, 0);
	//(d_sm.ndense+MUL-1)/MUL,32*MUL,d_sm.ktmax*sizeof(T),DIV,MUL,ATOMIC, v, b, nd, fname, 0);
	}
	// BENCH
	if(0){
	  hmvm_cuda_hybrid3_proxy<T>(d_v, d_b, d_sm.nlf, d_sm.ktmax,
								 d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt,
								 d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat,
								 d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense,
								 (d_sm.napprox+MUL-1)/MUL+(d_sm.ndense+MUL-1)/MUL,32*MUL,d_sm.ktmax*sizeof(T),DIV,MUL,ATOMIC, v, b, nd, fname, 1);
	}
  }
#endif
  // ######## ######## ######## ######## ######## ######## ######## ########
  hipFree(d_sm.ltmtx);
  hipFree(d_sm.ndl);
  hipFree(d_sm.ndt);
  hipFree(d_sm.nstrtl);
  hipFree(d_sm.nstrtt);
  hipFree(d_sm.kt);
  hipFree(d_sm.a1);
  hipFree(d_sm.a2);
  hipFree(d_sm.rowmat);
  hipFree(d_sm.rowmat_t);
  //hipFree(d_zaut);
  //hipFree(d_zbut);
  hipFree(d_b);
  hipFree(d_v);

  delete [] v; delete [] tmp; delete [] zero;
  //free(v); free(tmp); free(zero);
  printf("hmvm_cuda1: end\n");
}


// ######## ######## ######## ######## ######## ######## ######## ########
// template関数の実体化のための宣言
// ######## ######## ######## ######## ######## ######## ######## ########
template void hmvm_cuda1<float>(matrix2<float>  *mat2, float *b, int kernel, int dump_result);
template void hmvm_cuda1<double>(matrix2<double> *mat2, double *b, int kernel, int dump_result);
