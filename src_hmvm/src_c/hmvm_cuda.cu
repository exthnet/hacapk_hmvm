#include "hip/hip_runtime.h"
// -*- C++ -*-
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <omp.h>
#include <hip/hip_runtime_api.h>

#include "hacapk_c.h"

__global__ void hmvm_cudaD_kernel000000
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat, int ndense, int *dense);

__global__ void hmvm_cudaD
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense);
__global__ void hmvm_cudaD_block
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat,
 int napprox, int *approx, int ndense, int *dense);

template <int div>
__global__ void hmvm_cudaD_kernel00dd00
(double *d_zaut, double *d_zu, int nlf, int ktmax,
 int *_ltmtx, int *_ndt, int *_ndl, int *_nstrtl, int *_nstrtt, int *_kt, int *a1, int *a2, double *rowmat, int ndense, int *dense);

#define CHECK_DO(act,msg) {ret=act; if(ret!=hipSuccess){printf("%s failed\n",msg);exit(-1);};}

void hmvm_cuda1(matrix2 mat2, double *b, int kernel)
{
  const int L=5, M=5;
  FILE *F;
  matrix2 d_sm;
  int i, l, nd = mat2.nd;
  double d1, d2, dtimes[L+M], dmin, dmax, davg1, davg2;
  double *v=NULL, *tmp=NULL, *zero;
  double *d_b, *d_v, *d_zaut, *d_zbut;
  int ip;
  int len, offset=0;
  hipError_t ret;
  printf("hmvm_cuda1: begin\n");
  v=(double*)malloc(sizeof(double)*mat2.nd);
  tmp=(double*)malloc(sizeof(double)*mat2.ktmax);
  zero = (double*)malloc(sizeof(double)*mat2.ktmax);
  for(i=0;i<nd;i++){
	v[i] = 0.0;
  }
  for(i=0;i<mat2.ktmax;i++){
	zero[i] = 0.0;
  }
  CHECK_DO(hipMalloc((void**)&d_zaut, sizeof(double)*mat2.nd),"hipMalloc z_aut");
  CHECK_DO(hipMalloc((void**)&d_zbut, sizeof(double)*mat2.ktmax),"hipMalloc zbut");
  CHECK_DO(hipMalloc((void**)&d_b, sizeof(double)*mat2.nd),"hipMalloc d_b");
  CHECK_DO(hipMalloc((void**)&d_v, sizeof(double)*mat2.nd),"cudaMallod d_v");
  //for(i=0;i<mat2.nd;i++){d_b[i]=NULL;d_v[i]=NULL;}

  len = mat2.len;
  printf("total length = %d\n", len);
  // host alloc
  // device alloc
  d_sm.nd = mat2.nd;
  d_sm.nlf = mat2.nlf;
  d_sm.ktmax = mat2.ktmax;
  hipMalloc((void**)&d_sm.ltmtx, sizeof(int)*mat2.nlf);
  hipMalloc((void**)&d_sm.ndl, sizeof(int)*mat2.nlf);
  hipMalloc((void**)&d_sm.ndt, sizeof(int)*mat2.nlf);
  hipMalloc((void**)&d_sm.nstrtl, sizeof(int)*mat2.nlf);
  hipMalloc((void**)&d_sm.nstrtt, sizeof(int)*mat2.nlf);
  hipMalloc((void**)&d_sm.kt, sizeof(int)*mat2.nlf);
  hipMalloc((void**)&d_sm.a1, sizeof(int)*mat2.nlf);
  hipMalloc((void**)&d_sm.a2, sizeof(int)*mat2.nlf);
  hipMalloc((void**)&d_sm.rowmat,sizeof(double)*mat2.len);
  hipMalloc((void**)&d_sm.rowmat_t,sizeof(double)*mat2.len);
  // memcpy
  CHECK_DO(hipMemcpy(d_sm.ltmtx, mat2.ltmtx, sizeof(int)*mat2.nlf, hipMemcpyHostToDevice),"d_sm.ltmtx");
  CHECK_DO(hipMemcpy(d_sm.ndt, mat2.ndt, sizeof(int)*mat2.nlf, hipMemcpyHostToDevice),"d_sm.ndt");
  CHECK_DO(hipMemcpy(d_sm.ndl, mat2.ndl, sizeof(int)*mat2.nlf, hipMemcpyHostToDevice),"d_sm.ndl");
  CHECK_DO(hipMemcpy(d_sm.nstrtl, mat2.nstrtl, sizeof(int)*mat2.nlf, hipMemcpyHostToDevice),"d_sm.nstrtl");
  CHECK_DO(hipMemcpy(d_sm.nstrtt, mat2.nstrtt, sizeof(int)*mat2.nlf, hipMemcpyHostToDevice),"d_sm.nstrtt");
  CHECK_DO(hipMemcpy(d_sm.kt, mat2.kt, sizeof(int)*mat2.nlf, hipMemcpyHostToDevice),"d_sm.kt");
  CHECK_DO(hipMemcpy(d_sm.a1, mat2.a1, sizeof(int)*mat2.nlf, hipMemcpyHostToDevice),"d_sm.a1");
  CHECK_DO(hipMemcpy(d_sm.a2, mat2.a2, sizeof(int)*mat2.nlf, hipMemcpyHostToDevice),"d_sm.a2");
  CHECK_DO(hipMemcpy(d_sm.rowmat, mat2.rowmat, sizeof(double)*mat2.len, hipMemcpyHostToDevice),"d_sm.rowmat");
  CHECK_DO(hipMemcpy(d_sm.rowmat_t, mat2.rowmat_t, sizeof(double)*mat2.len, hipMemcpyHostToDevice),"d_sm.rowmat_t");

  // 分離
  printf("begin splitting\n");
  mat2.approx = (int*)malloc(sizeof(int)*mat2.nlf);
  mat2.dense  = (int*)malloc(sizeof(int)*mat2.nlf);
  mat2.napprox = mat2.ndense = 0;
  for(ip=0; ip<mat2.nlf; ip++){
	if(mat2.ltmtx[ip]==1){
	  mat2.approx[mat2.napprox++] = ip;
	}else{
	  mat2.dense[mat2.ndense++] = ip;
	}
  }
  hipMalloc((void**)&d_sm.approx,sizeof(int)*mat2.napprox);
  hipMalloc((void**)&d_sm.dense,sizeof(int)*mat2.ndense);
  CHECK_DO(hipMemcpy(d_sm.approx, mat2.approx, sizeof(int)*mat2.napprox, hipMemcpyHostToDevice),"d_sm.approx");
  CHECK_DO(hipMemcpy(d_sm.dense, mat2.dense, sizeof(int)*mat2.ndense, hipMemcpyHostToDevice),"d_sm.dense");
  d_sm.napprox = mat2.napprox;
  d_sm.ndense = mat2.ndense;
  printf("end splitting\n");

#if 0
  hmvm_cudaD_kernel00dd00<16><<<mat2.ndense,32>>>						\
  (d_v, d_b, mat2.nlf, mat2.ktmax, mat2.ltmtx, mat2.ndt, mat2.ndl, mat2.nstrtl, mat2.nstrtt, mat2.kt, mat2.a1, mat2.a2, mat2.rowmat, mat2.ndense, mat2.dense);\


	hmvm_cudaD_kernel000000<<<mat2.ndense,32>>>						\
	  (d_v, d_b, mat2.nlf, mat2.ktmax, mat2.ltmtx, mat2.ndt, mat2.ndl, mat2.nstrtl, mat2.nstrtt, mat2.kt, mat2.a1, mat2.a2, mat2.rowmat, mat2.ndense, mat2.dense); \

  //FUNCNAME(d_v, d_b, d_sm, a1, a2);									\

#endif

#define BENCH(FUNCNAME,B,T,S)											\
  printf("nd = %d\n", nd);												\
  hipMemcpy(d_v, v, sizeof(double)*nd, hipMemcpyHostToDevice);		\
  hipMemcpy(d_b, b, sizeof(double)*nd, hipMemcpyHostToDevice);		\
  FUNCNAME<<<B,T,S>>> \
  (d_v, d_b, d_sm.nlf, d_sm.ktmax, d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt, d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat, \
   d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense);					\
  hipDeviceSynchronize();												\
  hipMemcpy(v, d_v, sizeof(double)*nd, hipMemcpyDeviceToHost);		\
  printf("write to %s\n", fname);												\
  F = fopen(fname, "w");												\
  for(i=0;i<nd;i++)fprintf(F, "%E\n", v[i]);							\
  fclose(F);

#if 0
  hmvm_cudaD<<<1,1,d_sm.ktmax*sizeof(double)>>>							\
  (d_v, d_b, d_sm.nlf, d_sm.ktmax, d_sm.ltmtx, d_sm.ndt, d_sm.ndl, d_sm.nstrtl, d_sm.nstrtt, d_sm.kt, d_sm.a1, d_sm.a2, d_sm.rowmat, \
   d_sm.napprox, d_sm.approx, d_sm.ndense, d_sm.dense);					\

#endif

#if 0
  for(l=0;l<M+L;l++){													\
	for(i=0;i<nd;i++)v[i] = 0.0;										\
	d1 = omp_get_wtime();												\
	hipDeviceSynchronize();											\
	d2 = omp_get_wtime();												\
	dtimes[l] = d2-d1;													\
  }																		\
  dmin = 9999.99;         dmax = 0.0;									\
  davg1 = 0.0;            davg2 = 0.0;									\
  for(i=0;i<M+L;i++)davg1 += dtimes[i];									\
  for(i=M;i<M+L;i++){													\
	if(dmin>dtimes[i])dmin=dtimes[i];									\
	if(dmax<dtimes[i])dmax=dtimes[i];									\
	davg2 += dtimes[i];													\
  }																		\
  davg1 /= (M+L);         davg2 /= L;
#endif

  if(kernel==0)
  {
	int a1, a2;
	char name[8], fname[32];
	a1=a2=0;
	snprintf(name,8,"_%d_%d",a1,a2);
	snprintf(fname,32,"hmvm_cuda1%s.txt",name);
	printf("fname = %s\n", fname);
	BENCH(hmvm_cudaD,1,1,d_sm.ktmax*sizeof(double));
	printf("TIME %d hmvm_cuda1%s min %e max %e avg1 %e avg2 %e |", M+L, name, dmin, dmax, davg1, davg2);
	for(i=0;i<M+L;i++)printf(" %e", dtimes[i]);
	printf("\n");
  }
  if(kernel==1)
  {
	int a1, a2;
	char name[8], fname[32];
	a1=a2=0;
	snprintf(name,8,"_%d_%d",a1,a2);
	snprintf(fname,32,"hmvm_cuda1blk%s.txt",name);
	printf("fname = %s\n", fname);
	BENCH(hmvm_cudaD_block,d_sm.napprox+d_sm.ndense,1,d_sm.ktmax*sizeof(double));
	printf("TIME %d hmvm_cuda1%s min %e max %e avg1 %e avg2 %e |", M+L, name, dmin, dmax, davg1, davg2);
	for(i=0;i<M+L;i++)printf(" %e", dtimes[i]);
	printf("\n");
  }

  // ######## ######## ######## ######## ######## ######## ######## ########
  hipFree(d_sm.ltmtx);
  hipFree(d_sm.ndl);
  hipFree(d_sm.ndt);
  hipFree(d_sm.nstrtl);
  hipFree(d_sm.nstrtt);
  hipFree(d_sm.kt);
  hipFree(d_sm.a1);
  hipFree(d_sm.a2);
  hipFree(d_sm.rowmat);
  hipFree(d_sm.rowmat_t);
  hipFree(d_zaut);
  hipFree(d_zbut);
  hipFree(d_b);
  hipFree(d_v);

  free(v); free(tmp); free(zero);
  printf("hmvm_cuda1: end\n");
}
